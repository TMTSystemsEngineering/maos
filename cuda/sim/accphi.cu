#include "hip/hip_runtime.h"
/*
  Copyright 2009-2025 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "accphi.h"
#include "cudata.h"
/**
   Ray tracing from map to loc with boundary check. Real input
   This is memory bound. So increasing # of points processed does not help.
*/
__global__ void map2loc_linear(Real* restrict out, const Real* restrict in,
	const int nx, const int ny, KARG_COMMON){
	int step=blockDim.x*gridDim.x;
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=step){
		Real x=loc[i][0]*dxi+dispx;
		Real y=loc[i][1]*dyi+dispy;
		int ix=Z(floor)(x);
		int iy=Z(floor)(y);
		x=x-ix; y=y-iy;
		if(ix>=0&&ix<nx-1&&iy>=0&&iy<ny-1){
			Real tmp=((+in[iy*nx+ix]*(1.f-x)+in[iy*nx+ix+1]*x)*(1.f-y)
				+(+in[(iy+1)*nx+ix]*(1.f-x)+in[(iy+1)*nx+ix+1]*x)*y);
			add_valid(out[i], alpha, tmp);
		}
	}
}

/**
   Ray tracing from map to loc with boundary check. Complex input
*/
__global__ void map2loc_linear(Real* restrict out, const Comp* restrict in,
	const int nx, const int ny, KARG_COMMON){
	int step=blockDim.x*gridDim.x;
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=step){
		Real x=loc[i][0]*dxi+dispx;
		Real y=loc[i][1]*dyi+dispy;
		int ix=Z(floor)(x);
		int iy=Z(floor)(y);
		x=x-ix; y=y-iy;
		if(ix>=0&&ix<nx-1&&iy>=0&&iy<ny-1){
			Real tmp=((+in[iy*nx+ix].x*(1.f-x)+in[iy*nx+ix+1].x*x)*(1.f-y)
				+(+in[(iy+1)*nx+ix].x*(1.f-x)+in[(iy+1)*nx+ix+1].x*x)*y);
			add_valid(out[i], alpha, tmp);
		}
	}
}
/*
  Ray tracing from map to loc without boundary check. Real input.
*/
__global__ void map2loc_linear_nocheck(Real* restrict out, const Real* restrict in,
	const int nx, const int ny, KARG_COMMON){
	int step=blockDim.x*gridDim.x;
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=step){
		Real x=loc[i][0]*dxi+dispx;
		Real y=loc[i][1]*dyi+dispy;
		int ix=Z(floor)(x);
		int iy=Z(floor)(y);
		x=x-ix; y=y-iy;
		out[i]+=alpha*((in[iy*nx+ix]*(1-x)+in[iy*nx+ix+1]*x)*(1-y)
			+(in[(iy+1)*nx+ix]*(1-x)+in[(iy+1)*nx+ix+1]*x)*y);
	}
}
/*
  Ray tracing from map to loc with wrapping. Real input.
*/
__global__ void map2loc_linear_wrap(Real* restrict out, const Real* restrict in,
	const int nx, const int ny, KARG_COMMON){
	int step=blockDim.x*gridDim.x;
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=step){
		Real x=loc[i][0]*dxi+dispx;
		Real y=loc[i][1]*dyi+dispy;
		int ix=Z(floor)(x);
		int iy=Z(floor)(y);
		x=x-ix; y=y-iy;
		while(ix<0) ix+=nx;
		while(iy<0) iy+=ny;
		while(ix>nx-1) ix-=nx;
		while(iy>ny-1) iy-=ny;
		int ix1=(ix==nx-1)?0:(ix+1);
		int iy1=(iy==ny-1)?0:(iy+1);
		out[i]+=alpha*((in[iy*nx+ix]*(1.f-x)+in[iy*nx+ix1]*x)*(1.f-y)
			+(in[(iy1)*nx+ix]*(1.f-x)+in[(iy1)*nx+ix1]*x)*y);
	}
}

/*This is memory bound. So increasing # of points processed does not help. */
__global__ void map2loc_cubic(Real* restrict out, const Real* restrict in,
	const int nx, const int ny, KARG_COMMON, const Real* cc){
	int step=blockDim.x*gridDim.x;
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=step){
		Real x=loc[i][0]*dxi+dispx;
		Real y=loc[i][1]*dyi+dispy;
		int ix=Z(floor)(x); x=x-ix;
		int iy=Z(floor)(y); y=y-iy;
		Real fx[4], fy;
		Real sum=0;
		if(ix<1||ix>nx-3||iy<1||iy>ny-3){
			continue;/*out of range. */
		}
		/*cc need to be in device memory for sm_13 to work.*/
		fx[0]=(1.f-x)*(1.f-x)*(cc[3]+cc[4]*(1.f-x));
		fx[1]=cc[0]+x*x*(cc[1]+cc[2]*x);
		fx[2]=cc[0]+(1.f-x)*(1.f-x)*(cc[1]+cc[2]*(1.f-x));
		fx[3]=x*x*(cc[3]+cc[4]*x);

		fy=(1.f-y)*(1.f-y)*(cc[3]+cc[4]*(1.f-y));
#pragma unroll
		for(int kx=-1; kx<3; kx++){
			sum+=fx[kx+1]*fy*in[(iy-1)*nx+kx+ix];
		}

		fy=cc[0]+y*y*(cc[1]+cc[2]*y);
#pragma unroll
		for(int kx=-1; kx<3; kx++){
			sum+=fx[kx+1]*fy*in[iy*nx+kx+ix];
		}

		fy=cc[0]+(1.f-y)*(1.f-y)*(cc[1]+cc[2]*(1.f-y));
#pragma unroll
		for(int kx=-1; kx<3; kx++){
			sum+=fx[kx+1]*fy*in[(iy+1)*nx+kx+ix];
		}

		fy=y*y*(cc[3]+cc[4]*y);
#pragma unroll
		for(int kx=-1; kx<3; kx++){
			sum+=fx[kx+1]*fy*in[(iy+2)*nx+kx+ix];
		}
		add_valid(out[i], alpha, sum);
	}
}

/**
   Ray tracing of atm.
*/
void atm2loc(Real* phiout, const culoc_t& loc, Real hs, Real hc, Real thetax, Real thetay,
	Real misregx, Real misregy, Real dt, int isim, Real atmalpha0, hipStream_t stream){
	cumapcell& cuatm=cudata->atm;
	Real atmalpha=atmalpha0;
	if(Z(fabs)(atmalpha)<EPS) return;
	const int atm_dtrat=cuglobal->atm_dtrat;
	if(cuglobal->atmscale&&!atm_dtrat){
		atmalpha*=cuglobal->atmscale->p[isim];
	}
	int nps=atm_dtrat?cuglobal->atm_nps:cudata->atm.N();
	real wt=1;
	for(int jps=0; jps<nps; jps++){
		int ips;
		if(atm_dtrat){
			ips=atm_interp(&wt, jps, isim, atm_dtrat, cudata->atm.N(), cuglobal->atm_interp);
			/*ips=wrap_seq(isim/atm_dtrat+jps, cudata->atm.N());
			Real wt2=0;
			if(nps>1&&atm_interp){
				wt2=(real)(isim%atm_dtrat)/atm_dtrat;
				if(atm_interp==2){
					wt2=pow(sin(wt2*M_PI/2), 2);//smoother interp with sin^2 function
				}
			}*/
			//dbg("isim=%d, atmalpha=%g\n", isim, atmalpha);
		}else{
			ips=jps;
		}
		const Real dx=cuatm[ips].dx;
		const Real dy=cuatm[ips].dy;
		const Real ht=cuatm[ips].ht;
		const Real vx=cuatm[ips].vx;
		const Real vy=cuatm[ips].vy;
		const Real scale=1.f-(ht-hc)/(hs-hc);
		if(scale<0) continue;
		const Real dispx=(ht*thetax+scale*misregx-vx*dt*isim-cuatm[ips].ox)/dx;
		const Real dispy=(ht*thetay+scale*misregy-vy*dt*isim-cuatm[ips].oy)/dy;
		const int nloc=loc.Nloc();

#define COMM loc(),loc.Nloc(),scale/dx,scale/dy, dispx, dispy, atmalpha*wt
		if(cuglobal->atm_full){
			map2loc_linear_wrap<<<DIM(nloc, 256), 0, stream>>>
				(phiout, cuatm[ips](), cuatm[ips].nx, cuatm[ips].ny, COMM);
		} else{/*we are gauranteed. */
			//check boundary
			if(loc.xmin/dx*scale+dispx>=0&&loc.ymin/dy*scale+dispy>=0
				&&loc.xmax/dx*scale+dispx+1<cuatm[ips].nx&&loc.ymax/dy*scale+dispy+1<cuatm[ips].ny){
				map2loc_linear_nocheck<<<DIM(nloc, 256), 0, stream>>>
					(phiout, cuatm[ips](), cuatm[ips].nx, cuatm[ips].ny, COMM);
			} else{
				warning("Unexpected: need to check boundary. min=(%g, %g), max=(%g, %g), map: (%ld, %ld)\n",
					loc.xmin/dx+dispx, loc.ymin/dx+dispy,
					loc.xmax/dx+dispx+1, loc.ymax/dx+dispy+1, cuatm[ips].nx, cuatm[ips].ny);
				print_backtrace();
				map2loc_linear<<<DIM(nloc, 256), 0, stream>>>
					(phiout, cuatm[ips](), cuatm[ips].nx, cuatm[ips].ny, COMM);
			}
		}
#undef COMM
	}
}
void map2loc(const cumap_t& map, const culoc_t& loc, Real* phiout,
	Real alpha, Real dispx, Real dispy, Real scale, int wrap, hipStream_t stream){
	if(scale<0) return;
	dispx=(dispx-map.ox)/map.dx;
	dispy=(dispy-map.oy)/map.dy;
	const int nloc=loc.Nloc();
	if(map.cubic_cc){//128 is a good number for cubic. 
		if(wrap){
			error("Not supported\n");
		} else{
			map2loc_cubic<<<DIM(nloc, 128), 0, stream>>>
				(phiout, map(), map.nx, map.ny, loc(), loc.Nloc(), scale/map.dx, scale/map.dy, dispx, dispy, alpha,
					map.cubic_cc());
		}
	} else{
		if(wrap){
			map2loc_linear_wrap<<<DIM(nloc, 256), 0, stream>>>
				(phiout, map(), map.nx, map.ny, loc(), loc.Nloc(), scale/map.dx, scale/map.dy, dispx, dispy, alpha);
		} else{
			map2loc_linear<<<DIM(nloc, 256), 0, stream>>>
				(phiout, map(), map.nx, map.ny, loc(), loc.Nloc(), scale/map.dx, scale/map.dy, dispx, dispy, alpha);
		}
	}
}

/**
   Ray tracing of dm. use a different loc for each dm. so that distortion can be
   properly accounted for. Use the other version if no distortion.
*/
void mapcell2loc(Real* phiout, const Array<culoc_t>& locondm, const cumapcell& cudm, 
	Real hs, Real hc, Real thetax, Real thetay, Real misregx, Real misregy, Real alpha, hipStream_t stream){
	const Real theta=RSS(thetax, thetay);
	for(int idm=0; idm<cudm.N(); idm++){
		assert(cudm[idm].ny>1);//prevent accidentally pass in a vector
		const Real ht=cudm[idm].ht;
		const Real scale=1.-(ht-hc)/(hs-hc);
		map2loc(cudm[idm], locondm[idm], phiout, alpha*cos(theta*cudm[idm].dratio), ht*thetax+scale*misregx, ht*thetay+scale*misregy, scale, 0, stream);
	}
}
/**
   Ray tracing of dm.
*/
void mapcell2loc(Real* phiout, const culoc_t& locout, const cumapcell& cudm, 
	Real hs, Real hc, Real thetax, Real thetay, Real misregx, Real misregy, Real alpha, hipStream_t stream){
	const Real theta=RSS(thetax, thetay);
	for(int idm=0; idm<cudm.N(); idm++){
		assert(cudm[idm].ny>1);//prevent accidentally pass in a vector
		const Real ht=cudm[idm].ht;
		const Real scale=1.-(ht-hc)/(hs-hc);
		map2loc(cudm[idm], locout, phiout, alpha*cos(theta*cudm[idm].dratio), ht*thetax+scale*misregx, ht*thetay+scale*misregy, scale, 0, stream);
	}/*idm */
}
/**
   Convert NGS mode vector to aperture grid for science directions.  */
void ngsmod2loc(curmat& opd, Real(*restrict loc)[2],
	const ngsmod_t* ngsmod, const real* mod,
	real thetax, real thetay,
	real alpha, hipStream_t stream){
	if(ngsmod->nmod==2){
		curaddptt(opd, loc, 0, mod[0]*alpha, mod[1]*alpha, stream);
	} else{
		const Real ht=ngsmod->hdm;
		const Real scale=ngsmod->scale;

		Real focus=0, ps1=0, ps2=0, ps3=0, astigx=0, astigy=0;
		if(ngsmod->indfocus){
			focus+=mod[ngsmod->indfocus];
		}
		if(ngsmod->indps){
			if(!ngsmod->ahstfocus){
				focus+=mod[ngsmod->indps]*(1.f-scale);
			}
			ps1=mod[ngsmod->indps];
			ps2=mod[ngsmod->indps+1];
			ps3=mod[ngsmod->indps+2];
		}
		if(ngsmod->indastig){
			astigx=mod[ngsmod->indastig];
			astigy=mod[ngsmod->indastig+1];
		}

		add_ngsmod_do<<<DIM(opd.N(), 256), 0, stream>>>
			(opd(), loc, opd.N(),
				mod[0], mod[1], ps1, ps2, ps3, astigx, astigy, focus,
				thetax, thetay, scale, ht, alpha);
	}
}
