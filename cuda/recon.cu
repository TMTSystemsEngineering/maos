#include "hip/hip_runtime.h"
/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "wfs.h"
#include "recon.h"
#include "pcg.h"
#include "curmat.h"
#include "cucmat.h"
#include "accphi.h"
#undef TIMING
#define TIMING 0
#if !TIMING
#define TIC_test
#define tic_test
#define toc_test(A)
#else
#define TIC_test TIC
#define tic_test tic
#define toc_test(A) toc2(A);tic
#endif
__global__ static void saloc2ptr_do(int (*restrict saptr)[2], float (*restrict saloc)[2], 
				    int nsa, float ox, float oy, float dx, float dy){
    const int step=blockDim.x * gridDim.x;
    const float dx1=1./dx;
    const float dy1=1./dy;
    for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){
	saptr[isa][0]=(int)roundf((saloc[isa][0]-ox)*dx1);
	saptr[isa][1]=(int)roundf((saloc[isa][1]-oy)*dy1);
    }
}
W01_T *gpu_get_W01(dsp *R_W0, dmat *R_W1){
    if(!R_W0 || !R_W1){
	error("R0, R1 must not be empty\n");
    }
    W01_T *W01=(W01_T*)calloc(1, sizeof(W01_T));
    cp2gpu(&W01->W1, R_W1);
    {
	/*W0 of partially illuminates subaps are stored as sparse matrix in
	  GPU. W0 of fully illuminated subaps are not.*/
	spint *pp=R_W0->p;
	spint *pi=R_W0->i;
	double *px=R_W0->x;
	dsp *W0new=spnew(R_W0->m, R_W0->n, R_W0->nzmax);
	spint *pp2=W0new->p;
	spint *pi2=W0new->i;
	double *px2=W0new->x;
	int *full;
	hipHostMalloc(&full, R_W0->n*sizeof(int));
	//#define W0_BW 1
	double W1max=dmax(R_W1);
	double thres=W1max*(1.f-1e-6);
	W01->W0v=(float)(W1max*4./9.);//max of W0 is 4/9 of max of W1. 
	info("W0v=%g\n", W01->W0v);
	int count=0;
	int count2=0;
	for(int ic=0; ic<R_W0->n; ic++){
	    pp2[ic]=count;
	    if(R_W1->p[ic]>thres){
		full[count2]=ic;
		count2++;
	    }else{
		int nv=pp[ic+1]-pp[ic];
		memcpy(pi2+count, pi+pp[ic], sizeof(spint)*nv);
		memcpy(px2+count, px+pp[ic], sizeof(double)*nv);
		count+=nv;
	    }
	}
	pp2[R_W0->n]=count;
	W0new->nzmax=count;
	cp2gpu(&W01->W0p, W0new);
	cp2gpu(&W01->W0f, full, count2);
	W01->nW0f=count2;
	spfree(W0new);
	hipHostFree(full);
    }
    return W01;
}
/*
  The caller must specify current GPU.
*/
static void gpu_setup_recon_do(const PARMS_T *parms, POWFS_T *powfs, RECON_T *recon){
    if(!cudata->recon){
	cudata->recon=new curecon_t;
    }
    curecon_t *curecon=cudata->recon;
    if(parms->recon.mvm && (!parms->gpu.tomo || !parms->gpu.fit)){
	return; /*Use CPU to assemble MVM*/
    }
    if(parms->recon.alg!=0){
	error("Only MVR is implemented in GPU\n");
    }
    cuwloc_t *cupowfs=cudata->powfs;
    if((parms->gpu.tomo || parms->gpu.fit) && !parms->sim.idealfit){
	curecon->opdr=curcellnew(recon->npsr, 1, recon->xnx, recon->xny);
	curecon->opdr_vec=curcellnew(recon->npsr, 1);
	for(int ips=0; ips<recon->npsr; ips++){
	    curecon->opdr_vec->p[ips]=curecon->opdr->p[ips]->ref(1);
	}
    }
    if(parms->gpu.tomo || parms->gpu.fit){
	curecon->amap=new cugrid_t[parms->ndm];
	curecon->xmap=new cugrid_t[recon->npsr];
	for(int idm=0; idm<parms->ndm; idm++){
	    curecon->amap[idm].init(recon->amap[idm]);
	}
	for(int ipsr=0; ipsr<recon->npsr; ipsr++){
	    curecon->xmap[ipsr].init(recon->xmap[ipsr]);
	}
	if(parms->fit.cachedm){
	    curecon->acmap=new cumap_t[parms->ndm];
	    for(int idm=0; idm<parms->ndm; idm++){
		curecon->acmap[idm].init(recon->acmap[idm]);
	    }
	}
	if(parms->fit.cachex){
	    curecon->xcmap=new cugrid_t[recon->npsr];
	    for(int ipsr=0; ipsr<recon->npsr; ipsr++){
		curecon->xcmap[ipsr].init(recon->xcmap[ipsr]);
	    }
	}
	curecon->pmap.init(recon->pmap);
	curecon->fmap.init(recon->fmap);
    }
    if(parms->gpu.tomo){
	for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
	    if(parms->powfs[ipowfs].skip) continue;
	    int nsa=powfs[ipowfs].pts->nsa;
	    hipMalloc(&cupowfs[ipowfs].saptr, nsa*2*sizeof(int));
	    saloc2ptr_do<<<DIM(nsa,256)>>>
		(cupowfs[ipowfs].saptr, cupowfs[ipowfs].saloc, nsa, 
		 recon->pmap->ox, recon->pmap->oy, recon->pmap->dx, recon->pmap->dy);
	    if(recon->GP->p[ipowfs]){
		const int use_mat=parms->tomo.pos==2 ||parms->tomo.pos==1 ;
		if(use_mat){//normally true
		    dsp *GP=sptrans(recon->GP->p[ipowfs]);
		    spint *pp=GP->p;
		    spint *pi=GP->i;
		    double *px=GP->x;
		    //convert the max float to max 2 byte integer
		    double pxscale=floor(32767./maxabs(px, GP->nzmax));
		    int np1=parms->tomo.pos+1;
		    int np=np1*np1;
		    int zmax=parms->tomo.pos;
		    short2 *partxy=(short2*)calloc(sizeof(short2),np*nsa);//need to zero memory
		    int nsa=powfs[ipowfs].pts->nsa;
		    double dx1=1./recon->ploc->dx;
		    double dy1=1./recon->ploc->dy;
		    for(int ic=0; ic<GP->n; ic++){
			int isa=(ic<nsa)?ic:(ic-nsa);
			for(spint ir=pp[ic]; ir<pp[ic+1]; ir++){
			    int ix=pi[ir];
			    double lx=recon->ploc->locx[ix];
			    double ly=recon->ploc->locy[ix];
			    double sx=powfs[ipowfs].saloc->locx[isa];
			    double sy=powfs[ipowfs].saloc->locy[isa];
			    int zx=(int)round((lx-sx)*dx1);
			    int zy=(int)round((ly-sy)*dy1);
			    /**
			       When the points used to generate GP align well
			       with the subaperture edge, the coupled points are
			       confined within the subaperture.
			    */
			    if(zx<0 || zx>zmax || zy<0 || zy>zmax){
				warning("isa=%d, zxy=%d %d\n", isa, zx, zy);
			    }
			    if(zx<0) zx=0;
			    if(zx>zmax) zx=zmax;
			    if(zy<0) zy=0;
			    if(zy>zmax) zy=zmax;
			    if(ic<nsa){
				partxy[np*isa+zx+zy*np1].x+=(short)round(px[ir]*pxscale);
			    }else{
				partxy[np*isa+zx+zy*np1].y+=(short)round(px[ir]*pxscale);
			    }
			}
		    }
		    cupowfs[ipowfs].GPp=new cumat<int>(np, nsa);
		    hipMemcpy(cupowfs[ipowfs].GPp->p, partxy, sizeof(int)*np*nsa, hipMemcpyHostToDevice);
		    cupowfs[ipowfs].GPscale=1./pxscale;
		    free(partxy);
		    spfree(GP);
		}else{/*use sparse */
		    cp2gpu(&cupowfs[ipowfs].GP, recon->GP->p[ipowfs]);
		}
	    }else{
		error("GP is required\n");
	    }
	}
 
	curecon->l2c=(float*)calloc(recon->npsr, sizeof(float));
	for(int ips=0; ips<recon->npsr; ips++){
	    float tmp=laplacian_coef(recon->r0, recon->wt->p[ips], recon->xmap[ips]->dx)*0.25f;
	    curecon->l2c[ips]=tmp*tmp*TOMOSCALE;
	}
	if(parms->tomo.piston_cr){
	    curecon->zzi=(int*)calloc(recon->npsr, sizeof(int));
	    curecon->zzv=(float*)calloc(recon->npsr, sizeof(float));
	    for(int ips=0; ips<recon->npsr; ips++){
		double r0=recon->r0;
		double dx=recon->xloc[ips]->dx;
		double wt=recon->wt->p[ips];
		int icenter=loccenter(recon->xloc[ips]);
		curecon->zzi[ips]=icenter;
		curecon->zzv[ips]=pow(laplacian_coef(r0,wt,dx),2)*TOMOSCALE*1e-6;
	    }
	}
	curecon->neai=curcellnew(parms->nwfsr, 1);
	/*convert recon->saneai to our format. */
	for(int iwfs=0; iwfs<parms->nwfsr; iwfs++){
	    int ipowfs=parms->wfsr[iwfs].powfs;
	    int nsa=powfs[ipowfs].pts->nsa;
	    int iwfs0=parms->recon.glao?iwfs:parms->powfs[ipowfs].wfs[0];/*first wfs in this group. */
	    if(iwfs!=iwfs0 && recon->saneai->p[iwfs+iwfs*parms->nwfsr]->p
	       ==recon->saneai->p[iwfs0+iwfs0*parms->nwfsr]->p){
		curecon->neai->p[iwfs]=curecon->neai->p[iwfs0]->ref();
	    }else{
		dsp *nea=recon->saneai->p[iwfs+iwfs*parms->nwfsr];
		spint *pp=nea->p;
		spint *pi=nea->i;
		double *px=nea->x;
		
		float (*neai)[3]=(float(*)[3])calloc(3*nsa, sizeof(float));
		if(nea->n!=2*nsa) error("nea doesn't have 2nsa x 2nsa dimension\n");
		for(int ic=0; ic<nea->n; ic++){
		    for(spint ir=pp[ic]; ir<pp[ic+1]; ir++){
			int ix=pi[ir];
			int isa=ic<nsa?ic:ic-nsa;
			float val=(float)px[ir]*TOMOSCALE;
			if(ix==ic){/*diagonal part. */
			    if(ic==isa){/*x */
				neai[isa][0]=val;
			    }else{/*y */
				neai[isa][1]=val;
			    }
			}else if(ix==ic-nsa || ix==ic+nsa){/*cross part. symmetric. */
			    neai[isa][2]=val;
			}else{
			    error("saneai has invalid format\n");
			}
		    }
		}
		curecon->neai->p[iwfs]=curnew(3, nsa);
		DO(hipMemcpy(curecon->neai->p[iwfs]->p, neai, 3*nsa*sizeof(float), hipMemcpyHostToDevice));
		free(neai);
	    }
	}/*for iwfs */
	CUDA_SYNC_DEVICE;
	if(recon->PTT && !curecon->PTT){//for t/t proj in 1)uplink t/t 2) recon
	    cp2gpu(&curecon->PTT, recon->PTT);
	}
	if(parms->tomo.precond==1){
	    gpu_setup_recon_fdpcg(parms, recon);
	}
	if(parms->tomo.alg==0){//CBS
	    chol_convert(recon->RL.C, 0);
	    cp2gpu(&curecon->RCl, recon->RL.C->Cl);
	    cp2gpu(&curecon->RCp, recon->RL.C->Cp, recon->RL.C->Cl->m);
	    if(recon->RL.Up){
		cp2gpu(&curecon->RUp, recon->RL.Up);
		cp2gpu(&curecon->RVp, recon->RL.Vp);
	    }
	}else if(parms->tomo.alg==2){//SVD
	    cp2gpu(&curecon->RMI, recon->RL.MI);
	}

	const int nwfs=parms->nwfsr;
	int nxp=recon->pmap->nx;
	int nyp=recon->pmap->ny;
  
	int nxpw[nwfs], nypw[nwfs], ngw[nwfs];
	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    const int ipowfs = parms->wfsr[iwfs].powfs;
	    if(parms->powfs[ipowfs].skip){
		nxpw[iwfs]=0;
		nypw[iwfs]=0;
		ngw[iwfs]=0;
	    }else{
		nxpw[iwfs]=nxp;
		nypw[iwfs]=nyp;
		ngw[iwfs]=powfs[ipowfs].pts->nsa*2;
	    }
	}
	curecon->opdwfs=curcellnew(nwfs, 1, nxpw, nypw);
	curecon->grad=curcellnew(nwfs, 1, ngw, (int*)NULL);
	curecon->ttf=curnew(3*nwfs, 1);

	GPU_PROP_GRID_T *hxdata=new GPU_PROP_GRID_T[nwfs*recon->npsr];
	DO(hipMalloc(&curecon->hxdata, sizeof(GPU_PROP_GRID_T)*nwfs*recon->npsr));
	for(int ips=0; ips<recon->npsr; ips++){ 
	    const float ht=recon->ht->p[ips]; 
	    for(int iwfs=0; iwfs<nwfs; iwfs++){
		const int ipowfs = parms->wfsr[iwfs].powfs;
		if(!parms->powfs[ipowfs].skip){
		    const float hs = parms->powfs[ipowfs].hs; 
		    const float scale = 1.f - ht/hs; 
		    float dispx=parms->wfsr[iwfs].thetax*ht; 
		    float dispy=parms->wfsr[iwfs].thetay*ht; 
		    cugrid_t pmapscale=curecon->pmap.scale(scale);
		    gpu_prop_grid_prep(hxdata+iwfs+ips*nwfs, pmapscale, curecon->xmap[ips],
				       dispx, dispy, NULL); 
		    {
			float tmp=laplacian_coef(recon->r0, recon->wt->p[ips], recon->xmap[ips]->dx)*0.25f;
			hxdata[iwfs+ips*nwfs].l2c=tmp*tmp*TOMOSCALE;
			if(parms->tomo.piston_cr){
			    hxdata[iwfs+ips*nwfs].zzi=loccenter(recon->xloc[ips]);
			    hxdata[iwfs+ips*nwfs].zzv=tmp*tmp*TOMOSCALE*1e-6;
			}else{
			    hxdata[iwfs+ips*nwfs].zzi=-1;
			}
		    }
		}
		hxdata[iwfs+ips*nwfs].togpu(&curecon->hxdata[iwfs+ips*nwfs]);
	    }
	}
	delete [] hxdata;
	GPU_GP_T *gpdata=new GPU_GP_T[nwfs];

	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    const int ipowfs = parms->wfsr[iwfs].powfs;
	    if(parms->powfs[ipowfs].skip) continue;
	    if(parms->powfs[ipowfs].wfs[0]!=0){
		error("Check this case. We had assumption that this powfs is the first group.\n");
	    }
	    gpdata[iwfs].ipowfs=ipowfs;
	    gpdata[iwfs].nwfs=parms->powfs[ipowfs].nwfsr;
	    gpdata[iwfs].jwfs=parms->powfs[ipowfs].wfsind[iwfs];//wfs index in this group
	    gpdata[iwfs].saptr=cupowfs[ipowfs].saptr;
	    gpdata[iwfs].dsa=powfs[ipowfs].pts->dsa;
	    gpdata[iwfs].GPp=(short2*)cupowfs[ipowfs].GPp->p;
	    gpdata[iwfs].GPscale=cupowfs[ipowfs].GPscale;
	    gpdata[iwfs].pos=parms->tomo.pos;
	    if(curecon->PTT){
		gpdata[iwfs].PTT=curecon->PTT->p[iwfs+iwfs*nwfs]->p;
	    }
	    if(parms->powfs[ipowfs].dfrs){
		/*We only use the first diagonal block for each powfs. The
		  off diagonal is simply -0.2 times the diagonal block*/
		int iwfs0=parms->powfs[ipowfs].wfs[0];//first wfs
		int iwfs1=parms->powfs[ipowfs].wfs[1];//second wfs
		if(!curecon->PDF){
		    curecon->PDF=curcellnew(nwfs, 1);
		}
		if(iwfs==iwfs0){//not the first one.
		    cp2gpu(&curecon->PDF->p[iwfs], recon->PDF->p[iwfs1*nwfs+iwfs1]);
		}
		gpdata[iwfs].PDF=curecon->PDF->p[iwfs0]->p;//every one in this group.
		if(curecon->PTT){
		    /*coupling between TT and DF modes. 
		      We desire (I-DF*PDF)(I-TT*PTT)g=(I-TT*PTT-DF*PDF+DF*PDF*TT*PTT)g
		      So we first compute tt=PTT*g; df=PDF*g; then
		      g2=(I-TT*tt-DF*(df-(PDF*TT)*tt))
		      Here we record the values of PDF*TT
		    */
		    dcell *pdftt=NULL;
		    dcellmm(&pdftt, recon->PDF, recon->TT, "nn", 1);
		    if(!curecon->PDFTT){
			curecon->PDFTT=curcellnew(nwfs, 1);
		    }
		    if(iwfs==iwfs0){
			cp2gpu(&curecon->PDFTT->p[iwfs], pdftt->p[iwfs1*nwfs+iwfs1]);
		    }
		    gpdata[iwfs].PDFTT=curecon->PDFTT->p[iwfs0]->p;
		    dcellfree(pdftt);
		}
	    }
	    gpdata[iwfs].neai=(const float(*)[3])curecon->neai->p[iwfs]->p;
	    gpdata[iwfs].nsa=powfs[ipowfs].pts->nsa;
	    gpdata[iwfs].nxp=recon->pmap->nx;
	    gpdata[iwfs].dxp=recon->pmap->dx;
	    gpdata[iwfs].dyp=recon->pmap->dy;
	    gpdata[iwfs].oxp=recon->pmap->ox;
	    gpdata[iwfs].oyp=recon->pmap->oy;
	}
	DO(hipMalloc(&curecon->gpdata, sizeof(GPU_GP_T)*nwfs));
	DO(hipMemcpy(curecon->gpdata, gpdata, sizeof(GPU_GP_T)*nwfs, hipMemcpyHostToDevice));
	delete [] gpdata;
    }
    if(parms->gpu.fit){
	long npsr=recon->npsr;
	long ndir=parms->fit.nfit;
	long ndm=parms->ndm;
	if(parms->gpu.fit==1){ /*For fitting using sparse matrix*/
	    cp2gpu(&curecon->FR, &recon->FR);
	    cp2gpu(&curecon->FL, &recon->FL);
	    curecon->dmfit=curcellnew(ndm, 1, recon->anloc, (long*)NULL);
	    curecon->fitrhs=curcellnew(ndm, 1, recon->anloc, (long*)NULL);
	    curecon->dmfit_vec=curecon->dmfit;
	}else if(parms->gpu.fit==2){ /*For fitting using ray tracing*/
	    if(!recon->W0 || !recon->W1){
		error("W0, W1 is required\n");
	    }
	    if(parms->sim.idealfit){
		cp2gpu(&curecon->floc, recon->floc);
		curecon->nfloc=recon->floc->nloc;
	    }
	    curecon->W01=gpu_get_W01(recon->W0, recon->W1);
	    if(recon->fitNW){
		dmat *fitNW=dcell2m(recon->fitNW);
		cp2gpu(&curecon->fitNW, fitNW);
		dfree(fitNW);
		curecon->dotNW=curnew(curecon->fitNW->ny, 1);
	    }
	    if(recon->actslave){
		cp2gpu(&curecon->actslave, recon->actslave);
	    }
	    curecon->dmfit=curcellnew(ndm, 1, recon->anx, recon->any);
	    curecon->dmfit_vec=curcellnew(ndm, 1);
	    for(int idm=0; idm<ndm; idm++){
		curecon->dmfit_vec->p[idm]=curecon->dmfit->p[idm]->ref(1);
	    }
	    curecon->fitrhs=curcellnew(ndm, 1, recon->anx, recon->any);
	    if(parms->fit.cachedm){
		long acnx[ndm], acny[ndm];
		for(int idm=0; idm<ndm; idm++){
		    acnx[idm]=curecon->acmap[idm].nx;
		    acny[idm]=curecon->acmap[idm].ny;
		}
		curecon->dmcache=curcellnew(ndm, 1, acnx, acny);
	    }
	    if(parms->fit.cachex){
		long xcnx[npsr], xcny[npsr];
		for(int ips=0; ips<npsr; ips++){
		    xcnx[ips]=curecon->xcmap[ips].nx;
		    xcny[ips]=curecon->xcmap[ips].ny;
		}
		curecon->xcache=curcellnew(npsr, 1, xcnx, xcny);
	    } 
	    cp2gpu(&curecon->fitwt, recon->fitwt);
	    curecon->cubic_cc=curcellnew(ndm, 1);
	    for(int idm=0; idm<ndm; idm++){
		if(parms->dm[idm].cubic){
		    curecon->cubic_cc->p[idm]=gpu_dmcubic_cc(parms->dm[idm].iac);
		}
	    }
	    //xloc -> floc
	    GPU_PROP_GRID_T *hxpdata=new GPU_PROP_GRID_T[npsr*ndir];
	    //dm -> floc
	    GPU_PROP_GRID_T *hadata=new GPU_PROP_GRID_T[ndm*ndir];
	    DO(hipMalloc(&curecon->hxpdata, sizeof(GPU_PROP_GRID_T)*npsr*ndir));
	    DO(hipMalloc(&curecon->hadata, sizeof(GPU_PROP_GRID_T)*ndm*ndir));
	    //dm: amap->acmap
	    GPU_PROP_GRID_T *ha0data=NULL, *ha1data=NULL;
	    if(parms->fit.cachedm){
		ha0data=new GPU_PROP_GRID_T[ndm];
		ha1data=new GPU_PROP_GRID_T[ndm*ndir];
		DO(hipMalloc(&curecon->ha0data, sizeof(GPU_PROP_GRID_T)*ndm));
		DO(hipMalloc(&curecon->ha1data, sizeof(GPU_PROP_GRID_T)*ndm*ndir));
	    }
	    GPU_PROP_GRID_T *hxp0data=NULL, *hxp1data=NULL;
	    if(parms->fit.cachex){
		hxp0data=new GPU_PROP_GRID_T[npsr];
		hxp1data=new GPU_PROP_GRID_T[npsr*ndir];
		DO(hipMalloc(&curecon->hxp0data, sizeof(GPU_PROP_GRID_T)*npsr));
		DO(hipMalloc(&curecon->hxp1data, sizeof(GPU_PROP_GRID_T)*npsr*ndir));
	    }


	    for(int ipsr=0; ipsr<npsr; ipsr++){
		const float ht=recon->ht->p[ipsr];
		if(parms->fit.cachex){
		    gpu_prop_grid_prep(hxp0data+ipsr, curecon->xcmap[ipsr], curecon->xmap[ipsr],
				       0,0, NULL);
		    hxp0data[ipsr].togpu(&curecon->hxp0data[ipsr]);
		}
		for(int idir=0; idir<ndir; idir++){
		    const float hs=parms->fit.hs[idir];
		    const float thetax=(float)parms->fit.thetax[idir];
		    const float thetay=(float)parms->fit.thetay[idir];
		    const float scale=1.f-ht/hs;
		    cugrid_t fmapscale=curecon->fmap*scale;
		    gpu_prop_grid_prep(hxpdata+idir+ipsr*ndir, fmapscale, curecon->xmap[ipsr],
				       thetax*ht, thetay*ht, NULL);
		    hxpdata[idir+ipsr*ndir].togpu(&curecon->hxpdata[idir+ipsr*ndir]);
		    if(parms->fit.cachex){
			gpu_prop_grid_prep(hxp1data+idir+ipsr*ndir, fmapscale, curecon->xcmap[ipsr],
					   thetax*ht, thetay*ht, NULL);
			hxp1data[idir+ipsr*ndir].togpu(&curecon->hxp1data[idir+ipsr*ndir]);
		    }
		}
	    }
	    for(int idm=0; idm<ndm; idm++){
		const float ht=parms->dm[idm].ht;
		if(parms->fit.cachedm){
		    gpu_prop_grid_prep(ha0data+idm, curecon->acmap[idm], curecon->amap[idm],
				       0, 0, curecon->cubic_cc->p[idm]);
		    ha0data[idm].togpu(&curecon->ha0data[idm]);
		}
		for(int idir=0; idir<ndir; idir++){
		    const float hs=parms->fit.hs[idir];
		    const float thetax=(float)parms->fit.thetax[idir];
		    const float thetay=(float)parms->fit.thetay[idir];
		    const float scale=1.f-ht/hs;
		    cugrid_t fmapscale=curecon->fmap*scale;
		    if(parms->fit.cachedm){
			gpu_prop_grid_prep(ha1data+idir+idm*ndir, fmapscale, curecon->acmap[idm],
					   thetax*ht, thetay*ht, NULL);
			ha1data[idir+idm*ndir].togpu(&curecon->ha1data[idir+idm*ndir]);
		    }
		    gpu_prop_grid_prep(hadata+idir+idm*ndir, fmapscale, curecon->amap[idm],
				       thetax*ht, thetay*ht, curecon->cubic_cc->p[idm]);	
		    hadata[idir+idm*ndir].togpu(&curecon->hadata[idir+idm*ndir]);
		}
	    }
	    delete [] hxpdata;
	    delete [] hxp0data;
	    delete [] hxp1data;
	    delete [] hadata;
	    delete [] ha0data;
	    delete [] ha1data;
	}

	if(parms->fit.alg==0){
	    chol_convert(recon->FL.C, 0);
	    cp2gpu(&curecon->FCl, recon->FL.C->Cl);
	    cp2gpu(&curecon->FCp, recon->FL.C->Cp, recon->FL.C->Cl->m);
	    if(recon->FL.Up){
		cp2gpu(&curecon->FUp, recon->FL.Up);
		cp2gpu(&curecon->FVp, recon->FL.Vp);
	    }
	}else if(parms->fit.alg==2){
	    cp2gpu(&curecon->FMI, recon->FL.MI);
	}
	const int nfit=parms->fit.nfit;
	long fnx[nfit],fny[nfit];
	for(int ifit=0; ifit<nfit; ifit++){
	    fnx[ifit]=recon->fmap->nx;
	    fny[ifit]=recon->fmap->ny;
	}
	curecon->opdfit=curcellnew(nfit, 1, fnx, fny);
	curecon->opdfit2=curcellnew(nfit, 1, fnx, fny);
	curecon->opdfitv=curnew(recon->fmap->nx*recon->fmap->ny, nfit, curecon->opdfit->m->p, 0);
	curecon->opdfit2v=curnew(recon->fmap->nx*recon->fmap->ny, nfit, curecon->opdfit2->m->p, 0);
	curecon->pis=curnew(1, parms->fit.nfit);
    }
 
    if(recon->RFlgsx){
	cp2gpu(&curecon->RFlgsx, recon->RFlgsx);
    }
    if(recon->RFngsx){
	cp2gpu(&curecon->RFngsx, recon->RFngsx);
    }
    if(recon->RFdfx){
	cp2gpu(&curecon->RFdfx, recon->RFdfx);
    }
    if(recon->GXL){
	cp2gpu(&curecon->GXL, recon->GXL);
    }
    gpu_print_mem("recon init");
}
void gpu_setup_recon(const PARMS_T *parms, POWFS_T *powfs, RECON_T *recon){
    if(parms->recon.mvm && parms->gpu.tomo && parms->gpu.fit && !parms->load.mvm){
	for(int igpu=0; igpu<NGPU; igpu++){
	    gpu_set(igpu);
	    gpu_setup_recon_do(parms, powfs, recon);
	}
    }else{
	gpu_set(gpu_recon);
	gpu_setup_recon_do(parms, powfs, recon);
    }
}
/*Copy FDPCG information to GPU. This may be done every time step if prediction is used.*/
static void gpu_setup_recon_fdpcg_do(const PARMS_T *parms, RECON_T *recon){
    if(!parms->tomo.precond==1) return;
    curecon_t *curecon=cudata->recon;
    FDPCG_T *fdpcg=recon->fdpcg;
    if(!fdpcg){
	return;
    }
    int bs=fdpcg->bs;
    int nb=(fdpcg->nbx/2+1)*fdpcg->nby;//half frequency range
    if(curecon->fdpcg){//already initialized. Just update Mb
	int nxsave=fdpcg->Mbinv->nx;
	fdpcg->Mbinv->nx=nb;
	cp2gpu(&curecon->fdpcg->Mb, fdpcg->Mbinv);
	fdpcg->Mbinv->nx=nxsave;
	return;
    }
    cufdpcg_t *cufd=curecon->fdpcg=new cufdpcg_t;
    cufd->scale=fdpcg->scale;
    cp2gpu(&cufd->perm, fdpcg->permhf, nb*bs);
    //copy only needed blocks to gpu
    int nxsave=fdpcg->Mbinv->nx;
    fdpcg->Mbinv->nx=nb;
    cp2gpu(&cufd->Mb, fdpcg->Mbinv);
    fdpcg->Mbinv->nx=nxsave;
    int nps=recon->npsr;
    int count=0;
    int osi=-1;
    int start[nps];
    for(int ips=0; ips<nps; ips++){
	/*group layers with the same os together in a batch fft.*/
	if(osi != parms->atmr.os[ips]){
	    start[count]=ips;
	    osi = parms->atmr.os[ips];
	    count++;
	}
    }
    cufd->fft=(hipfftHandle*)calloc(count, sizeof(hipfftHandle));
    cufd->ffti=(hipfftHandle*)calloc(count, sizeof(hipfftHandle));
    cufd->fftnc=count;
    cufd->fftips=(int*)calloc(count+1, sizeof(int));
    for(int ic=0; ic<count; ic++){
	cufd->fftips[ic]=start[ic];
    }
    cufd->fftips[count]=nps;
    for(int ic=0; ic<count; ic++){
	int ncomp[2];
	/*Notice the reverse in specifying dimensions. THe first element is outmost rank.*/
	ncomp[0]=recon->xny[start[ic]];
	ncomp[1]=recon->xnx[start[ic]];

	int nembed[2];
	nembed[0]=recon->xnx[start[ic]]*recon->xny[start[ic]];
	nembed[1]=recon->xnx[start[ic]];
	DO(hipfftPlanMany(&cufd->fft[ic], 2, ncomp, 
			 nembed, 1, ncomp[0]*ncomp[1], 
			 nembed, 1, ncomp[0]*ncomp[1], 
			 HIPFFT_R2C, cufd->fftips[ic+1]-cufd->fftips[ic]));
	DO(hipfftPlanMany(&cufd->ffti[ic], 2, ncomp, 
			 nembed, 1, ncomp[0]*ncomp[1], 
			 nembed, 1, ncomp[0]*ncomp[1],
			 HIPFFT_C2R, cufd->fftips[ic+1]-cufd->fftips[ic]));
	DO(hipfftSetStream(cufd->ffti[ic], curecon->cgstream));

	DO(hipfftSetStream(cufd->fft[ic], curecon->cgstream));
    }
    cufd->xhat1=cuccellnew(recon->npsr, 1, recon->xnx, recon->xny);
    {
	int nby=256/bs;//number of blocks in each grid
	int nbz=nb/nby;//number of grids to launch.
	while(nb!=nbz*nby){
	    nby--;
	    nbz=nb/nby;
	}
	cufd->nby=nby;
	cufd->nbz=nbz;
    }
    /* notice: performance may be improved by using
       R2C FFTs instead of C2C. Need to update perm
       and Mbinv to use R2C.*/
    GPU_FDPCG_T *fddata=new GPU_FDPCG_T[nps];
    for(int ips=0; ips<nps; ips++){
	fddata[ips].nx=recon->xnx[ips];
	fddata[ips].ny=recon->xny[ips];
	if(cufd->scale){
	    fddata[ips].scale=1.f/sqrtf((float)(recon->xnx[ips]*recon->xny[ips]));
	}else{
	    fddata[ips].scale=1.f;
	}
    }
    hipMalloc(&curecon->fddata, sizeof(GPU_FDPCG_T)*nps);
    hipMemcpy(curecon->fddata, fddata, sizeof(GPU_FDPCG_T)*nps, hipMemcpyHostToDevice);
    delete [] fddata;
}
void gpu_setup_recon_fdpcg(const PARMS_T *parms, RECON_T *recon){
    if(parms->recon.mvm && parms->gpu.tomo && parms->gpu.fit && !parms->load.mvm){
	for(int igpu=0; igpu<NGPU; igpu++){
	    gpu_set(igpu);
	    gpu_setup_recon_fdpcg_do(parms, recon);
	}
    }else{
	gpu_set(gpu_recon);
	gpu_setup_recon_fdpcg_do(parms, recon);
    }
}
void gpu_recon_free_do(){
    curecon_t *curecon=cudata->recon;
    if(!curecon) return;
    curcellfree(curecon->neai);
    curcellfree(curecon->opdwfs);
    curcellfree(curecon->grad); 
    curcellfree(curecon->opdr); 
    curcellfree(curecon->opdr_vec); 
    delete curecon->fdpcg;
    if(curecon->dmfit_vec!=curecon->dmfit){
	curcellfree(curecon->dmfit_vec);
    }else{
	curecon->dmfit_vec=NULL;
    }
    curcellfree(curecon->dmfit);
    free(curecon->l2c);
    free(curecon->zzi);
    free(curecon->zzv);
    if(curecon->W01){
	W01_T *W01=curecon->W01;
	curfree(W01->W1);
	delete W01->W0p;
	hipFree(W01->W0f);
    }
    curcellfree(curecon->opdfit);
    curcellfree(curecon->opdfit2);
    curfree(curecon->pis);
    hipFree(curecon->floc);
    curcellfree(curecon->fitNW);
    delete curecon->actslave;
    delete curecon->RCl;
    hipFree(curecon->RCp);
    curfree(curecon->RUp);
    curfree(curecon->RVp);
    curfree(curecon->RMI);
    delete curecon->FCl;
    hipFree(curecon->FCp);
    curfree(curecon->FUp);
    curfree(curecon->FVp);
    curfree(curecon->FMI);
    curcellfree(curecon->GXL);
    //    delete curecon;
}
void gpu_recon_free(){
    for(int igpu=0; igpu<NGPU; igpu++){
	gpu_set(igpu);
	if(cudata->recon){
	    gpu_recon_free_do();
	}
    }
}
void gpu_setup_recon_mvm(const PARMS_T *parms, RECON_T *recon, POWFS_T *powfs){
    /*The following routine assemble MVM and put in recon->MVM*/
    if(parms->recon.mvm==1){
	gpu_setup_recon_mvm_trans(parms, recon, powfs);
    }else{
	gpu_setup_recon_mvm_direct(parms, recon, powfs);
    }
    for(int igpu=0; igpu<NGPU; igpu++){
	gpu_set(igpu);
	gpu_recon_free_do();
	CUDA_SYNC_DEVICE;
    }///for GPU
    if(!parms->sim.mvmport){
	gpu_set(gpu_recon);
	curecon_t *curecon=cudata->recon;
	cp2gpu(&curecon->MVM, recon->MVM);
    }
    gpu_print_mem("MVM");
}
void gpu_setup_recon_predict_do(const PARMS_T *parms, RECON_T *recon){
    if(!parms->gpu.tomo || !parms->tomo.predict){
	return;
    }
    curecon_t *curecon=cudata->recon;
    const int nwfs=parms->nwfsr;
    GPU_PROP_GRID_T *hxdata=new GPU_PROP_GRID_T[nwfs*recon->npsr];
    for(int ips=0; ips<recon->npsr; ips++){ 
	const float ht=recon->ht->p[ips]; 
	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    const int ipowfs = parms->wfsr[iwfs].powfs;
	    if(parms->powfs[ipowfs].skip) continue;
	    const float hs = parms->powfs[ipowfs].hs; 
	    const float scale = 1.f - ht/hs; 
	    float dispx=parms->wfsr[iwfs].thetax*ht; 
	    float dispy=parms->wfsr[iwfs].thetay*ht; 
	    if(parms->tomo.predict){ 
		int ips0=parms->atmr.indps[ips]; 
		dispx+=cudata->atm[ips0].vx*parms->sim.dt*2; 
		dispy+=cudata->atm[ips0].vy*parms->sim.dt*2; 
	    } 
	    cugrid_t pmapscale=curecon->pmap*scale;
	    gpu_prop_grid_prep(hxdata+iwfs+ips*nwfs,  pmapscale, curecon->xmap[ips],
			       dispx, dispy, NULL); 
	    {
		float tmp=laplacian_coef(recon->r0, recon->wt->p[ips], recon->xmap[ips]->dx)*0.25f;
		hxdata[iwfs+ips*nwfs].l2c=tmp*tmp*TOMOSCALE;
		if(parms->tomo.piston_cr){
		    hxdata[iwfs+ips*nwfs].zzi=loccenter(recon->xloc[ips]);
		    hxdata[iwfs+ips*nwfs].zzv=tmp*tmp*TOMOSCALE*1e-6;
		}else{
		    hxdata[iwfs+ips*nwfs].zzi=-1;
		}
	    }
	}
    }
    if(!curecon->hxdata){
	DO(hipMalloc(&curecon->hxdata, sizeof(GPU_PROP_GRID_T)*nwfs*recon->npsr));
    }
    DO(hipMemcpy(curecon->hxdata, hxdata, sizeof(GPU_PROP_GRID_T)*nwfs*recon->npsr, hipMemcpyHostToDevice));
    delete [] hxdata;
}
void gpu_setup_recon_predict(const PARMS_T *parms, RECON_T *recon){
    if(parms->recon.mvm && parms->gpu.tomo && parms->gpu.fit && !parms->load.mvm){
	for(int igpu=0; igpu<NGPU; igpu++){
	    gpu_set(igpu);
	    gpu_setup_recon_predict_do(parms, recon);
	}
    }else{
	gpu_set(gpu_recon);
	gpu_setup_recon_predict_do(parms, recon);
    }
}
/*update reconstruction parameters after slodar.*/
void gpu_update_recon(const PARMS_T *parms, RECON_T *recon){
    gpu_set(gpu_recon);
    TO_IMPLEMENT;//copy to GPU struct (hxtdata)
    curecon_t *curecon=cudata->recon;
    for(int ips=0; ips<recon->npsr; ips++){
	float tmp=laplacian_coef(recon->r0, recon->wt->p[ips], recon->xmap[ips]->dx)*0.25f;
	curecon->l2c[ips]=tmp*tmp*TOMOSCALE;
    }
    if(parms->tomo.piston_cr){
	for(int ips=0; ips<recon->npsr; ips++){
	    double r0=recon->r0;
	    double dx=recon->xloc[ips]->dx;
	    double wt=recon->wt->p[ips];
	    int icenter=loccenter(recon->xloc[ips]);
	    curecon->zzi[ips]=icenter;
	    curecon->zzv[ips]=pow(laplacian_coef(r0,wt,dx),2)*TOMOSCALE*1e-6;
	}
    }
}
void gpu_recon_reset(const PARMS_T *parms){/*reset warm restart.*/
    gpu_set(gpu_recon);
    curecon_t *curecon=cudata->recon;
    curcellzero(curecon->opdr, 0);
    curcellzero(curecon->dmfit, 0);
    if(curecon->dm_wfs){
	for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	    curcellzero(curecon->dm_wfs[iwfs], 0);
	}
    }
    if(curecon->dm_evl){
	for(int ievl=0; ievl<parms->evl.nevl; ievl++){
	    curcellzero(curecon->dm_evl[ievl], 0);
	}
    }
    for(int igpu=0; igpu<NGPU; igpu++){
	gpu_set(igpu);
	if(cudata->dm_wfs){
	    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
		cudata->dm_wfs[iwfs].p.zero();
	    }
	}
	if(cudata->dm_evl){
	    for(int ievl=0; ievl<parms->evl.nevl; ievl++){
		cudata->dm_evl[ievl].p.zero();
	    }
	}
	CUDA_SYNC_DEVICE;
    }
}

void gpu_tomo(SIM_T *simu){
    gpu_set(gpu_recon);
    curecon_t *curecon=cudata->recon;
    TIC_test;tic_test;
    const PARMS_T *parms=simu->parms;
    RECON_T *recon=simu->recon;
    curecon->reconisim=simu->reconisim;
#if 0
    gpu_tomo_test(simu);
#endif
    toc_test("Before gradin");
    cp2gpu(&curecon->gradin, parms->tomo.psol?simu->gradlastol:simu->gradlastcl);
    toc_test("Gradin");
    hipProfilerStart();
    curcell *opdrsave=NULL;
    warning_once("remove opdrsave after debugging\n");
    curcellcp(&opdrsave, curecon->opdr, curecon->cgstream);
    simu->cgres->p[0]->p[simu->reconisim]=
	gpu_tomo_do(parms, recon, curecon->opdr, NULL, curecon->gradin, curecon->cgstream);
    //Sanity check the result
    float opdrmax=curcellmax(curecon->opdr, curecon->cgstream);
    if(curecon->opdr->nx>1 && opdrmax>4e-6){
	simu->status->warning=2;
	info("opdrmax=%g\n", opdrmax);
	curcellwrite(curecon->gradin, "dbg_gradin_%d", simu->reconisim);
	curcellwrite(opdrsave, "dbg_opdrlast_%d", simu->reconisim);
	curcellwrite(curecon->opdr, "dbg_opdr_%d", simu->reconisim);
	curcellcp(&curecon->opdr, opdrsave, curecon->cgstream);
	extern int pcg_save;
	pcg_save=1;
	double newres=gpu_tomo_do(parms, recon, curecon->opdr, NULL, curecon->gradin, curecon->cgstream);
	pcg_save=0;
	curcellwrite(curecon->opdr, "dbg_opdrredo_%d", simu->reconisim);
	info("oldres=%g. newres=%g\n", simu->cgres->p[0]->p[simu->reconisim], newres);
    }
    curfree(opdrsave);
    if(!parms->gpu.fit || parms->save.opdr || (recon->moao && !parms->gpu.moao)){
	cp2cpu(&simu->opdr, 0, curecon->opdr_vec, 1, curecon->cgstream);
    }
    if(parms->recon.split==2){
	curcell *gngsmvst=NULL;
	curcellmm(&gngsmvst, 1, curecon->GXL, curecon->opdr_vec, "nn", 1./parms->sim.dtrat_lo, curecon->cgstream);
	add2cpu(&simu->gngsmvst, gngsmvst, curecon->cgstream);
	curfree(gngsmvst);
    }
    if(parms->dbg.deltafocus){
	curcell *tmp1=NULL;
	curcellmm(&tmp1, 1, curecon->RFdfx, curecon->opdr_vec, "nn", 1, curecon->cgstream);
	scell *tmp=NULL;
	cp2cpu(&tmp, tmp1, curecon->cgstream);
	curcellfree(tmp1);
	if(tmp->nx!=1 || tmp->ny!=1 || tmp->p[0]->nx!=1 || tmp->p[0]->ny!=1){
	    error("Wrong format");
	}
	curecon->cgstream.sync();
	simu->deltafocus=tmp->p[0]->p[0];
	scellfree(tmp); 
    }else{
	curecon->cgstream.sync();
    }
    hipProfilerStop();
    toc_test("Tomo");
}

void gpu_fit(SIM_T *simu){
    TIC_test;tic_test;
    gpu_set(gpu_recon);
    curecon_t *curecon=cudata->recon;
    const PARMS_T *parms=simu->parms;
    const RECON_T *recon=simu->recon;
    if(!parms->gpu.tomo){
	cp2gpu(&curecon->opdr_vec, simu->opdr);
    }
#if 0
    gpu_fit_test(simu);
#endif
    toc_test("Before FitR");
    curcell *fitsave=NULL;//for debugging purpose.
    warning_once("remove fitsave after debugging\n");
    curcellcp(&fitsave, curecon->dmfit, curecon->cgstream);
    simu->cgres->p[1]->p[simu->reconisim]=
	gpu_fit_do(parms, recon, curecon->dmfit, curecon->opdr, curecon->cgstream);
    curecon->cgstream.sync();
    if(simu->reconisim>0 && simu->cgres->p[1]->p[simu->reconisim]>simu->cgres->p[1]->p[simu->reconisim-1]*2){
	simu->status->warning=2;
	curcellwrite(curecon->gradin, "dbg_gradin_%d", simu->reconisim);
	curcellwrite(curecon->dmfit, "dbg_dmfit_%d", simu->reconisim);
	curcellwrite(curecon->opdr, "dbg_opdr_%d", simu->reconisim);
	curcellwrite(fitsave, "dbg_dmfitlast_%d", simu->reconisim);
	curcellcp(&curecon->dmfit, fitsave, curecon->cgstream);
	double newres=gpu_fit_do(parms, recon, curecon->dmfit, curecon->opdr, curecon->cgstream);
	curcellwrite(curecon->dmfit, "dbg_dmfitredo_%d", simu->reconisim);
	info("oldres=%g newres=%g\n", simu->cgres->p[1]->p[simu->reconisim], newres);
    }
    cp2cpu(&simu->dmfit, 0, curecon->dmfit_vec, 1, curecon->cgstream);
    curfree(fitsave);
    /*Don't free opdr. Needed for warm restart in tomo.*/
    toc_test("Fit");
}
void gpu_recon_mvm(SIM_T *simu){
    const PARMS_T *parms=simu->parms;
    gpu_set(gpu_recon);
    curecon_t *curecon=cudata->recon;
    cp2gpu(&curecon->gradin, parms->tomo.psol?simu->gradlastol:simu->gradlastcl);
    curcellmm(&curecon->dmfit_vec, 0., curecon->MVM, curecon->gradin,"nn", 1., curecon->cgstream);
    cp2cpu(&simu->dmerr, 0., curecon->dmfit_vec, 1., curecon->cgstream);
    curecon->cgstream.sync();
}
