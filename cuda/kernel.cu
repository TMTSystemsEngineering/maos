#include "hip/hip_runtime.h"
/*
  Copyright 2009-2012 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "kernel.h"
/**
   A few kernels.
*/


__global__ void set_do(float *a, float alpha, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=alpha;
    }
}

__global__ void add_ptt_do(float *restrict opd, float (*restrict loc)[2], 
			   int n, float pis, float tx, float ty){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	opd[i]+=pis+loc[i][0]*tx+loc[i][1]*ty;
    }
}

__global__ void add_ngsmod_do(float *restrict opd, float (*restrict loc)[2], int n, 
			      float m0, float m1, float m2, float m3, float m4,
			      float thetax, float thetay, float scale, float ht, float MCC_fcp, float alpha
			      ){
    float scale1=1.f-scale;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	float x=loc[i][0];
	float y=loc[i][1];
	float xy=x*y;
	float x2=x*x;
	float y2=y*y;
	opd[i]+= alpha*(+x*m0
			+y*m1
			+m2*((x2+y2-MCC_fcp)*scale1-2*scale*ht*(thetax*x+thetay*y))
			+m3*((x2-y2)*scale1 - 2*scale*ht*(thetax*x-thetay*y))
			+m4*(xy*scale1-scale*ht*(thetay*x+thetax*y)));
    }
}
__global__ void inn_do(float *restrict res, const float *a, const float *b, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i]*b[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	atomicAdd(res, sb[0]);
    }
}
