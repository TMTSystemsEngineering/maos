#include "hip/hip_runtime.h"
/*
  Copyright 2009, 2010, 2011 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "kernel.h"
/**
   A few kernels.
*/


__global__ void set_do(float *a, float alpha, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=alpha;
    }
}

__global__ void show_do(float *a, int nx, int ny){
    const int stepx=blockDim.x * gridDim.x;
    const int stepy=blockDim.y * gridDim.y;
    for(int iy=blockIdx.y * blockDim.y + threadIdx.y; iy<ny; iy+=stepy){
	for(int ix=blockIdx.x * blockDim.x + threadIdx.x; ix<nx; ix+=stepx){
	    printf("a(%d,%d)=%g\n", ix, iy, a[ix+iy*nx]);
	}
    }
}
__global__ void add_ptt_do(float *restrict opd, float (*restrict loc)[2], 
			   int n, float pis, float tx, float ty){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	opd[i]+=pis+loc[i][0]*tx+loc[i][1]*ty;
    }
}

__global__ void add_ngsmod_do(float *restrict opd, float (*restrict loc)[2], int n, 
			      float m0, float m1, float m2, float m3, float m4,
			      float thetax, float thetay, float scale, float ht, float MCC_fcp, float alpha
			      ){
    float scale1=1.f-scale;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	float x=loc[i][0];
	float y=loc[i][1];
	float xy=x*y;
	float x2=x*x;
	float y2=y*y;
	opd[i]+= alpha*(+x*m0
			+y*m1
			+m2*((x2+y2-MCC_fcp)*scale1-2*scale*ht*(thetax*x+thetay*y))
			+m3*((x2-y2)*scale1 - 2*scale*ht*(thetax*x-thetay*y))
			+m4*(xy*scale1-scale*ht*(thetay*x+thetax*y)));
    }
}
