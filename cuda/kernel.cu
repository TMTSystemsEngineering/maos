#include "hip/hip_runtime.h"
/*
  Copyright 2009-2012 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "kernel.h"
/**
   A few kernels.
*/
/*somehow I must test both CUDA_ARCH existance and version.*/


__global__ void set_do(float *a, float alpha, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=alpha;
    }
}
__global__ void scale_do(float *restrict in, int n, float alpha){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	in[i]*=alpha;
    }
}

__global__ void add_ptt_do(float *restrict opd, float (*restrict loc)[2], 
			   int n, float pis, float tx, float ty){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	opd[i]+=pis+loc[i][0]*tx+loc[i][1]*ty;
    }
}

__global__ void add_ngsmod_do(float *restrict opd, float (*restrict loc)[2], int n, 
			      float m0, float m1, float m2, float m3, float m4,
			      float thetax, float thetay, float scale, float ht, float MCC_fcp, float alpha
			      ){
    float scale1=1.f-scale;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	float x=loc[i][0];
	float y=loc[i][1];
	float xy=x*y;
	float x2=x*x;
	float y2=y*y;
	opd[i]+= alpha*(+x*m0
			+y*m1
			+m2*((x2+y2-MCC_fcp)*scale1-2*scale*ht*(thetax*x+thetay*y))
			+m3*((x2-y2)*scale1 - 2*scale*ht*(thetax*x-thetay*y))
			+m4*(xy*scale1-scale*ht*(thetay*x+thetax*y)));
    }
}

/**
   add a vector to another, scaled by alpha and beta. all in device memory.
   a=a*alpha+b*beta;
*/
__global__ void add_do(float *restrict a, float *alpha1, float alpha2, 
		       const float *restrict b, float *beta1, float beta2, int n){
    float alpha=alpha1?*alpha1*alpha2:alpha2;
    float beta=beta1?*beta1*beta2:beta2;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=a[i]*alpha+b[i]*beta;
    }
}

__global__ void add_do(float *restrict a, const float *restrict b, float *beta1, float beta2, int n){
    float beta=beta1?*beta1*beta2:beta2;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]+=b[i]*beta;
    }
}
__global__ void add_do(float *restrict a, float *alpha1, float alpha2, const float *restrict b,  int n){
    float alpha=alpha1?*alpha1*alpha2:alpha2;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=a[i]*alpha+b[i];
    }
}

/**
   add a beta to a vector. 
*/
__global__ void add_do(float *vec, float beta, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	vec[i]+=beta;
    }
}

__global__ void addcabs2_do(float *restrict a, float alpha, 
			    const fcomplex *restrict b, float beta, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=a[i]*alpha+CABS2(b[i])*beta;
    }
}

/*reduction routines*/

__global__ void sum_do(float *restrict res, const float *a, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	atomicAdd(res, sb[0]);
    }
}
__global__ void max_do(float *restrict res, const float *a, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	if(sb[threadIdx.x]<a[i]) sb[threadIdx.x]=a[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    if(sb[threadIdx.x]<sb[threadIdx.x+step]){
		sb[threadIdx.x]=sb[threadIdx.x+step];
	    }
	}
    }
    if(threadIdx.x==0){
	*res=sb[0];
    }
}
/**
   tmp=sum(a.*b) res_rep=tmp, res_add+=tmp*/
__global__ void inn_do(float *res_rep, float *res_add, const float *a, const float *b, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i]*b[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	if(res_rep) *res_rep=sb[0];
	if(res_add) atomicAdd(res_add, sb[0]);
    }
}
