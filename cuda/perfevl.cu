#include "hip/hip_runtime.h"
/*
  Copyright 2009-2015 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "gpu.h"
#include "utils.h"
#include "accphi.h"
#include "cucmat.h"
#include "kernel.h"
#include "cudata.h"
#include "perf.h"
#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define ctoc(A)
#else
#define ctoc(A) toc2(A)
#endif
/** 
    save aper_locs, aper_amp to GPU.
*/
const int TT_NBX=128;//Number of thread in a block. (for reduction).
__global__ static void calc_ptt_do( Real *cc,
				    const Real (*restrict loc)[2], 
				    const int nloc,
				    const Real *restrict phi,
				    const Real *restrict amp){
    __shared__ Real ccb[4][TT_NBX];
    for(int i=0; i<4; i++){
	ccb[i][threadIdx.x]=0.f;
    }
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<nloc; i+=blockDim.x * gridDim.x){
	const Real tmp=phi[i]*amp[i];
	ccb[0][threadIdx.x]+=tmp*phi[i];
	ccb[1][threadIdx.x]+=tmp;
	ccb[2][threadIdx.x]+=tmp*loc[i][0];
	ccb[3][threadIdx.x]+=tmp*loc[i][1];
    }
    for(int step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    for(int i=0; i<4; i++){
		ccb[i][threadIdx.x]+=ccb[i][threadIdx.x+step];
	    }
	}
    }
    if(threadIdx.x<4){
	atomicAdd(&cc[threadIdx.x], ccb[threadIdx.x][0]);
    }
}

__global__ static void calc_ngsmod_do( Real *cc,
				       const Real (*restrict loc)[2], 
				       const int nloc,
				       const Real *restrict phi,
				       const Real *restrict amp){
    __shared__ Real ccb[7][TT_NBX];
    for(int i=0; i<7; i++){
	ccb[i][threadIdx.x]=0.f;
    }
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<nloc; i+=blockDim.x * gridDim.x){
	const Real tmp=phi[i]*amp[i];
	const Real x=loc[i][0];
	const Real y=loc[i][1];
	ccb[0][threadIdx.x]+=tmp*phi[i];
	ccb[1][threadIdx.x]+=tmp;
	ccb[2][threadIdx.x]+=tmp*x;
	ccb[3][threadIdx.x]+=tmp*y;
	ccb[4][threadIdx.x]+=tmp*x*x;
	ccb[5][threadIdx.x]+=tmp*y*y;
	ccb[6][threadIdx.x]+=tmp*x*y;
    }
    for(int step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
#pragma unroll
	    for(int i=0; i<7; i++){
		ccb[i][threadIdx.x]+=ccb[i][threadIdx.x+step];
	    }
	}
    }
    __syncthreads();
    if(threadIdx.x<7){
	atomicAdd(&cc[threadIdx.x], ccb[threadIdx.x][0]);
    }
}
/*
  Let M be the modal matrix of pistion/tip/tilt. Calculate M'*diag(amp)*phi
  where amp is the amptliude weighting.  */
static int calc_ptt_post(double *rmsout, double *coeffout, 
		     const double ipcc, const dmat *imcc,
		     Real *ccb){
    double coeff[3];
    double tot=ccb[0];
    coeff[0]=ccb[1]; coeff[1]=ccb[2]; coeff[2]=ccb[3]; 
    if(coeffout){
	dmulvec3(coeffout, imcc, coeff);
    }
    int ans=0;
    if(rmsout){
	double pis=ipcc*coeff[0]*coeff[0];/*piston mode variance */
	double ptt=dwdot3(coeff, imcc, coeff);/*p/t/t mode variance. */
	rmsout[0]=tot-pis;/*PR */
	rmsout[1]=ptt-pis;/*TT */
	rmsout[2]=tot-ptt;/*PTTR*/
	if(tot*1.01<pis || tot*1.01<ptt){//sanity check. allow round off error
	    warning("tot=%g, pis=%g, ptt=%g\n", tot, pis, ptt);
	    ans=1;
	}
    }
    return ans;
}
static int calc_ngsmod(double *pttr_out, double *pttrcoeff_out,
		       double *ngsmod_out, int nmod,
		       double MCC_fcp, double ht, double scale,
		       double thetax, double thetay,
		       const double ipcc, const dmat *imcc,
		       const PARMS_T *parms,
		       Real *ccb){
    double tot=(double)ccb[0];
    double coeff[6];//convert to double
    coeff[0]=ccb[1]; coeff[1]=ccb[2]; 
    coeff[2]=ccb[3]; coeff[3]=ccb[4];
    coeff[4]=ccb[5]; coeff[5]=ccb[6];
    
    if(pttrcoeff_out){//p/t/t
	memset(pttrcoeff_out, 0, sizeof(double)*3);
	dmulvec(pttrcoeff_out, imcc, coeff, 1);
    }
    int ans=0;
    if(pttr_out){
	//compute TT removed wavefront variance as a side product 
	double pis=ipcc*coeff[0]*coeff[0];
	double ptt=dwdot3(coeff, imcc, coeff);
	pttr_out[0]=tot-pis;//PR
	pttr_out[1]=ptt-pis;//TT
	pttr_out[2]=tot-ptt;//PTTR
	if(tot<pis || tot<ptt || ptt<pis || pis<0){
	    warning("tot=%g, pis=%g, ptt=%g\n", tot, pis, ptt);
	    ans=1;
	}
    }
    //don't use +=. need locking
    ngsmod_out[0]=coeff[1];
    ngsmod_out[1]=coeff[2];
    const double scale1=1.-scale;
    if(nmod>=5){
	if(parms->sim.ahstfocus){
	    ngsmod_out[2]=(-2*scale*ht*(thetax*coeff[1]+thetay*coeff[2]));
	}else{
	    ngsmod_out[2]=(scale1*(coeff[3]+coeff[4]-coeff[0]*MCC_fcp)
			   -2*scale*ht*(thetax*coeff[1]+thetay*coeff[2]));
	}
	ngsmod_out[3]=(scale1*(coeff[3]-coeff[4])
		       -2*scale*ht*(thetax*coeff[1]-thetay*coeff[2]));
	ngsmod_out[4]=(scale1*(coeff[5])
		       -scale*ht*(thetay*coeff[1]+thetax*coeff[2]));
	if(nmod>5){
	    ngsmod_out[5]=(coeff[3]+coeff[4]-coeff[0]*MCC_fcp);
	}
    }
    return ans;
}



__global__ static void 
strehlcomp_do(Comp *strehlc, 
	      const Real *opd, const Real *amp, const int nloc, const Real kk){
    extern __shared__ Real sbx[];
    Real *sby=sbx+blockDim.x;
    sbx[threadIdx.x]=0;
    sby[threadIdx.x]=0;
    Real s,c;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<nloc; i+=blockDim.x * gridDim.x){
	Z(sincos)(kk*opd[i], &s, &c);
	sbx[threadIdx.x]+=amp[i]*c;
	sby[threadIdx.x]+=amp[i]*s;
    }
    for(int step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sbx[threadIdx.x]+=sbx[threadIdx.x+step];
	    sby[threadIdx.x]+=sby[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	if(strehlc){
	    atomicAdd((Real*)strehlc, sbx[0]);
	    atomicAdd((Real*)strehlc+1, sby[0]);
	}
	//donot try to accumuate x*x+y*y. that is not correct because of many blocks.
    }
}
/**
   Compute complex PSF and return.
*/
static void psfcomp(curmat *iopdevl, int nwvl, int ievl, int nloc, hipStream_t stream){
    LOCK(cudata->perf->mutex);/*wvf is allocated per GPU.*/
    for(int iwvl=0; iwvl<nwvl; iwvl++){
	cucmat *psf=cudata->perf->psfs->p[iwvl];
	if(cuperf_t::psfsize[iwvl]==1){
	    strehlcomp_do<<<REDUCE(nloc), DIM_REDUCE*sizeof(Comp),stream>>>
		(psf->p, iopdevl->p, cudata->perf->amp, nloc, 2.*M_PI/cuperf_t::wvls[iwvl]);
	}else{
	    cucmat *wvf=cudata->perf->wvf->p[iwvl];
	    cuczero(wvf, stream);
	    embed_wvf_do<<<DIM(iopdevl->nx,256),0,stream>>>
		(wvf->p, iopdevl->p, cudata->perf->amp, cudata->perf->embed[iwvl], nloc, cuperf_t::wvls[iwvl]);
	    CUFFT(cuperf_t::plan[iwvl+nwvl*ievl], wvf->p, HIPFFT_FORWARD);
	    if(cuperf_t::psfsize[iwvl]<cuperf_t::nembed[iwvl]){
		corner2center_do<<<DIM2(psf->nx,psf->ny,16),0,stream>>>
		    (psf->p, psf->nx, psf->ny, wvf->p, wvf->nx, wvf->ny);
	    }else{
		fftshift_do<<<DIM2(psf->nx,psf->ny,16),0,stream>>>
		    (psf->p, psf->nx, psf->ny);
	    }
	}
    }
    UNLOCK(cudata->perf->mutex);
}
/**
   Compute only PSF and add to result.
*/
static void psfcomp_r(curmat **psf, curmat *iopdevl, int nwvl, int ievl, int nloc, int atomic, hipStream_t stream){
    LOCK(cudata->perf->mutex);/*wvf is allocated per GPU.*/
    for(int iwvl=0; iwvl<nwvl; iwvl++){
	cucmat *wvf=cudata->perf->wvf->p[iwvl];
	cuczero(wvf, stream);
	if(!psf[iwvl]) psf[iwvl]=curnew(cuperf_t::psfsize[iwvl], cuperf_t::psfsize[iwvl]);
	if(cuperf_t::psfsize[iwvl]==1){
	    strehlcomp_do<<<REDUCE(nloc), DIM_REDUCE*sizeof(Real)*2,stream>>>
		(wvf->p, iopdevl->p, cudata->perf->amp, nloc, 2.*M_PI/cuperf_t::wvls[iwvl]);
	    //do abs2.
	    addcabs2_do<<<1,1,0,stream>>>(psf[iwvl]->p, 1.f, wvf->p, 1.f, 1);
	}else{
	    embed_wvf_do<<<DIM(iopdevl->nx,256),0,stream>>>
		(wvf->p, iopdevl->p, cudata->perf->amp, cudata->perf->embed[iwvl], nloc, cuperf_t::wvls[iwvl]);
	    CUFFT(cuperf_t::plan[iwvl+nwvl*ievl], wvf->p, HIPFFT_FORWARD);
	    if(atomic){
		corner2center_abs2_atomic_do<<<DIM2((psf[iwvl])->nx,(psf[iwvl])->ny,16),0,stream>>>
		    ((psf[iwvl])->p, (psf[iwvl])->nx, (psf[iwvl])->ny, wvf->p, wvf->nx, wvf->ny);
	    }else{
		corner2center_abs2_do<<<DIM2((psf[iwvl])->nx,(psf[iwvl])->ny,16),0,stream>>>
		    ((psf[iwvl])->p, (psf[iwvl])->nx, (psf[iwvl])->ny, wvf->p, wvf->nx, wvf->ny);
	    }
	}
    }
    UNLOCK(cudata->perf->mutex);
}
#define PERFEVL_WFE_GPU(cc,ccb)						\
    if((parms->recon.split && recon->ngsmod->nmod==2)			\
       || (!parms->recon.split && parms->evl.nmod==3)){			\
	hipMemsetAsync(cc, 0, 4*sizeof(Real), stream);			\
	calc_ptt_do<<<DIM(nloc, TT_NBX), 0, stream>>>			\
	    (cc, cudata->perf->locs->p, nloc, iopdevl->p, cudata->perf->amp); \
	hipMemcpyAsync(ccb, cc, 4*sizeof(Real), hipMemcpyDeviceToHost, stream); \
    }else if(parms->recon.split){					\
	hipMemsetAsync(cc, 0, 7*sizeof(Real), stream);			\
	calc_ngsmod_do<<<DIM(nloc,TT_NBX),0,stream>>>			\
	    (cc, cudata->perf->locs->p, nloc, iopdevl->p, cudata->perf->amp);\
	hipMemcpyAsync(ccb, cc, 7*sizeof(Real), hipMemcpyDeviceToHost, stream); \
    }

#define PERFEVL_WFE_CPU(ans, pclep, pclmp, cleNGSmp, ccb)		\
    if(nmod!=3){							\
	TO_IMPLEMENT;/*mode decomposition. */				\
    }									\
    int ans=0;								\
    if(parms->recon.split){						\
	double *pcleNGSmp=COLUMN(cleNGSmp->p[ievl], isim);		\
	ans=calc_ngsmod(nmod==3?pclep:0, nmod==3?pclmp:0,		\
			pcleNGSmp,recon->ngsmod->nmod,			\
			recon->ngsmod->aper_fcp, recon->ngsmod->ht,	\
			recon->ngsmod->scale, thetax, thetay,		\
			aper->ipcc, aper->imcc,	parms, ccb);		\
    }else{								\
	ans=calc_ptt_post(pclep, pclmp, aper->ipcc, aper->imcc, ccb);	\
    }

/**
   Performance evaluation. Designed to replace perfevl_ievl in maos/perfevl.c
*/
void gpu_perfevl_queue(thread_t *info){
    TIC;tic;
    SIM_T *simu=(SIM_T*)info->data;
    const PARMS_T *parms=simu->parms;
    const APER_T *aper=simu->aper;
    const RECON_T *recon=simu->recon;
    const int isim=simu->isim;
    const int imoao=parms->evl.moao;
    const int nloc=aper->locs->nloc;
    const int nwvl=parms->evl.nwvl;
    for(int ievl=info->start; ievl<info->end; ievl++){
	gpu_set(cudata_t::evlgpu[ievl]);
	//info2("thread %ld gpu %d ievl %d start\n", thread_id(), cudata->igpu, ievl);
	const int do_psf_cov=(parms->evl.psfmean || parms->evl.psfhist || parms->evl.cov) 
	    && isim>=parms->evl.psfisim && parms->evl.psf->p[ievl]!=0;
	const int save_evlopd=parms->save.evlopd>0 && ((isim+1)%parms->save.evlopd)==0;
	const double thetax=parms->evl.thetax->p[ievl];
	const double thetay=parms->evl.thetay->p[ievl];
 
	hipStream_t stream=cuperf_t::stream[ievl];
	hipblasHandle_t handle=cuperf_t::handle[ievl];
	curmat *iopdevl=cuperf_t::opd->p[ievl];
	// iopdevl must be in device memory. 6 times slower if in host memory.
	if(cuperf_t::surf && cuperf_t::surf->p[ievl]){
	    curcp(&iopdevl, cuperf_t::surf->p[ievl], stream);
	}else{
	    curset(iopdevl, 0, stream);
	}
	if(parms->sim.idealevl){
	    gpu_dm2loc(iopdevl->p, cudata->perf->locs_dm[ievl], cudata->dmproj, cudata->ndm,
		       parms->evl.hs->p[ievl], thetax, thetay, 0,0,1, stream);
	}else if(simu->atm && !parms->sim.wfsalias){
	    gpu_atm2loc(iopdevl->p, cudata->perf->locs, parms->evl.hs->p[ievl], thetax, thetay, 
			0,0,parms->sim.dt,isim, 1, stream);
	}
	if(simu->telws){//Wind shake 
	    Real tt=simu->telws->p[isim];
	    Real angle=simu->winddir?simu->winddir->p[0]:0;
	    curaddptt(iopdevl, cudata->perf->locs->p, 0, tt*cosf(angle), tt*sinf(angle), stream);
	}
	if(save_evlopd){
	    cellarr_cur(simu->save->evlopdol[ievl], isim, iopdevl, stream);
	}
	if(parms->plot.run){
	    drawopdamp_gpu("OL", aper->locs, iopdevl, stream, aper->amp1->p, NULL,
			   "Science Open Loop OPD", "x (m)", "y (m)", "OL %d", ievl);
	}
	PERFEVL_WFE_GPU(cuperf_t::cc_ol->p[ievl]->p, cuperf_t::ccb_ol[ievl]);
	if((parms->evl.psfmean  || parms->evl.cov)
	   && isim>=parms->evl.psfisim 
	   &&((parms->evl.psfol==1 && ievl==parms->evl.indoa)
	      ||(parms->evl.psfol==2 && parms->evl.psf->p[ievl]))){
	    //calculate Openloop PSF. we also test psfisim to synchronize with psfcl.
	    curmat *opdcopy=NULL;
	    curmv(cuperf_t::coeff->p[ievl]->p, 0, cudata->perf->imcc, 
		  cuperf_t::cc_ol->p[ievl]->p, 'n', 1, handle);
	    curcp(&opdcopy, iopdevl, stream);
	    if(parms->evl.pttr->p[ievl]){//remove piston/tip/tilt
		curaddptt(opdcopy, cudata->perf->locs->p, cuperf_t::coeff->p[ievl]->p, -1,-1,-1,stream);
	    }else{//remove piston only
		curaddptt(opdcopy, cudata->perf->locs->p, cuperf_t::coeff->p[ievl]->p, -1, 0, 0, stream);
	    }
	    if(parms->evl.cov){
		if(parms->gpu.psf){
		    curmm(&cudata->perf->opdcovol, 1, opdcopy, opdcopy, "nt", 1, handle);
		    curadd(&cudata->perf->opdmeanol, 1, opdcopy, 1, stream);
		}else{
		    dmat *tmp=NULL;
		    cp2cpu(&tmp, opdcopy, stream);
		    dmm(&simu->evlopdcovol, 1,tmp, tmp, "nt", 1);
		    dadd(&simu->evlopdmeanol, 1, tmp, 1);
		    dfree(tmp);
		}
	    }
	    if(parms->evl.psfmean){
		psfcomp_r(cudata->perf->psfol->p, opdcopy, nwvl, ievl, nloc, parms->evl.psfol==2?1:0, stream);
		if(opdcopy!=iopdevl){
		    curfree(opdcopy);
		}
		if(!parms->gpu.psf){ //need to move psf from GPU to CPU for accumulation.
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			add2cpu(&simu->evlpsfolmean->p[iwvl], 1, cudata->perf->psfol->p[iwvl], 1, stream);
			curzero(cudata->perf->psfol->p[iwvl]); //do not accumulate in gpu.
		    }
		}
	    }
	}
	if(parms->sim.evlol) continue;
	if(parms->evl.tomo){
	    TO_IMPLEMENT;
	}else{
	    gpu_dm2loc(iopdevl->p, cudata->perf->locs_dm[ievl], cudata->dmreal, cudata->ndm, 
		       parms->evl.hs->p[ievl], thetax, thetay,
		       0,0,-1, stream);
	    if(simu->ttmreal){
		curaddptt(iopdevl, cudata->perf->locs->p, 0, -simu->ttmreal->p[0], -simu->ttmreal->p[1], stream);
	    }
	    if(imoao!=-1){
		gpu_dm2loc(iopdevl->p, &cudata->perf->locs, cudata->dm_evl[ievl], 1,
			   INFINITY, 0, 0, 0, 0, -1, stream);
	    }
	}
	if(save_evlopd){
	    cellarr_cur(simu->save->evlopdcl[ievl], isim, iopdevl, stream);
	}

	if(parms->plot.run){
	    drawopdamp_gpu("CL", aper->locs,iopdevl, stream , aper->amp1->p, NULL,
			   "Science Closed loop OPD", "x (m)", "y (m)", "CL %d", ievl);
	}
	PERFEVL_WFE_GPU(cuperf_t::cc_cl->p[ievl]->p, cuperf_t::ccb_cl[ievl]);
	if(do_psf_cov && parms->evl.psfngsr->p[ievl]!=2){//also do normal one.
	    curmv(cuperf_t::coeff->p[ievl]->p, 0, cudata->perf->imcc, 
		  cuperf_t::cc_cl->p[ievl]->p, 'n', 1, handle);
	    if(parms->evl.pttr->p[ievl]){
		curaddptt(iopdevl, cudata->perf->locs->p, cuperf_t::coeff->p[ievl]->p, -1, -1, -1, stream);
	    }else{
		curaddptt(iopdevl, cudata->perf->locs->p, cuperf_t::coeff->p[ievl]->p, -1, 0, 0, stream);
	    }
	    if(parms->evl.cov){
		if(parms->gpu.psf){
		    curmm(&cuperf_t::opdcov->p[ievl], 1, iopdevl, iopdevl, "nt", 1, handle);
		    curadd(&cuperf_t::opdmean->p[ievl], 1, iopdevl, 1, stream);
		}else{
		    dmat *tmp=NULL;
		    cp2cpu(&tmp, iopdevl, stream);
		    dmm(&simu->evlopdcov->p[ievl], 1,tmp, tmp, "nt", 1);
		    dadd(&simu->evlopdmean->p[ievl], 1, tmp, 1);
		    dfree(tmp);
		}
	    }//opdcov 
	    if(parms->evl.psfhist || parms->evl.psfmean){
		if(parms->evl.psfhist){
		    //Compute complex. 
		    psfcomp(iopdevl, nwvl, ievl, nloc, stream);
		    cellarr_cuccell(simu->save->evlpsfhist[ievl], isim, cudata->perf->psfs, stream);
		    if(parms->evl.psfmean){
			for(int iwvl=0; iwvl<nwvl; iwvl++){
			    curaddcabs2(cuperf_t::psfcl->p+iwvl+nwvl*ievl, 1, 
					cudata->perf->psfs->p[iwvl], 1, stream);
			}
		    }
		}else if(parms->evl.psfmean){
		    psfcomp_r(cuperf_t::psfcl->p+nwvl*ievl, iopdevl, nwvl, ievl, nloc, 0, stream);
		}
		if(!parms->gpu.psf){
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			add2cpu(&simu->evlpsfmean->p[iwvl+ievl*nwvl], 1, cuperf_t::psfcl->p[iwvl+ievl*nwvl], 1, stream);
			curzero(cuperf_t::psfcl->p[iwvl+ievl*nwvl]); 
		    }
		}
	    }
	}
	//info2("thread %ld gpu %d ievl %d queued\n", thread_id(), cudata->igpu, ievl);
	ctoc("queued");
    }//for ievl
}
void gpu_perfevl_sync(thread_t *info){
    TIC;tic;
    SIM_T *simu=(SIM_T*)info->data;
    const PARMS_T *parms=simu->parms;
    const int isim=simu->isim;
    const APER_T *aper=simu->aper;
    const RECON_T *recon=simu->recon;
    const int nmod=parms->evl.nmod;
    for(int ievl=info->start; ievl<info->end; ievl++){
	/*lock the mutex because iopdevl, evlwvf is allocated per GPU.*/
	gpu_set(cudata_t::evlgpu[ievl]);
	hipStream_t stream=cuperf_t::stream[ievl];
	const double thetax=parms->evl.thetax->p[ievl];
	const double thetay=parms->evl.thetay->p[ievl];
	/*Setup pointers for easy usage */
	double *polmp=COLUMN(simu->olmp->p[ievl], isim);
	double *pclmp=COLUMN(simu->clmp->p[ievl], isim);
	double *polep=COLUMN(simu->olep->p[ievl], isim);
	double *pclep=COLUMN(simu->clep->p[ievl], isim);
	CUDA_SYNC_STREAM;
	PERFEVL_WFE_CPU(ans1, polep, polmp, simu->oleNGSmp, cuperf_t::ccb_ol[ievl]);
	PERFEVL_WFE_CPU(ans2, pclep, pclmp, simu->cleNGSmp, cuperf_t::ccb_cl[ievl]);
	if(ans1 || ans2){
	    warning("Perfevl fails, redo\n");
	    gpu_perfevl_queue(info);
	    gpu_perfevl_sync(info);
	}
	//info2("thread %ld gpu %d ievl %d end\n", thread_id(), cudata->igpu, ievl);
    }//for ievl
    ctoc("done");
}
/**
   Compute the PSF or OPDCOV for NGS mode removed opd.
*/
void gpu_perfevl_ngsr(SIM_T *simu, double *cleNGSm){
    const PARMS_T *parms=simu->parms;
    const APER_T *aper=simu->aper;
    const int nloc=aper->locs->nloc;
    const int nwvl=parms->evl.nwvl;
    for(int ievl=0; ievl<parms->evl.nevl; ievl++){
	if(parms->evl.psfngsr->p[ievl]==0){
	    continue;
	}
	warning("Compare with CPU code to verify accuracy. Need to verify focus mode\n");
	gpu_set(cudata_t::evlgpu[ievl]);
	curmat *iopdevl=cuperf_t::opd->p[ievl];
	hipStream_t stream=cuperf_t::stream[ievl];
	hipblasHandle_t handle=cuperf_t::handle[ievl];
	gpu_ngsmod2science(iopdevl, cudata->perf->locs->p, simu->recon->ngsmod, cleNGSm, 
			   parms->evl.thetax->p[ievl], parms->evl.thetay->p[ievl],
			   -1, stream);
	if(parms->evl.pttr->p[ievl]){
	    double ptt[3];
	    calc_ptt_do<<<DIM(nloc,TT_NBX), 0, stream>>>	
		(cuperf_t::cc_cl->p[ievl]->p, cudata->perf->locs->p, nloc, iopdevl->p, cudata->perf->amp); 
	    hipMemcpyAsync(cuperf_t::ccb_cl[ievl], cuperf_t::cc_cl->p[ievl]->p, 
			    4*sizeof(Real), hipMemcpyDeviceToHost, stream); 
	    calc_ptt_post(NULL, ptt,  aper->ipcc, aper->imcc, cuperf_t::ccb_cl[ievl]);
	    curaddptt(iopdevl, cudata->perf->locs->p, -ptt[0], -ptt[1], -ptt[2], stream);
	}
	if(parms->evl.cov){
	    if(parms->gpu.psf){
		curmm(&cuperf_t::opdcov_ngsr->p[ievl], 1, iopdevl, iopdevl, "nt", 1, handle);
		curadd(&cuperf_t::opdmean_ngsr->p[ievl], 1, iopdevl, 1, stream);
	    }else{
		dmat *tmp=NULL;
		cp2cpu(&tmp, iopdevl, stream);
		dmm(&simu->evlopdcov_ngsr->p[ievl], 1,tmp, tmp, "nt", 1);
		dadd(&simu->evlopdmean_ngsr->p[ievl], 1, tmp, 1);
		dfree(tmp);
	    }
	}/*opdcov */
	if(parms->evl.psfhist||parms->evl.psfmean){
	    if(parms->evl.psfhist){
		/*Compute complex. */
		psfcomp(iopdevl, nwvl, ievl, nloc, stream);
		cellarr_cuccell(simu->save->evlpsfhist_ngsr[ievl], simu->isim, cudata->perf->psfs, stream);
		if(parms->evl.psfmean){
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			curaddcabs2(cuperf_t::psfcl_ngsr->p+iwvl+nwvl*ievl, 1, 
				    cudata->perf->psfs->p[iwvl], 1, stream);
		    }
		}
	    }else if(parms->evl.psfmean){
		psfcomp_r(cuperf_t::psfcl_ngsr->p+nwvl*ievl, iopdevl, nwvl, ievl, nloc, 0, stream);
	    }
	    if(!parms->gpu.psf){
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    add2cpu(&simu->evlpsfmean_ngsr->p[iwvl+ievl*nwvl], 1, cuperf_t::psfcl_ngsr->p[iwvl+ievl*nwvl], 1, stream);
		    curzero(cuperf_t::psfcl_ngsr->p[iwvl+ievl*nwvl]); 
		}
	    }
	}
	CUDA_SYNC_STREAM;
    }
}
void gpu_perfevl_save(SIM_T *simu){
    const PARMS_T *parms=simu->parms;
    if(!parms->evl.nevl) return;
    const int isim=simu->isim;
    if(parms->evl.psfmean && CHECK_SAVE(parms->evl.psfisim, parms->sim.end, isim, parms->evl.psfmean)){
	info2("Step %d: Output PSF\n", isim);
	const int nwvl=parms->evl.nwvl;
	if(cudata->perf->psfol){
	    /*copy the PSF accumulated in all the GPUs to CPU.*/
	    X(cell) *temp=X(cellnew)(nwvl, 1);
	    X(cell) *temp2=X(cellnew)(nwvl, 1);
	    double scale=1./(double)(simu->isim+1-parms->evl.psfisim);
	    if(parms->evl.psfol==2){
		scale=scale/parms->evl.npsf;
	    }
	    for(int im=0; im<NGPU; im++){
		gpu_set(im);
		cp2cpu(&temp2, cudata->perf->psfol, 0);
		hipStreamSynchronize(0);
		X(celladd)(&temp, 1, temp2, scale);
	    }
	    for(int iwvl=0; iwvl<nwvl; iwvl++){
		if(!temp || !temp->p[iwvl]) continue;
		temp->p[iwvl]->header=evl_header(simu->parms, simu->aper, -1, iwvl);
		cellarr_mat(simu->save->evlpsfolmean, isim*nwvl+iwvl, temp->p[iwvl]);
		free(temp->p[iwvl]->header); temp->p[iwvl]->header=NULL;
	    }
	    X(cellfree)(temp);
	    X(cellfree)(temp2);
	}
	if(cuperf_t::psfcl){
	    double scale=1./(double)(simu->isim+1-parms->evl.psfisim);
	    for(int ievl=0; ievl<parms->evl.nevl; ievl++){
		if(!parms->evl.psf->p[ievl] || parms->evl.psfngsr->p[ievl]==2) continue;
		gpu_set(cudata_t::evlgpu[ievl]);
		hipStream_t stream=cuperf_t::stream[ievl];
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    curmat *pp=cuperf_t::psfcl->p[iwvl+nwvl*ievl];
		    curscale(pp, scale, stream);
		    if(!pp->header){
			pp->header=evl_header(simu->parms, simu->aper, ievl, iwvl);
		    }
		    cellarr_cur(simu->save->evlpsfmean[ievl], isim*nwvl+iwvl, pp, stream);
		    curscale(pp, 1.f/scale, stream);
		}
	    }
	}
	if(cuperf_t::psfcl_ngsr){
	    double scale=1./(double)(simu->isim+1-parms->evl.psfisim);
	    for(int ievl=0; ievl<parms->evl.nevl; ievl++){
		if(!parms->evl.psf->p[ievl] || !parms->evl.psfngsr->p[ievl]) continue;
		gpu_set(cudata_t::evlgpu[ievl]);
		hipStream_t stream=cuperf_t::stream[ievl];
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    curmat *pp=cuperf_t::psfcl_ngsr->p[iwvl+nwvl*ievl];
		    curscale(pp, scale, stream);
		    if(!pp->header){
			pp->header=evl_header(simu->parms, simu->aper, ievl, iwvl);
		    }
		    cellarr_cur(simu->save->evlpsfmean_ngsr[ievl], isim*nwvl+iwvl, pp, stream);
		    curscale(pp, 1.f/scale, stream);
		}
	    }
	}
    }
    if(parms->evl.cov && CHECK_SAVE(parms->evl.psfisim, parms->sim.end, isim, parms->evl.cov)){
	info2("Step %d: Output opdcov\n", isim);
	double scale=1./(isim+1-parms->evl.psfisim);
	for(int ievl=0; ievl<parms->evl.nevl; ievl++){
	    if(!parms->evl.psf->p[ievl]|| parms->evl.psfngsr->p[ievl]==2) continue;
	    gpu_set(cudata_t::evlgpu[ievl]);
	    hipStream_t stream=cuperf_t::stream[ievl];
	    curmat *pp;
	    {
		pp=cuperf_t::opdcov->p[ievl];
		curscale(pp, scale, stream);
		cellarr_cur(simu->save->evlopdcov[ievl], isim, pp, stream);
		curscale(pp, 1./scale, stream);
	    }
	    {
		pp=cuperf_t::opdmean->p[ievl];
		curscale(pp, scale, stream);
		cellarr_cur(simu->save->evlopdmean[ievl], isim, pp, stream);
		curscale(pp, 1./scale, stream);
	    }
	}
	for(int ievl=0; ievl<parms->evl.nevl; ievl++){
	    if(!parms->evl.psf->p[ievl]|| !parms->evl.psfngsr->p[ievl]) continue;
	    gpu_set(cudata_t::evlgpu[ievl]);
	    hipStream_t stream=cuperf_t::stream[ievl];
	    curmat *pp;
	    {
		pp=cuperf_t::opdcov_ngsr->p[ievl];
		curscale(pp, scale, stream);
		cellarr_cur(simu->save->evlopdcov_ngsr[ievl], isim, pp, stream);
		curscale(pp, 1./scale, stream);
	    }
	    {
		pp=cuperf_t::opdmean_ngsr->p[ievl];
		curscale(pp, scale, stream);
		cellarr_cur(simu->save->evlopdmean_ngsr[ievl], isim, pp, stream);
		curscale(pp, 1./scale, stream);
	    }
	}
	if(parms->evl.psfol){
	    if(parms->evl.psfol==2){
		scale=scale/parms->evl.npsf;
	    }
	    {
		X(mat) *temp=NULL;
		X(mat) *temp2=NULL;
		for(int im=0; im<NGPU; im++){
		    gpu_set(im);
		    cp2cpu(&temp2, cudata->perf->opdcovol, 0);
		    hipStreamSynchronize(0);
		    X(add)(&temp, 1, temp2, scale);
		}
		cellarr_mat(simu->save->evlopdcovol, isim, temp);
		X(free)(temp);
		X(free)(temp2);
	    }
	    {
		X(mat) *temp=NULL;
		X(mat) *temp2=NULL;
		for(int im=0; im<NGPU; im++){
		    gpu_set(im);
		    cp2cpu(&temp2, cudata->perf->opdmeanol, 0);
		    hipStreamSynchronize(0);
		    X(add)(&temp, 1, temp2, scale);
		}
		cellarr_mat(simu->save->evlopdmeanol, isim, temp);
		X(free)(temp);
		X(free)(temp2);
	    }
	}
    }
}
