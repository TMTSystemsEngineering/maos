#include <hip/hip_runtime.h>
#include <stdlib.h>
const int NG1D=64;
const int NG2D=8;
const int WRAP_SIZE=32; /*The wrap size is currently always 32 */
const int REDUCE_WRAP=4;
const int REDUCE_WRAP_LOG2=2;
const int DIM_REDUCE=WRAP_SIZE*REDUCE_WRAP; /*dimension to use in reduction. */
const int REDUCE_STRIDE=WRAP_SIZE+WRAP_SIZE/2+1;
extern "C" __global__ void sum_do(float * res, const float *a, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	atomicAdd(res, sb[0]);
    }
}
/*
  In each block, we first do the reduction in each warp. This avoid syncthreads and if test. Then we copy results from each wrap to the first wrap and do the reduction again.
*/
extern "C" __global__ void sum2_do(float * res, const float *a, const int n){
    __shared__ float sb[REDUCE_WRAP*REDUCE_STRIDE];
    const int idx=threadIdx.x;
    const int wrap=idx/WRAP_SIZE; //which wrap
    const int jdx=(WRAP_SIZE-1) & idx;//index within this wrap
    volatile float *s=sb+REDUCE_STRIDE*wrap+jdx+WRAP_SIZE/2;
    s[-16]=0;
    //Read in vector from global mem
    register float sum=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + idx; i<n; i+=step){
	sum+=a[i];
	}
    s[0]=sum;
    //Handle each wrap without sync
#pragma unroll
    for(int i=0; i<5; i++){
	int offset=1<<i;
	sum += s[-offset];//every thread retrives current value
	s[0] = sum;//every thread write new value.
    }
    __syncthreads();//synchronize different wraps*/
    if(idx<REDUCE_WRAP){//use a few threads for reduce
	float sum2=sb[REDUCE_STRIDE * idx + WRAP_SIZE/2 + WRAP_SIZE - 1];
	//reuse sb for size of REDUCE_WRAP+REDUCE_WRAP/2;
	sb[idx]=0;
	volatile float *s2 = sb + REDUCE_WRAP/2 + idx;
	s2[0]=sum2;
#pragma unroll	
	for(int i=0; i<REDUCE_WRAP_LOG2; i++){
	    int offset=1<<i;
	    sum2+=s2[-offset];
	    s2[0]=sum2;
	}
	if(idx+1==REDUCE_WRAP){
	    atomicAdd(res, sum2);
	}
    }
}
