#include "hip/hip_runtime.h"
/*
  Copyright 2009, 2010, 2011 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipfft/hipfft.h>
#include "wfs.h"
static void gpu_pts2cuwloc(cuwloc_t *wloc, pts_t *pts, loc_t *loc){
    wloc->nxsa=pts->nx;
    wloc->nsa=pts->nsa;
    wloc->dx=pts->dx;
    wloc->nloc=loc->nloc;
    gpu_loc2dev(&wloc->pts, (loc_t*)pts);
    gpu_loc2dev(&wloc->loc, loc);
}
int *wfsgpu=NULL;/*assign GPU to wfs statically. */
/**
   Initialize other arrays
*/
void gpu_wfsgrad_init(const PARMS_T *parms, const POWFS_T *powfs){
    wfsgpu=(int*)calloc(parms->nwfs, sizeof(int));
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	wfsgpu[iwfs]=gpu_next();
    }
    for(int im=0; im<NGPU; im++){
	gpu_set(im);
	cudata->powfs=(cuwloc_t*)calloc(parms->npowfs, sizeof(cuwloc_t));
	cudata->wfs=(cuwfs_t*)calloc(parms->nwfs, sizeof(cuwfs_t));
	cuwloc_t *cupowfs=cudata->powfs;
	
	/*DO(hipSetDeviceFlags(hipDeviceScheduleBlockingSync)); */
	DO(hipsparseCreateMatDescr(&cudata->wfsspdesc));
	hipsparseSetMatType(cudata->wfsspdesc, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(cudata->wfsspdesc, HIPSPARSE_INDEX_BASE_ZERO);
	
	/* Setup information that are same for wfs in each powfs*/
	for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
	    if(parms->powfs[ipowfs].nwfs==0) continue;
	    pts_t *pts=powfs[ipowfs].pts;
	    loc_t *loc=powfs[ipowfs].loc;
	    gpu_pts2cuwloc(&cupowfs[ipowfs], pts, loc);
	    gpu_loc2dev(&cupowfs[ipowfs].saloc, powfs[ipowfs].saloc);
	    cupowfs[ipowfs].dsa=pts->dsa;
	    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		pts=powfs[ipowfs].llt->pts;
		loc=powfs[ipowfs].llt->loc;
		cupowfs[ipowfs].llt=(cuwloc_t*)calloc(1, sizeof(cuwloc_t));
		gpu_pts2cuwloc(cupowfs[ipowfs].llt, pts, loc);
	    }
	    /*cupowfs[ipowfs].skip=parms->powfs[ipowfs].skip; */
	}
    }

    /* setup information that maybe different for wfs in same powfs due to
       misregistration or NCPA.*/
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwloc_t *cupowfs=cudata->powfs;
	cuwfs_t *cuwfs=cudata->wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int nsa=powfs[ipowfs].pts->nsa;
	int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
	int iwfs0=parms->powfs[ipowfs].wfs[0];/*first wfs in this group. */
	/*imcc for ztilt. */
	STREAM_NEW(cuwfs[iwfs].stream);
	DO(hipsparseCreate(&cuwfs[iwfs].sphandle));
	DO(cusparseSetKernelStream(cuwfs[iwfs].sphandle, cuwfs[iwfs].stream));
	DO(hipblasCreate(&cuwfs[iwfs].handle));
	DO(hipblasSetStream(cuwfs[iwfs].handle, cuwfs[iwfs].stream));
	if(powfs[ipowfs].saimcc){
	    if(powfs[ipowfs].nsaimcc>1 || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		hipHostMalloc(&cuwfs[iwfs].imcc, nsa*sizeof(void*));
		for(int isa=0; isa<nsa; isa++){
		    cuwfs[iwfs].imcc[isa]=NULL;
		    gpu_dmat2dev((float**)&(cuwfs[iwfs].imcc[isa]),
				 powfs[ipowfs].saimcc[powfs[ipowfs].nsaimcc>1?wfsind:0]->p[isa]);
		}
	    }else{
		cuwfs[iwfs].imcc=cuwfs[iwfs0].imcc;
	    }
	}
	cuwfs[iwfs].powfs=cupowfs+ipowfs;
	hipDeviceSynchronize();
	/*GS0 for gtilt. */
	if(powfs[ipowfs].GS0){
	    if(powfs[ipowfs].GS0->nx>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		dsp *t=sptrans(powfs[ipowfs].GS0->p[powfs[ipowfs].GS0->nx>1?wfsind:0]);
		gpu_sp2dev(&cuwfs[iwfs].GS0t, t);
		spfree(t);
	    }else{
		cuwfs[iwfs].GS0t=cuwfs[iwfs0].GS0t;
	    }
	}
	/*wfs amplitude map on loc */
	if(powfs[ipowfs].nlocm>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
	    gpu_dbl2dev(&cuwfs[iwfs].amp, powfs[ipowfs].realamp[powfs[ipowfs].nlocm>1?wfsind:0], powfs[ipowfs].loc->nloc);
	}else{
	    cuwfs[iwfs].amp=cuwfs[iwfs0].amp;
	}
	

	dmat *nea=powfs[ipowfs].neasim->p[wfsind];
	if(nea){
	    gpu_dmat2dev(&cuwfs[iwfs].neasim, nea);
	}

	/* * Now start physical optics setup * */

	if(parms->powfs[ipowfs].usephy||parms->powfs[ipowfs].psfout||parms->powfs[ipowfs].pistatout){
	    /*If there is llt. */
	    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		if(powfs[ipowfs].llt->ncpa){
		    if(powfs[ipowfs].llt->ncpa->nx>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			gpu_dmat2dev(&cuwfs[iwfs].lltncpa, powfs[ipowfs].llt->ncpa->p[powfs[ipowfs].llt->ncpa->nx>1?wfsind:0]);
		    }else{
			cuwfs[iwfs].lltncpa=cuwfs[iwfs0].lltncpa;
		    }
		}
		if(wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    hipHostMalloc(&cuwfs[iwfs].lltimcc, 1*sizeof(void*));
		    cuwfs[iwfs].lltimcc[0]=NULL;
		    gpu_dmat2dev((float**)&cuwfs[iwfs].lltimcc[0], powfs[ipowfs].llt->imcc->p[0]);
		    gpu_dmat2dev((float**)&cuwfs[iwfs].lltamp, powfs[ipowfs].llt->amp);
		}else{
		    cuwfs[iwfs].lltimcc=cuwfs[iwfs0].lltimcc;
		    cuwfs[iwfs].lltamp=cuwfs[iwfs0].lltamp;
		}
	    }
	    hipDeviceSynchronize();

	    /*CUFFTW is row major. */
	    int npsf=powfs[ipowfs].pts->nx*parms->powfs[ipowfs].embfac;/*size of fft */
	    int npsf2[2]={npsf, npsf};
	    const int ncompx=powfs[ipowfs].ncompx;
	    const int ncompy=powfs[ipowfs].ncompy;
	    const int ncompm=MAX(ncompx, ncompy);
	    int ncomp[2]={ncompx, ncompy};
	    int ncompm2[2]={ncompm, ncompm};
	    /*
	      int inembed[2]; inembed[0]=nx; inembed[1]=nx;
	      int istride=1;
	      int idist=nx*nx;
	      DO(hipfftPlanMany(&cuwfs[iwfs].plan, 2, nr, 
	      inembed, istride, idist, 
	      inembed, istride, idist, 
	      HIPFFT_C2C, nsa));
	    */
	    /*limit the number of subapertures in each batch to less than 1024
	      to save memory. The speed is actually a tiny bit faster for NFIRAOS.*/
	    cuwfs[iwfs].msa=nsa>1024?((int)ceil((float)nsa/(float)(nsa/800))):nsa;
	    if(hipfftPlanMany(&cuwfs[iwfs].plan1, 2, npsf2, NULL, 1, 0, NULL, 1, 0, 
			     HIPFFT_C2C, cuwfs[iwfs].msa)){
		error("CUFFT plan failed\n");
	    }
	    hipfftSetStream(cuwfs[iwfs].plan1, cuwfs[iwfs].stream);
	    if(ncompx==npsf && ncompy==npsf){
		cuwfs[iwfs].plan2=cuwfs[iwfs].plan1;
	    }else{
		if(hipfftPlanMany(&cuwfs[iwfs].plan2, 2, ncomp, NULL, 1, 0, NULL, 1, 0, 
				 HIPFFT_C2C, cuwfs[iwfs].msa)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].plan2, cuwfs[iwfs].stream);
	    }
	    if(ncompm==ncompx && ncompm==ncompy){
		cuwfs[iwfs].plan3=cuwfs[iwfs].plan2;
	    }else{
		if(hipfftPlanMany(&cuwfs[iwfs].plan3, 2, ncompm2, NULL, 1, 0, NULL, 1, 0, 
				 HIPFFT_C2C, cuwfs[iwfs].msa)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].plan3, cuwfs[iwfs].stream);
	    }

	    if(parms->powfs[ipowfs].llt){
		int nlpsf=powfs[ipowfs].llt->pts->nx*parms->powfs[ipowfs].embfac;
		int nlpsf2[2]={nlpsf, nlpsf};
		if(hipfftPlanMany(&cuwfs[iwfs].lltplan_wvf, 2, nlpsf2, NULL, 1,0, NULL, 1, 0, 
				 HIPFFT_C2C, 1)){
		    error("CUFFT plan failed\n");
		    hipfftSetStream(cuwfs[iwfs].lltplan_wvf, cuwfs[iwfs].stream);
		}
		if(npsf==nlpsf){
		    cuwfs[iwfs].lltplan_otf=cuwfs[iwfs].lltplan_wvf;
		}else{
		    if(hipfftPlanMany(&cuwfs[iwfs].lltplan_otf, 2, npsf2, NULL, 1, 0, NULL, 1, 0, 
				     HIPFFT_C2C, 1)){
			error("CUFFT plan failed\n");
		    }
		    hipfftSetStream(cuwfs[iwfs].lltplan_otf, cuwfs[iwfs].stream);
		}
	    }
	    /*DTF. */
	    if(parms->powfs[ipowfs].usephy){
		if(parms->powfs[ipowfs].llt && parms->powfs[ipowfs].llt->n>1 
		   || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    /*Need one per wfs in this powfs, or the first wfs. */
		    int nwvl=parms->powfs[ipowfs].nwvl;
		    cuwfs[iwfs].dtf=(cudtf_t*)calloc(nwvl, sizeof(cudtf_t));
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			int notfused=!powfs[ipowfs].dtf[iwvl].fused;
			if(notfused){
			    cudaCallocHostBlock(cuwfs[iwfs].dtf[iwvl].nominal, nsa*sizeof(void*));
			}
			/*cudaCallocHostBlock(cuwfs[iwfs].dtf[iwvl].si, nsa*sizeof(void*)); */
			int multi_nominal=(powfs[ipowfs].dtf[iwvl].si->nx==nsa);
			for(int isa=0; isa<nsa; isa++){
			    if(multi_nominal || isa==0){
				if(notfused){
				    gpu_cmat2dev(&cuwfs[iwfs].dtf[iwvl].nominal[isa], 
						 powfs[ipowfs].dtf[iwvl].nominal->p[isa+nsa*(powfs[ipowfs].dtf[iwvl].nominal->ny>1?wfsind:0)]);
				}
			    }else{
				cuwfs[iwfs].dtf[iwvl].nominal[isa]=cuwfs[iwfs].dtf[iwvl].nominal[0];
			    }
			}
		    
			if(parms->powfs[ipowfs].llt){
			    cudaCallocHostBlock(cuwfs[iwfs].dtf[iwvl].etf, nsa*sizeof(void*));
			    cmat *(*petf)[nsa]=NULL;
			    if(powfs[ipowfs].etfsim[iwvl].p1){
				petf=(cmat *(*)[nsa])powfs[ipowfs].etfsim[iwvl].p1->p;
				cuwfs[iwfs].dtf[iwvl].etfis1d=1;
			    }else{
				petf=(cmat *(*)[nsa])powfs[ipowfs].etfsim[iwvl].p2->p;
				cuwfs[iwfs].dtf[iwvl].etfis1d=0;
			    }
			    for(int isa=0; isa<nsa; isa++){
				gpu_cmat2dev(&cuwfs[iwfs].dtf[iwvl].etf[isa], petf[parms->powfs[ipowfs].llt->n>1?wfsind:0][isa]);
			    }
			}
		    }/*for iwvl. */
		    if(parms->powfs[ipowfs].llt){
			gpu_dmat2dev(&cuwfs[iwfs].srot, powfs[ipowfs].srot->p[parms->powfs[ipowfs].llt->n>1?wfsind:0]);
		    }
		}else{
		    cuwfs[iwfs].dtf  = cuwfs[iwfs0].dtf;
		    cuwfs[iwfs].srot = cuwfs[iwfs0].srot;
		}
		/*Matched filter */
		if(parms->powfs[ipowfs].phytypesim==1){
		    if(powfs[ipowfs].intstat->mtche->ny>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cudaCallocHostBlock(cuwfs[iwfs].mtche, nsa*sizeof(void*));
			dmat **mtche=powfs[ipowfs].intstat->mtche->p+nsa*(powfs[ipowfs].intstat->mtche->ny>1?wfsind:0);
			for(int isa=0; isa<nsa; isa++){
			    gpu_dmat2dev((float**)&cuwfs[iwfs].mtche[isa], mtche[isa]);
			}
		    }else{
			cuwfs[iwfs].mtche=cuwfs[iwfs0].mtche;
		    }
		}
		if(powfs[ipowfs].bkgrnd){
		    if(powfs[ipowfs].bkgrnd->ny==1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cudaCallocHostBlock(cuwfs[iwfs].bkgrnd2, nsa*sizeof(void*));
			dmat **bkgrnd=powfs[ipowfs].bkgrnd->p+nsa*(powfs[ipowfs].bkgrnd->ny==1?wfsind:0);
			for(int isa=0; isa<nsa; isa++){
			    gpu_dmat2dev((float**)&cuwfs[iwfs].bkgrnd2[isa], bkgrnd[isa]);
			}
		    }else{
			cuwfs[iwfs].bkgrnd2=cuwfs[iwfs0].bkgrnd2;
		    }
		}
		if(powfs[ipowfs].bkgrndc){
		    if(powfs[ipowfs].bkgrndc->ny==1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cudaCallocHostBlock(cuwfs[iwfs].bkgrnd2c, nsa*sizeof(void*));
			dmat **bkgrndc=powfs[ipowfs].bkgrndc->p+nsa*(powfs[ipowfs].bkgrndc->ny==1?wfsind:0);
			for(int isa=0; isa<nsa; isa++){
			    gpu_dmat2dev((float**)&cuwfs[iwfs].bkgrnd2c[isa], bkgrndc[isa]);
			}
		    }else{
			cuwfs[iwfs].bkgrnd2c=cuwfs[iwfs0].bkgrnd2c;
		    }	
		}
	    }
	}/*if phy */
	CUDA_SYNC_DEVICE;
	gpu_print_mem("wfs init");
    }/*for iwfs */
}
void gpu_wfs_init_sim(const PARMS_T *parms, POWFS_T *powfs){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwfs_t *cuwfs=cudata->wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int nsa=powfs[ipowfs].pts->nsa;
	if(parms->powfs[ipowfs].phystep!=0
	   ||parms->save.gradgeom[iwfs]
	   ||parms->powfs[ipowfs].pistatout){
	    /*gradacc for geom wfs accumulation */
	    curfree(cuwfs[iwfs].gradacc);
	    cuwfs[iwfs].gradacc=curnew(nsa*2,1);
	}
	if(parms->powfs[ipowfs].usephy){
	    curcellfree(cuwfs[iwfs].ints);
	    cuwfs[iwfs].ints=curcellnew(nsa,1,powfs[ipowfs].pixpsax,powfs[ipowfs].pixpsay);
	    if(parms->powfs[ipowfs].noisy){
		curfree(cuwfs[iwfs].neareal);
		cuwfs[iwfs].neareal=curnew(nsa*4,1);
	    }
	}
	if(parms->powfs[ipowfs].pistatout){
	    if(parms->powfs[ipowfs].pistatstc){
		error("pistatstc is not supported yet.\n");
	    }
	    curcellfree(cuwfs[iwfs].pistatout);
	    const int notfx=powfs[ipowfs].ncompx;/*necessary size to build detector image. */
	    const int notfy=powfs[ipowfs].ncompy;
	    const int npsf=MAX(notfx,notfy);
	    cuwfs[iwfs].pistatout=curcellnew(nsa, parms->powfs[ipowfs].nwvl, npsf, npsf);
	}
	CUDA_SYNC_DEVICE;
    }
}
void gpu_wfssurf2gpu(const PARMS_T *parms, POWFS_T *powfs){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(GPUS[wfsgpu[iwfs]]);
	cuwfs_t *cuwfs=cudata->wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
	if(powfs[ipowfs].ncpa){
	    gpu_dmat2cu(&cuwfs[iwfs].opdadd, powfs[ipowfs].ncpa->p[wfsind]);
	}else{
	    curzero(cuwfs[iwfs].opdadd, cuwfs[iwfs].stream);
	}
	if(powfs[ipowfs].opdadd && powfs[ipowfs].opdadd->p[wfsind]){
	    curmat *temp=NULL;
	    gpu_dmat2cu(&temp, powfs[ipowfs].opdadd->p[wfsind]);
	    curadd(&cuwfs[iwfs].opdadd, 1, temp, 1, cuwfs[iwfs].stream);
	    curfree(temp);
	    dfree(powfs[ipowfs].opdadd->p[wfsind]);/*no longer need it in CPU memory. */
	}
    }
}
__global__ static void setup_rand(curandStat *rstat, int seed){
    int id=threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &rstat[id]);
}
/**
   Seed the random number genrator
*/
void gpu_wfsgrad_seeding(const PARMS_T *parms, const POWFS_T *powfs, rand_t *rstat){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(GPUS[wfsgpu[iwfs]]);
	cuwfs_t *cuwfs=cudata->wfs;
	int seed=lrand(rstat);/*don't put this after continue. */
	int ipowfs=parms->wfs[iwfs].powfs;
	if(!parms->powfs[ipowfs].noisy) continue;
	int nsa=powfs[ipowfs].pts->nsa*2;
	if(nsa<RAND_THREAD){
	    cuwfs[iwfs].custatt=nsa;
	    cuwfs[iwfs].custatb=1;
	}else if(nsa<RAND_THREAD*RAND_BLOCK){
	    cuwfs[iwfs].custatt=RAND_THREAD;
	    cuwfs[iwfs].custatb=nsa/RAND_THREAD+(nsa%RAND_THREAD)?1:0;
	}else{
	    cuwfs[iwfs].custatt=RAND_THREAD;
	    cuwfs[iwfs].custatb=RAND_BLOCK;
	}
	hipMalloc(&cuwfs[iwfs].custat, (cuwfs[iwfs].custatt*cuwfs[iwfs].custatb)*sizeof(curandStat));
	setup_rand<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt>>>(cuwfs[iwfs].custat, seed);
    }
    CUDA_SYNC_DEVICE;
    gpu_print_mem("wfs seeding");
}
