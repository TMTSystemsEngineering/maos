#include "hip/hip_runtime.h"
/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include "gpu.h"
}
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include "cucmat.h"
#include "wfs.h"
#include "cudata.h"

/**
   Reshape each cell into column vector and concatenate each cell into a column
*/
static cmat *concat_ccell_as_vector(ccell *input){
    if(input->ny!=1){
	error("Invalid format\n");
    }
    int npix=input->p[0]->nx*input->p[0]->ny;
    int nsa=input->nx;
    cmat *output=cnew(npix, nsa);
    for(long isa=0; isa<nsa; isa++){
	memcpy(output->p+isa*npix, input->p[isa]->p, sizeof(dcomplex)*npix);
    }
    return output;
}
/**
   Reshape each cell into column vector and concatenate each cell into a column
*/
static dmat *concat_dcell_as_vector(dcell *input){
    if(input->ny!=1){
	error("Invalid format\n");
    }
    int npix=input->p[0]->nx*input->p[0]->ny;
    int nsa=input->nx;
    dmat *output=dnew(npix, nsa);
    for(long isa=0; isa<nsa; isa++){
	memcpy(output->p+isa*npix, input->p[isa]->p, sizeof(double)*npix);
    }
    return output;
}
static void etf2gpu(cucmat **cuetf, ETF_T *etf, int icol, int *etfis1d){
    ccell *etfc=0;
    if(etf->p1){
	etfc=ccellsub(etf->p1, 0, 0, icol, 1);
	*etfis1d=1;
    }else{
	etfc=ccellsub(etf->p2, 0, 0, icol, 1);
	*etfis1d=0;
    }
    cmat *etfm=concat_ccell_as_vector(etfc);
    cp2gpu(cuetf, etfm);
    cfree(etfm);
    ccellfree(etfc);
}
/**
   Initialize or update etf.
*/
void gpu_wfsgrad_update_etf(const PARMS_T *parms, const POWFS_T *powfs){
    const int *wfsgpu=cudata_t::wfsgpu;
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwfs_t *cuwfs=cudata_t::wfs;
	const int ipowfs=parms->wfs[iwfs].powfs;
	const int nwvl=parms->powfs[ipowfs].nwvl;
	const int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
	const int iwfs0=parms->powfs[ipowfs].wfs->p[0];
	if(parms->powfs[ipowfs].usephy||parms->powfs[ipowfs].psfout||parms->powfs[ipowfs].pistatout){
	    if(parms->powfs[ipowfs].usephy){
		if(parms->powfs[ipowfs].llt && parms->powfs[ipowfs].llt->n>1 || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    if(parms->powfs[ipowfs].llt){
			for(int iwvl=0; iwvl<nwvl; iwvl++){
			    int icol=parms->powfs[ipowfs].llt->n>1?wfsind:0;
			    if(powfs[ipowfs].etfsim){
				etf2gpu(&cuwfs[iwfs].dtf[iwvl].etf, &powfs[ipowfs].etfsim[iwvl], icol, &cuwfs[iwfs].dtf[iwvl].etfis1d);
			    }
			    if(powfs[ipowfs].etfsim2){
				etf2gpu(&cuwfs[iwfs].dtf[iwvl].etf2, &powfs[ipowfs].etfsim2[iwvl], icol, &cuwfs[iwfs].dtf[iwvl].etfis1d);
			    }
			}
		    }
		}
	    }
	}
    }
}
void gpu_wfsgrad_update_mtche(const PARMS_T *parms, const POWFS_T *powfs){
    const int *wfsgpu=cudata_t::wfsgpu;
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwfs_t *cuwfs=cudata_t::wfs;
	const int ipowfs=parms->wfs[iwfs].powfs;
	const int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
	const int iwfs0=parms->powfs[ipowfs].wfs->p[0];
	const int nsa=powfs[ipowfs].pts->nsa;
	if(parms->powfs[ipowfs].usephy){
	    if(parms->powfs[ipowfs].phytypesim==1){
		if(powfs[ipowfs].intstat->mtche->ny>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    int icol=powfs[ipowfs].intstat->mtche->ny>1?wfsind:0;
		    dcell *mtchec=dcellsub(powfs[ipowfs].intstat->mtche, 0, 0, icol, 1);
		    dmat *mtche=concat_dcell_as_vector(mtchec);
		    dcellfree(mtchec);
		    if(iwfs!=iwfs0 && cuwfs[iwfs].mtche==cuwfs[iwfs0].mtche){
			info("Reset mtche to 0\n");
			cuwfs[iwfs].mtche=0;
			cuwfs[iwfs].i0sum=0;
		    }
		    cp2gpu(&cuwfs[iwfs].mtche, mtche);
		    dfree(mtche);
		    cp2gpu(&cuwfs[iwfs].i0sum, powfs[ipowfs].intstat->i0sum->p+nsa*icol, nsa, 1);
		}else{
		    cuwfs[iwfs].mtche=cuwfs[iwfs0].mtche;
		    cuwfs[iwfs].i0sum=cuwfs[iwfs0].i0sum;
		}
	    }
	}
    }
}
/**
   Initialize or update mtched filter
*/

/**
   Initialize other arrays
*/
void gpu_wfsgrad_init(const PARMS_T *parms, const POWFS_T *powfs){
    const int *wfsgpu=cudata_t::wfsgpu;
    cudata_t::wfs=(cuwfs_t*)calloc(parms->nwfs, sizeof(cuwfs_t));
    for(int im=0; im<NGPU; im++){
	gpu_set(im);
	cudata->powfs=(cuwloc_t*)calloc(parms->npowfs, sizeof(cuwloc_t));
	cuwloc_t *cupowfs=cudata->powfs;
	/* Setup information that are same for wfs in each powfs*/
	for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
	    if(parms->powfs[ipowfs].nwfs==0) continue;
	    pts_t *pts=powfs[ipowfs].pts;
	    loc_t *loc=powfs[ipowfs].loc;
	    cupowfs[ipowfs].pts=new cupts_t(pts);
	    cupowfs[ipowfs].loc=new culoc_t(loc);
	    cupowfs[ipowfs].saloc=new culoc_t(powfs[ipowfs].saloc);
	
	    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		pts=powfs[ipowfs].llt->pts;
		loc=powfs[ipowfs].llt->loc;
		cupowfs[ipowfs].llt=new cullt_t;
		cupowfs[ipowfs].llt->pts=new cupts_t(pts);
		cupowfs[ipowfs].llt->loc=new culoc_t(loc);
	    }
	    /*cupowfs[ipowfs].skip=parms->powfs[ipowfs].skip; */
	    if(parms->powfs[ipowfs].fieldstop){
		cp2gpu(&cupowfs[ipowfs].embed, powfs[ipowfs].fieldstop->embed->p[0]->p, powfs[ipowfs].loc->nloc, 1);
		cupowfs[ipowfs].nembed=powfs[ipowfs].fieldstop->nembed->p[0];
		cp2gpu(&cupowfs[ipowfs].fieldstop, powfs[ipowfs].fieldstop->fieldmask->p[0]);
	    }
	}
    }

    /* setup information that maybe different for wfs in same powfs due to
       misregistration or NCPA.*/
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwloc_t *cupowfs=cudata->powfs;
	cuwfs_t *cuwfs=cudata_t::wfs;
	const int ipowfs=parms->wfs[iwfs].powfs;
	const int nsa=powfs[ipowfs].pts->nsa;
	const int nwvl=parms->powfs[ipowfs].nwvl;
	const int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
	const int iwfs0=parms->powfs[ipowfs].wfs->p[0];
	const int nwfsp=parms->powfs[ipowfs].nwfs;
	const int ndm=parms->ndm;
	/*imcc for ztilt. */
	cuwfs[iwfs].stream=new stream_t;
	cuwfs[iwfs].loc_dm=new culoc_t*[ndm];
	for(int idm=0; idm<ndm; idm++){
	    if(powfs[ipowfs].loc_dm){
		cuwfs[iwfs].loc_dm[idm]=new culoc_t(powfs[ipowfs].loc_dm->p[wfsind+idm*nwfsp]);
	    }else{
		cuwfs[iwfs].loc_dm[idm]=new culoc_t(powfs[ipowfs].loc);
	    }
	}
	if(powfs[ipowfs].loc_tel){
	    cuwfs[iwfs].loc_tel=new culoc_t(powfs[ipowfs].loc_tel->p[wfsind]);
	}else{
	    cuwfs[iwfs].loc_tel=new culoc_t(powfs[ipowfs].loc);
	}
	cuwfs[iwfs].phiout=curnew(powfs[ipowfs].loc->nloc, 1);

	if(parms->powfs[ipowfs].fieldstop){
	    DO(hipfftPlan2d(&cuwfs[iwfs].plan_fs, cupowfs[ipowfs].nembed, cupowfs[ipowfs].nembed, FFT_T_C2C));
	    hipfftSetStream(cuwfs[iwfs].plan_fs, *cuwfs[iwfs].stream);
	}
	if(powfs[ipowfs].saimcc){
	    if(powfs[ipowfs].saimcc->nx>1 || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		Real *imcc[nsa];
		for(int isa=0; isa<nsa; isa++){
		    imcc[isa]=NULL;
		    cp2gpu(&(imcc[isa]),
			   powfs[ipowfs].saimcc->p[powfs[ipowfs].saimcc->nx>1?wfsind:0]->p[isa]);
		}
		DO(hipMalloc(&cuwfs[iwfs].imcc, nsa*sizeof(void*)));
		DO(hipMemcpy(cuwfs[iwfs].imcc, imcc, nsa*sizeof(void*), hipMemcpyHostToDevice));
	    }else{
		cuwfs[iwfs].imcc=cuwfs[iwfs0].imcc;
	    }
	}
	cuwfs[iwfs].powfs=cupowfs+ipowfs;
	hipDeviceSynchronize();
	/*GS0 for gtilt. */
	if(powfs[ipowfs].GS0){
	    dsp *t=powfs[ipowfs].GS0->p[powfs[ipowfs].GS0->nx>1?wfsind:0];
	    cuwfs[iwfs].GS0=new cusp(t, 1);
	}
	/*wfs amplitude map on loc */
	cp2gpu(&cuwfs[iwfs].amp, powfs[ipowfs].realamp->p[wfsind]);
	dmat *nea=powfs[ipowfs].neasim->p[wfsind];
	if(nea){
	    cp2gpu(&cuwfs[iwfs].neasim, nea);
	}

	/* * Now start physical optics setup * */

	if(parms->powfs[ipowfs].usephy||parms->powfs[ipowfs].psfout||parms->powfs[ipowfs].pistatout){
	    /*If there is llt. */
	    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		if(powfs[ipowfs].llt->ncpa){
		    if(powfs[ipowfs].llt->ncpa->nx>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cp2gpu(&cuwfs[iwfs].lltncpa, 
			       powfs[ipowfs].llt->ncpa->p[powfs[ipowfs].llt->ncpa->nx>1?wfsind:0]);
		    }else{
			cuwfs[iwfs].lltncpa=cuwfs[iwfs0].lltncpa;
		    }
		}
		if(wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    DO(hipHostMalloc(&cuwfs[iwfs].lltimcc, 1*sizeof(void*)));
		    cuwfs[iwfs].lltimcc[0]=NULL;
		    cp2gpu((Real**)&cuwfs[iwfs].lltimcc[0], powfs[ipowfs].llt->imcc->p[0]);
		    cp2gpu((Real**)&cuwfs[iwfs].lltamp, powfs[ipowfs].llt->amp);
		}else{
		    cuwfs[iwfs].lltimcc=cuwfs[iwfs0].lltimcc;
		    cuwfs[iwfs].lltamp=cuwfs[iwfs0].lltamp;
		}
	    }
	    hipDeviceSynchronize();

	    /*CUFFTW is row major. */
	    int nwvf=powfs[ipowfs].pts->nx*parms->powfs[ipowfs].embfac;/*size of fft */
	    int nwvf2[2]={nwvf, nwvf};
	    const int ncompx=powfs[ipowfs].ncompx;
	    const int ncompy=powfs[ipowfs].ncompy;
	    const int notf=MAX(ncompx, ncompy);
	    int ncomp2[2]={ncompx, ncompy};
	    int notf2[2]={notf, notf};
	    /*limit the number of subapertures in each batch to less than 1024
	      to save memory. The speed is actually a tiny bit faster for NFIRAOS.*/
	    cuwfs[iwfs].msa=nsa>1024?((int)ceil((Real)nsa/(Real)(nsa/800))):nsa;
	    if(hipfftPlanMany(&cuwfs[iwfs].plan1, 2, nwvf2, NULL, 1, 0, NULL, 1, 0, 
			     FFT_T_C2C, cuwfs[iwfs].msa)){
		error("CUFFT plan failed\n");
	    }
	    hipfftSetStream(cuwfs[iwfs].plan1, *cuwfs[iwfs].stream);

	    if(notf==nwvf){
		cuwfs[iwfs].plan2=cuwfs[iwfs].plan1;
	    }else{
		if(hipfftPlanMany(&cuwfs[iwfs].plan2, 2, notf2, NULL, 1, 0, NULL, 1, 0, 
				 FFT_T_C2C, cuwfs[iwfs].msa)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].plan2, *cuwfs[iwfs].stream);
	    }
	    if(notf==ncompx && notf==ncompy){
		cuwfs[iwfs].plan3=cuwfs[iwfs].plan2;
	    }else{
		if(hipfftPlanMany(&cuwfs[iwfs].plan3, 2, ncomp2, NULL, 1, 0, NULL, 1, 0, 
				 FFT_T_C2C, cuwfs[iwfs].msa)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].plan3, *cuwfs[iwfs].stream);
	    }

	    if(parms->powfs[ipowfs].llt){
		int nlwvf=powfs[ipowfs].llt->pts->nx*parms->powfs[ipowfs].embfac;
		int nlwvf2[2]={nlwvf, nlwvf};
		if(hipfftPlanMany(&cuwfs[iwfs].lltplan_wvf, 2, nlwvf2, NULL, 1,0, NULL, 1, 0, 
				 FFT_T_C2C, 1)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].lltplan_wvf, *cuwfs[iwfs].stream);
		if(notf==nlwvf){
		    cuwfs[iwfs].lltplan_otf=cuwfs[iwfs].lltplan_wvf;
		}else{
		    if(hipfftPlanMany(&cuwfs[iwfs].lltplan_otf, 2, notf2, NULL, 1, 0, NULL, 1, 0, 
				     FFT_T_C2C, 1)){
			error("CUFFT plan failed\n");
		    }
		    hipfftSetStream(cuwfs[iwfs].lltplan_otf, *cuwfs[iwfs].stream);
		}
	    }
	    /*DTF. */
	    if(parms->powfs[ipowfs].usephy){
		if(parms->powfs[ipowfs].llt && parms->powfs[ipowfs].llt->n>1 
		   || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    /*Need one per wfs in this powfs, or the first wfs. */
		    cuwfs[iwfs].dtf=(cudtf_t*)calloc(nwvl, sizeof(cudtf_t));
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			int notfused=!powfs[ipowfs].dtf[iwvl].fused;
			if(notfused){
			    int icol=powfs[ipowfs].dtf[iwvl].nominal->ny>1?wfsind:0;
			    ccell *nominalc=ccellsub(powfs[ipowfs].dtf[iwvl].nominal, 0, 0, icol, 1);
			    cmat *nominal=concat_ccell_as_vector(nominalc);
			    ccellfree(nominalc);
			    cp2gpu(&cuwfs[iwfs].dtf[iwvl].nominal, nominal);
			    cfree(nominal);
			}
			//ETF moved to gpu_wfsgrad_update_etf();
		    }/*for iwvl. */
		    if(parms->powfs[ipowfs].llt){
			cp2gpu(&cuwfs[iwfs].srot, powfs[ipowfs].srot->p[parms->powfs[ipowfs].llt->n>1?wfsind:0]);
		    }
		}else{
		    cuwfs[iwfs].dtf  = cuwfs[iwfs0].dtf;
		    cuwfs[iwfs].srot = cuwfs[iwfs0].srot;
		}
		/*Matched filter */
		if(parms->powfs[ipowfs].phytypesim==1){
		    //Separated with gpu_wfsgrad_upate_mtche();
		}else if(parms->powfs[ipowfs].phytypesim==2){/*cog*/
		    if(powfs[ipowfs].intstat->cogcoeff->nx>1 || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cp2gpu(&cuwfs[iwfs].cogcoeff, 
			       powfs[ipowfs].intstat->cogcoeff->p[powfs[ipowfs].intstat->cogcoeff->nx>1?wfsind:0]->p, nsa*2, 1);
		    }else{
			cuwfs[iwfs].cogcoeff=cuwfs[iwfs0].cogcoeff;
		    }
		}
		if(powfs[ipowfs].bkgrnd){
		    if(powfs[ipowfs].bkgrnd->ny==1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cudaCallocHostBlock(cuwfs[iwfs].bkgrnd2, nsa*sizeof(void*));
			dmat **bkgrnd=powfs[ipowfs].bkgrnd->p+nsa*(powfs[ipowfs].bkgrnd->ny==1?wfsind:0);
			for(int isa=0; isa<nsa; isa++){
			    cp2gpu((Real**)&cuwfs[iwfs].bkgrnd2[isa], bkgrnd[isa]);
			}
		    }else{
			cuwfs[iwfs].bkgrnd2=cuwfs[iwfs0].bkgrnd2;
		    }
		}
		if(powfs[ipowfs].bkgrndc){
		    if(powfs[ipowfs].bkgrndc->ny==1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cudaCallocHostBlock(cuwfs[iwfs].bkgrnd2c, nsa*sizeof(void*));
			dmat **bkgrndc=powfs[ipowfs].bkgrndc->p+nsa*(powfs[ipowfs].bkgrndc->ny==1?wfsind:0);
			for(int isa=0; isa<nsa; isa++){
			    cp2gpu((Real**)&cuwfs[iwfs].bkgrnd2c[isa], bkgrndc[isa]);
			}
		    }else{
			cuwfs[iwfs].bkgrnd2c=cuwfs[iwfs0].bkgrnd2c;
		    }	
		}
		if(parms->powfs[ipowfs].dither){
		    cuwfs[iwfs].dither=new dither_t(nsa,powfs[ipowfs].pixpsax,powfs[ipowfs].pixpsay);
		}
	    }
	    const int msa=cuwfs[iwfs].msa;
	    cuwfs[iwfs].wvf=cucnew(nwvf*nwvf,msa);
	    if(nwvf!=notf){
		cuwfs[iwfs].psf=cucnew(notf*notf,msa);
	    }
	    if(parms->powfs[ipowfs].radrot || ncompx!=notf || ncompy!=notf){
		cuwfs[iwfs].otf=cucnew(ncompx*ncompy,msa);
	    }
	    if(parms->powfs[ipowfs].psfout){
		const int wvf_n=notf/2+2;
		cuwfs[iwfs].wvfout=cuccellnew(nsa, nwvl, wvf_n, wvf_n);
		cuwfs[iwfs].psfout=cucnew(notf*notf, msa);
	    }
	    if(parms->powfs[ipowfs].pistatout){
		cuwfs[iwfs].psfstat=cucnew(notf*notf, msa);
	    }
	    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		int nlx=powfs[ipowfs].llt->pts->nx;
		int nlwvf=nlx*parms->powfs[ipowfs].embfac;
		cuwfs[iwfs].lltopd=curnew(nlx, nlx);
		if(parms->powfs[ipowfs].pistatout || parms->sim.uptideal){
		    DO(hipHostMalloc(&cuwfs[iwfs].lltg, 2*sizeof(Real)));
		}
		cuwfs[iwfs].lltwvf=cucnew(nlwvf, nlwvf);
		if(nlwvf!=notf){
		    cuwfs[iwfs].lltotfc=cucnew(notf, notf);
		}
	    }
	
	
	}/*if phy */
	CUDA_SYNC_DEVICE;
	gpu_print_mem("wfs init");
    }/*for iwfs */
    gpu_wfsgrad_update_etf(parms, powfs);
    gpu_wfsgrad_update_mtche(parms, powfs);
}
void gpu_wfs_init_sim(const PARMS_T *parms, POWFS_T *powfs){
    int *wfsgpu=cudata_t::wfsgpu;
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwfs_t *cuwfs=cudata_t::wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int nsa=powfs[ipowfs].pts->nsa;
	//gradacc is used for accumulation in geom mode and for output in phy mode
	initzero(&cuwfs[iwfs].gradacc, nsa*2, 1);
	initzero(&cuwfs[iwfs].gradcalc,nsa*2,1);
	if(parms->powfs[ipowfs].usephy || parms->powfs[ipowfs].dither){
	    if(!cuwfs[iwfs].ints){
		cuwfs[iwfs].ints=curcellnew(nsa,1,powfs[ipowfs].pixpsax,powfs[ipowfs].pixpsay);
	    }else{
		curcellzero(cuwfs[iwfs].ints);
	    }
	}
	if(parms->powfs[ipowfs].pistatout){
	    if(parms->powfs[ipowfs].pistatstc){
		error("pistatstc is not supported yet.\n");
	    }
	    if(!cuwfs[iwfs].pistatout){
		const int notfx=powfs[ipowfs].ncompx;/*necessary size to build detector image. */
		const int notfy=powfs[ipowfs].ncompy;
		const int npsf=MAX(notfx,notfy);
		cuwfs[iwfs].pistatout=curcellnew(nsa, parms->powfs[ipowfs].nwvl, npsf, npsf);
	    }else{
		curcellzero(cuwfs[iwfs].pistatout);
	    }
	}
	CUDA_SYNC_DEVICE;
    }
}
void gpu_wfssurf2gpu(const PARMS_T *parms, POWFS_T *powfs){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(cudata_t::wfsgpu[iwfs]);
	cuwfs_t *cuwfs=cudata_t::wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
	if(powfs[ipowfs].opdadd && powfs[ipowfs].opdadd->p[wfsind]){
	    cp2gpu(&cuwfs[iwfs].opdadd, powfs[ipowfs].opdadd->p[wfsind]);
	    dfree(powfs[ipowfs].opdadd->p[wfsind]);/*no longer need it in CPU memory. */
	}
    }
}
__global__ static void setup_rand(hiprandState *rstat, int seed){
    int id=threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &rstat[id]);
}
/**
   Seed the random number genrator
*/
void gpu_wfsgrad_seeding(const PARMS_T *parms, const POWFS_T *powfs, rand_t *rstat){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(cudata_t::wfsgpu[iwfs]);
	cuwfs_t *cuwfs=cudata_t::wfs;
	int seed=lrand(rstat);/*don't put this after continue. */
	int ipowfs=parms->wfs[iwfs].powfs;
	if(!parms->powfs[ipowfs].noisy) continue;
	int nsa=powfs[ipowfs].pts->nsa*2;
	if(nsa<RAND_THREAD){
	    cuwfs[iwfs].custatt=nsa;//number of threads
	    cuwfs[iwfs].custatb=1;//number of blocks
	}else if(nsa<RAND_THREAD*RAND_BLOCK){
	    cuwfs[iwfs].custatt=RAND_THREAD;
	    cuwfs[iwfs].custatb=nsa/RAND_THREAD+(nsa%RAND_THREAD)?1:0;
	}else{
	    cuwfs[iwfs].custatt=RAND_THREAD;
	    cuwfs[iwfs].custatb=RAND_BLOCK;
	}
	DO(hipMalloc(&cuwfs[iwfs].custat, (cuwfs[iwfs].custatt*cuwfs[iwfs].custatb)*sizeof(hiprandState)));
	setup_rand<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt>>>(cuwfs[iwfs].custat, seed);
    }
    CUDA_SYNC_DEVICE;
    gpu_print_mem("wfs seeding");
}
