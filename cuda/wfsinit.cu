#include "hip/hip_runtime.h"
/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipfft/hipfft.h>
#include "wfs.h"
#include "cudata.h"

int *wfsgpu=NULL;/*assign GPU to wfs statically. */
/**
   Initialize other arrays
*/
void gpu_wfsgrad_init(const PARMS_T *parms, const POWFS_T *powfs){
    wfsgpu=(int*)calloc(parms->nwfs, sizeof(int));
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	wfsgpu[iwfs]=gpu_next();
	if(NGPU>4 && wfsgpu[iwfs]==gpu_recon){
	    wfsgpu[iwfs]=gpu_next();
	}
    }
    for(int im=0; im<NGPU; im++){
	gpu_set(im);
	cudata->powfs=(cuwloc_t*)calloc(parms->npowfs, sizeof(cuwloc_t));
	cudata->wfs=(cuwfs_t*)calloc(parms->nwfs, sizeof(cuwfs_t));
	cuwloc_t *cupowfs=cudata->powfs;
	
	/* Setup information that are same for wfs in each powfs*/
	for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
	    if(parms->powfs[ipowfs].nwfs==0) continue;
	    pts_t *pts=powfs[ipowfs].pts;
	    loc_t *loc=powfs[ipowfs].loc;
	    cupowfs[ipowfs].pts=new cupts_t(pts);
	    cupowfs[ipowfs].loc=new culoc_t(loc);
	    cupowfs[ipowfs].saloc=new culoc_t(powfs[ipowfs].saloc);
	
	    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		pts=powfs[ipowfs].llt->pts;
		loc=powfs[ipowfs].llt->loc;
		cupowfs[ipowfs].llt=new cullt_t;
		cupowfs[ipowfs].llt->pts=new cupts_t(pts);
		cupowfs[ipowfs].llt->loc=new culoc_t(loc);
	    }
	    /*cupowfs[ipowfs].skip=parms->powfs[ipowfs].skip; */
	    if(parms->powfs[ipowfs].fieldstop){
		cp2gpu(&cupowfs[ipowfs].embed, powfs[ipowfs].embed, powfs[ipowfs].loc->nloc, 1);
		cupowfs[ipowfs].nembed=powfs[ipowfs].nembed;
		cp2gpu(&cupowfs[ipowfs].fieldstop, powfs[ipowfs].fieldstop);
	    }
	}
    }

    /* setup information that maybe different for wfs in same powfs due to
       misregistration or NCPA.*/
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwloc_t *cupowfs=cudata->powfs;
	cuwfs_t *cuwfs=cudata->wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int nsa=powfs[ipowfs].pts->nsa;
	int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
	int iwfs0=parms->powfs[ipowfs].wfs[0];
	/*imcc for ztilt. */
	cuwfs[iwfs].stream=new stream_t;
	if(parms->powfs[ipowfs].fieldstop){
	    DO(hipfftPlan2d(&cuwfs[iwfs].plan_fs, cupowfs[ipowfs].nembed, cupowfs[ipowfs].nembed, HIPFFT_C2C));
	    hipfftSetStream(cuwfs[iwfs].plan_fs, *cuwfs[iwfs].stream);
	}
	if(powfs[ipowfs].saimcc){
	    if(powfs[ipowfs].nsaimcc>1 || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		void *imcc[nsa];
		for(int isa=0; isa<nsa; isa++){
		    imcc[isa]=NULL;
		    cp2gpu((float**)&(imcc[isa]),
			   powfs[ipowfs].saimcc[powfs[ipowfs].nsaimcc>1?wfsind:0]->p[isa]);
		}
		hipMalloc(&cuwfs[iwfs].imcc, nsa*sizeof(void*));
		hipMemcpy(cuwfs[iwfs].imcc, imcc, nsa*sizeof(void*), hipMemcpyHostToDevice);
	    }else{
		cuwfs[iwfs].imcc=cuwfs[iwfs0].imcc;
	    }
	}
	cuwfs[iwfs].powfs=cupowfs+ipowfs;
	hipDeviceSynchronize();
	/*GS0 for gtilt. */
	if(powfs[ipowfs].GS0){
	    if(powfs[ipowfs].GS0->nx>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		dsp *t=powfs[ipowfs].GS0->p[powfs[ipowfs].GS0->nx>1?wfsind:0];
		cuwfs[iwfs].GS0=new cusp(t, 1);
	    }else{
		cuwfs[iwfs].GS0=cuwfs[iwfs0].GS0;
	    }
	}
	/*wfs amplitude map on loc */
	if(powfs[ipowfs].nlocm>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
	    cp2gpu(&cuwfs[iwfs].amp, powfs[ipowfs].realamp[powfs[ipowfs].nlocm>1?wfsind:0], powfs[ipowfs].loc->nloc, 1);
	}else{
	    cuwfs[iwfs].amp=cuwfs[iwfs0].amp;
	}
	dmat *nea=powfs[ipowfs].neasim->p[wfsind];
	if(nea){
	    cp2gpu(&cuwfs[iwfs].neasim, nea);
	}

	/* * Now start physical optics setup * */

	if(parms->powfs[ipowfs].usephy||parms->powfs[ipowfs].psfout||parms->powfs[ipowfs].pistatout){
	    /*If there is llt. */
	    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		if(powfs[ipowfs].llt->ncpa){
		    if(powfs[ipowfs].llt->ncpa->nx>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cp2gpu(&cuwfs[iwfs].lltncpa, powfs[ipowfs].llt->ncpa->p[powfs[ipowfs].llt->ncpa->nx>1?wfsind:0]);
		    }else{
			cuwfs[iwfs].lltncpa=cuwfs[iwfs0].lltncpa;
		    }
		}
		if(wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    hipHostMalloc(&cuwfs[iwfs].lltimcc, 1*sizeof(void*));
		    cuwfs[iwfs].lltimcc[0]=NULL;
		    cp2gpu((float**)&cuwfs[iwfs].lltimcc[0], powfs[ipowfs].llt->imcc->p[0]);
		    cp2gpu((float**)&cuwfs[iwfs].lltamp, powfs[ipowfs].llt->amp);
		}else{
		    cuwfs[iwfs].lltimcc=cuwfs[iwfs0].lltimcc;
		    cuwfs[iwfs].lltamp=cuwfs[iwfs0].lltamp;
		}
	    }
	    hipDeviceSynchronize();

	    /*CUFFTW is row major. */
	    int nwvf=powfs[ipowfs].pts->nx*parms->powfs[ipowfs].embfac;/*size of fft */
	    int nwvf2[2]={nwvf, nwvf};
	    const int ncompx=powfs[ipowfs].ncompx;
	    const int ncompy=powfs[ipowfs].ncompy;
	    const int notf=MAX(ncompx, ncompy);
	    int ncomp2[2]={ncompx, ncompy};
	    int notf2[2]={notf, notf};
	    /*limit the number of subapertures in each batch to less than 1024
	      to save memory. The speed is actually a tiny bit faster for NFIRAOS.*/
	    cuwfs[iwfs].msa=nsa>1024?((int)ceil((float)nsa/(float)(nsa/800))):nsa;
	    if(hipfftPlanMany(&cuwfs[iwfs].plan1, 2, nwvf2, NULL, 1, 0, NULL, 1, 0, 
			     HIPFFT_C2C, cuwfs[iwfs].msa)){
		error("CUFFT plan failed\n");
	    }
	    hipfftSetStream(cuwfs[iwfs].plan1, *cuwfs[iwfs].stream);

	    if(notf==nwvf){
		cuwfs[iwfs].plan2=cuwfs[iwfs].plan1;
	    }else{
		if(hipfftPlanMany(&cuwfs[iwfs].plan2, 2, notf2, NULL, 1, 0, NULL, 1, 0, 
				 HIPFFT_C2C, cuwfs[iwfs].msa)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].plan2, *cuwfs[iwfs].stream);
	    }
	    if(notf==ncompx && notf==ncompy){
		cuwfs[iwfs].plan3=cuwfs[iwfs].plan2;
	    }else{
		if(hipfftPlanMany(&cuwfs[iwfs].plan3, 2, ncomp2, NULL, 1, 0, NULL, 1, 0, 
				 HIPFFT_C2C, cuwfs[iwfs].msa)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].plan3, *cuwfs[iwfs].stream);
	    }

	    if(parms->powfs[ipowfs].llt){
		int nlwvf=powfs[ipowfs].llt->pts->nx*parms->powfs[ipowfs].embfac;
		int nlwvf2[2]={nlwvf, nlwvf};
		if(hipfftPlanMany(&cuwfs[iwfs].lltplan_wvf, 2, nlwvf2, NULL, 1,0, NULL, 1, 0, 
				 HIPFFT_C2C, 1)){
		    error("CUFFT plan failed\n");
		}
		hipfftSetStream(cuwfs[iwfs].lltplan_wvf, *cuwfs[iwfs].stream);
		if(notf==nlwvf){
		    cuwfs[iwfs].lltplan_otf=cuwfs[iwfs].lltplan_wvf;
		}else{
		    if(hipfftPlanMany(&cuwfs[iwfs].lltplan_otf, 2, notf2, NULL, 1, 0, NULL, 1, 0, 
				     HIPFFT_C2C, 1)){
			error("CUFFT plan failed\n");
		    }
		    hipfftSetStream(cuwfs[iwfs].lltplan_otf, *cuwfs[iwfs].stream);
		}
	    }
	    /*DTF. */
	    if(parms->powfs[ipowfs].usephy){
		if(parms->powfs[ipowfs].llt && parms->powfs[ipowfs].llt->n>1 
		   || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
		    /*Need one per wfs in this powfs, or the first wfs. */
		    int nwvl=parms->powfs[ipowfs].nwvl;
		    cuwfs[iwfs].dtf=(cudtf_t*)calloc(nwvl, sizeof(cudtf_t));
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			int notfused=!powfs[ipowfs].dtf[iwvl].fused;
			if(notfused){
			    fcomplex *nominal[nsa];
			    /*cudaCallocHostBlock(cuwfs[iwfs].dtf[iwvl].si, nsa*sizeof(void*)); */
			    int multi_nominal=(powfs[ipowfs].dtf[iwvl].si->nx==nsa);
			    for(int isa=0; isa<nsa; isa++){
				if(multi_nominal || isa==0){
				    if(notfused){
					nominal[isa]=NULL;
					cp2gpu(&nominal[isa], powfs[ipowfs].dtf[iwvl].nominal->p[isa+nsa*(powfs[ipowfs].dtf[iwvl].nominal->ny>1?wfsind:0)]);
				    }
				}else{
				    nominal[isa]=nominal[0];
				}
			    }
			    hipMalloc(&cuwfs[iwfs].dtf[iwvl].nominal, nsa*sizeof(void*));
			    hipMemcpy(cuwfs[iwfs].dtf[iwvl].nominal, nominal, nsa*sizeof(void*), hipMemcpyHostToDevice);
			}
			if(parms->powfs[ipowfs].llt){
			    fcomplex *etf[nsa];
			    cmat *(*petf)[nsa]=NULL;
			    if(powfs[ipowfs].etfsim[iwvl].p1){
				petf=(cmat *(*)[nsa])powfs[ipowfs].etfsim[iwvl].p1->p;
				cuwfs[iwfs].dtf[iwvl].etfis1d=1;
			    }else{
				petf=(cmat *(*)[nsa])powfs[ipowfs].etfsim[iwvl].p2->p;
				cuwfs[iwfs].dtf[iwvl].etfis1d=0;
			    }
			    cmat **petfi=petf[parms->powfs[ipowfs].llt->n>1?wfsind:0];
			    int ncx=petfi[0]->nx;
			    int ncy=petfi[0]->ny;
			    /* Coping many small arrays is slow due to
			       overhead. We first copy the array to a single
			       array then assign pointers. 2012-07-06*/
			    fcomplex *temp=(fcomplex*)malloc(sizeof(fcomplex)*nsa*ncx*ncy);
			    fcomplex *tempi;
			    fcomplex *temp2;
			    DO(hipMalloc(&temp2, sizeof(fcomplex)*nsa*ncx*ncy));
			    for(int isa=0; isa<nsa; isa++){
				tempi=temp+isa*ncx*ncy;
				etf[isa]=temp2+isa*ncx*ncy;
				for(int i=0; i<ncx*ncy; i++){
				    tempi[i]=make_hipFloatComplex(creal(petfi[isa]->p[i]),
								 cimag(petfi[isa]->p[i]));
				}
			    }
			    DO(hipMemcpy(temp2, temp, sizeof(fcomplex)*nsa*ncx*ncy, hipMemcpyHostToDevice));
			    free(temp);
			    hipMalloc(&cuwfs[iwfs].dtf[iwvl].etf, nsa*sizeof(void*));
			    hipMemcpy(cuwfs[iwfs].dtf[iwvl].etf, etf, nsa*sizeof(void*), hipMemcpyHostToDevice);
			}
		    }/*for iwvl. */
		    if(parms->powfs[ipowfs].llt){
			cp2gpu(&cuwfs[iwfs].srot, powfs[ipowfs].srot->p[parms->powfs[ipowfs].llt->n>1?wfsind:0]);
		    }
		}else{
		    cuwfs[iwfs].dtf  = cuwfs[iwfs0].dtf;
		    cuwfs[iwfs].srot = cuwfs[iwfs0].srot;
		}
		/*Matched filter */
		if(parms->powfs[ipowfs].phytypesim==1){
		    if(powfs[ipowfs].intstat->mtche->ny>1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			dmat **mtche=powfs[ipowfs].intstat->mtche->p+nsa*(powfs[ipowfs].intstat->mtche->ny>1?wfsind:0);
			float *mtche2[nsa];
			int ncx=mtche[0]->nx;
			int ncy=mtche[0]->ny;
			/* Coping many small arrays is slow due to
			   overhead. We first copy the array to a single
			   array then assign pointers. 2012-07-06*/
			float *temp=(float*)malloc(sizeof(float)*nsa*ncx*ncy);
			float *tempi;
			float *temp2;
			hipMalloc(&temp2, sizeof(float)*nsa*ncx*ncy);
			for(int isa=0; isa<nsa; isa++){
			    mtche2[isa]=temp2+isa*ncx*ncy;
			    tempi=temp+isa*ncx*ncy;
			    for(int i=0; i<ncx*ncy; i++){
				tempi[i]=(float)(mtche[isa]->p[i]);
			    }
			}
			hipMemcpy(temp2, temp, nsa*ncx*ncy*sizeof(float), hipMemcpyHostToDevice);
			free(temp);
			hipMalloc(&cuwfs[iwfs].mtche, nsa*sizeof(void*));
			hipMemcpy(cuwfs[iwfs].mtche, mtche2, nsa*sizeof(void*),hipMemcpyHostToDevice);
			cp2gpu(&cuwfs[iwfs].i0sum, powfs[ipowfs].intstat->i0sum->p+nsa*(powfs[ipowfs].intstat->i0sum->ny>1?wfsind:0), nsa, 1);
		    }else{
			cuwfs[iwfs].mtche=cuwfs[iwfs0].mtche;
			cuwfs[iwfs].i0sum=cuwfs[iwfs0].i0sum;
		    }
		}else if(parms->powfs[ipowfs].phytypesim==2){/*cog*/
		    if(powfs[ipowfs].intstat->cogcoeff->nx>1 || wfsind==0 || wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cp2gpu(&cuwfs[iwfs].cogcoeff, 
			       powfs[ipowfs].intstat->cogcoeff->p[powfs[ipowfs].intstat->cogcoeff->nx>1?wfsind:0]->p, nsa*2, 1);
		    }else{
			cuwfs[iwfs].cogcoeff=cuwfs[iwfs0].cogcoeff;
		    }
		}
		if(powfs[ipowfs].bkgrnd){
		    if(powfs[ipowfs].bkgrnd->ny==1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cudaCallocHostBlock(cuwfs[iwfs].bkgrnd2, nsa*sizeof(void*));
			dmat **bkgrnd=powfs[ipowfs].bkgrnd->p+nsa*(powfs[ipowfs].bkgrnd->ny==1?wfsind:0);
			for(int isa=0; isa<nsa; isa++){
			    cp2gpu((float**)&cuwfs[iwfs].bkgrnd2[isa], bkgrnd[isa]);
			}
		    }else{
			cuwfs[iwfs].bkgrnd2=cuwfs[iwfs0].bkgrnd2;
		    }
		}
		if(powfs[ipowfs].bkgrndc){
		    if(powfs[ipowfs].bkgrndc->ny==1 || wfsind==0|| wfsgpu[iwfs]!=wfsgpu[iwfs0]){
			cudaCallocHostBlock(cuwfs[iwfs].bkgrnd2c, nsa*sizeof(void*));
			dmat **bkgrndc=powfs[ipowfs].bkgrndc->p+nsa*(powfs[ipowfs].bkgrndc->ny==1?wfsind:0);
			for(int isa=0; isa<nsa; isa++){
			    cp2gpu((float**)&cuwfs[iwfs].bkgrnd2c[isa], bkgrndc[isa]);
			}
		    }else{
			cuwfs[iwfs].bkgrnd2c=cuwfs[iwfs0].bkgrnd2c;
		    }	
		}
	    }
	}/*if phy */
	CUDA_SYNC_DEVICE;
	gpu_print_mem("wfs init");
    }/*for iwfs */
}
void gpu_wfs_init_sim(const PARMS_T *parms, POWFS_T *powfs){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
	cuwfs_t *cuwfs=cudata->wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int nsa=powfs[ipowfs].pts->nsa;
	if(parms->powfs[ipowfs].phystep!=0
	   ||parms->save.gradgeom[iwfs]
	   ||parms->powfs[ipowfs].pistatout){
	    /*gradacc for geom wfs accumulation */
	    curfree(cuwfs[iwfs].gradacc);
	    cuwfs[iwfs].gradacc=curnew(nsa*2,1);
	}
	if(parms->powfs[ipowfs].usephy){
	    curcellfree(cuwfs[iwfs].ints);
	    cuwfs[iwfs].ints=curcellnew(nsa,1,powfs[ipowfs].pixpsax,powfs[ipowfs].pixpsay);
	}
	if(parms->powfs[ipowfs].pistatout){
	    if(parms->powfs[ipowfs].pistatstc){
		error("pistatstc is not supported yet.\n");
	    }
	    curcellfree(cuwfs[iwfs].pistatout);
	    const int notfx=powfs[ipowfs].ncompx;/*necessary size to build detector image. */
	    const int notfy=powfs[ipowfs].ncompy;
	    const int npsf=MAX(notfx,notfy);
	    cuwfs[iwfs].pistatout=curcellnew(nsa, parms->powfs[ipowfs].nwvl, npsf, npsf);
	}
	CUDA_SYNC_DEVICE;
    }
}
void gpu_wfssurf2gpu(const PARMS_T *parms, POWFS_T *powfs){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);
	cuwfs_t *cuwfs=cudata->wfs;
	int ipowfs=parms->wfs[iwfs].powfs;
	int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
	if(powfs[ipowfs].opdadd && powfs[ipowfs].opdadd->p[wfsind]){
	    cp2gpu(&cuwfs[iwfs].opdadd, powfs[ipowfs].opdadd->p[wfsind]);
	    dfree(powfs[ipowfs].opdadd->p[wfsind]);/*no longer need it in CPU memory. */
	}
	if(powfs[ipowfs].gradoff && powfs[ipowfs].gradoff->p[wfsind]){
	    cp2gpu(&cuwfs[iwfs].gradoff, powfs[ipowfs].gradoff->p[wfsind]);
	}	
    }
}
__global__ static void setup_rand(hiprandState *rstat, int seed){
    int id=threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &rstat[id]);
}
/**
   Seed the random number genrator
*/
void gpu_wfsgrad_seeding(const PARMS_T *parms, const POWFS_T *powfs, rand_t *rstat){
    for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
	gpu_set(wfsgpu[iwfs]);
	cuwfs_t *cuwfs=cudata->wfs;
	int seed=lrand(rstat);/*don't put this after continue. */
	int ipowfs=parms->wfs[iwfs].powfs;
	if(!parms->powfs[ipowfs].noisy) continue;
	int nsa=powfs[ipowfs].pts->nsa*2;
	if(nsa<RAND_THREAD){
	    cuwfs[iwfs].custatt=nsa;
	    cuwfs[iwfs].custatb=1;
	}else if(nsa<RAND_THREAD*RAND_BLOCK){
	    cuwfs[iwfs].custatt=RAND_THREAD;
	    cuwfs[iwfs].custatb=nsa/RAND_THREAD+(nsa%RAND_THREAD)?1:0;
	}else{
	    cuwfs[iwfs].custatt=RAND_THREAD;
	    cuwfs[iwfs].custatb=RAND_BLOCK;
	}
	hipMalloc(&cuwfs[iwfs].custat, (cuwfs[iwfs].custatt*cuwfs[iwfs].custatb)*sizeof(hiprandState));
	setup_rand<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt>>>(cuwfs[iwfs].custat, seed);
    }
    CUDA_SYNC_DEVICE;
    gpu_print_mem("wfs seeding");
}
