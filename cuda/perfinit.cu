/*
  Copyright 2009-2018 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "accphi.h"
#include "cucmat.h"
#include "kernel.h"
#include "cudata.h"
#include "perf.h"

cuarray<int> cuperf_t::nembed;
cuarray<int> cuperf_t::psfsize;
cuarray<Real> cuperf_t::wvls;
cuarray<stream_t> cuperf_t::stream;
cuarray<hipfftHandle> cuperf_t::plan;
int cuperf_t::nevl;
curcell cuperf_t::surf;
curcell cuperf_t::opd;
curcell cuperf_t::psfcl;
curcell cuperf_t::psfcl_ngsr;
curcell cuperf_t::opdcov;
curcell cuperf_t::opdcov_ngsr;
curcell cuperf_t::opdmean;
curcell cuperf_t::opdmean_ngsr;
curcell cuperf_t::cc_cl;
curcell cuperf_t::cc_ol;
curcell cuperf_t::coeff;
Real **cuperf_t::ccb_cl=0;
Real **cuperf_t::ccb_ol=0;
pthread_mutex_t cuperf_t::perfmutex=PTHREAD_MUTEX_INITIALIZER;
cuperf_t::~cuperf_t(){
    //The static members are shared across devices. Need to lock mutex before reinitializing.
    lock_t tmp(perfmutex);
    surf=curcell();
    opd=curcell();
    psfcl=curcell();
    psfcl_ngsr=curcell();
    opdcov=curcell();
    opdcov_ngsr=curcell();
    opdmean=curcell();
    opdmean_ngsr=curcell();
    cc_ol=curcell();
    cc_cl=curcell();
    coeff=curcell();
    if(cuperf_t::ccb_cl){
	for(int ievl=0; ievl<nevl; ievl++){
	    free4async(cuperf_t::ccb_cl[ievl]);
	    free4async(cuperf_t::ccb_ol[ievl]);
	}
        free(cuperf_t::ccb_cl); cuperf_t::ccb_cl=0;
	free(cuperf_t::ccb_ol); cuperf_t::ccb_ol=0;
    }
}
/**
   Initialize perfevl
*/
void gpu_perfevl_init(const PARMS_T *parms, APER_T *aper){
    if(!parms->gpu.evl){
	return;
    }
    const int nevl=parms->evl.nevl;
    const int nwvl=parms->evl.nwvl;
    /*The following lives in CPU. */
    if(parms->evl.psfmean || parms->evl.psfhist){
	cuperf_t::nembed =cuarray<int>(nwvl, 1);
	cuperf_t::psfsize=cuarray<int>(nwvl, 1);
	cuperf_t::wvls   =cuarray<Real>(nwvl, 1);
    
	for(int iwvl=0; iwvl<nwvl; iwvl++){
	    cuperf_t::nembed[iwvl]=(int)aper->embed->nembed->p[iwvl];
	    cuperf_t::psfsize[iwvl]=parms->evl.psfsize->p[iwvl];
	    cuperf_t::wvls[iwvl]=parms->evl.wvl->p[iwvl];
	}
    }
    /*The following lives in GPU. */
    for(int im=0; im<NGPU; im++){
	gpu_set(im);
	cudata->perf.locs=culoc_t(aper->locs);
	cp2gpu(cudata->perf.amp, aper->amp);
	cp2gpu(cudata->perf.imcc, aper->imcc);
	if(parms->evl.psfmean || parms->evl.psfhist){
	    cudata->perf.embed    = (int**) calloc(nwvl, sizeof(int*));
	    for(int iwvl=0; iwvl<nwvl; iwvl++){
		cp2gpu(&cudata->perf.embed[iwvl], aper->embed->embed->p[iwvl]->p, aper->locs->nloc, 1);
	    }
	}
    }/*for igpu */
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cudata_t::evlgpu[ievl]);
	if(!cudata->perf.locs_dm){
	    cudata->perf.locs_dm=cuarray<cuarray<culoc_t> >(nevl, 1);
	}
	cudata->perf.locs_dm[ievl]=cuarray<culoc_t>(parms->ndm,1);
	for(int idm=0; idm<parms->ndm; idm++){
	    loc_t *loc_dm;
	    if(aper->locs_dm && aper->locs_dm->p[ievl+idm*nevl]){
		loc_dm=aper->locs_dm->p[ievl+idm*nevl];
	    }else{
		loc_dm=aper->locs;
	    }
	    cudata->perf.locs_dm[ievl][idm]=culoc_t(loc_dm);
	}
    }
    cuperf_t::stream=cuarray<stream_t>(nevl, 1);
    if(parms->evl.psfmean || parms->evl.psfhist){
	cuperf_t::plan  = cuarray<hipfftHandle>(nwvl*nevl,1);
    }
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cudata_t::evlgpu[ievl]);
	//STREAM_NEW(cuperf_t::stream[ievl]);
	//Use stream created per GPU in order to share resource within GPU between different evl dir.
	cuperf_t::stream[ievl]=cudata->perf_stream;
	if(parms->evl.psfmean || parms->evl.psfhist){
	    for(int iwvl=0; iwvl<nwvl; iwvl++){
		DO(hipfftPlan2d(&cuperf_t::plan[iwvl+nwvl*ievl],cuperf_t::nembed[iwvl],
			       cuperf_t::nembed[iwvl],FFT_T_C2C));
		DO(hipfftSetStream(cuperf_t::plan[iwvl+nwvl*ievl], cuperf_t::stream[ievl]));
	    }/*for iwvl */
	}
    }
    cuperf_t::nevl=nevl;
    cuperf_t::opd=curcell(nevl,1);
    cuperf_t::cc_cl=curcell(nevl, 1);
    cuperf_t::cc_ol=curcell(nevl, 1);
    cuperf_t::coeff=curcell(nevl, 1);
    cuperf_t::ccb_ol=(Real**)malloc(sizeof(Real*)*nevl);
    cuperf_t::ccb_cl=(Real**)malloc(sizeof(Real*)*nevl);
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cudata_t::evlgpu[ievl]);
	cuperf_t::ccb_ol[ievl]=(Real*)malloc4async(sizeof(Real)*7);
	cuperf_t::ccb_cl[ievl]=(Real*)malloc4async(sizeof(Real)*7);
	cuperf_t::cc_cl[ievl]=curmat(7,1);
	cuperf_t::cc_ol[ievl]=curmat(7,1);
	cuperf_t::coeff[ievl]=curmat(7,1);
	cuperf_t::opd[ievl]=curmat(aper->locs->nloc, 1);
    }
    if(!parms->sim.evlol){
	if(parms->evl.cov && parms->gpu.psf){
	    cuperf_t::opdcov=curcell(nevl, 1);
	    cuperf_t::opdmean=curcell(nevl, 1);
	    cuperf_t::opdcov_ngsr=curcell(nevl, 1);
	    cuperf_t::opdmean_ngsr=curcell(nevl, 1);
	}
	if(parms->evl.psfmean || parms->evl.psfhist){
	    cuperf_t::psfcl = curcell(nwvl, parms->evl.nevl);
	    cuperf_t::psfcl_ngsr = curcell(nwvl, parms->evl.nevl);
	}
    }
    if(aper->opdadd){
	cuperf_t::surf=curcell(nevl, 1);
	for(int ievl=0; ievl<nevl; ievl++){
	    gpu_set(cudata_t::evlgpu[ievl]);
	    cp2gpu(cuperf_t::surf[ievl], aper->opdadd->p[ievl]);
	}
    }
    gpu_print_mem("perf init");
}
/*
  Initialize simulation data. Seed dependent. Create for the first seed and zero for the next.
*/
void gpu_perfevl_init_sim(const PARMS_T *parms, APER_T *aper){
    const int nevl=parms->evl.nevl;
    const int nwvl=parms->evl.nwvl;
    int nloc=aper->locs->nloc;
    if(!parms->gpu.evl){
	return;
    }
    /*first open loop ones are on every GPU.*/
    if(parms->evl.psfol){
	for(int im=0; im<NGPU; im++){
	    gpu_set(im);
	    if(parms->evl.cov && parms->gpu.psf){ /*do OL opd cov*/
		initzero(cudata->perf.opdcovol, nloc, nloc);
		initzero(cudata->perf.opdmeanol, nloc, 1);
	    }
	    if(parms->evl.psfmean || parms->evl.psfhist){
		if(cudata->perf.psfol){
		    cuzero(cudata->perf.psfol);
		}else{
		    cudata->perf.psfol=curcell(nwvl,1);
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			cudata->perf.psfol[iwvl]=curmat(cudata->perf.psfsize[iwvl], 
							cudata->perf.psfsize[iwvl]);
		    }
		}
	    }
	}
    }

    if(parms->evl.cov && parms->gpu.psf && !parms->sim.evlol){
	for(int ievl=0; ievl<nevl; ievl++){
	    if(parms->evl.psf->p[ievl]==0){
		continue;
	    }
	    gpu_set(cudata_t::evlgpu[ievl]);
	    if(parms->evl.psfngsr->p[ievl]){
		initzero(cuperf_t::opdcov_ngsr[ievl], nloc,nloc);
		initzero(cuperf_t::opdmean_ngsr[ievl], nloc,1);
	    }
	    if(parms->evl.psfngsr->p[ievl]!=2){
		initzero(cuperf_t::opdcov[ievl],nloc,nloc);
		initzero(cuperf_t::opdmean[ievl],nloc,1);
	    }
	}
    }
	
    if(parms->evl.psfmean || parms->evl.psfhist){
	for(int im=0; im<NGPU; im++){
	    gpu_set(im);
	    /**
	       Using mutex to protect psf data that is allocated per GPU does
	       not work because the process happens in streams that is outside
	       of the lock. Streams are not allocated per GPU.
	    
	       Even if same stream is used for different EVL directions on the
	       same GPU. It is still not ok to share the perf.wvf or perf.psfs as different
	       EVL directions are running in different threads and will step
	       onto each other.
	     */
	    /*
	    if(!cudata->perf.wvf){
		cudata->perf.wvf=cuccell(nwvl, 1);
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    if(0 && !parms->evl.psfhist && iwvl>0 && cuperf_t::nembed[iwvl] == cuperf_t::nembed[iwvl-1]){
			cudata->perf.wvf[iwvl]=cudata->perf.wvf[iwvl-1];
		    }else{
			cudata->perf.wvf[iwvl]=cucmat(cuperf_t::nembed[iwvl], cuperf_t::nembed[iwvl]);
		    }
		}
		}*/
	    if(parms->evl.psfhist){
		warning("Need to revise how the PSF history is stored. Per GPU storage is only valid if EVL operations are seralized within each GPU\n");
		cudata->perf.psfs=cuccell(nwvl, 1);
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    if(0 && cuperf_t::psfsize[iwvl]<cuperf_t::nembed[iwvl]){
			cudata->perf.psfs[iwvl]=cucmat(cuperf_t::psfsize[iwvl], cuperf_t::psfsize[iwvl]);
		    }else{
			cudata->perf.psfs[iwvl]=cudata->perf.wvf[iwvl];
		    }
		}
		}
	}
	if(!parms->sim.evlol){
	    for(int ievl=0; ievl<nevl; ievl++){
		if(parms->evl.psf->p[ievl]==0){
		    continue;
		}
		gpu_set(cudata_t::evlgpu[ievl]);
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    if(parms->evl.psfngsr->p[ievl]){
			initzero(cuperf_t::psfcl_ngsr[iwvl+nwvl*ievl], 
				 cuperf_t::psfsize[iwvl], cuperf_t::psfsize[iwvl]);
		    }
		    if(parms->evl.psfngsr->p[ievl]!=2){
			initzero(cuperf_t::psfcl[iwvl+nwvl*ievl],
				 cuperf_t::psfsize[iwvl], cuperf_t::psfsize[iwvl]);
		    }
		}	
	    }
	}
    }
}
