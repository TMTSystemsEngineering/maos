/*
  Copyright 2009-2015 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "accphi.h"
#include "cucmat.h"
#include "kernel.h"
#include "cudata.h"
#include "perf.h"

int   *cuperf_t::nembed=0;
int   *cuperf_t::psfsize=0;
Real *cuperf_t::wvls=0;    
hipStream_t    *cuperf_t::stream=0;
hipblasHandle_t  *cuperf_t::handle=0;
hipfftHandle     *cuperf_t::plan=0;

curcell *cuperf_t::surf=0;
curcell *cuperf_t::opd=0;
curcell *cuperf_t::psfcl=0;
curcell *cuperf_t::psfcl_ngsr=0;
curcell *cuperf_t::opdcov=0;
curcell *cuperf_t::opdcov_ngsr=0;
curcell *cuperf_t::opdmean=0;
curcell *cuperf_t::opdmean_ngsr=0;
curcell *cuperf_t::cc_cl=0;
curcell *cuperf_t::cc_ol=0;
curcell *cuperf_t::coeff=0;
Real **cuperf_t::ccb_cl=0;
Real **cuperf_t::ccb_ol=0;

/**
   Initialize perfevl
*/
void gpu_perfevl_init(const PARMS_T *parms, APER_T *aper){
    if(!parms->gpu.evl){
	return;
    }
    const int nevl=parms->evl.nevl;
    const int nwvl=parms->evl.nwvl;
    /*The following lives in CPU. */
    if(parms->evl.psfmean || parms->evl.psfhist){
	cuperf_t::nembed =(int*)  calloc(nwvl, sizeof(int));
	cuperf_t::psfsize=(int*)  calloc(nwvl, sizeof(int));
	cuperf_t::wvls   =(Real*)calloc(nwvl, sizeof(Real));
    
	for(int iwvl=0; iwvl<nwvl; iwvl++){
	    cuperf_t::nembed[iwvl]=(int)aper->embed->nembed->p[iwvl];
	    cuperf_t::psfsize[iwvl]=parms->evl.psfsize->p[iwvl];
	    cuperf_t::wvls[iwvl]=parms->evl.wvl->p[iwvl];
	}
    }
    /*The following lives in GPU. */
    for(int im=0; im<NGPU; im++){
	gpu_set(im);
	cudata->perf=new cuperf_t;
	pthread_mutex_init(&cudata->perf->mutex, 0);
	cudata->perf->locs=new culoc_t(aper->locs);
	cp2gpu(&cudata->perf->amp, aper->amp);
	cp2gpu(&cudata->perf->imcc, aper->imcc);
	if(parms->evl.psfmean || parms->evl.psfhist){
	    cudata->perf->embed    = (int**) calloc(nwvl, sizeof(int*));
	    for(int iwvl=0; iwvl<nwvl; iwvl++){
		cp2gpu(&cudata->perf->embed[iwvl], aper->embed->embed->p[iwvl]->p, aper->locs->nloc, 1);
	    }
	}
    }/*for igpu */
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cudata_t::evlgpu[ievl]);
	if(!cudata->perf->locs_dm){
	    cudata->perf->locs_dm=new culoc_t**[nevl];
	}
	cudata->perf->locs_dm[ievl]=new culoc_t*[parms->ndm];
	for(int idm=0; idm<parms->ndm; idm++){
	    loc_t *loc_dm;
	    if(aper->locs_dm && aper->locs_dm->p[ievl+idm*nevl]){
		loc_dm=aper->locs_dm->p[ievl+idm*nevl];
	    }else{
		loc_dm=aper->locs;
	    }
	    cudata->perf->locs_dm[ievl][idm]=new culoc_t(loc_dm);
	}
    }
    cuperf_t::stream=(hipStream_t*)calloc(nevl, sizeof(hipStream_t));
    cuperf_t::handle=(hipblasHandle_t*)calloc(nevl, sizeof(hipblasHandle_t));
    if(parms->evl.psfmean || parms->evl.psfhist){
	cuperf_t::plan  = (hipfftHandle*)calloc(nwvl*nevl, sizeof(hipfftHandle));
    }
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cudata_t::evlgpu[ievl]);
	STREAM_NEW(cuperf_t::stream[ievl]);
	hipblasCreate(&cuperf_t::handle[ievl]);
	hipblasSetStream(cuperf_t::handle[ievl], cuperf_t::stream[ievl]);
	if(parms->evl.psfmean || parms->evl.psfhist){
	    for(int iwvl=0; iwvl<nwvl; iwvl++){
		if(iwvl>0 && cuperf_t::nembed[iwvl]==cuperf_t::nembed[0]){
		    cuperf_t::plan[iwvl+nwvl*ievl]=cuperf_t::plan[0+nwvl*ievl];
		}else{
		    DO(hipfftPlan2d(&cuperf_t::plan[iwvl+nwvl*ievl],cuperf_t::nembed[iwvl],
				   cuperf_t::nembed[iwvl],FFT_T_C2C));
		    DO(hipfftSetStream(cuperf_t::plan[iwvl+nwvl*ievl], cuperf_t::stream[ievl]));
		}
	    }/*for iwvl */
	}
    }
    cuperf_t::opd=curcellnew(nevl,1);
    cuperf_t::cc_cl=curcellnew(nevl, 1);
    cuperf_t::cc_ol=curcellnew(nevl, 1);
    cuperf_t::coeff=curcellnew(nevl, 1);
    cuperf_t::ccb_ol=(Real**)malloc(sizeof(Real*)*nevl);
    cuperf_t::ccb_cl=(Real**)malloc(sizeof(Real*)*nevl);
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cudata_t::evlgpu[ievl]);
	cuperf_t::ccb_ol[ievl]=(Real*)malloc4async(sizeof(Real)*7);
	cuperf_t::ccb_cl[ievl]=(Real*)malloc4async(sizeof(Real)*7);
	cuperf_t::cc_cl->p[ievl]=curnew(7, 1);
	cuperf_t::cc_ol->p[ievl]=curnew(7, 1);
	cuperf_t::coeff->p[ievl]=curnew(7, 1);
	cuperf_t::opd->p[ievl]=curnew(aper->locs->nloc, 1);
    }
    if(!parms->sim.evlol){
	if(parms->evl.cov && parms->gpu.psf){
	    cuperf_t::opdcov=curcellnew(nevl, 1);
	    cuperf_t::opdmean=curcellnew(nevl, 1);
	    cuperf_t::opdcov_ngsr=curcellnew(nevl, 1);
	    cuperf_t::opdmean_ngsr=curcellnew(nevl, 1);
	}
	if(parms->evl.psfmean || parms->evl.psfhist){
	    cuperf_t::psfcl = curcellnew(nwvl, parms->evl.nevl);
	    cuperf_t::psfcl_ngsr = curcellnew(nwvl, parms->evl.nevl);
	}
    }
    if(aper->opdadd){
	cuperf_t::surf=curcellnew(nevl, 1);
	for(int ievl=0; ievl<nevl; ievl++){
	    gpu_set(cudata_t::evlgpu[ievl]);
	    cp2gpu(&cuperf_t::surf->p[ievl], aper->opdadd->p[ievl]);
	}
    }
    gpu_print_mem("perf init");
}
/*
  Initialize simulation data. Seed dependent. Create for the first seed and zero for the next.
*/
void gpu_perfevl_init_sim(const PARMS_T *parms, APER_T *aper){
    const int nevl=parms->evl.nevl;
    const int nwvl=parms->evl.nwvl;
    int nloc=aper->locs->nloc;
    if(!parms->gpu.evl){
	return;
    }
    /*first open loop ones are on every GPU.*/
    if(parms->evl.psfol){
	for(int im=0; im<NGPU; im++){
	    gpu_set(im);
	    if(parms->evl.cov && parms->gpu.psf){ /*do OL opd cov*/
		initzero(&cudata->perf->opdcovol, nloc, nloc);
		initzero(&cudata->perf->opdmeanol, nloc, 1);
	    }
	    if(parms->evl.psfmean || parms->evl.psfhist){
		if(cudata->perf->psfol){
		    curcellzero(cudata->perf->psfol);
		}else{
		    cudata->perf->psfol=curcellnew(nwvl,1);
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			cudata->perf->psfol->p[iwvl]=curnew(cudata->perf->psfsize[iwvl], 
							    cudata->perf->psfsize[iwvl]);
		    }
		}
	    }
	}
    }

    if(parms->evl.cov && parms->gpu.psf && !parms->sim.evlol){
	for(int ievl=0; ievl<nevl; ievl++){
	    if(parms->evl.psf->p[ievl]==0){
		continue;
	    }
	    gpu_set(cudata_t::evlgpu[ievl]);
	    if(parms->evl.psfngsr->p[ievl]){
		initzero(&cuperf_t::opdcov_ngsr->p[ievl], nloc,nloc);
		initzero(&cuperf_t::opdmean_ngsr->p[ievl], nloc,1);
	    }
	    if(parms->evl.psfngsr->p[ievl]!=2){
		initzero(&cuperf_t::opdcov->p[ievl],nloc,nloc);
		initzero(&cuperf_t::opdmean->p[ievl],nloc,1);
	    }
	}
    }
	
    if(parms->evl.psfmean || parms->evl.psfhist){
	for(int im=0; im<NGPU; im++){
	    gpu_set(im);
	    if(!cudata->perf->wvf){/*temporary. no need to zero*/
		cudata->perf->wvf=cuccellnew(nwvl, 1);
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    if(!parms->evl.psfhist && iwvl>0 && cuperf_t::nembed[iwvl] == cuperf_t::nembed[iwvl-1]){
			cudata->perf->wvf->p[iwvl]=cucref(cudata->perf->wvf->p[iwvl-1]);
		    }else{
			cudata->perf->wvf->p[iwvl]=cucnew(cuperf_t::nembed[iwvl], cuperf_t::nembed[iwvl]);
		    }
		}
	    }
	    if(parms->evl.psfmean){
		cudata->perf->psfs=cuccellnew(nwvl, 1);
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    if(cuperf_t::psfsize[iwvl]<cuperf_t::nembed[iwvl]){
			cudata->perf->psfs->p[iwvl]=cucnew(cuperf_t::psfsize[iwvl], cuperf_t::psfsize[iwvl]);
		    }else{
			cudata->perf->psfs->p[iwvl]=cucref(cudata->perf->wvf->p[iwvl]);
		    }
		}
	    }
	}
	if(!parms->sim.evlol){
	    for(int ievl=0; ievl<nevl; ievl++){
		if(parms->evl.psf->p[ievl]==0){
		    continue;
		}
		gpu_set(cudata_t::evlgpu[ievl]);
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    if(parms->evl.psfngsr->p[ievl]){
			initzero(&cuperf_t::psfcl_ngsr->p[iwvl+nwvl*ievl], 
				 cuperf_t::psfsize[iwvl], cuperf_t::psfsize[iwvl]);
		    }
		    if(parms->evl.psfngsr->p[ievl]!=2){
			initzero(&cuperf_t::psfcl->p[iwvl+nwvl*ievl],
				 cuperf_t::psfsize[iwvl], cuperf_t::psfsize[iwvl]);
		    }
		}	
	    }
	}
    }
    CUDA_SYNC_DEVICE;
}
