/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "cudata.h"
int nstream=0;
#define MEM_RESERVE 200000000
int gpu_recon;/**<GPU for reconstruction*/
int NGPU=0;
int* GPUS=NULL;
cudata_t *cudata_all=NULL;/*for all GPU. */

#ifdef __APPLE__
pthread_key_t cudata_key;
static __attribute((constructor)) void init(){
    pthread_key_create(&cudata_key, NULL);
}
#else
__thread cudata_t *cudata=NULL;/*for current thread and current GPU */
#endif

int *cudata_t::evlgpu=0;
int *cudata_t::wfsgpu=0;
cuwfs_t *cudata_t::wfs=0;
/**
   Get GPU info.
*/
void gpu_info(){
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    info("name=%s\n"
	 "TotalGlobalMem=%d\n"
	 "SharedMemPerBlock=%d\n"
	 "regsPerBlock=%d\n"
	 "warpSize=%d",
	 prop.name,
	 (int)prop.totalGlobalMem,
	 (int)prop.sharedMemPerBlock,
	 prop.regsPerBlock,
	 prop.warpSize);
}
/**
   Print memory consumption.
*/
void gpu_print_mem(const char *msg){
    size_t fr, tot;
    hipDeviceSynchronize();
    DO(hipMemGetInfo(&fr, &tot));
    info2("GPU (%d) mem used %ld MB (%s)\n",GPUS[cudata-cudata_all],(long)(tot-fr)/1024/1024, msg);
}
/**
   Get available memory.
*/
long gpu_get_mem(void){
    size_t fr, tot;
    DO(hipMemGetInfo(&fr, &tot));
    return (long)fr;
}
/**
   Get available memory.
*/
static long gpu_get_idle_mem(void){
    size_t fr, tot;
    if(hipMemGetInfo(&fr, &tot)){
	fr=0; tot=0;
    }
    if(tot-fr>MEM_RESERVE){//GPU used by some other process. do not use it.
	return 0;
    }else{
	return (long)fr;
    }
}
static int cmp_long2_descend(const long *a, const long *b){
    return b[1]>a[1]?1:0;
}
static int cmp_long2_ascend(const long *a, const long *b){
    return a[1]>b[1]?1:0;
}
/**
   Initialize GPU. Return 1 if success.
   if gpus is not null, it is of length ngpu. gpus specifies gpu index to use.
   if gpus is null, ngpu specifies number of gpus to use. all if 0.

   when mix Tesla and GTX cards, the ordering the GPUs may be different in CUDA
   and NVML, causing the selection of GPUs to fail. Do not use NVML 
*/
int gpu_init(int *gpus, int ngpu){
    int ans, ngpu_tot=0;//total number of GPUs.
    if((ans=hipGetDeviceCount(&ngpu_tot)) || ngpu_tot==0){//no GPUs available.
	info2("No GPUs available. ans=%d\n", ans);
	return 0;
    }
    char fnlock[PATH_MAX];
    snprintf(fnlock, PATH_MAX, "%s/gpu.lock", TEMP);
    int fdlock=lock_file(fnlock, 1, 0);
    /*
      Create a mapping between CUDA device ordering and NVML ordering (nvidia-smi)
    */
    long gmap[ngpu_tot][2];
    for(int ig=0; ig<ngpu_tot; ig++){
	gmap[ig][0]=ig;
	hipDeviceProp_t properties;
	if(!hipSetDevice(ig) && !hipGetDeviceProperties(&properties, ig)){
	    gmap[ig][1]=properties.pciBusID;
	}else{
	    error("Error getting information for GPU %d\n", ig);
	}
    }
    qsort(gmap, ngpu_tot, sizeof(long)*2, (int(*)(const void*, const void *))cmp_long2_ascend);
    
    NGPU=0;
    /*
      User specified exact GPUs to use. We check every entry. 
      If <0 is found, do not use any GPU.
      If >=ngpu_tot is found, skip the GPU and print warning.
      If duplicates are found, use only once.
     */
    if(gpus && ngpu>0){
	if(!GPUS) GPUS=(int*)malloc(ngpu*sizeof(int));
	for(int ig=0; ig<ngpu; ig++){
	    if(gpus[ig]<0){
		info2("CUDA is disabled by user.\n");
		free(GPUS); GPUS=NULL; 
		NGPU=0;
		goto end;
	    }else{
		if(gpus[ig]>=ngpu_tot){
		    warning("GPU %d: not exist\n", gpus[ig]);
		}else{
		    GPUS[NGPU++]=gmap[gpus[ig]][0];
		    /* Enable the following to disallow use GPUs in multiple threads
		      int j;
		    for(j=0; j<NGPU; j++){
			if(GPUS[j]==gpus[ig]){
			    warning2("Skip GPU %d: duplicated\n", gpus[ig]);
			    break;
			}
		    }
		    if(j==NGPU){
			GPUS[NGPU++]=gpus[ig];
			}*/
		}
	    }
	}
    }else{
	int repeat=0;
	if(ngpu<=0){
	    repeat=0;
	    ngpu=ngpu_tot;
	}
	GPUS=(int*)calloc(ngpu, sizeof(int));
	register_deinit(NULL, GPUS);
	/*For each GPU, query the available memory.*/
	long (*gpu_info)[2]=(long(*)[2])calloc(2*ngpu_tot, sizeof(long));
	int gpu_valid_count;
	do{
	    for(int jg=0; jg<ngpu_tot; jg++){
		int ig=gmap[jg][0];
		gpu_info[ig][0]=ig;
		if(!hipSetDevice(ig)){
		    //this allocates context and create a CPU thread for this GPU.
		    gpu_info[ig][1]=gpu_get_idle_mem();
		}
	    }
	    /*sort so that gpus with higest memory is in the front.*/
	    qsort(gpu_info, ngpu_tot, sizeof(long)*2, (int(*)(const void*, const void *))cmp_long2_descend);
	    gpu_valid_count=0;
	    for(int ig=0; ig<ngpu_tot; ig++){
		if(gpu_info[ig][1]>=MEM_RESERVE){
		    gpu_valid_count++;
		}
		info2("GPU %d has mem %.1f GB\n", (int)gpu_info[ig][0], gpu_info[ig][1]/1024/1024/1024.);
	    }
	}while(0);//while(gpu_valid_count<ngpu && gpu_valid_count<ngpu_tot && sleep(60));

	for(int i=0, ig=0; i<ngpu; i++, ig++){
	    if(ig==ngpu_tot || gpu_info[ig][1]<MEM_RESERVE){
		if(repeat){
		    ig=0; //reset to beginning.
		}else{
		    break; //stop
		}
	    }
	    GPUS[NGPU++]=(int)gpu_info[ig][0];
	}
	free(gpu_info);
    }
    if(NGPU) {
	gpu_recon=0;/*first gpu in GPUS*/
	cudata_all=new cudata_t[NGPU];
	register_deinit(NULL, cudata_all);
	info2("Using GPU");
	for(int i=0; GPUS && i<NGPU; i++){
	    info2(" %d", GPUS[i]);
	    gpu_set(i);
	    //Reserve memory in GPU so the next maos will not pick this GPU.
	    DO(hipMalloc(&cudata->reserve, MEM_RESERVE));
	}
	info2("\n");
    }
 end:
    close(fdlock);
    return NGPU;
}

/**
   Clean up device.
*/
void gpu_cleanup(void){
    for(int ig=0; ig<NGPU; ig++){
	hipSetDevice(GPUS[ig]);
	hipDeviceReset();
    }
}
