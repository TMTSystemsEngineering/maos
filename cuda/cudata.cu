#include "hip/hip_runtime.h"
/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "cudata.h"
int nstream=0;
#define MEM_RESERVE 200000000
int NGPU=0;
int* GPUS=NULL;
cudata_t *cudata_all=NULL;/*for all GPU. */

#ifdef __APPLE__
pthread_key_t cudata_key;
static __attribute((constructor)) void init(){
    pthread_key_create(&cudata_key, NULL);
}
#else
__thread cudata_t *cudata=NULL;/*for current thread and current GPU */
#endif
int cudata_t::recongpu=0;
int *cudata_t::evlgpu=0;
int *cudata_t::wfsgpu=0;
cuwfs_t *cudata_t::wfs=0;
/**
   Get GPU info.
*/
void gpu_info(){
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    info("name=%s\n"
	 "TotalGlobalMem=%d\n"
	 "SharedMemPerBlock=%d\n"
	 "regsPerBlock=%d\n"
	 "warpSize=%d",
	 prop.name,
	 (int)prop.totalGlobalMem,
	 (int)prop.sharedMemPerBlock,
	 prop.regsPerBlock,
	 prop.warpSize);
}
/**
   Print memory consumption.
*/
void gpu_print_mem(const char *msg){
    size_t fr, tot;
    hipDeviceSynchronize();
    DO(hipMemGetInfo(&fr, &tot));
    info2("GPU (%d) mem used %ld MB (%s)\n",cudata->igpu,(long)(tot-fr)/1024/1024, msg);
}
/**
   Get available memory.
*/
long gpu_get_mem(void){
    size_t fr, tot;
    DO(hipMemGetInfo(&fr, &tot));
    return (long)fr;
}
/**
   Get available memory.
*/
static long gpu_get_free_mem(int igpu){
    size_t fr=0, tot=0;
    int ans;
    if((ans=hipMemGetInfo(&fr, &tot))){
	warning2("hipMemGetInfo failed with error %d\n", ans);
    }
    info2("GPU%2d has %.1fGB free, %.1fGB total device memory.\n", 
	  igpu, fr*9.3e-10, tot*9.3e-10);
    return (long)fr;
}
static int cmp_long2_descend(const long *a, const long *b){
    if(b[1]>a[1]){
	return 1;
    }else if(a[1]>b[1]){
	return -1;
    }else{
	return 0;
    }
}
static int cmp_long2_ascend(const long *a, const long *b){
    if(b[1]<a[1]){
	return 1;
    }else if(a[1]<b[1]){
	return -1;
    }else{
	return 0;
    }
}
struct task_t{
    float timing;/*based on empirical data*/
    int *dest;
};
static int task_cmp(const task_t *a, const task_t *b){
    if(b->timing > a->timing){
	return 1;
    }else if(b->timing < a->timing){
	return -1;
    }else{
	return 0;
    }
}
/**
   Initialize GPU. Return 1 if success.
   if gpus is not null, it is of length ngpu. gpus specifies gpu index to use.
   if gpus is null, ngpu specifies number of gpus to use. all if 0.

   when mix Tesla and GTX cards, the ordering the GPUs may be different in CUDA
   and NVML, causing the selection of GPUs to fail. Do not use NVML 
*/
int gpu_init(int *gpus, int ngpu, const PARMS_T *parms){
    if(gpus && ngpu>0){
	for(int ig=0; ig<ngpu; ig++){
	    if(gpus[ig]<0){
		info2("CUDA is disabled by user.\n");
		return 0;
	    }
	}
    }
    int ans, ngpu_tot=0;//total number of GPUs.
    if((ans=hipGetDeviceCount(&ngpu_tot)) || ngpu_tot==0){//no GPUs available.
	info2("No GPUs available. ans=%d\n", ans);
	return 0;
    }
    long mem_minimum=0;
    if(parms){
	if(parms->gpu.evl || parms->gpu.wfs){
	    const int nps=parms->atm.nps;
	    long nxn=parms->atm.nxn;
	    long nyn=parms->atm.nyn;
	    mem_minimum+=sizeof(Real)*nps*nxn*nyn;
	}
	if(parms->gpu.evl){
	    mem_minimum+=sizeof(Real)*parms->evl.nevl*pow(parms->aper.d/parms->evl.dx, 2);
	}
	if(parms->gpu.wfs){
	    for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
		mem_minimum+=sizeof(Real)*parms->powfs[ipowfs].nwfs*pow(parms->aper.d/parms->powfs[ipowfs].dx, 2)*4;
	    }
	}
	if(parms->gpu.tomo || parms->gpu.fit){
	    mem_minimum+=sizeof(Real)*parms->atmr.nps*pow(parms->aper.d*parms->tomo.pos/parms->atmr.dx, 2)*4;
	}
	if(mem_minimum==0){//gpu is disabled
	    return 0;
	}else{
	    info2("CUDA: minimum memory requirement is %.1fGB\n", mem_minimum/(double)(1024*1024*1024));
	}
    }
    char fnlock[PATH_MAX];
    snprintf(fnlock, PATH_MAX, "%s/gpu.lock", TEMP);
    int fdlock=lock_file(fnlock, 1, 0);
    /*
      Create a mapping between CUDA device ordering and NVML ordering (nvidia-smi).
    */
    long gmap[ngpu_tot][2];
    for(int ig=0; ig<ngpu_tot; ig++){
	gmap[ig][0]=ig;
	hipDeviceProp_t properties;
	if(!hipSetDevice(ig) && !hipGetDeviceProperties(&properties, ig)){
	    gmap[ig][1]=properties.pciBusID;
	}else{
	    error("Error getting information for GPU %d\n", ig);
	}
    }
    qsort(gmap, ngpu_tot, sizeof(long)*2, (int(*)(const void*, const void *))cmp_long2_ascend);
    //now gmap[igpu] is the cuda index of the nvml index igpu.
    NGPU=0;
    /*
      User specified exact GPUs to use. We check every entry. 
      If <0 is found, do not use any GPU.
      If >=ngpu_tot is found, skip the GPU and print warning.
      If duplicates are found, use only once.
     */
    if(gpus && ngpu>0){
	if(!GPUS) GPUS=(int*)malloc(ngpu*sizeof(int));
	for(int ig=0; ig<ngpu; ig++){
	    if(gpus[ig]<0){
		info2("CUDA is disabled by user.\n");
		free(GPUS); GPUS=NULL; 
		NGPU=0;
		goto end;
	    }else{
		if(gpus[ig]>=ngpu_tot){
		    warning("GPU %d: not exist\n", gpus[ig]);
		}else{
		    GPUS[NGPU++]=gmap[gpus[ig]][0];
		}
	    }
	}
    }else{
	int repeat=0;
	if(ngpu<=0){
	    repeat=0;
	    ngpu=ngpu_tot;
	}
	GPUS=(int*)calloc(ngpu, sizeof(int));//stores CUDA index
	register_deinit(NULL, GPUS);
	/*For each GPU, query the available memory.*/
	long (*gpu_info)[2]=(long(*)[2])calloc(2*ngpu_tot, sizeof(long));
	int gpu_valid_count;
	do{
	    gpu_valid_count=0;
	    for(int jg=0; jg<ngpu_tot; jg++){//jg: nvml index. ig: cuda index
		int ig=gmap[jg][0];
		gpu_info[ig][0]=ig;
		if(!hipSetDevice(ig)){
		    //this allocates context and create a CPU thread for this GPU.
		    gpu_info[ig][1]=gpu_get_free_mem(ig);
		    if(gpu_info[ig][1]>=mem_minimum){
			gpu_valid_count++;
		    }
		}
	    }
	}while(0);
	if(gpu_valid_count){
	    /*sort so that gpus with higest memory is in the front.*/
	    qsort(gpu_info, ngpu_tot, sizeof(long)*2, (int(*)(const void*, const void *))cmp_long2_descend);
	    for(int i=0, ig=0; i<ngpu; i++, ig++){//ig: cuda index
		if(ig<ngpu_tot && gpu_info[ig][1]>=mem_minimum){
		    GPUS[NGPU++]=(int)gpu_info[ig][0];
		}else if(ig==ngpu_tot || gpu_info[ig][1]<mem_minimum){
		    if(NGPU && repeat){
			ig=0; //reset to beginning.
		    }else{
			break; //stop
		    }
		}
	    }
	}
	free(gpu_info);
    }
    if(NGPU) {
	cudata_all=new cudata_t[NGPU];
	register_deinit(NULL, cudata_all);
	info2("Using GPU");
	for(int i=0; GPUS && i<NGPU; i++){
	    gpu_set(i);
	    for(int j=0; j<NGPU; j++){
		if(j!=i){
		    hipDeviceEnablePeerAccess(j, 0);
		}
	    }
	    for(int j=0; j<ngpu_tot; j++){
		if(GPUS[i]==gmap[j][0]){
		    cudata->igpu=j;
		    break;
		}
	    }
	    info2(" %d", cudata->igpu);
	    //Reserve memory in GPU so the next maos will not pick this GPU.
	    DO(hipMalloc(&cudata->reserve, MEM_RESERVE));
	}
	info2("\n");
	if(parms){
	    /*Assign task to gpu evenly based on empirical data to maximum GPU
	     * usage. We first gather together all the tasks and assign a timing
	     * in ms to each. Sort all the tasks in descend order and then
	     * iteratively assign each task to the minimally used GPU*/
	    cudata_t::evlgpu=(int*)calloc(parms->evl.nevl, sizeof(int));
	    cudata_t::wfsgpu=(int*)calloc(parms->nwfs, sizeof(int));
	    int ntask=0;
	    if(parms->gpu.tomo || parms->gpu.fit) ntask++;
	    if(parms->gpu.evl) ntask+=parms->evl.nevl;
	    if(parms->gpu.wfs) ntask+=parms->nwfs;
	    if(ntask==0){
		delete [] cudata_all;
		free(GPUS);
		NGPU=0;GPUS=0;
		return NGPU;
	    }
	    struct task_t *tasks=(task_t*)calloc(ntask, sizeof(task_t));
	    //recon
	    int count=0;
	    if(parms->gpu.tomo || parms->gpu.fit){
		tasks[count].timing=20;//ms
		tasks[count].dest=&cudata_t::recongpu;
		count++;
	    }
	    //evl
	    for(int ievl=0; parms->gpu.evl && ievl<parms->evl.nevl; ievl++){
		tasks[count].timing=4.7;//ms
		tasks[count].dest=cudata_t::evlgpu+ievl;
		count++;
	    }
	    //wfs
	    for(int iwfs=0; parms->gpu.wfs && iwfs<parms->nwfs; iwfs++){
		const int ipowfs=parms->wfs[iwfs].powfs;
		tasks[count].timing=parms->powfs[ipowfs].usephy?17:1.5;
		tasks[count].dest=cudata_t::wfsgpu+iwfs;
		count++;
	    }
	    qsort(tasks, count, sizeof(task_t), (int(*)(const void*, const void *))task_cmp);
	    float timtot[NGPU];
	    for(int igpu=0; igpu<NGPU; igpu++){
		timtot[igpu]=0;
	    }
	    for(int it=0; it<count; it++){
		int min_gpu=0; /*current gpu with minimum task*/
		float min_val=INFINITY;
		for(int igpu=0; igpu<NGPU; igpu++){
		    if(timtot[igpu]<min_val){
			min_val=timtot[igpu];
			min_gpu=igpu;
		    }
		}
		*(tasks[it].dest)=min_gpu;
		timtot[min_gpu]+=tasks[it].timing;
	    }
	    if(parms->sim.nthread>NGPU && (parms->gpu.tomo || parms->gpu.fit) && parms->gpu.evl && parms->gpu.wfs){
		((PARMS_T*)parms)->sim.nthread=NGPU+1;
		NTHREAD=NGPU+1;
		info2("Reset nthread to %d\n", NTHREAD);
	    }
	    free(tasks);
	}
    }
 end:
    close(fdlock);
    return NGPU;
}

/**
   Clean up device.
*/
void gpu_cleanup(void){
    for(int ig=0; ig<NGPU; ig++){
	hipSetDevice(GPUS[ig]);
	hipDeviceReset();
    }
}
