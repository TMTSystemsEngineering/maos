#include "hip/hip_runtime.h"
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
#include "utils.h"
}
static hipChannelFormatDesc channelDesc=hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
int nstream=0;
/**
   Get GPU info.
*/
void gpu_info(){
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    info("name=%s\n"
	 "TotalGlobalMem=%d\n"
	 "SharedMemPerBlock=%d\n"
	 "regsPerBlock=%d\n"
	 "warpSize=%d",
	 prop.name,
	 (int)prop.totalGlobalMem,
	 (int)prop.sharedMemPerBlock,
	 prop.regsPerBlock,
	 prop.warpSize);
}


/**
   Copy map_t to cumap_t. if type==1, use hipArray, otherwise use float array.
*/
void map2gpu(map_t **source, int nps, cumap_t *dest, int type){
    if(nps==0) return;
    if(dest->nlayer!=0 && dest->nlayer!=nps){
	error("Mismatch. nlayer=%d, nps=%d\n", dest->nlayer, nps);
    }
    dest->nlayer=nps;
    int nx0=source[0]->nx;
    int ny0=source[0]->ny;
    if(!dest->vx){
	if(type==1){
	    hipChannelFormatDesc channelDesc=hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
	    DO(hipMalloc3DArray(&dest->ca, &channelDesc, make_hipExtent(nx0, ny0, nps), 
				 hipArrayLayered));
	}else{
	    DO(hipHostMalloc(&(dest->p), nps*sizeof(float*)));
	    //memory in device.
	    for(int ips=0; ips<nps; ips++){
		DO(hipMalloc(&(dest->p[ips]), nx0*ny0*sizeof(float)));
	    }
	}
        dest->vx=new float[nps];
	dest->vy=new float[nps]; 
	dest->ht=new float[nps];
	dest->ox=new float[nps];
	dest->oy=new float[nps];
	dest->dx=new float[nps];
	dest->nx=new int[nps];
	dest->ny=new int[nps];
    }
    
    float *tmp=NULL;
    if(type==1) tmp=new float[nx0*ny0*nps];
    
    for(int ips=0; ips<nps; ips++){
	int nx=source[ips]->nx;
	int ny=source[ips]->ny;
	if(type==1 && (nx!=nx0 || ny!=ny0)){
	    error("Only support map_t arrays of the same size if type==1\n");
	}
	dest->vx[ips]=source[ips]->vx;
	dest->vy[ips]=source[ips]->vy;
	dest->ht[ips]=source[ips]->h;
	dest->ox[ips]=source[ips]->ox;
	dest->oy[ips]=source[ips]->oy;
	dest->dx[ips]=source[ips]->dx;
	dest->nx[ips]=source[ips]->nx;
	dest->ny[ips]=source[ips]->ny;
	if(type==1){//hipArray
	    for(long ix=0; ix<(long)nx0*(long)ny0; ix++){
		tmp[ips*nx0*ny0+ix]=(float)source[ips]->p[ix];
	    }
	}else{//Flat memory
	    gpu_dbl2dev(&dest->p[ips], source[ips]->p, nx*ny);
	    /*
	      float *tmp2=new float[nx*ny];
	      for(int iloc=0; iloc<nx*ny; iloc++){
	      tmp2[iloc]=(float)source[ips]->p[iloc];
	      }
	      hipMemcpy(dest->p[ips], tmp2, nx*ny*sizeof(float), hipMemcpyHostToDevice);
	      free(tmp2);*/
	}
    }
    if(type==1){
	struct hipMemcpy3DParms par={0};
	par.srcPos = make_hipPos(0,0,0);
	par.dstPos = make_hipPos(0,0,0);
	par.dstArray=dest->ca;
	par.srcPtr = make_hipPitchedPtr(tmp, nx0*sizeof(float), nx0, ny0);
	par.extent = make_hipExtent(nx0, ny0, nps);
	par.kind   = hipMemcpyHostToDevice;
	DO(hipMemcpy3D(&par));
	delete [] tmp;
    }
}

/*
  Convert a host dsp array to GPU sprase array. Both are in CSC format. 
*/
void sp2gpu(cusp_t *dest, dsp *src){
    dest->nx=src->m;
    dest->ny=src->n;
    dest->nzmax=src->nzmax;
    dest->p=NULL; dest->i=NULL; dest->x=NULL;
#if MYSPARSE == 1
    gpu_spint2int(&dest->p, src->p, src->n+1);
    gpu_spint2dev(&dest->i, src->i, src->nzmax);
    gpu_dbl2dev(&dest->x, src->x, src->nzmax);
#else
    gpu_spint2dev(&dest->p, src->p, src->n+1);
    gpu_spint2dev(&dest->i, src->i, src->nzmax);
    gpu_dbl2dev(&dest->x, src->x, src->nzmax);
#endif
}
__global__ void cuspmul_do(float *y, cusp_t *A, float *x, float alpha){
    int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<A->ny; i+=step){
	for(int j=A->p[i]; j<A->p[i+1]; j++){
	    atomicAdd(&y[A->i[j]], A->x[j]*x[i]*alpha);
	}
    }
}
/*
  y=A*x where A is sparse. x, y are vectors. Slow for GS0.
*/
void cuspmul(float *y, cusp_t *A, float *x, float alpha, hipStream_t stream){
    cuspmul_do<<<A->nx/256, 256, 0, stream>>>(y,A,x,alpha);
}

/*
  y=A'*x where A is sparse. x, y are vectors
*/
__global__ void cusptmul_do(float *y, int icol, cusp_t *A, float *x, float alpha){
    __shared__ float val;
    if(threadIdx.x==0) val=0;
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    int j=i+A->p[icol];
    atomicAdd(&val, A->x[j]*x[A->i[j]]);
    if(threadIdx.x==0) y[icol]+=val*alpha;
}
/*
  Does not work yet. Try to launch a block for each column and n items in each block.
*/
void cusptmul(float *y, cusp_t *A, float *x, float alpha, hipStream_t stream){
    for(int i=0; i<A->ny; i++){
	cusptmul_do<<<1, A->p[i+1]-A->p[i], 0, stream>>>(y,i,A,x,alpha);
    }
}
__global__ static void calc_ptt_do( float *cc,
				    const float (*restrict loc)[2], 
				    const int nloc,
				    const float *restrict phi,
				    const float *restrict amp){
    __shared__ float ccb[4];//for each block.
    if(threadIdx.x==0){
	ccb[0]=ccb[1]=ccb[2]=ccb[3]=0.f;
    }
    float cci[4]={0.f,0.f,0.f,0.f};//for each thread
    int step=blockDim.x * gridDim.x; 
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<nloc; i+=step){
	float tmp=phi[i]*amp[i];
	cci[0]+=tmp;
	cci[1]+=tmp*loc[i][0];
	cci[2]+=tmp*loc[i][1];
	cci[3]+=tmp*phi[i];
    }
    //Add results to shared value in each block.
    atomicAdd(&ccb[0], cci[0]);
    atomicAdd(&ccb[1], cci[1]);
    atomicAdd(&ccb[2], cci[2]);
    atomicAdd(&ccb[3], cci[3]);
    __syncthreads();//Wait until all threads in this block is done.
    if(threadIdx.x==0){//This is the first thread of a block. add block result to global.
	atomicAdd(&cc[0], ccb[0]);
	atomicAdd(&cc[1], ccb[1]);
	atomicAdd(&cc[2], ccb[2]);
	atomicAdd(&cc[3], ccb[3]);
    }
}
__global__ static void calc_ngsmod_do( float *cc,
				       const float (*restrict loc)[2], 
				       const int nloc,
				       const float *restrict phi,
				       const float *restrict amp){
    int step=blockDim.x * gridDim.x; 
    float cci[7]={0,0,0,0,0,0,0};//for each thread
    __shared__ float ccb[7];//for each block.
    if(threadIdx.x==0){
	ccb[0]=ccb[1]=ccb[2]=ccb[3]=ccb[4]=ccb[5]=ccb[6]=0;
    }
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<nloc; i+=step){
	float tmp=phi[i]*amp[i];
	cci[0]+=tmp;
	cci[1]+=tmp*loc[i][0];
	cci[2]+=tmp*loc[i][1];
	cci[3]+=tmp*loc[i][0]*loc[i][0];
	cci[4]+=tmp*loc[i][1]*loc[i][1];
	cci[5]+=tmp*loc[i][0]*loc[i][1];
	cci[6]+=tmp*phi[i];
    }
    //Add results to shared value in each block.
    atomicAdd(&ccb[0], cci[0]);
    atomicAdd(&ccb[1], cci[1]);
    atomicAdd(&ccb[2], cci[2]);
    atomicAdd(&ccb[3], cci[3]);
    atomicAdd(&ccb[4], cci[4]);
    atomicAdd(&ccb[5], cci[5]);
    atomicAdd(&ccb[6], cci[6]);
    __syncthreads();//Wait until all threads in this block is done.
    if(threadIdx.x==0){//This is the first thread of a block. add block result to global.
	atomicAdd(&cc[0], ccb[0]);
	atomicAdd(&cc[1], ccb[1]);
	atomicAdd(&cc[2], ccb[2]);
	atomicAdd(&cc[3], ccb[3]);
	atomicAdd(&cc[4], ccb[4]);
	atomicAdd(&cc[5], ccb[5]);
	atomicAdd(&cc[6], ccb[6]);
    }
}
/*
  Let M be the modal matrix of pistion/tip/tilt. Calculate M'*diag(amp)*phi
  where amp is the amptliude weighting.  */
void gpu_calc_ptt(double *rmsout, double *coeffout, 
		  const double ipcc, const dmat *imcc,
		  const float (*restrict loc)[2], 
		  const int nloc,
		  const float *restrict phi,
		  const float *restrict amp,
		  hipStream_t stream
		  ){
    //sum with 16 blocks, each with 256 threads.
    float *cc;
    cudaCallocHost(cc, 4*sizeof(float));
    calc_ptt_do<<<16, 256, 0, stream>>>(cc, loc, nloc, phi, amp);
    CUDA_SYNC_STREAM;
    double coeff[3], tot;
    coeff[0]=cc[0]; coeff[1]=cc[1]; coeff[2]=cc[2]; tot=cc[3];
    hipFree(cc);
    if(coeffout){
	dmulvec3(coeffout, imcc, coeff);
    }
    if(rmsout){
	double pis=ipcc*coeff[0]*coeff[0];//piston mode variance
	double ptt=dwdot3(coeff, imcc, coeff);//p/t/t mode variance.
	rmsout[0]=tot-pis;//PR
	rmsout[1]=ptt-pis;//TT
	rmsout[2]=tot-ptt;//PTTR	
    }
}
void gpu_calc_ngsmod(double *pttr_out, double *pttrcoeff_out,
		     double *ngsmod_out, int nmod,
		     double MCC_fcp, double ht, double scale,
		     double thetax, double thetay,
		     const double ipcc, const dmat *imcc,
		     const float (*restrict loc)[2], 
		     const int nloc,
		     const float *restrict phi,
		     const float *restrict amp,
		     hipStream_t stream){
    float *cc;
    double tot=0;
    cudaCallocHost(cc, 7*sizeof(float));
    cc[0]=cc[1]=cc[2]=cc[3]=cc[4]=cc[5]=cc[6]=0;
    if(nmod==2){//single DM.
	calc_ptt_do<<<16,256,0,stream>>>(cc, loc, nloc, phi, amp);
    }else if(nmod==5){//AHST mode
	calc_ngsmod_do<<<16,256,0,stream>>>(cc, loc, nloc, phi, amp);
    }else{
	TO_IMPLEMENT;
    }
    CUDA_SYNC_STREAM;
    tot=cc[nmod==2?3:6];

    double coeff[6];
    coeff[0]=cc[0]; coeff[1]=cc[1]; 
    coeff[2]=cc[2]; coeff[3]=cc[3];
    coeff[4]=cc[4]; coeff[5]=cc[5];
    
    hipFree(cc); 
    if(pttrcoeff_out){
	memset(pttrcoeff_out, 0, sizeof(double)*3);
	dmulvec(pttrcoeff_out, imcc, coeff, 1);
    }
    if(pttr_out){
	//compute TT removed wavefront variance as a side product
	double pis=ipcc*coeff[0]*coeff[0];
	double ptt=dwdot3(coeff, imcc, coeff);
	pttr_out[0]=tot-pis;//PR
	pttr_out[1]=ptt-pis;//TT
	pttr_out[2]=tot-ptt;//PTTR
    }
    //don't use +=. need locking
    ngsmod_out[0]=coeff[1];
    ngsmod_out[1]=coeff[2];
    const double scale1=1.-scale;
    if(nmod==5){
	ngsmod_out[2]=(scale1*(coeff[3]+coeff[4]-coeff[0]*MCC_fcp)
		       -2*scale*ht*(thetax*coeff[1]+thetay*coeff[2]));
	ngsmod_out[3]=(scale1*(coeff[3]-coeff[4])
		       -2*scale*ht*(thetax*coeff[1]-thetay*coeff[2]));
	ngsmod_out[4]=(scale1*(coeff[5])
		       -scale*ht*(thetay*coeff[1]+thetax*coeff[2]));
    }
}
/**
   Convert a source loc_t to device memory.
*/
void gpu_loc2dev(float (* restrict *dest)[2], loc_t *src){
    float (*tmp)[2]=(float(*)[2])malloc(src->nloc*2*sizeof(float));
    for(int iloc=0; iloc<src->nloc; iloc++){
	tmp[iloc][0]=(float)src->locx[iloc];
	tmp[iloc][1]=(float)src->locy[iloc];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, src->nloc*2*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, src->nloc*2*sizeof(float),hipMemcpyDefault));
    free(tmp);
}
/**
   Convert double array to device memory (float)
*/
void gpu_dbl2dev(float * restrict *dest, double *src, int n){
    float *tmp=(float*)malloc(n*sizeof(float));
    for(int i=0; i<n; i++){
	tmp[i]=(float)src[i];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, n*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, n*sizeof(float),hipMemcpyDefault));
    free(tmp);
}
/**
   Convert double array to device memory (float)
*/
void gpu_dbl2flt(float * restrict *dest, double *src, int n){
    if(!*dest){
	hipHostMalloc((float**)dest, n*sizeof(float));
    }
    for(int i=0; i<n; i++){
	(*dest)[i]=(float)src[i];
    }
}
/**
   Convert long array to device int
*/
void gpu_long2dev(int * restrict *dest, long *src, int n){
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(long)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyDefault));
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((long)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyDefault));
	free(tmp);
    }
}
/**
   Convert long array to device int
*/
void gpu_spint2dev(int * restrict *dest, spint *src, int n){
    info("sizeof(spint)=%ld\n", sizeof(spint));
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(spint)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyDefault));
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((spint)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyDefault));
	free(tmp);
    }
}
/**
   Convert long array to device int
*/
void gpu_spint2int(int * restrict *dest, spint *src, int n){
    info("sizeof(spint)=%ld\n", sizeof(spint));

    if(!*dest){
	DO(hipHostMalloc((int**)dest, n*sizeof(int)));
    }

    for(int i=0; i<n; i++){
	(*dest)[i]=(int)src[i];
    }
}
/**
   Convert device (float) array to host double.
*/
void gpu_dev2dbl(double * restrict *dest, float *src, int n){
    TIC;tic;
    float *tmp=(float*)malloc(n*sizeof(float));
    DO(hipMemcpy(tmp, src, n*sizeof(float), hipMemcpyDefault));
    if(!*dest){
	*dest=(double*)malloc(sizeof(double)*n);
    }
    double *restrict p=*dest;
    for(int i=0; i<n; i++){
	p[i]=tmp[i];
    }
    free(tmp);
}
/**
   scale vector by alpha.
*/
__global__ void fscale_do(float *v, int n, float alpha){
    int step=blockDim.x * gridDim.x; 
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	v[i]*=alpha;
    }
}

/*
  Write float on gpu to file
*/
void gpu_writeflt(float *p, int nx, int ny, const char *format, ...){
    format2fn;
    float *tmp=(float*)malloc(nx*ny*sizeof(float));
    hipMemcpy(tmp, p, nx*ny*sizeof(float), hipMemcpyDefault);
    writeflt(tmp,nx,ny,"%s",fn);
    free(tmp);
}
/*
  Write float on gpu to file
*/
void gpu_writeint(int *p, int nx, int ny, const char *format, ...){
    format2fn;
    int *tmp=(int*)malloc(nx*ny*sizeof(int));
    hipMemcpy(tmp, p, nx*ny*sizeof(int), hipMemcpyDefault);
    writeint(tmp,nx,ny,"%s",fn);
    free(tmp);
}
