#include "hip/hip_runtime.h"
#include "utils.h"
#include "curmat.h"
#include <pthread.h>
static hipChannelFormatDesc channelDesc=hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
pthread_mutex_t cufft_mutex=PTHREAD_MUTEX_INITIALIZER;
extern hipsparseMatDescr_t cuspdesc;

int nstream=0;
int NG1D=64; /**<Optimum number of blocks. Twice of multi processors.*/
int NG2D=8; /**<Optimum number of blocks. Twice of multi processors.*/

/**
   Get GPU info.
*/
void gpu_info(){
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    info("name=%s\n"
	 "TotalGlobalMem=%d\n"
	 "SharedMemPerBlock=%d\n"
	 "regsPerBlock=%d\n"
	 "warpSize=%d",
	 prop.name,
	 (int)prop.totalGlobalMem,
	 (int)prop.sharedMemPerBlock,
	 prop.regsPerBlock,
	 prop.warpSize);
}
/**
   Print memory consumption.
*/
void gpu_print_mem(const char *msg){
    size_t fr, tot;
    DO(hipMemGetInfo(&fr, &tot));
    info2("GPU mem used %5lu MiB (%s)\n",(tot-fr)/1024/1024, msg);
}
/**
   Get available memory.
*/
size_t gpu_get_mem(void){
    size_t fr, tot;
    DO(hipMemGetInfo(&fr, &tot));
    return fr;
}
/**
   Initialize GPU. Return 1 if success.
 */
int gpu_init(int igpu){
    if(igpu<0) return 0;
    if(igpu==INT_MAX){//automatic
	int ngpu=0;
	DO(hipGetDeviceCount(&ngpu));
	switch(ngpu){
	case 0:
	    warning2("No GPU is available for computing\n");
	    igpu=-1;
	    break;
	case 1:{//1 device. check whethere it is emulation.
	    hipDeviceProp_t prop;
	    hipGetDeviceProperties(&prop, 0);
	    if(prop.major==9999){
		warning("The only device is in emulation mode. Won't use it\n");
		igpu=-1;
	    }else{
		igpu=0;
	    }
	}
	    break;
	default:{//There are multiple devices.
	    hipDeviceProp_t prop;
	    prop.canMapHostMemory=1;
	    prop.major=2;
	    int ans=hipChooseDevice(&igpu, &prop);
	    if(ans==hipErrorInvalidValue){
		warning("hipChooseDevice Failed. Will not use GPU computing.\n");
		igpu=-1;
	    }
	}
	    break;
	}
    }
    if(igpu>-1){
	int ans=hipSetDevice(igpu);
	switch(ans){
	case hipSuccess:{
	    hipDeviceProp_t prop;
	    hipGetDeviceProperties(&prop, igpu);
	    NG1D=prop.multiProcessorCount*2;
	    NG2D=(int)round(sqrt((double)NG1D));
	    if(0){
		int nn=16777216;
		//int nn=1024;
		curmat *temp=curnew(nn,1);
		curset(temp,1,0);
		curmat *res=curnew(1,1);
		TIC;tic;
		cursum2(res->p, temp, 0);
		hipStreamSynchronize(0);
		toc("inn %d", nn);
		curshow(res, 0);
		CUDA_SYNC_DEVICE;
		hipStreamSynchronize(0);
		exit(0);
	    }
	}return 1;
	case hipErrorInvalidDevice:
	    error2("Invalid GPU device %d\n", igpu);
	    _exit(1);
	    break;
	case hipErrorSetOnActiveProcess:
	    warning2("Error set on active process\n");
	    break;
	default:
	    error2("Unknown error\n");
	    _exit(1);
	}
    }
    gpu_print_mem("gpu init");
    return 0;
}
/**
   Clean up device.
*/
void gpu_cleanup(void){
    hipDeviceReset();
}
/**
   Copy map_t to cumap_t. if type==1, use hipArray, otherwise use float
array. Allow multiple calling to override the data.  */
void gpu_map2dev(cumap_t **dest0, map_t **source, int nps, int type){
    if(nps==0) return;
    if(!*dest0){
	*dest0=(cumap_t*)calloc(1, sizeof(cumap_t));
    }else if((*dest0)->nlayer!=nps){
	error("Mismatch. nlayer=%d, nps=%d\n", (*dest0)->nlayer, nps);
    }
    cumap_t *dest=*dest0;
    dest->nlayer=nps;
    int nx0=source[0]->nx;
    int ny0=source[0]->ny;
    if(!dest->vx){//data is not initialized.
	if(type==1){//all layers must be same size.
	    hipChannelFormatDesc channelDesc=hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
	    DO(hipMalloc3DArray(&dest->ca, &channelDesc, make_hipExtent(nx0, ny0, nps), 
				 hipArrayLayered));
	}else{
	    DO(hipHostMalloc(&(dest->p), nps*sizeof(float*)));
	    //memory in device.
	    for(int ips=0; ips<nps; ips++){
		DO(hipMalloc(&(dest->p[ips]), source[ips]->nx*source[ips]->ny*sizeof(float)));
	    }
	}
        dest->vx=new float[nps];
	dest->vy=new float[nps]; 
	dest->ht=new float[nps];
	dest->ox=new float[nps];
	dest->oy=new float[nps];
	dest->dx=new float[nps];
	dest->nx=new int[nps];
	dest->ny=new int[nps];
	for(int ips=0; ips<nps; ips++){
	    if(type==1 && source[ips]->nx!=nx0 && source[ips]->ny!=ny0){
		error("Only support map_t arrays of the same size if type==1\n");
	    }
	    dest->nx[ips]=source[ips]->nx;
	    dest->ny[ips]=source[ips]->ny;
	}
    }
    
    float *tmp=NULL;
    if(type==1) tmp=new float[nx0*ny0*nps];
    
    for(int ips=0; ips<nps; ips++){
	int nx=source[ips]->nx;
	int ny=source[ips]->ny;
	dest->vx[ips]=source[ips]->vx;
	dest->vy[ips]=source[ips]->vy;
	dest->ht[ips]=source[ips]->h;
	dest->ox[ips]=source[ips]->ox;
	dest->oy[ips]=source[ips]->oy;
	dest->dx[ips]=source[ips]->dx;

	if(type==1){//hipArray
	    for(long ix=0; ix<(long)nx0*(long)ny0; ix++){
		tmp[ips*nx0*ny0+ix]=(float)source[ips]->p[ix];
	    }
	}else{//Flat memory
	    gpu_dbl2dev(&dest->p[ips], source[ips]->p, nx*ny);
	}
    }
    if(type==1){
	struct hipMemcpy3DParms par={0};
	par.srcPos = make_hipPos(0,0,0);
	par.dstPos = make_hipPos(0,0,0);
	par.dstArray=dest->ca;
	par.srcPtr = make_hipPitchedPtr(tmp, nx0*sizeof(float), nx0, ny0);
	par.extent = make_hipExtent(nx0, ny0, nps);
	par.kind   = hipMemcpyHostToDevice;
	DO(hipMemcpy3D(&par));
	delete [] tmp;
    }
    CUDA_SYNC_DEVICE;
}

/*
  Convert a host dsp array to GPU sprase array. Both are in CSC format. 
*/
void gpu_sp2dev(cusp **dest0, dsp *src){
    if(!*dest0) *dest0=(cusp*)calloc(1, sizeof(cusp));
    cusp *dest=*dest0;
    dest->nx=src->m;
    dest->ny=src->n;
    dest->nzmax=src->nzmax;
    dest->p=NULL; dest->i=NULL; dest->x=NULL;
#if MYSPARSE == 1
    gpu_spint2int(&dest->p, src->p, src->n+1);
    gpu_spint2dev(&dest->i, src->i, src->nzmax);
    gpu_dbl2dev(&dest->x, src->x, src->nzmax);
#else
    gpu_spint2dev(&dest->p, src->p, src->n+1);
    gpu_spint2dev(&dest->i, src->i, src->nzmax);
    gpu_dbl2dev(&dest->x, src->x, src->nzmax);
#endif
}
void gpu_spcell2dev(cuspcell **dest0, spcell *src){
    if(!*dest0){
	*dest0=cuspcellnew(src->nx, src->ny);
    }
    for(int i=0; i<src->nx*src->ny; i++){
	gpu_sp2dev(&(*dest0)->p[i], src->p[i]);
    }
}
__global__ void cuspmul_do(float *y, cusp *A, float *x, float alpha){
    int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<A->ny; i+=step){
	for(int j=A->p[i]; j<A->p[i+1]; j++){
	    atomicAdd(&y[A->i[j]], A->x[j]*x[i]*alpha);
	}
    }
}
/*
  y=A*x where A is sparse. x, y are vectors. Slow for GS0.
*/
void cuspmul(float *y, cusp *A, float *x, float alpha, 
#if MYSPARSE ==1
	     hipStream_t stream
#else
	     hipsparseHandle_t handle
#endif
	     ){
#if MYSPARSE ==1
    cuspmul_do<<<A->nx/256, 256, 0, stream>>>(y,A,x,alpha);
#else
    int status=hipsparseScsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, 
			      A->ny, A->nx, alpha, cuspdesc,
			      A->x, A->p, A->i, x, 1.f, y);
    if(status!=0){
	error("hipsparseScsrmv failed with status %d\n", status);
    }
#endif
}

/*
  y=A'*x where A is sparse. x, y are vectors
*/
__global__ void cusptmul_do(float *y, int icol, cusp *A, float *x, float alpha){
    __shared__ float val;
    if(threadIdx.x==0) val=0;
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    int j=i+A->p[icol];
    atomicAdd(&val, A->x[j]*x[A->i[j]]);
    if(threadIdx.x==0) y[icol]+=val*alpha;
}
/*
  Does not work right yet. Try to launch a block for each column and n items in each block.
*/
void cusptmul(float *y, cusp *A, float *x, float alpha, 
#if MYSPARSE ==1
	     hipStream_t stream
#else
	     hipsparseHandle_t handle
#endif
	      ){
#if MYSPARSE == 1
    for(int i=0; i<A->ny; i++){
	cusptmul_do<<<1, A->p[i+1]-A->p[i], 0, stream>>>(y,i,A,x,alpha);
    }
    warning("Not working correctly yet\n");
#else
    int status=hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			      A->ny, A->nx, alpha, cuspdesc,
			      A->x, A->p, A->i, x, 1.f, y);
    if(status!=0){
	error("hipsparseScsrmv failed with status %d\n", status);
    }
#endif
}
__global__ static void calc_ptt_do( float *cc,
				    const float (*restrict loc)[2], 
				    const int nloc,
				    const float *restrict phi,
				    const float *restrict amp){
    __shared__ float ccb[4];//for each block.
    if(threadIdx.x<4){
	ccb[threadIdx.x]=0.f;
    }
    __syncthreads();
    float cci[4]={0.f,0.f,0.f,0.f};//for each thread
    int step=blockDim.x * gridDim.x; 
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<nloc; i+=step){
	float tmp=phi[i]*amp[i];
	cci[0]+=tmp;
	cci[1]+=tmp*loc[i][0];
	cci[2]+=tmp*loc[i][1];
	cci[3]+=tmp*phi[i];
    }
    //Add results to shared value in each block.
    atomicAdd(&ccb[0], cci[0]);
    atomicAdd(&ccb[1], cci[1]);
    atomicAdd(&ccb[2], cci[2]);
    atomicAdd(&ccb[3], cci[3]);
    __syncthreads();//Wait until all threads in this block is done.
    if(threadIdx.x<4){//This is the first thread of a block. add block result to global.
	atomicAdd(&cc[threadIdx.x], ccb[threadIdx.x]);
    }
}
__global__ static void calc_ngsmod_do( float *cc,
				       const float (*restrict loc)[2], 
				       const int nloc,
				       const float *restrict phi,
				       const float *restrict amp){
    int step=blockDim.x * gridDim.x; 
    float cci[7]={0,0,0,0,0,0,0};//for each thread
    __shared__ float ccb[7];//for each block.
    if(threadIdx.x<7){
	ccb[threadIdx.x]=0.f;
    }
    __syncthreads();
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<nloc; i+=step){
	float tmp=phi[i]*amp[i];
	cci[0]+=tmp;
	cci[1]+=tmp*loc[i][0];
	cci[2]+=tmp*loc[i][1];
	cci[3]+=tmp*loc[i][0]*loc[i][0];
	cci[4]+=tmp*loc[i][1]*loc[i][1];
	cci[5]+=tmp*loc[i][0]*loc[i][1];
	cci[6]+=tmp*phi[i];
    }
    //Add results to shared value in each block.
    atomicAdd(&ccb[0], cci[0]);
    atomicAdd(&ccb[1], cci[1]);
    atomicAdd(&ccb[2], cci[2]);
    atomicAdd(&ccb[3], cci[3]);
    atomicAdd(&ccb[4], cci[4]);
    atomicAdd(&ccb[5], cci[5]);
    atomicAdd(&ccb[6], cci[6]);
    __syncthreads();//Wait until all threads in this block is done.
    if(threadIdx.x<7){//This is the first thread of a block. add block result to global.
	atomicAdd(&cc[threadIdx.x], ccb[threadIdx.x]);
    }
}
/*
  Let M be the modal matrix of pistion/tip/tilt. Calculate M'*diag(amp)*phi
  where amp is the amptliude weighting.  */
void gpu_calc_ptt(double *rmsout, double *coeffout, 
		  const double ipcc, const dmat *imcc,
		  const float (*restrict loc)[2], 
		  const int nloc,
		  const float *restrict phi,
		  const float *restrict amp,
		  hipStream_t stream
		  ){
        //sum with 16 blocks, each with 256 threads.
    float *cc;
    cudaCalloc(cc, 4*sizeof(float), stream);
    calc_ptt_do<<<16, 256, 0, stream>>>(cc, loc, nloc, phi, amp);
    CUDA_SYNC_STREAM;
    float ccb[4];
    hipMemcpy(ccb, cc, 4*sizeof(float), hipMemcpyDefault);
    hipFree(cc); cc=NULL;
    double coeff[3], tot;
    coeff[0]=ccb[0]; coeff[1]=ccb[1]; coeff[2]=ccb[2]; tot=ccb[3];
    if(coeffout){
	dmulvec3(coeffout, imcc, coeff);
    }
    if(rmsout){
	double pis=ipcc*coeff[0]*coeff[0];//piston mode variance
	double ptt=dwdot3(coeff, imcc, coeff);//p/t/t mode variance.
	rmsout[0]=tot-pis;//PR
	rmsout[1]=ptt-pis;//TT
	rmsout[2]=tot-ptt;//PTTR	
    }
}
void gpu_calc_ngsmod(double *pttr_out, double *pttrcoeff_out,
		     double *ngsmod_out, int nmod,
		     double MCC_fcp, double ht, double scale,
		     double thetax, double thetay,
		     const double ipcc, const dmat *imcc,
		     const float (*restrict loc)[2], 
		     const int nloc,
		     const float *restrict phi,
		     const float *restrict amp,
		     hipStream_t stream){
    float *cc;
    double tot=0;
    cudaCalloc(cc, 7*sizeof(float), stream);
    if(nmod==2){//single DM.
	calc_ptt_do<<<16,256,0,stream>>>(cc, loc, nloc, phi, amp);
    }else if(nmod==5){//AHST mode
	calc_ngsmod_do<<<16,256,0,stream>>>(cc, loc, nloc, phi, amp);
    }else{
	TO_IMPLEMENT;
    }
    CUDA_SYNC_STREAM;
    float ccb[7];
    hipMemcpy(ccb, cc, 7*sizeof(float), hipMemcpyDefault);
    hipFree(cc); 
    tot=ccb[nmod==2?3:6];
    
    double coeff[6];
    coeff[0]=ccb[0]; coeff[1]=ccb[1]; 
    coeff[2]=ccb[2]; coeff[3]=ccb[3];
    coeff[4]=ccb[4]; coeff[5]=ccb[5];
    
    if(pttrcoeff_out){
	memset(pttrcoeff_out, 0, sizeof(double)*3);
	dmulvec(pttrcoeff_out, imcc, coeff, 1);
    }
    if(pttr_out){
	//compute TT removed wavefront variance as a side product
	double pis=ipcc*coeff[0]*coeff[0];
	double ptt=dwdot3(coeff, imcc, coeff);
	pttr_out[0]=tot-pis;//PR
	pttr_out[1]=ptt-pis;//TT
	pttr_out[2]=tot-ptt;//PTTR
    }
    //don't use +=. need locking
    ngsmod_out[0]=coeff[1];
    ngsmod_out[1]=coeff[2];
    const double scale1=1.-scale;
    if(nmod==5){
	ngsmod_out[2]=(scale1*(coeff[3]+coeff[4]-coeff[0]*MCC_fcp)
		       -2*scale*ht*(thetax*coeff[1]+thetay*coeff[2]));
	ngsmod_out[3]=(scale1*(coeff[3]-coeff[4])
		       -2*scale*ht*(thetax*coeff[1]-thetay*coeff[2]));
	ngsmod_out[4]=(scale1*(coeff[5])
		       -scale*ht*(thetay*coeff[1]+thetax*coeff[2]));
    }
}
/**
   Convert a source loc_t to device memory.
*/
void gpu_loc2dev(float (* restrict *dest)[2], loc_t *src){
    float (*tmp)[2]=(float(*)[2])malloc(src->nloc*2*sizeof(float));
    for(int iloc=0; iloc<src->nloc; iloc++){
	tmp[iloc][0]=(float)src->locx[iloc];
	tmp[iloc][1]=(float)src->locy[iloc];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, src->nloc*2*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, src->nloc*2*sizeof(float),hipMemcpyDefault));
    hipDeviceSynchronize();
    free(tmp);
}
/**
   Convert double array to device memory (float)
*/
void gpu_dbl2dev(float * restrict *dest, double *src, int n){
    if(!src) return;
    float *tmp=(float*)malloc(n*sizeof(float));
    for(int i=0; i<n; i++){
	tmp[i]=(float)src[i];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, n*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, n*sizeof(float),hipMemcpyDefault));
    hipDeviceSynchronize();
    free(tmp);
}
/**
   Convert double array to device memory (float)
*/
void gpu_cmp2dev(fcomplex * restrict *dest, dcomplex *src, int n){
    if(!src) return;
    fcomplex *tmp=(fcomplex*)malloc(n*sizeof(fcomplex));
    for(int i=0; i<n; i++){
	tmp[i]=(make_hipFloatComplex)(hipCreal(src[i]), hipCimag(src[i]));
    }
    if(!*dest){
	DO(hipMalloc((fcomplex**)dest, n*sizeof(fcomplex)));
    }
    DO(hipMemcpy(*dest, tmp, n*sizeof(fcomplex),hipMemcpyDefault));
    hipDeviceSynchronize();
    free(tmp);
}
/**
   Convert dmat array to device memory.
*/
void gpu_dmat2dev(float * restrict *dest, dmat *src){
    if(!src) return;
    gpu_dbl2dev(dest, src->p, src->nx*src->ny);
}
/**
   Convert dmat array to curmat
*/
void gpu_dmat2cu(curmat *restrict *dest, dmat *src){
    if(!src){
	dzero(*dest);
	return;
    }
    if(!*dest){
	*dest=curnew(src->nx, src->ny);
    }
    gpu_dbl2dev(&(*dest)->p, src->p, src->nx*src->ny);
}
/**
   Convert dcell to curcell
*/
void gpu_dcell2cu(curcell *restrict *dest, dcell *src){
    if(!src) {
	dzero(*dest);
	return;
    }
    if(!*dest) {
	*dest=curcellnew(src->nx, src->ny);
    }else if((*dest)->nx!=src->nx || (*dest)->ny!=src->ny){
	error("Mismatch: %dx%d vs %ldx%ld\n", 
	      (*dest)->nx, (*dest)->ny, src->nx, src->ny);
    }
    for(int i=0; i<src->nx*src->ny; i++){
	gpu_dmat2cu(&(*dest)->p[i], src->p[i]);
    }
}
/**
   Convert dmat array to device memory.
*/
void gpu_cmat2dev(fcomplex * restrict *dest, cmat *src){
    if(src){
	gpu_cmp2dev(dest, (dcomplex*)src->p, src->nx*src->ny);
    }
}
/**
   Convert double array to device memory (float)
*/
void gpu_dbl2flt(float * restrict *dest, double *src, int n){
    if(!src) return;
    if(!*dest){
	hipHostMalloc((float**)dest, n*sizeof(float));
    }
    for(int i=0; i<n; i++){
	(*dest)[i]=(float)src[i];
    }
}
/**
   Convert long array to device int
*/
void gpu_long2dev(int * restrict *dest, long *src, int n){
    if(!src) return;
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(long)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyDefault));
	hipDeviceSynchronize();
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((long)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyDefault));
	hipDeviceSynchronize();
	free(tmp);
    }
}
void gpu_int2dev(int *restrict *dest, int *src, int n){
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    DO(hipMemcpy(*dest, src, sizeof(int)*n, hipMemcpyDefault));
}
/**
   Convert long array to device int
*/
void gpu_spint2dev(int * restrict *dest, spint *src, int n){
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(spint)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyDefault));
	hipDeviceSynchronize();
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((spint)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyDefault));
	hipDeviceSynchronize();
	free(tmp);
    }
}
/**
   Convert long array to device int
*/
void gpu_spint2int(int * restrict *dest, spint *src, int n){
    info("sizeof(spint)=%ld\n", sizeof(spint));

    if(!*dest){
	DO(hipHostMalloc((int**)dest, n*sizeof(int)));
    }

    for(int i=0; i<n; i++){
	(*dest)[i]=(int)src[i];
    }
}
/**
   Convert device (float) array to host double.
*/
void gpu_dev2dbl(double * restrict *dest, float *src, int n, hipStream_t stream){
    TIC;tic;
    float *tmp=(float*)malloc(n*sizeof(float));
    DO(hipMemcpyAsync(tmp, src, n*sizeof(float), hipMemcpyDeviceToHost, stream));
    CUDA_SYNC_STREAM;
    if(!*dest){
	*dest=(double*)malloc(sizeof(double)*n);
    }
    double *restrict p=*dest;
    for(int i=0; i<n; i++){
	p[i]=tmp[i];
    }
    free(tmp);
}
/**
   scale vector by alpha.
*/
__global__ void fscale_do(float *v, int n, float alpha){
    int step=blockDim.x * gridDim.x; 
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	v[i]*=alpha;
    }
}

/*
  Write float on gpu to file
*/
void gpu_writeflt(float *p, int nx, int ny, const char *format, ...){
    format2fn;
    float *tmp=(float*)malloc(nx*ny*sizeof(float));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(float), hipMemcpyDefault);
    hipDeviceSynchronize();
    writeflt(tmp,nx,ny,"%s",fn);
    free(tmp);
}

/*
  Write float on gpu to file
*/
void gpu_writefcmp(fcomplex *p, int nx, int ny, const char *format, ...){
    format2fn;
    fcomplex *tmp=(fcomplex*)malloc(nx*ny*sizeof(fcomplex));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(fcomplex), hipMemcpyDefault);
    hipDeviceSynchronize();
    writefcmp((float complex*)tmp,nx,ny,"%s",fn);
    free(tmp);
}
/*
  Write float on gpu to file
*/
void gpu_writeint(int *p, int nx, int ny, const char *format, ...){
    format2fn;
    int *tmp=(int*)malloc(nx*ny*sizeof(int));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(int), hipMemcpyDefault);
    hipDeviceSynchronize();
    writeint(tmp,nx,ny,"%s",fn);
    free(tmp);
}

/**
   Compute the dot product of two vectors
*/
/*
__global__ static void dot_do(float *res, const float *restrict a, const float *restrict b, const int n){
    float sumt=0;
    __shared__ float sumb;
    if(threadIdx.x == 0) sumb=0;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sumt+=a[i]*b[i];
	}
    atomicAdd(&sumb, sumt);
    __syncthreads();
    if(threadIdx.x==0){
	atomicAdd(res, sumb);
    }
}
float gpu_dot(const float *restrict a, const float *restrict b, const int n, hipStream_t stream){
    float *res=NULL;
    cudaCallocHost(res, 1*sizeof(float), stream);
    dot_do<<<MAX(MIN(n/256, 32), 1), MIN(n, 256), 0, stream>>>(res, a, b, n);
    CUDA_SYNC_STREAM;
    float result=res[0];
    DO(hipHostFree(res));
    return result;
    }*/

void gpu_muv2dev(cumuv_t *out, MUV_T *in){
    if(!in->M) error("in->M should not be NULL\n");
    spcell *Mt=spcelltrans(in->M);
    gpu_spcell2dev(&(out)->Mt, Mt);
    gpu_dcell2cu(&(out)->U, in->U);
    gpu_dcell2cu(&(out)->V, in->V);
    spcellfree(Mt);
}
void gpu_cur2d(dmat **out, const curmat *in, hipStream_t stream){
    if(!*out) *out=dnew(in->nx, in->ny);
    gpu_dev2dbl(&(*out)->p, in->p, in->nx*in->ny, stream);
}
void gpu_curcell2d(dcell **out, const curcell *in, hipStream_t stream){
    if(!*out) *out=dcellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	gpu_cur2d(&(*out)->p[i], in->p[i], stream);
    }
}

void gpu_cur2s(smat **out, const curmat *in, hipStream_t stream){
    if(!*out) *out=snew(in->nx, in->ny);
    DO(hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(float), hipMemcpyDeviceToHost, stream));
}


void gpu_cuc2z(zmat **out, const cucmat *in, hipStream_t stream){
    if(!*out) *out=znew(in->nx, in->ny);
    DO(hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(fcomplex), hipMemcpyDeviceToHost, stream));
}

void gpu_curcell2s(scell **out, const curcell *in, hipStream_t stream){
    if(!*out) *out=scellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	gpu_cur2s(&(*out)->p[i], in->p[i], stream);
    }
}

void gpu_cuccell2z(zcell **out, const cuccell *in, hipStream_t stream){
    if(!*out) *out=zcellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	gpu_cuc2z(&(*out)->p[i], in->p[i], stream);
    }
}
void cellarr_cur(struct cellarr *ca, const curmat *A, hipStream_t stream){
    smat *tmp=NULL;
    gpu_cur2s(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_smat(ca, tmp);
    sfree(tmp);
}

void cellarr_cuc(struct cellarr *ca, const cucmat *A, hipStream_t stream){
    zmat *tmp=NULL;
    gpu_cuc2z(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_zmat(ca, tmp);
    zfree(tmp);
}
