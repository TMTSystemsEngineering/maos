#include "hip/hip_runtime.h"
/*
  Copyright 2009-2012 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "curmat.h"
#include "cucmat.h"
#include <pthread.h>
const char *cufft_str[]={
    "success", 
    "invalid plan",
    "allocation failed",
    "",
    "invalid value",
    "internal errlr",
    "exec failed (error elsewhere caused cufft to fail)",
    "setup failed"
    "invalid size"
};

#if CUDA_VERSION < 4010
pthread_mutex_t cufft_mutex=PTHREAD_MUTEX_INITIALIZER;
#endif
int gpu_recon;/**<GPU for reconstruction*/
int NGPU=0;
int* GPUS=NULL;
int nstream=0;
cudata_t **cudata_all=NULL;/*for all GPU. */
static hipsparseMatDescr_t spdesc=NULL;
#ifdef __APPLE__
pthread_key_t cudata_key;
#else
__thread cudata_t *cudata=NULL;/*for current thread and current GPU */
#endif

static __attribute((constructor)) void init(){
#ifdef __APPLE__
    pthread_key_create(&cudata_key, NULL);
#endif
    DO(hipsparseCreateMatDescr(&spdesc));
    hipsparseSetMatType(spdesc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(spdesc, HIPSPARSE_INDEX_BASE_ZERO);
}
/**
   Get GPU info.
*/
void gpu_info(){
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    info("name=%s\n"
	 "TotalGlobalMem=%d\n"
	 "SharedMemPerBlock=%d\n"
	 "regsPerBlock=%d\n"
	 "warpSize=%d",
	 prop.name,
	 (int)prop.totalGlobalMem,
	 (int)prop.sharedMemPerBlock,
	 prop.regsPerBlock,
	 prop.warpSize);
}
/**
   Print memory consumption.
*/
void gpu_print_mem(const char *msg){
    size_t fr, tot;
    hipDeviceSynchronize();
    DO(hipMemGetInfo(&fr, &tot));
    info2("GPU mem used %'lu B (%s)\n",(tot-fr), msg);
}
/**
   Get available memory.
*/
size_t gpu_get_mem(void){
    size_t fr, tot;
    DO(hipMemGetInfo(&fr, &tot));
    return fr;
}
/**
   Initialize GPU. Return 1 if success.
 */
int gpu_init(int *gpus, int ngpu){
    if(gpus && gpus[0]<0){
	info2("CUDA is disabled by user.\n");
	return 0;
    }
    int free_gpus=0;
    if(!gpus || ngpu==0){/*automatic. Use all GPUs. */
	free_gpus=1;
	if(hipGetDeviceCount(&ngpu) || ngpu==0){
	    return 0;
	}
	gpus=(int*)calloc(ngpu, sizeof(int));
	for(int i=0; i<ngpu; i++){
	    gpus[i]=i;
	}
    }
    /* check usability of GPUs.*/
    GPUS=(int*)calloc(ngpu ,sizeof(int));
    for(int im=0; im<ngpu; im++){
	int ig=gpus[im];
	hipDeviceProp_t prop={0};
	if(hipGetDeviceProperties(&prop, ig)){
	    warning2("Skip GPU %d: not supporting CUDA.\n", ig);
	}else if(prop.major!=9999){
	    if(prop.major>=1.3 || prop.totalGlobalMem>500000000){/*require minimum of 500M */
		GPUS[NGPU]=ig;
		NGPU++;
	    }else{
		warning2("Skip GPU %d: insufficient memory\n", ig);
	    }
	}
    }
    if(free_gpus) {
	free(gpus); 
	gpus=NULL;
    }
     if(NGPU) {
	cudata_all=(cudata_t**)calloc(NGPU, sizeof(cudata_t*));
	for(int im=0; im<NGPU; im++){
	    cudata_all[im]=(cudata_t*)calloc(1, sizeof(cudata_t));
	}
    }
    gpu_recon=0;
    if(!NGPU){
	warning("no gpu is available\n");
	return 0;
    }else{
	return 1;
    }
}

/**
   Clean up device.
*/
void gpu_cleanup(void){
    for(int ig=0; ig<NGPU; ig++){
	hipSetDevice(GPUS[ig]);
	hipDeviceReset();
    }
}
/**
   Convert double array to device memory (float)
*/

void cp2gpu(float * restrict *dest, double *src, int n){
    if(!src) return;
    float *tmp=(float*)malloc(n*sizeof(float));
    for(int i=0; i<n; i++){
	tmp[i]=(float)src[i];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, n*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, n*sizeof(float),hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    free(tmp);
    }
/**
   Convert double array to device memory (float)
*/

void cp2gpu(fcomplex * restrict *dest, dcomplex *restrict src, int n){
    if(!src) return;
    fcomplex *tmp=(fcomplex*)malloc(n*sizeof(fcomplex));
    for(int i=0; i<n; i++){
	tmp[i]=(make_hipFloatComplex)(hipCreal(src[i]), hipCimag(src[i]));
    }
    if(!*dest){
	DO(hipMalloc((fcomplex**)dest, n*sizeof(fcomplex)));
    }
    DO(hipMemcpy(*dest, tmp, n*sizeof(fcomplex),hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    free(tmp);
}

/**
   Copy map_t to cumap_t. if type==1, use hipArray, otherwise use float
array. Allow multiple calling to override the data.  */
void cp2gpu(cumap_t ***dest0, map_t **source, int nps){
    if(nps==0) return;
    if(!*dest0){
	*dest0=new cumap_t*[nps];
	for(int ips=0; ips<nps; ips++){
	    (*dest0)[ips]=new cumap_t(source[ips]->nx, source[ips]->ny);
	}	
    }
    cumap_t **dest=*dest0;
    for(int ips=0; ips<nps; ips++){
	dest[ips]->vx=source[ips]->vx;
	dest[ips]->vy=source[ips]->vy;
	dest[ips]->ht=source[ips]->h;
	dest[ips]->ox=source[ips]->ox;
	dest[ips]->oy=source[ips]->oy;
	dest[ips]->dx=source[ips]->dx;
	int nx=source[ips]->nx;
	int ny=source[ips]->ny;
	cp2gpu(&dest[ips]->p, source[ips]->p, nx*ny);
    }
    CUDA_SYNC_DEVICE;
}

/*
  Convert a host dsp array to GPU sprase array. Both are in CSC format. 
*/
void cp2gpu(cusp **dest0, dsp *src){
    if(!*dest0) *dest0=(cusp*)calloc(1, sizeof(cusp));
    cusp *dest=*dest0;
    dest->nx=src->m;
    dest->ny=src->n;
    dest->nzmax=src->nzmax;
    dest->p=NULL; dest->i=NULL; dest->x=NULL;
    cp2gpu(&dest->p, src->p, src->n+1);
    cp2gpu(&dest->i, src->i, src->nzmax);
    cp2gpu(&dest->x, src->x, src->nzmax);
}
void cp2gpu(cuspcell **dest0, spcell *src){
    if(!src) return;
    if(!*dest0){
	*dest0=cuspcellnew(src->nx, src->ny);
    }
    for(int i=0; i<src->nx*src->ny; i++){
	cp2gpu(&(*dest0)->p[i], src->p[i]);
    }
}
__global__ void cuspmul_do(float *y, cusp *A, float *x, float alpha){
    int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<A->ny; i+=step){
	for(int j=A->p[i]; j<A->p[i+1]; j++){
	    atomicAdd(&y[A->i[j]], A->x[j]*x[i]*alpha);
	}
    }
}
/*
  y=A*x where A is sparse. x, y are vectors. Slow for GS0.
*/
void cuspmul(float *y, cusp *A, float *x, float alpha, 
#if MYSPARSE ==1
	     hipStream_t stream
#else
	     hipsparseHandle_t handle
#endif
	     ){
#if MYSPARSE ==1
    cuspmul_do<<<DIM(A->nx, 256), 0, stream>>>(y,A,x,alpha);
#else
    int status=hipsparseScsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, 
			      A->ny, A->nx, alpha, spdesc,
			      A->x, A->p, A->i, x, 1.f, y);
    if(status!=0){
	error("hipsparseScsrmv failed with status %d\n", status);
    }
#endif
}

/*
  y=A'*x where A is sparse. x, y are vectors
*/
__global__ void cusptmul_do(float *y, int icol, cusp *A, float *x, float alpha){
    __shared__ float val;
    if(threadIdx.x==0) val=0;
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    int j=i+A->p[icol];
    atomicAdd(&val, A->x[j]*x[A->i[j]]);
    if(threadIdx.x==0) y[icol]+=val*alpha;
}
/*
  Does not work right yet. Try to launch a block for each column and n items in each block.
*/
void cusptmul(float *y, cusp *A, float *x, float alpha, 
#if MYSPARSE ==1
	     hipStream_t stream
#else
	     hipsparseHandle_t handle
#endif
	      ){
#if MYSPARSE == 1
    for(int i=0; i<A->ny; i++){
	cusptmul_do<<<1, A->p[i+1]-A->p[i], 0, stream>>>(y,i,A,x,alpha);
    }
    warning("Not working correctly yet\n");
#else
    int status=hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			      A->ny, A->nx, alpha, spdesc,
			      A->x, A->p, A->i, x, 1.f, y);
    if(status!=0){
	error("hipsparseScsrmv failed with status %d\n", status);
    }
#endif
}

/**
   Convert a source loc_t to device memory.
*/
void cp2gpu(float (* restrict *dest)[2], loc_t *src){
    float (*tmp)[2]=(float(*)[2])malloc(src->nloc*2*sizeof(float));
    for(int iloc=0; iloc<src->nloc; iloc++){
	tmp[iloc][0]=(float)src->locx[iloc];
	tmp[iloc][1]=(float)src->locy[iloc];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, src->nloc*2*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, src->nloc*2*sizeof(float),hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    free(tmp);
}

/**
   Convert dmat array to device memory.
*/
void cp2gpu(float * restrict *dest, dmat *src){
    if(!src) return;
    cp2gpu(dest, src->p, src->nx*src->ny);
}
/**
   Convert dmat array to curmat
*/
void cp2gpu(curmat *restrict *dest, dmat *src){
    if(!src){
	dzero(*dest);
	return;
    }
    if(!*dest){
	*dest=curnew(src->nx, src->ny);
    }else{
	assert(src->nx*src->ny==(*dest)->nx*(*dest)->ny);
    }
    cp2gpu(&(*dest)->p, src->p, src->nx*src->ny);
}
/*
  convert cmat to cucmat
*/
void cp2gpu(cucmat *restrict *dest, cmat *src){
    if(!src){
	czero(*dest);
	return;
    }
    if(!*dest){
	*dest=cucnew(src->nx, src->ny);
    }else{
	assert(src->nx*src->ny==(*dest)->nx*(*dest)->ny);
    }
    cp2gpu(&(*dest)->p, (dcomplex*)src->p, (int)(src->nx*src->ny));
}
/**
   Convert dcell to curcell
*/
void cp2gpu(curcell *restrict *dest, dcell *src){
    if(!src) {
	dzero(*dest);
	return;
    }
    if(!*dest) {
	*dest=curcellnew(src->nx, src->ny);
    }else if((*dest)->nx!=src->nx || (*dest)->ny!=src->ny){
	error("Mismatch: %dx%d vs %ldx%ld\n", 
	      (*dest)->nx, (*dest)->ny, src->nx, src->ny);
    }
    for(int i=0; i<src->nx*src->ny; i++){
	cp2gpu(&(*dest)->p[i], src->p[i]);
    }
}
/**
   Convert dmat array to device memory.
*/
void cp2gpu(fcomplex * restrict *dest, cmat *src){
    if(src){
	cp2gpu(dest, (dcomplex*)src->p, src->nx*src->ny);
    }
}
/**
   Convert double array to device memory (float)
*/
void dbl2flt(float * restrict *dest, double *src, int n){
    if(!src) return;
    if(!*dest){
	hipHostMalloc((float**)dest, n*sizeof(float));
    }
    for(int i=0; i<n; i++){
	(*dest)[i]=(float)src[i];
    }
}
/**
   Convert long array to device int
*/
void cp2gpu(int * restrict *dest, long *src, int n){
    if(!src) return;
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(long)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((long)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	free(tmp);
    }
}
void cp2gpu(int *restrict *dest, int *src, int n){
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    DO(hipMemcpy(*dest, src, sizeof(int)*n, hipMemcpyHostToDevice));
}
/**
   Convert long array to device int
*/
void cp2gpu(int * restrict *dest, spint *src, int n){
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(spint)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((spint)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	free(tmp);
    }
}
/**
   Convert device (float) array and add to host double.
   dest = alpha * dest + beta *src;
*/
void cp2cpu(double * restrict *dest, double alpha, float *src, double beta, int n, hipStream_t stream){
    float *tmp=(float*)malloc4async(n*sizeof(float));
    DO(hipMemcpyAsync(tmp, src, n*sizeof(float), hipMemcpyDeviceToHost, stream));
    if(!*dest){
	*dest=(double*)malloc(sizeof(double)*n);
    }
    double *restrict p=*dest;
    CUDA_SYNC_STREAM;
    for(int i=0; i<n; i++){
	p[i]=p[i]*alpha+beta*tmp[i];
    }
    free4async(tmp);
}
/*
  Write float on gpu to file
*/
void gpu_write(float *p, int nx, int ny, const char *format, ...){
    format2fn;
    float *tmp=(float*)malloc(nx*ny*sizeof(float));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(float), hipMemcpyDeviceToHost);
    writeflt(tmp,nx,ny,"%s",fn);
    free(tmp);
}

/*
  Write float on gpu to file
*/
void gpu_write(fcomplex *p, int nx, int ny, const char *format, ...){
    format2fn;
    fcomplex *tmp=(fcomplex*)malloc(nx*ny*sizeof(fcomplex));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(fcomplex), hipMemcpyDeviceToHost);
    writefcmp((float complex*)tmp,nx,ny,"%s",fn);
    free(tmp);
}
/*
  Write float on gpu to file
*/
void gpu_write(int *p, int nx, int ny, const char *format, ...){
    format2fn;
    int *tmp=(int*)malloc(nx*ny*sizeof(int));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    writeint(tmp,nx,ny,"%s",fn);
    free(tmp);
}

void cp2gpu(cumuv_t *out, MUV_T *in){
    if(!in->M) error("in->M should not be NULL\n");
    spcell *Mt=spcelltrans(in->M);
    cp2gpu(&(out)->Mt, Mt);
    cp2gpu(&(out)->U, in->U);
    cp2gpu(&(out)->V, in->V);
    spcellfree(Mt);
}
void cp2cpu(dmat **out, double alpha, const curmat *in, double beta, hipStream_t stream){
    if(!in){
	if(*out) dzero(*out);
	return;
    }
    if(!*out) {
	*out=dnew(in->nx, in->ny);
    }else{
	assert((*out)->nx*(*out)->ny==in->nx*in->ny);
    }
    cp2cpu(&(*out)->p, alpha, in->p, beta, in->nx*in->ny, stream);
}
void cp2cpu(dcell **out, double alpha, const curcell *in, double beta, hipStream_t stream){
    if(!in){
	if(*out) dcellzero(*out);
	return;
    }
    if(!*out) *out=dcellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	cp2cpu(&(*out)->p[i], alpha, in->p[i], beta, stream);
    }
}

void cp2cpu(smat **out, const curmat *in, hipStream_t stream){
    if(!in) {
	if(*out) szero(*out);
	return;
    }
    if(!*out) *out=snew(in->nx, in->ny);
    DO(hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(float), hipMemcpyDeviceToHost, stream));
    if(in->header) (*out)->header=strdup(in->header);
}


void cp2cpu(zmat **out, const cucmat *in, hipStream_t stream){
    if(!in){
	if(*out) zzero(*out);
	return;
    }
    if(!*out) *out=znew(in->nx, in->ny);
    DO(hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(fcomplex), hipMemcpyDeviceToHost, stream));
    if(in->header) (*out)->header=strdup(in->header);
}

void cp2cpu(scell **out, const curcell *in, hipStream_t stream){
    if(!in){
	if(*out) scellzero(*out);
	return;
    }
    if(!*out) *out=scellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	cp2cpu(&(*out)->p[i], in->p[i], stream);
    }
}

void cp2cpu(zcell **out, const cuccell *in, hipStream_t stream){
    if(!in){
	if(*out) zcellzero(*out);
	return;
    }
    if(!*out) *out=zcellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	cp2cpu(&(*out)->p[i], in->p[i], stream);
    }
}
void cellarr_cur(struct cellarr *ca, const curmat *A, hipStream_t stream){
    smat *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_smat(ca, tmp);
    sfree(tmp);
}

void cellarr_cuc(struct cellarr *ca, const cucmat *A, hipStream_t stream){
    zmat *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_zmat(ca, tmp);
    zfree(tmp);
}

void cellarr_curcell(struct cellarr *ca, const curcell *A, hipStream_t stream){
    scell *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_scell(ca, tmp);
    scellfree(tmp);
}

void cellarr_cuccell(struct cellarr *ca, const cuccell *A, hipStream_t stream){
    zcell *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_zcell(ca, tmp);
    zcellfree(tmp);
}
