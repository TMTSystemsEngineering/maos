#include "hip/hip_runtime.h"
/*
  Copyright 2009-2012 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "curmat.h"
#include "cucmat.h"
#include <errno.h>
#include <pthread.h>
#if defined(HAS_NVML) && HAS_NVML==1
extern "C"{
    /*taken from nvml.h*/
    typedef struct nvmlDevice_st* nvmlDevice_t;
    typedef struct nvmlMemory_st 
    {
	unsigned long long total; //!< Total installed FB memory (in bytes)
	unsigned long long free; //!< Unallocated FB memory (in bytes)
	unsigned long long used; //!< Allocated FB memory (in bytes). Note that the driver/GPU always sets aside a small amount of memory for bookkeeping
    } nvmlMemory_t;
    int nvmlDeviceGetHandleByIndex(unsigned int index, nvmlDevice_t *device);
    int nvmlDeviceGetMemoryInfo(nvmlDevice_t device, nvmlMemory_t *memory);
    int nvmlInit();
    int nvmlShutdown();
}
#endif
const char *cufft_str[]={
    "success", 
    "invalid plan",
    "allocation failed",
    "",
    "invalid value",
    "internal errlr",
    "exec failed (error elsewhere caused cufft to fail)",
    "setup failed"
    "invalid size"
};

#if CUDA_VERSION < 4010
pthread_mutex_t cufft_mutex=PTHREAD_MUTEX_INITIALIZER;
#endif
int gpu_recon;/**<GPU for reconstruction*/
int NGPU=0;
int* GPUS=NULL;
int nstream=0;
cudata_t *cudata_all=NULL;/*for all GPU. */
static hipsparseMatDescr_t spdesc=NULL;
#ifdef __APPLE__
pthread_key_t cudata_key;
#else
__thread cudata_t *cudata=NULL;/*for current thread and current GPU */
#endif

static __attribute((constructor)) void init(){
#ifdef __APPLE__
    pthread_key_create(&cudata_key, NULL);
#endif
    DO(hipsparseCreateMatDescr(&spdesc));
    hipsparseSetMatType(spdesc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(spdesc, HIPSPARSE_INDEX_BASE_ZERO);
}
/**
   Get GPU info.
*/
void gpu_info(){
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    info("name=%s\n"
	 "TotalGlobalMem=%d\n"
	 "SharedMemPerBlock=%d\n"
	 "regsPerBlock=%d\n"
	 "warpSize=%d",
	 prop.name,
	 (int)prop.totalGlobalMem,
	 (int)prop.sharedMemPerBlock,
	 prop.regsPerBlock,
	 prop.warpSize);
}
/**
   Print memory consumption.
*/
void gpu_print_mem(const char *msg){
    size_t fr, tot;
    hipDeviceSynchronize();
    DO(hipMemGetInfo(&fr, &tot));
    info2("GPU (%d) mem used %'lu MB (%s)\n",(int)(cudata-cudata_all),(tot-fr)/1024/1024, msg);
}
/**
   Get available memory.
*/
long gpu_get_mem(void){
    size_t fr, tot;
    DO(hipMemGetInfo(&fr, &tot));
    return (long)fr;
}
static int cmp_gpu_info(const long *a, const long *b){
    return (int)(b[1]-a[1]);
}
/**
   Initialize GPU. Return 1 if success.
   if gpus is not null, it is of length ngpu. gpus specifies gpu index to use.
   if gpus is null, ngpu specifies number of gpus to use. all if 0.
*/
int gpu_init(int *gpus, int ngpu){
    int ans, ngpu_tot=0;//total number of GPUs.
    if((ans=hipGetDeviceCount(&ngpu_tot)) || ngpu_tot==0){//no GPUs available.
	info2("No GPUs available. ans=%d\n", ans);
	return 0;
    }
    NGPU=0;
    /*
      User specified exact GPUs to use. We check every entry. 
      If <0 is found, do not use any GPU.
      If >=ngpu_tot is found, skip the GPU.
      If duplicates are found, use only once.
     */
    if(gpus && ngpu>0){
	if(!GPUS) GPUS=(int*)malloc(ngpu*sizeof(int));
	for(int ig=0; ig<ngpu; ig++){
	    if(gpus[ig]<0){
		info2("CUDA is disabled by user.\n");
		free(GPUS); GPUS=NULL; 
		return 0;
	    }else{
		if(gpus[ig]>=ngpu_tot){
		    warning2("Skip GPU %d: not exist\n", gpus[ig]);
		}else{
		    GPUS[NGPU++]=gpus[ig];
		    /* Enable the following to disallow use GPUs in multiple threads
		      int j;
		    for(j=0; j<NGPU; j++){
			if(GPUS[j]==gpus[ig]){
			    warning2("Skip GPU %d: duplicated\n", gpus[ig]);
			    break;
			}
		    }
		    if(j==NGPU){
			GPUS[NGPU++]=gpus[ig];
			}*/
		}
	    }
	}
    }else{
	int repeat=1;
	if(ngpu<=0){
	    repeat=0;
	    ngpu=ngpu_tot;
	}
	GPUS=(int*)calloc(ngpu, sizeof(int));
	register_deinit(NULL, GPUS);
	/*For each GPU, query the available memory.*/
	long (*gpu_info)[2]=(long(*)[2])calloc(2*ngpu_tot, sizeof(long));
#if defined(HAS_NVML) && HAS_NVML==1
	nvmlDevice_t dev;
	nvmlMemory_t mem;
	if(nvmlInit()){
	    warning("nvml init failed\n");
	}
#endif
	for(int ig=0; ig<ngpu_tot; ig++){
	    gpu_info[ig][0]=ig;
#if defined(HAS_NVML) && HAS_NVML==1
	    if(nvmlDeviceGetHandleByIndex(ig, &dev) == 0 
	       && nvmlDeviceGetMemoryInfo(dev, &mem) == 0){
		gpu_info[ig][1]=mem.free;
	    }else
#endif
		{
		    hipSetDevice(ig);//this allocates context.
		    gpu_info[ig][1]=gpu_get_mem();
		    //hipDeviceReset(); We already started simulation. Do not reset.
		}
	}
#if defined(HAS_NVML) && HAS_NVML==1
	nvmlShutdown();
#endif
	/*sort so that gpus with higest memory is in the front.*/
	qsort(gpu_info, ngpu_tot, sizeof(long)*2, (int(*)(const void*, const void *))cmp_gpu_info);
	for(int igpu=0; igpu<ngpu_tot; igpu++){
	    info2("GPU %d has mem %.1f GB\n", igpu, gpu_info[igpu][1]/1024/1024/1024.);
	}
	for(int i=0, igpu=0; i<ngpu; i++, igpu++){
	    if(igpu==ngpu_tot || gpu_info[igpu][1]<500000000){
		if(repeat){
		    igpu=0; //reset to beginning.
		}else{
		    break; //stop
		}
	    }
	    GPUS[NGPU++]=(int)gpu_info[igpu][0];
	}
	free(gpu_info);
    }
    if(NGPU) {
	gpu_recon=0;/*first gpu in GPUS*/
	cudata_all=(cudata_t*)calloc(NGPU, sizeof(cudata_t));
	register_deinit(NULL, cudata_all);
	info2("Using GPU");
	for(int i=0; GPUS && i<NGPU; i++){
	    info2(" %d", GPUS[i]);
	    gpu_set(i);
	    //Reserve memory in GPU so the next maos will not pick this GPU.
	    DO(hipMalloc(&cudata->reserve, 500000000));
	}
	info2("\n");
    }
    return NGPU;
}

/**
   Clean up device.
*/
void gpu_cleanup(void){
    for(int ig=0; ig<NGPU; ig++){
	hipSetDevice(GPUS[ig]);
	hipDeviceReset();
    }
}
/**
   Convert double array to device memory (float)
*/

void cp2gpu(float * restrict *dest, double *src, int n){
    if(!src) return;
    float *tmp=(float*)malloc(n*sizeof(float));
    for(int i=0; i<n; i++){
	tmp[i]=(float)src[i];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, n*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, n*sizeof(float),hipMemcpyHostToDevice));
    free(tmp);
}
/**
   Convert double array to device memory (float)
*/

void cp2gpu(fcomplex * restrict *dest, dcomplex *restrict src, int n){
    if(!src) return;
    fcomplex *tmp=(fcomplex*)malloc(n*sizeof(fcomplex));
    for(int i=0; i<n; i++){
	tmp[i]=(make_hipFloatComplex)(hipCreal(src[i]), hipCimag(src[i]));
    }
    if(!*dest){
	DO(hipMalloc((fcomplex**)dest, n*sizeof(fcomplex)));
    }
    DO(hipMemcpy(*dest, tmp, n*sizeof(fcomplex),hipMemcpyHostToDevice));
    free(tmp);
}

/**
   Copy map_t to cumap_t. if type==1, use hipArray, otherwise use float
   array. Allow multiple calling to override the data.  */
void cp2gpu(cumap_t ***dest0, map_t **source, int nps){
    if(nps==0) return;
    if(!*dest0){
	*dest0=new cumap_t*[nps];
	for(int ips=0; ips<nps; ips++){
	    (*dest0)[ips]=new cumap_t(source[ips]->nx, source[ips]->ny);
	}	
    }
    cumap_t **dest=*dest0;
    for(int ips=0; ips<nps; ips++){
	dest[ips]->vx=source[ips]->vx;
	dest[ips]->vy=source[ips]->vy;
	dest[ips]->ht=source[ips]->h;
	dest[ips]->ox=source[ips]->ox;
	dest[ips]->oy=source[ips]->oy;
	dest[ips]->dx=source[ips]->dx;
	int nx=source[ips]->nx;
	int ny=source[ips]->ny;
	cp2gpu(&dest[ips]->p, source[ips]->p, nx*ny);
    }
    CUDA_SYNC_DEVICE;
}

/*
  Convert a host dsp array to GPU sprase array. Both are in CSC format. 
*/
void cp2gpu(cusp **dest0, dsp *src){
    if(!src) return;
    if(!*dest0) *dest0=(cusp*)calloc(1, sizeof(cusp));
    cusp *dest=*dest0;
    dest->nx=src->m;
    dest->ny=src->n;
    dest->nzmax=src->nzmax;
    dest->p=NULL; dest->i=NULL; dest->x=NULL;
    cp2gpu(&dest->p, src->p, src->n+1);
    cp2gpu(&dest->i, src->i, src->nzmax);
    cp2gpu(&dest->x, src->x, src->nzmax);
}
void cp2gpu(cuspcell **dest0, spcell *src){
    if(!src) return;
    if(!*dest0){
	*dest0=cuspcellnew(src->nx, src->ny);
    }
    for(int i=0; i<src->nx*src->ny; i++){
	cp2gpu(&(*dest0)->p[i], src->p[i]);
    }
}
__global__ void cuspmul_do(float *y, cusp *A, float *x, float alpha){
    int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<A->ny; i+=step){
	for(int j=A->p[i]; j<A->p[i+1]; j++){
	    atomicAdd(&y[A->i[j]], A->x[j]*x[i]*alpha);
	}
    }
}
static const char *scsrmv_err[]={
    "Success",
    "Not initialized",
    "Allocation failed",
    "Invalid value",
    "Archtecture mismatch",
    "Mapping error",
    "Execution failed",
    "Internal error",
    "Matrix type not supported"
};
/*
  y=A*x where A is sparse. x, y are vectors. Slow for GS0.
*/
void cuspmul(float *y, cusp *A, float *x, float alpha, 
#if MYSPARSE ==1
	     hipStream_t stream
#else
	     hipsparseHandle_t handle
#endif
	     ){
#if MYSPARSE ==1
    cuspmul_do<<<DIM(A->nx, 256), 0, stream>>>(y,A,x,alpha);
#else
    int status=hipsparseScsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, 
			      A->ny, A->nx, alpha, spdesc,
			      A->x, A->p, A->i, x, 1.f, y);
    if(status!=0){
	error("hipsparseScsrmv failed with status '%s'\n", scsrmv_err[status]);
    }
#endif
}

/*
  y=A'*x where A is sparse. x, y are vectors
*/
__global__ void cusptmul_do(float *y, int icol, cusp *A, float *x, float alpha){
    __shared__ float val;
    if(threadIdx.x==0) val=0;
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    int j=i+A->p[icol];
    atomicAdd(&val, A->x[j]*x[A->i[j]]);
    if(threadIdx.x==0) y[icol]+=val*alpha;
}
/*
  Does not work right yet. Try to launch a block for each column and n items in each block.
*/
void cusptmul(float *y, cusp *A, float *x, float alpha, 
#if MYSPARSE ==1
	      hipStream_t stream
#else
	      hipsparseHandle_t handle
#endif
	      ){
#if MYSPARSE == 1
    for(int i=0; i<A->ny; i++){
	cusptmul_do<<<1, A->p[i+1]-A->p[i], 0, stream>>>(y,i,A,x,alpha);
    }
    warning("Not working correctly yet\n");
#else
    int status=hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			      A->ny, A->nx, alpha, spdesc,
			      A->x, A->p, A->i, x, 1.f, y);
    if(status!=0){
	error("hipsparseScsrmv failed with status %d\n", status);
    }
#endif
}

/**
   Convert a source loc_t to device memory.
*/
void cp2gpu(float (* restrict *dest)[2], loc_t *src){
    float (*tmp)[2]=(float(*)[2])malloc(src->nloc*2*sizeof(float));
    for(int iloc=0; iloc<src->nloc; iloc++){
	tmp[iloc][0]=(float)src->locx[iloc];
	tmp[iloc][1]=(float)src->locy[iloc];
    }
    if(!*dest){
	DO(hipMalloc((float**)dest, src->nloc*2*sizeof(float)));
    }
    DO(hipMemcpy(*dest, tmp, src->nloc*2*sizeof(float),hipMemcpyHostToDevice));
    free(tmp);
}

/**
   Convert dmat array to device memory.
*/
void cp2gpu(float * restrict *dest, dmat *src){
    if(!src) return;
    cp2gpu(dest, src->p, src->nx*src->ny);
}
/**
   Convert dmat array to curmat
*/
void cp2gpu(curmat *restrict *dest, dmat *src){
    if(!src){
	curzero(*dest);
	return;
    }
    float *pdest=NULL;
    if(*dest){
	pdest=(*dest)->p;
	assert(src->nx*src->ny==(*dest)->nx*(*dest)->ny);
    }
    cp2gpu(&pdest, src->p, src->nx*src->ny);
    if(!*dest){
	*dest=curnew(src->nx, src->ny, pdest);
    }
}
void cp2gpu(curmat *restrict *dest, float *src, int nx, int ny, hipStream_t stream){
    if(!src){
	curzero(*dest);
	return;
    }
    float *pdest=NULL;
    if(*dest){
	pdest=(*dest)->p;
	assert(nx*ny==(*dest)->nx*(*dest)->ny);
    }else{
	hipMalloc(&pdest, nx*ny*sizeof(float));
    }
    if(stream){
	DO(hipMemcpyAsync(pdest, src, nx*ny*sizeof(float),hipMemcpyHostToDevice, stream));
    }else{
	DO(hipMemcpy(pdest, src, nx*ny*sizeof(float),hipMemcpyHostToDevice));
    }
    if(!*dest){
	*dest=curnew(nx, ny, pdest);
    }
}
/*
  convert cmat to cucmat
*/
void cp2gpu(cucmat *restrict *dest, cmat *src){
    if(!src){
	czero(*dest);
	return;
    }
    if(!*dest){
	*dest=cucnew(src->nx, src->ny);
    }else{
	assert(src->nx*src->ny==(*dest)->nx*(*dest)->ny);
    }
    cp2gpu(&(*dest)->p, (dcomplex*)src->p, (int)(src->nx*src->ny));
}
/**
   Convert dcell to curcell
*/
void cp2gpu(curcell *restrict *dest, dcell *src){
    if(!src) {
	dzero(*dest);
	return;
    }
    if(!*dest) {
	long nc=src->nx*src->ny;
	long nx[nc];
	long ny[nc];
	for(long i=0; i<nc; i++){
	    if(src->p[i]){
		nx[i]=src->p[i]->nx;
		ny[i]=src->p[i]->ny;
	    }else{
		nx[i]=0;
		ny[i]=0;
	    }
	}
	*dest=curcellnew(src->nx, src->ny, nx, ny);
    }else if((*dest)->nx!=src->nx || (*dest)->ny!=src->ny){
	error("Mismatch: %ldx%ld vs %ldx%ld\n", 
	      (*dest)->nx, (*dest)->ny, src->nx, src->ny);
    }
    for(int i=0; i<src->nx*src->ny; i++){
	cp2gpu(&(*dest)->p[i], src->p[i]);
    }
}
/**
   Convert dcell to curcell
*/
void cp2gpu(cuccell *restrict *dest, ccell *src){
    if(!src) {
	dzero(*dest);
	return;
    }
    if(!*dest) {
	long nc=src->nx*src->ny;
	long nx[nc];
	long ny[nc];
	for(long i=0; i<nc; i++){
	    if(src->p[i]){
		nx[i]=src->p[i]->nx;
		ny[i]=src->p[i]->ny;
	    }else{
		nx[i]=0;
		ny[i]=0;
	    }
	}
	*dest=cuccellnew(src->nx, src->ny, nx, ny);
    }else if((*dest)->nx!=src->nx || (*dest)->ny!=src->ny){
	error("Mismatch: %ldx%ld vs %ldx%ld\n", 
	      (*dest)->nx, (*dest)->ny, src->nx, src->ny);
    }
    for(int i=0; i<src->nx*src->ny; i++){
	cp2gpu(&(*dest)->p[i], src->p[i]);
    }
}
/**
   Convert dmat array to device memory.
*/
void cp2gpu(fcomplex * restrict *dest, cmat *src){
    if(src){
	cp2gpu(dest, (dcomplex*)src->p, src->nx*src->ny);
    }
}
/**
   Convert double array to device memory (float)
*/
void dbl2flt(float * restrict *dest, double *src, int n){
    if(!src) return;
    if(!*dest){
	hipHostMalloc((float**)dest, n*sizeof(float));
    }
    for(int i=0; i<n; i++){
	(*dest)[i]=(float)src[i];
    }
}
/**
   Convert long array to device int
*/
void cp2gpu(int * restrict *dest, long *src, int n){
    if(!src) return;
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(long)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((long)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	free(tmp);
    }
}
void cp2gpu(int *restrict *dest, int *src, int n){
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    DO(hipMemcpy(*dest, src, sizeof(int)*n, hipMemcpyHostToDevice));
}
/**
   Convert long array to device int
*/
void cp2gpu(int * restrict *dest, spint *src, int n){
    if(!*dest){
	DO(hipMalloc((int**)dest, n*sizeof(int)));
    }
    if(sizeof(spint)==sizeof(int)){
	DO(hipMemcpy(*dest, src, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
    }else{
	int *tmp=(int*)malloc(sizeof(int)*n);
	for(int i=0; i<n; i++){
	    tmp[i]=(int)src[i];
	    if((spint)tmp[i]!=src[i]){
		error("Overflow occured\n");
	    }
	}
	DO(hipMemcpy(*dest, tmp, n*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	free(tmp);
    }
}
/**
   Convert device (float) array and add to host double.
   dest = alpha * dest + beta *src;
*/
void cp2cpu(double * restrict *dest, double alpha, float *src, double beta, int n, 
	    hipStream_t stream, pthread_mutex_t *mutex){
    float *tmp=(float*)malloc4async(n*sizeof(float));
    DO(hipMemcpyAsync(tmp, src, n*sizeof(float), hipMemcpyDeviceToHost, stream));
    if(!*dest){
	*dest=(double*)malloc(sizeof(double)*n);
    }
    double *restrict p=*dest;
    CUDA_SYNC_STREAM;
    if(mutex) LOCK(*mutex);
    for(int i=0; i<n; i++){
	p[i]=p[i]*alpha+beta*tmp[i];
    }
    if(mutex) UNLOCK(*mutex);
    free4async(tmp);
}
/*
  Write float on gpu to file
*/
void gpu_write(float *p, int nx, int ny, const char *format, ...){
    format2fn;
    float *tmp=(float*)malloc(nx*ny*sizeof(float));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(float), hipMemcpyDeviceToHost);
    writeflt(tmp,nx,ny,"%s",fn);
    free(tmp);
}

/*
  Write float on gpu to file
*/
void gpu_write(fcomplex *p, int nx, int ny, const char *format, ...){
    format2fn;
    fcomplex *tmp=(fcomplex*)malloc(nx*ny*sizeof(fcomplex));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(fcomplex), hipMemcpyDeviceToHost);
    writefcmp((float complex*)tmp,nx,ny,"%s",fn);
    free(tmp);
}
/*
  Write float on gpu to file
*/
void gpu_write(int *p, int nx, int ny, const char *format, ...){
    format2fn;
    int *tmp=(int*)malloc(nx*ny*sizeof(int));
    hipDeviceSynchronize();
    hipMemcpy(tmp, p, nx*ny*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    writeint(tmp,nx,ny,"%s",fn);
    free(tmp);
}

void cp2gpu(cumuv_t *out, MUV_T *in){
    if(!in->M) error("in->M should not be NULL\n");
    spcell *Mt=spcelltrans(in->M);
    cp2gpu(&(out)->Mt, Mt);
    cp2gpu(&(out)->U, in->U);
    cp2gpu(&(out)->V, in->V);
    spcellfree(Mt);
}
void cp2cpu(dmat **out, double alpha, const curmat *in, double beta, hipStream_t stream, pthread_mutex_t *mutex){
    if(!in){
	if(*out) dzero(*out);
	return;
    }
    if(!*out) {
	*out=dnew(in->nx, in->ny);
    }else{
	assert((*out)->nx*(*out)->ny==in->nx*in->ny);
    }
    cp2cpu(&(*out)->p, alpha, in->p, beta, in->nx*in->ny, stream, mutex);
}
void cp2cpu(dcell **out, double alpha, const curcell *in, double beta, hipStream_t stream, pthread_mutex_t *mutex){
    if(!in){
	if(*out) dcellzero(*out);
	return;
    }
    if(!*out) *out=dcellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	cp2cpu(&(*out)->p[i], alpha, in->p[i], beta, stream, mutex);
    }
}
void cp2cpu(smat **out, const curmat *in, hipStream_t stream){
    if(!in) {
	if(*out) szero(*out);
	return;
    }
    if(!*out) *out=snew(in->nx, in->ny);
    DO(hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(float), hipMemcpyDeviceToHost, stream));
    if(in->header) (*out)->header=strdup(in->header);
}


void cp2cpu(zmat **out, const cucmat *in, hipStream_t stream){
    if(!in){
	if(*out) zzero(*out);
	return;
    }
    if(!*out) *out=znew(in->nx, in->ny);
    DO(hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(fcomplex), hipMemcpyDeviceToHost, stream));
    if(in->header) (*out)->header=strdup(in->header);
}

void cp2cpu(scell **out, const curcell *in, hipStream_t stream){
    if(!in){
	if(*out) scellzero(*out);
	return;
    }
    if(!*out) *out=scellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	cp2cpu(&(*out)->p[i], in->p[i], stream);
    }
}

void cp2cpu(zcell **out, const cuccell *in, hipStream_t stream){
    if(!in){
	if(*out) zcellzero(*out);
	return;
    }
    if(!*out) *out=zcellnew(in->nx, in->ny);
    for(int i=0; i<in->nx*in->ny; i++){
	cp2cpu(&(*out)->p[i], in->p[i], stream);
    }
}
void cellarr_cur(struct cellarr *ca, const curmat *A, hipStream_t stream){
    smat *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_smat(ca, tmp);
    sfree(tmp);
}

void cellarr_cuc(struct cellarr *ca, const cucmat *A, hipStream_t stream){
    zmat *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_zmat(ca, tmp);
    zfree(tmp);
}

void cellarr_curcell(struct cellarr *ca, const curcell *A, hipStream_t stream){
    scell *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_scell(ca, tmp);
    scellfree(tmp);
}

void cellarr_cuccell(struct cellarr *ca, const cuccell *A, hipStream_t stream){
    zcell *tmp=NULL;
    cp2cpu(&tmp,A,stream);
    CUDA_SYNC_STREAM;
    cellarr_zcell(ca, tmp);
    zcellfree(tmp);
}
