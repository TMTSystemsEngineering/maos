#include "hip/hip_runtime.h"
/*
  Copyright 2009-2015 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include "wfs.h"
#include "cudata.h"
#include "cucmat.h"
#ifdef __cplusplus
extern "C"{
#endif
#include "../maos/pywfs.h"
#include "../maos/sim.h"
#ifdef __cplusplus
}
#endif

__global__ static void
pywfs_grad_do(Real *grad, Real *ints, Real *saa, Real *isum, Real *goff, Real gain, int nsa){
    Real alpha0=gain*nsa/(*isum);
    for(int i=threadIdx.x + blockIdx.x * blockDim.x; i<nsa; i+=blockDim.x * gridDim.x){
	Real alpha=alpha0/saa[i];
	grad[i]=(-ints[i]+ints[i+nsa]-ints[nsa*2+i]+ints[nsa*3+i])*alpha-goff[i];
	grad[i+nsa]=(-ints[i]-ints[i+nsa]+ints[nsa*2+i]+ints[nsa*3+i])*alpha-goff[i+nsa];
    }
}
void pywfs_grad(curmat *grad, /**<[out] gradients*/
		const curmat *ints, /**<[in] Intensity*/
		const curmat *saa,  /**<[in] Subaperture normalized area*/
		curmat *isum, /**<[out] Sum intensity*/
		const curmat *goff, /**<[in] Gradient of flat wavefront*/
		Real gain,   /**<[in] Gain*/
		hipStream_t stream){
    cursum2(isum->p, ints, stream);//sum of ints
    pywfs_grad_do<<<DIM(ints->nx, 256), 0, stream>>>
	(grad->p, ints->p, saa->p, isum->p, goff->p, gain, ints->nx);
}
void pywfs_ints(curmat *ints, curmat *phiout, cuwfs_t *cuwfs, Real siglev, hipStream_t stream){
    //Pyramid WFS
    cupowfs_t *cupowfs=cuwfs->powfs;
    PYWFS_T *pywfs=cupowfs->pywfs;
    cuzero(cuwfs->pypsf, stream);
    locfft_t *locfft=pywfs->locfft;
    const int nwvl=locfft->wvl->nx;
    Real pos_r=pywfs->modulate; 
    Real dx=locfft->loc->dx;
    long nembed=locfft->nembed->p[0];
    long nembed2=nembed/2;
    long ncomp=pywfs->nominal->nx;
    long ncomp2=ncomp/2;
    int pos_n=pywfs->modulpos;
    if(pos_r<=0){
	pos_n=1;
    }
    cucmat *otf=cuwfs->pyotf;
    for(int iwvl=0; iwvl<nwvl; iwvl++){
	cucmat *wvf=cuwfs->pywvf->p[iwvl];
	Real alpha=pywfs->wvlwts->p[iwvl]/(ncomp*ncomp*pos_n);
	Real wvl=locfft->wvl->p[iwvl];
	cuzero(wvf, stream);
	embed_wvf_do<<<DIM(phiout->nx,256),0,stream>>>
	    (wvf->p, phiout->p, cuwfs->amp, cupowfs->embed[iwvl], phiout->nx, wvl);
	CUFFT(cuwfs->plan_fs, wvf->p, HIPFFT_FORWARD);
	fftshift_do<<<DIM2(wvf->nx, wvf->nx, 16),0,stream>>>
	    (wvf->p, wvf->nx, wvf->ny);
	Real otfnorm=1./(sqrt(locfft->ampnorm)*locfft->nembed->p[iwvl]);
	cucscale(wvf, otfnorm, stream);
	Real dtheta=locfft->wvl->p[iwvl]/(dx*nembed);
	for(int ipos=0; ipos<pos_n; ipos++){
	    Real theta=2*M_PI*ipos/pos_n;
	    Real posx=cos(theta)*pos_r;
	    Real posy=sin(theta)*pos_r;
	    long offy=(long)round(posy/dtheta);
	    long offy2=nembed2+offy-ncomp2;
	    long iy0=MAX(-offy2, 0);
	    long ny2=MIN(ncomp+offy2, nembed)-offy2-iy0;

	    long offx=(long)round(posx/dtheta);
	    long offx2=nembed/2+offx-ncomp2;
	    long ix0=MAX(-offx2, 0);
	    long nx2=MIN(ncomp+offx2, nembed)-offx2-ix0;
	    cuzero(otf, stream);
	    cwm_do<<<DIM2(nx2, ny2,16),0,stream>>>
		(otf->p+ix0+iy0*ncomp, 
		 cupowfs->pyramid->p[iwvl]->p+ix0+iy0*ncomp, 
		 wvf->p+ix0+offx2+(iy0+offy2)*nembed,
		 ncomp, nembed, nx2, ny2);
	    CUFFT(cuwfs->plan_py, otf->p, HIPFFT_BACKWARD);
	    curaddcabs2(&cuwfs->pypsf, 1, otf, alpha, stream);
	}
	embed_do<<<DIM(ncomp*ncomp, 256),0,stream>>>
	    (otf->p, cuwfs->pypsf->p, ncomp*ncomp);
	CUFFT(cuwfs->plan_py, otf->p, HIPFFT_FORWARD);
	cwm_do<<<DIM(ncomp*ncomp, 256), 0, stream>>>
	    (otf->p, cupowfs->pynominal->p, ncomp*ncomp);
	CUFFT(cuwfs->plan_py, otf->p, HIPFFT_BACKWARD);
	//Use ray tracing for si
	Real dx2=dx*nembed/ncomp;
	const int nsa=cupowfs->saloc->nloc;
	for(int iy=0; iy<2; iy++){
	    for(int ix=0; ix<2; ix++){
		Real* pout=ints->p+(ix+iy*2)*nsa;
		prop_linear<<<DIM(nsa, 256), 0, stream>>>
		    (pout, otf->p, otf->nx, otf->ny, 
		     cupowfs->saloc->p, cupowfs->saloc->nloc,
		     1./dx2, 1./dx2, 
		     (((ix-0.5)*ncomp2)-(-ncomp2+0.5)),
		     (((iy-0.5)*ncomp2)-(-ncomp2+0.5)), 
		     (Real)nsa/(ncomp*ncomp)*siglev);
	    }
	}
    }
}
dsp *gpu_pywfs_mkg(const PARMS_T *parms, const POWFS_T *powfs, loc_t *aloc, int iwfs, int idm){
    gpu_set(cudata_t::wfsgpu[iwfs]);
    cuwfs_t *cuwfs=cudata_t::wfs+iwfs;
    cupowfs_t *cupowfs=cuwfs->powfs;
    PYWFS_T *pywfs=cupowfs->pywfs;
    stream_t &stream=*cuwfs->stream;
    mapcell *dmrealsq=(mapcell*)cellnew(parms->ndm,1);
    dcell *dmreal=dcellnew(parms->ndm, 1);
    Real siglev=100;//irrelevant in noise free case.
    for(int i=0; i<parms->ndm; i++){
	dmrealsq->p[i]=mapnew2(aloc->map);
	dmreal->p[i]=dnew(aloc->nloc,1);
    }
    gpu_dmreal2gpu(dmrealsq, &parms->dm[idm]);
    Real poke=1e-6;
    Real poke1=1./poke;
    curmat *phiout=curnew(pywfs->locfft->loc->nloc,1);
    const int nsa=cupowfs->saloc->nloc;
    curmat *ints=curnew(nsa,4);
    curmat *grad=curnew(nsa*2,1);
    curmat *grad0=curnew(nsa*2,1);
    dmat *gradc=dnew(nsa*2,1);

    curmat *opd0=0;
    cp2gpu(&opd0, pywfs->atm);
    if(opd0) curadd(&phiout, 1, opd0, 1, stream);
    cuzero(ints, stream);
    pywfs_ints(ints, phiout, cuwfs, siglev, stream);
    pywfs_grad(grad0, ints, cupowfs->saa, cuwfs->isum, cupowfs->pyoff, pywfs->gain, stream);
    dsp *gg=dspnew(nsa*2, aloc->nloc, nsa*2*aloc->nloc);
    int count=0;
    TIC;tic;
    for(int iloc=0; iloc<aloc->nloc; iloc++){
	dmreal->p[idm]->p[iloc]=poke;
	if(iloc>0){
	    dmreal->p[idm]->p[iloc-1]=0;
	}
	loc_embed(dmrealsq->p[idm], aloc, dmreal->p[idm]->p);
	gpu_dmreal2gpu(dmrealsq, 0);
	if(opd0){
	    curcp(&phiout, opd0, stream);
	}else{
	    cuzero(phiout, stream);
	}
	int ipowfs=parms->wfs[iwfs].powfs;
	gpu_dm2loc(phiout->p, cuwfs->loc_dm, cudata->dmreal, cudata->ndm,
		   parms->powfs[ipowfs].hs, parms->wfs[iwfs].thetax, parms->wfs[iwfs].thetay, 
		   0, 0, 1, stream);
	cuzero(ints, stream);
	pywfs_ints(ints, phiout, cuwfs, siglev, stream);
	pywfs_grad(grad, ints, cupowfs->saa, cuwfs->isum, cupowfs->pyoff, pywfs->gain,stream);
	curadd(&grad, 1, grad0, -1, stream);
	dzero(gradc);
	cp2cpu(&gradc, grad, stream);
	gg->p[iloc]=count;
	const Real thres=dmaxabs(gradc)*EPS;
	for(int ig=0; ig<gradc->nx; ig++){
	    if(fabs(gradc->p[ig])>thres){
		gg->x[count]=gradc->p[ig]*poke1;
		gg->i[count]=ig;
		count++;
	    }
	}
	if(iloc%10==0){
	    Real ts=myclockd()-tk;
	    info2("%d of %ld. %.2f of %.2f seconds\n", iloc, aloc->nloc, ts, ts/(iloc+1)*aloc->nloc);
	}
    }
    gg->p[aloc->nloc]=count;
    dspsetnzmax(gg, count);
    cufree(grad0);
    cufree(opd0);
    cufree(grad);
    cufree(ints);
    dfree(gradc);
    cufree(phiout);
    cellfree(dmreal);
    cellfree(dmrealsq);
    return gg;
}
