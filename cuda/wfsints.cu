#include "hip/hip_runtime.h"
/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include "gpu.h"
}
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include "cucmat.h"
#include "wfs.h"
#include "cudata.h"
/*
  Timing results for TMT NFIRAOS case per LGS WFS:
  Embedding takes about 1 ms.
  FFT takes about 2 ms
  CCWM takes about 1ms
  realpart takes about 1ms

  Total takes about 12 ms.
*/

#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define ctoc(A)
#else
#define ctoc(A) CUDA_SYNC_STREAM; toc2(A)
#endif

/**
   Embed amp*exp(2*pi*i*opd). input is nxin*nxin, output is nxout*nxout;
*/
__global__ static void sa_embed_wvf_do(Comp *restrict wvf, 
				    const Real *restrict opd, const Real *restrict amp, 
				    const Real wvl, const int nxin, const int nxout){
    const int isa=blockIdx.x;
    const int pad=(nxout-nxin)>>1;
    const int skipin=isa*nxin*nxin;
    const int skipout=isa*nxout*nxout+pad;
    const Real pi2l=2.f*M_PI/wvl;
    for(int iy=threadIdx.y; iy<nxin; iy+=blockDim.y){
	const int skipin2=skipin+iy*nxin;
	const int skipout2=skipout+(iy+pad)*nxout;
	for(int ix=threadIdx.x; ix<nxin; ix+=blockDim.x){
	    /*test sinpi later */
	    Real s,c;
	    Z(sincos)(pi2l*opd[skipin2+ix], &s, &c);
	    wvf[skipout2+ix].x=amp[skipin2+ix]*c;
	    wvf[skipout2+ix].y=amp[skipin2+ix]*s;
	}
    }
}

/**
   Embed or crop an array to another array. Preserve corner.
*/
__global__ static void sa_cpcorner_do(Comp *restrict out, int noutx,  int nouty,
				   const Comp *restrict in, int ninx, int niny){
    int nx,ny;
    ny=MIN(niny, nouty)>>1;
    nx=MIN(ninx, noutx)>>1;
    const int isa=blockIdx.x;
    out+=isa*noutx*nouty;
    in+=isa*ninx*niny;
    for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    out[iy*noutx+ix]                   = in[iy*ninx+ix];
	    out[iy*noutx+(noutx-1-ix)]         = in[iy*ninx+(ninx-1-ix)];
	    out[(nouty-1-iy)*noutx+(noutx-1-ix)] = in[(niny-1-iy)*ninx+(ninx-1-ix)];
	    out[(nouty-1-iy)*noutx+(ix)]       = in[(niny-1-iy)*ninx+(ix)];
	}
    }
}

/**
   Embed or crop an array to another array. Preserve center. 
*/
__global__ void sa_cpcenter_do(Comp *restrict out, int noutx, int nouty,
			    const Comp *restrict in, int ninx, int niny, Real scale){
    int nx, ny, nskipoutx, nskipouty, nskipinx, nskipiny;
    if(noutx<ninx){
	nx=noutx;
	nskipoutx=0;
	nskipinx=(ninx-noutx)>>1;
    }else{
	nx=ninx;
	nskipoutx=(noutx-ninx)>>1;
	nskipinx=0;
    }
    if(nouty<niny){
	ny=nouty;
	nskipouty=0;
	nskipiny=(niny-nouty)>>1;
    }else{
	ny=niny;
	nskipouty=(nouty-niny)>>1;
	nskipiny=0;
    }
    const int isa=blockIdx.x;
    out+=isa*noutx*nouty+nskipouty*(noutx)+nskipoutx;
    in+=isa*ninx*niny+nskipiny*(ninx)+nskipinx;
    for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    out[iy*noutx+ix].x=scale*Z(hipCreal)(in[iy*ninx+ix]);
	    out[iy*noutx+ix].y=scale*Z(hipCimag)(in[iy*ninx+ix]);
	}
    }
}
/**
   abs2 to real.
*/
__global__ static void sa_abs2real_do(Comp *wvf, const int nx, Real alpha){
    const int isa=blockIdx.x;
    wvf+=nx*nx*isa;
    for(int iy=threadIdx.y; iy<nx; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    Real r=Z(hipCreal)(wvf[iy*nx+ix]);
	    Real i=Z(hipCimag)(wvf[iy*nx+ix]);
	    wvf[iy*nx+ix].x=(r*r+i*i)*alpha;
	    wvf[iy*nx+ix].y=0;
	}
    }
}
/**
   FFT Shift.
*/
__global__ static void sa_fftshift_do(Comp *wvf, const int nx, const int ny){
    const int isa=blockIdx.x;
    wvf+=nx*ny*isa;
    int nx2=nx>>1;
    int ny2=ny>>1;
    for(int iy=threadIdx.y; iy<ny2; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx2; ix+=blockDim.x){
	    Comp tmp;
	    tmp=wvf[ix+iy*nx];
	    wvf[ix+iy*nx]=wvf[(ix+nx2)+(iy+ny2)*nx];
	    wvf[(ix+nx2)+(iy+ny2)*nx]=tmp;
	    tmp=wvf[ix+(iy+ny2)*nx];
	    wvf[ix+(iy+ny2)*nx]=wvf[(ix+nx2)+iy*nx];
	    wvf[(ix+nx2)+iy*nx]=tmp;
	}
    }
}
/**
   FFT Shift from complex to real.
*/
__global__ static void sa_acc_real_fftshift_do(Real *restrict out, const Comp *restrict wvf, 
					    int nx, int ny, Real alpha){
    const int isa=blockIdx.x;
    wvf+=nx*ny*isa;
    out+=nx*ny*isa;
    int nx2=nx>>1;
    int ny2=ny>>1;
    for(int iy=threadIdx.y; iy<ny2; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx2; ix+=blockDim.x){
	    out[ix+iy*nx]+=alpha*Z(hipCreal)(wvf[(ix+nx2)+(iy+ny2)*nx]);
	    out[(ix+nx2)+(iy+ny2)*nx]+=alpha*Z(hipCreal)(wvf[ix+iy*nx]);
	    out[ix+(iy+ny2)*nx]+=alpha*Z(hipCreal)(wvf[(ix+nx2)+iy*nx]);
	    out[(ix+nx2)+iy*nx]+=alpha*Z(hipCreal)(wvf[ix+(iy+ny2)*nx]);
	}
    }
}
/**
   Rotate and embed.
 */
__global__ static void sa_embed_rot_do(Comp *restrict out, const int noutx, const int nouty,
				    const Comp *restrict in, const int ninx, const int niny, const Real* srot){
    const int isa=blockIdx.x;
    out+=isa*noutx*nouty;
    in+=isa*ninx*niny;
    Real theta=srot[isa];
    Real sx, cx;
    Z(sincos)(theta, &sx, &cx);
    const int niny2=niny/2;
    const int ninx2=ninx/2;
    const int nouty2=nouty/2;
    const int noutx2=noutx/2;

    for(int iy=threadIdx.y; iy<nouty; iy+=blockDim.y){
	int y0=iy-nouty2;
	for(int ix=threadIdx.x; ix<noutx; ix+=blockDim.x){
	    int x0=ix-noutx2;
	    Real x=(cx*x0-sx*y0)+ninx2;
	    Real y=(sx*x0+cx*y0)+niny2;
	    int jx=floorf(x); x=x-jx;
	    int jy=floorf(y); y=y-jy;
	    if(jx>=0 && jx<ninx-1 && jy>=0 && jy<niny-1){
		out[iy*noutx+ix].x=((Z(hipCreal)(in[jy*ninx+jx])*(1-x)
				     +Z(hipCreal)(in[jy*ninx+jx+1])*x)*(1-y)
				    +(Z(hipCreal)(in[(jy+1)*ninx+jx])*(1-x)
				      +Z(hipCreal)(in[(jy+1)*ninx+jx+1])*x)*y);
		out[iy*noutx+ix].y=0;
	    }
	}
    }
} 
/**
   Multiple each OTF with another. 
*/
__global__ static void sa_ccwm_do(Comp *otf, const int notfx, const int notfy, 
				  Comp **lotfcs, int each){
    const int isa=blockIdx.x;
    otf+=notfx*notfy*isa;
    const Comp *restrict lotfc=each?(Comp*)lotfcs:lotfcs[isa];
    for(int iy=threadIdx.y; iy<notfy; iy+=blockDim.y){
	const int skip=iy*notfx;
	Comp *restrict otf2=otf+skip;
	const Comp *restrict lotfc2=lotfc+skip; 
	for(int ix=threadIdx.x; ix<notfx; ix+=blockDim.x){
	    otf2[ix]=Z(hipCmul)(otf2[ix], lotfc2[ix]);
	}
    }
}
/**
   Multiple an otf with another 1-d otf along each column
*/
__global__ static void sa_ccwmcol_do(Comp *otf, const int notfx, const int notfy,
				  Comp *const *etfs, int each){
    const int isa=blockIdx.x;
    otf+=notfy*notfx*isa;
    const Comp *restrict etf=etfs[each?0:isa];
    for(int iy=threadIdx.y; iy<notfy; iy+=blockDim.y){
	Comp *restrict otf2=otf+iy*notfx;
	for(int ix=threadIdx.x; ix<notfx; ix+=blockDim.x){
	    otf2[ix]=Z(hipCmul)(otf2[ix], etf[ix]);
	}
    }
}
/**
   Take the real part and accumulate to output
*/
__global__ static void sa_acc_real_do(Real *out, const Comp*restrict in, int ninx, int niny, Real alpha){
    const int isa=blockIdx.x;
    in+=isa*ninx*niny;
    out+=isa*ninx*niny;
    for(int iy=threadIdx.y; iy<niny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<ninx; ix+=blockDim.x){
	    out[ix+iy*ninx]+=Z(hipCreal)(in[ix+iy*ninx])*alpha;
	}
    }
}
/**
   Do the role of si. input psfr is sampled with notfx*notfy, with sampling dtheta.
   Output ints is sampled with pixpsax*pixpsay, at pixtheta.
*/
__global__ static void sa_si_rot_do(Real *restrict ints, int pixpsax, int pixpsay, 
				 int pixoffx, int pixoffy, Real pixthetax, Real pixthetay,
				 const Comp *restrict otf, Real dtheta, int notfx, int notfy,
				 const Real *restrict srot, Real alpha){
    Real pxo=-(pixpsax*0.5-0.5+pixoffx)*pixthetax;
    Real pyo=-(pixpsay*0.5-0.5+pixoffy)*pixthetay;
    int isa=blockIdx.x;
    Real dispx=notfx/2;
    Real dispy=notfy/2;
    Real dtheta1=1.f/dtheta;
    Real sx=0, cx=1.f;
    if(srot){
	Z(sincos)(srot[isa], &sx, &cx);
    }
    ints+=isa*pixpsax*pixpsay;
    otf+=isa*notfx*notfy;
    for(int iy=threadIdx.y; iy<pixpsay; iy+=blockDim.y){
	Real y0=iy*pixthetay+pyo;
	for(int ix=threadIdx.x; ix<pixpsax; ix+=blockDim.x){
	    Real x0=ix*pixthetax+pxo;
	    Real x=(cx*x0-sx*y0)*dtheta1+dispx;
	    Real y=(sx*x0+cx*y0)*dtheta1+dispy;
	    int jx=Z(floor)(x); x=x-jx;
	    int jy=Z(floor)(y); y=y-jy;
	    if(jx>=0 && jx<notfx-1 && jy>=0 && jy<notfy-1){
		ints[iy*pixpsax+ix]+=alpha*((Z(hipCreal)(otf[jy*notfx+jx])*(1.-x)
					     +Z(hipCreal)(otf[jy*notfx+jx+1])*x)*(1.-y)
					    +(Z(hipCreal)(otf[(jy+1)*notfx+jx])*(1.-x)
					      +Z(hipCreal)(otf[(jy+1)*notfx+jx+1])*x)*y);
	    }
	}
    }
}

/**
   Add tip/tilt to the OTF for each subaps. exp(-2*pi*sx/nx)*exp(-2*pi*sy/ny).
   peak of otf is in corner.
 */
__global__ static void sa_add_otf_tilt_corner_do(Comp *restrict otf, int nx, int ny, 
					      Real *restrict gx, Real *restrict gy, Real gscale){
    int isa=blockIdx.x;
    Real sx=gx[isa]*gscale;
    Real sy=gy[isa]*gscale;
    Comp *restrict otfi=otf+isa*nx*ny;
    for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    Real phase=-2.f*M_PI*(+(Real)(ix-(ix*2>=nx?nx:0))/(Real)nx*sx
				   +(Real)(iy-(iy*2>=ny?ny:0))/(Real)ny*sy);
	    Real s,c;
	    Z(sincos)(phase, &s, &c);
	    Comp otfii=otfi[ix+iy*nx];
	    otfi[ix+iy*nx].x=Z(hipCreal)(otfii)*c - Z(hipCimag)(otfii)*s;
	    otfi[ix+iy*nx].y=Z(hipCreal)(otfii)*s + Z(hipCimag)(otfii)*c;
	}
    }
}
/**
   Do physical wfs images in GPU. please check wfsints() in CPU code for comments.
*/
void gpu_wfsints(SIM_T *simu, Real *phiout, curmat *gradref, int iwfs, int isim, hipStream_t stream){
    TIC;tic;
    cuwloc_t *cupowfs=cudata->powfs;
    cuwfs_t *cuwfs=cudata->wfs;
    const PARMS_T *parms=simu->parms;
    const POWFS_T *powfs=simu->powfs;
    const int ipowfs=parms->wfs[iwfs].powfs;
    const int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
    const Real hs=parms->powfs[ipowfs].hs;
    const Real dtisim=parms->sim.dt*isim;
    const int nsa=powfs[ipowfs].pts->nsa;
    const int ncompx=powfs[ipowfs].ncompx;/*necessary size to build detector image. */
    const int ncompy=powfs[ipowfs].ncompy;
    const int notf=MAX(ncompx,ncompy);
    const int nx=powfs[ipowfs].pts->nx;
    const int nwvf=nx*parms->powfs[ipowfs].embfac;
    const int pixpsax=powfs[ipowfs].pixpsax;
    const int pixpsay=powfs[ipowfs].pixpsay;
    const Real pixthetax=parms->powfs[ipowfs].radpixtheta;
    const Real pixthetay=parms->powfs[ipowfs].pixtheta;
    const Real siglev=parms->wfs[iwfs].siglevsim;
    const Real *restrict const srot1=parms->powfs[ipowfs].radrot?cuwfs[iwfs].srot:NULL;
    const int multi_dtf=(parms->powfs[ipowfs].llt&&!parms->powfs[ipowfs].radrot 
			 && parms->powfs[ipowfs].radpix);
    const Real *restrict const srot2=multi_dtf?cuwfs[iwfs].srot:NULL;
    const int nwvl=parms->powfs[ipowfs].nwvl;
    curcell *restrict const ints=cuwfs[iwfs].ints;
    curmat *lltopd=NULL;
    curcell *pistatout=NULL;
    if(parms->powfs[ipowfs].pistatout && isim>=parms->powfs[ipowfs].pistatstart){
	pistatout=cuwfs[iwfs].pistatout;
    }
    cuccell *wvfout=NULL;
    const int wvf_n=notf/2+2;//was notf/2
    if(parms->powfs[ipowfs].psfout){
	wvfout=cuwfs[iwfs].wvfout;
    }
    Real norm_psf=sqrt(powfs[ipowfs].areascale)/((Real)powfs[ipowfs].pts->nx*nwvf);
    Real norm_pistat=norm_psf*norm_psf/((Real)notf*notf);
    Real norm_ints=siglev*norm_psf*norm_psf/((Real)ncompx*ncompy);
    /* Do msa subapertures in a batch to avoid using too much memory.*/
    Comp *psf, *wvf, *otf, *psfstat=NULL;

    Comp *lotfc=NULL;
    Comp *lwvf=NULL;
    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
	int nlx=powfs[ipowfs].llt->pts->nx;
	lltopd=cuwfs[iwfs].lltopd;
	if(cuwfs[iwfs].lltncpa){
	    curcp(&lltopd, cuwfs[iwfs].lltncpa, stream);
	}else{
	    curzero(lltopd, stream);
	}
	const int illt=parms->powfs[ipowfs].llt->i[wfsind];
	const double thetaxl=parms->wfs[iwfs].thetax-parms->powfs[ipowfs].llt->ox[illt]/hs;
	const double thetayl=parms->wfs[iwfs].thetay-parms->powfs[ipowfs].llt->oy[illt]/hs;
	gpu_atm2loc(lltopd->p, cupowfs[ipowfs].llt->loc,
		    hs, thetaxl, thetayl, 
		    parms->powfs[ipowfs].llt->misreg[0], 
		    parms->powfs[ipowfs].llt->misreg[1], 
		    dtisim, 1, stream);
	Real ttx=0,tty=0;
	if((simu->uptreal && simu->uptreal->p[iwfs]) ||pistatout||parms->sim.uptideal){
	    if(pistatout||parms->sim.uptideal){
		//warning("Remove tip/tilt in uplink ideally\n");
		Real *lltg=cuwfs[iwfs].lltg;
		lltg[0]=lltg[1]=0;
		cuztilt(lltg, lltopd->p, 1, 
			cupowfs[ipowfs].llt->pts->dxsa, 
			cupowfs[ipowfs].llt->pts->nxsa, cuwfs[iwfs].lltimcc,
			cupowfs[ipowfs].llt->pts->p, cuwfs[iwfs].lltamp, 1.f, stream);
		CUDA_SYNC_STREAM;
		ttx=-lltg[0];
		tty=-lltg[1];
	    }else{
		ttx=simu->uptreal->p[iwfs]->p[0];
		tty=simu->uptreal->p[iwfs]->p[1];
	    }
	    /* copy uptreal to output  */
	    PDMAT(simu->uptcmds->p[iwfs], puptcmds);
	    puptcmds[isim][0]=ttx;
	    puptcmds[isim][1]=tty;
	}/*if uptreal */
	if(simu->telws){
	    Real tt=simu->telws->p[isim];
	    Real angle=simu->winddir?simu->winddir->p[0]:0;
	    ttx+=tt*cosf(angle)*parms->powfs[ipowfs].llt->ttrat;
	    tty+=tt*sinf(angle)*parms->powfs[ipowfs].llt->ttrat;
	}
	if(ttx!=0 && tty!=0){
	    /* add tip/tilt to opd  */
	    const double dx=powfs[ipowfs].llt->pts->dx;
	    const double ox=powfs[ipowfs].llt->pts->origx[0];
	    const double oy=powfs[ipowfs].llt->pts->origy[0];
	    add_tilt_do<<<1, dim3(16,16), 0, stream>>>(lltopd->p, nlx, nlx, ox, oy, dx, ttx, tty);
	}
	ctoc("llt opd");
	int nlwvf=nlx*parms->powfs[ipowfs].embfac;
	lwvf=cuwfs[iwfs].lltwvf->p;
	if(nlwvf != notf){
	    lotfc=cuwfs[iwfs].lltotfc->p;
	}else{
	    lotfc=lwvf;
	}
	if(parms->save.wfsopd[iwfs]){
	    cellarr_cur(simu->save->wfslltopd[iwfs], isim, lltopd, stream);
	}
    }/*if has llt */

    /* Now begin physical optics preparation*/
    int isotf=(lltopd || pistatout);
    int msa=cuwfs[iwfs].msa;/* number of subaps to process at each time.*/
    wvf=cuwfs[iwfs].wvf->p;
    if(nwvf==notf){
	psf=wvf;
    }else{
	psf=cuwfs[iwfs].psf->p;
    }
    if(srot1 || ncompx!=notf || ncompy!=notf){
	otf=cuwfs[iwfs].otf->p;
	if(isotf){/*There is an additional pair of FFT.*/
	    norm_ints/=((Real)notf*notf);
	}
    }else{
	otf=psf;
    }

    if(pistatout){
	psfstat=cuwfs[iwfs].psfstat->p;
    }
    /* Now begin physical optics  */
    for(int iwvl=0; iwvl<nwvl; iwvl++){
	Real wvl=parms->powfs[ipowfs].wvl[iwvl];
	Real dtheta=wvl/(nwvf*powfs[ipowfs].pts->dx);
	if(lltopd){ /*First calculate LOTF */
	    int nlx=powfs[ipowfs].llt->pts->nx;
	    int nlwvf=nlx*parms->powfs[ipowfs].embfac;
	    hipMemsetAsync(lwvf, 0, sizeof(Comp)*nlwvf*nlwvf, stream);
	    if(lotfc!=lwvf){
		hipMemsetAsync(lotfc, 0, sizeof(Comp)*notf*notf, stream);
	    }
	    sa_embed_wvf_do<<<1,dim3(16,16),0,stream>>>
		(lwvf, lltopd->p, cuwfs[iwfs].lltamp, wvl, nlx, nlwvf);
	    /*Turn to PSF. peak in corner */
	    CUFFT(cuwfs[iwfs].lltplan_wvf, lwvf, HIPFFT_FORWARD);
	    sa_abs2real_do<<<1,dim3(16,16),0,stream>>>(lwvf, nlwvf, 1./(Real)(nlwvf*nlwvf));
	    /*Turn to OTF. peak in corner*/
	    /*Use backward to make lotfc the conjugate of otf. peak is in corner. */
	    CUFFT(cuwfs[iwfs].lltplan_wvf, lwvf, HIPFFT_BACKWARD);
	    if(lwvf!=lotfc){
		sa_cpcorner_do<<<1, dim3(16,16),0,stream>>>(lotfc, notf, notf, lwvf, nlwvf, nlwvf);
	    }
	}
	ctoc("llt otf");

	for(int isa=0; isa<nsa; isa+=msa){
	    int ksa=MIN(msa, nsa-isa);/*total number of subapertures left to do. */
	    /*embed amp/opd to wvf */
	    hipMemsetAsync(wvf, 0, sizeof(Comp)*ksa*nwvf*nwvf, stream);
	    if(psf!=wvf){
		hipMemsetAsync(psf, 0, sizeof(Comp)*ksa*notf*notf, stream);
	    }
	    sa_embed_wvf_do<<<ksa, dim3(16,16),0,stream>>>
		(wvf, phiout+isa*nx*nx, cuwfs[iwfs].amp+isa*nx*nx, wvl, nx, nwvf);
	    ctoc("embed");
	    /* turn to complex psf, peak in corner */
	    CUFFT(cuwfs[iwfs].plan1, wvf, HIPFFT_FORWARD);
	    /* copy big psf to smaller psf covering detector focal plane. */
	    if(psf!=wvf){
		sa_cpcorner_do<<<ksa, dim3(16,16),0,stream>>>
		    (psf, notf, notf, wvf, nwvf, nwvf);
	    }
	    //gpu_write(psf, notf, notf*ksa, "psf_out_1");
	    ctoc("psf");
	    if(wvfout){
		curzero(cuwfs[iwfs].psfout, stream);
		CUFFT2(cuwfs[iwfs].plan2, psf, cuwfs[iwfs].psfout->p, HIPFFT_BACKWARD);
		sa_cpcenter_do<<<ksa,dim3(16,16),0,stream>>>
		    (wvfout->p[isa+nsa*iwvl]->p, wvf_n, wvf_n, 
		     cuwfs[iwfs].psfout->p, notf, notf, norm_psf/(notf*notf));
	    }
	    /* abs2 part to real, peak in corner */
	    sa_abs2real_do<<<ksa,dim3(16,16),0,stream>>>(psf, notf, 1);
	    ctoc("abs2real");
	    //gpu_write(psf, notf, notf*ksa, "psf_out_2");
	    if(isotf){
		/* turn to otf. peak in corner */
		CUFFT(cuwfs[iwfs].plan2, psf, HIPFFT_FORWARD);
		ctoc("fft to otf");
		if(pistatout){
		    hipMemcpyAsync(psfstat, psf, sizeof(Comp)*notf*notf*ksa, 
				    MEMCPY_D2D, stream);
		    if(parms->powfs[ipowfs].pistatout==1){
			sa_add_otf_tilt_corner_do<<<ksa,dim3(16,16),0,stream>>>
			    (psfstat, notf,notf, gradref->p+isa, gradref->p+nsa+isa, -1.f/dtheta);
		    }
		    CUFFT(cuwfs[iwfs].plan2, psfstat, HIPFFT_BACKWARD);/*back to PSF. peak in corner*/
		    if(parms->sim.skysim){/*want peak in corner*/
			sa_acc_real_do<<<ksa,dim3(16,16),0,stream>>>
			    (pistatout->p[isa+nsa*iwvl]->p, psfstat, notf, notf, norm_pistat);
		    }else{/*want peak in center*/
			sa_acc_real_fftshift_do<<<ksa,dim3(16,16),0,stream>>>
			    (pistatout->p[isa+nsa*iwvl]->p, psfstat, notf, notf, norm_pistat);
		    }
		}
		if(lltopd){/*multiply with uplink otf. */
		    sa_ccwm_do<<<ksa,dim3(16,16),0,stream>>>(psf, notf, notf, (Comp**)lotfc, 1);
		    ctoc("ccwm with lotfc");
		}
		/* is OTF now. */
	    }
	    //gpu_write(psf, notf, notf*ksa, "psf_out_3");
	    ctoc("before ints");
	    if(ints){
		if(!isotf || otf!=psf){/*rotate PSF, or turn to OTF first time. */
		    if(isotf){/*srot1 is true. turn otf back to psf for rotation. */
			CUFFT(cuwfs[iwfs].plan2, psf, HIPFFT_BACKWARD);
			ctoc("fft to psf");
		    }
		    if(otf!=psf){
			hipMemsetAsync(otf, 0, sizeof(Comp)*ksa*ncompx*ncompy, stream);
		    }
		    if(srot1){/*rotate and embed psf*/
			sa_fftshift_do<<<ksa, dim3(16,16),0,stream>>>
			    (psf, notf, notf);/*shift to center */
			sa_embed_rot_do<<<ksa, dim3(16,16), 0, stream>>>
			    (otf, ncompx, ncompy, psf, notf, notf, srot1?srot1+isa:NULL);
			sa_fftshift_do<<<ksa, dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy);/*shift back to corner */
		    }else if(otf!=psf){/*copy the psf corner*/
			sa_cpcorner_do<<<ksa, dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy, psf, notf, notf);
			ctoc("cpcorner");
		    }
		    /*Turn PSF to OTF. */
		    CUFFT(cuwfs[iwfs].plan3, otf,HIPFFT_FORWARD);
		    ctoc("fft to otf");
		}
		/*now we have otf. multiply with etf, dtf. */
		if(cuwfs[iwfs].dtf[iwvl].etf){
		    if(cuwfs[iwfs].dtf[iwvl].etfis1d){
			sa_ccwmcol_do<<<ksa,dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy, cuwfs[iwfs].dtf[iwvl].etf+isa, 0);
		    }else{
			ctoc("before ccwm");
			sa_ccwm_do<<<ksa,dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy, cuwfs[iwfs].dtf[iwvl].etf+isa, 0);
			ctoc("ccwm");
		    }
		}
		/*multiple with nominal */
		if(cuwfs[iwfs].dtf[iwvl].nominal){
		    sa_ccwm_do<<<ksa,dim3(16,16),0,stream>>>
			(otf, ncompx, ncompy, cuwfs[iwfs].dtf[iwvl].nominal+isa, 0);
		    ctoc("nominal");
		}
		/*back to spatial domain. */
		CUFFT(cuwfs[iwfs].plan3, otf, HIPFFT_BACKWARD);
		ctoc("fft");
		sa_si_rot_do<<<ksa, dim3(16,16),0,stream>>>
		    (ints->p[isa]->p, pixpsax, pixpsay, 
		     parms->powfs[ipowfs].pixoffx, parms->powfs[ipowfs].pixoffy,
		     pixthetax, pixthetay, otf, dtheta, ncompx, ncompy, srot2?srot2+isa:NULL, 
		     norm_ints*parms->wfs[iwfs].wvlwts[iwvl]);
		ctoc("final");
	    }/*if ints. */
	}/*for isa block loop */
    }/*for iwvl */
    if(parms->powfs[ipowfs].psfout){
	cellarr_cuccell(simu->save->wfspsfout[iwfs], isim, wvfout, stream);
    }
}
