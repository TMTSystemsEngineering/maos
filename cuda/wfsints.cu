#include "hip/hip_runtime.h"
/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipfft/hipfft.h>
#include "cucmat.h"
#include "wfs.h"
/*
  Timing results for TMT NFIRAOS case per LGS WFS:
  Embedding takes about 1 ms.
  FFT takes about 2 ms
  CCWM takes about 1ms
  realpart takes about 1ms

  Total takes about 12 ms.
*/

#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define ctoc(A)
#else
#define ctoc(A) CUDA_SYNC_STREAM; toc2(A)
#endif

/**
   Embed amp*exp(2*pi*i*opd). input is nxin*nxin, output is nxout*nxout;
*/
__global__ static void sa_embed_wvf_do(fcomplex *restrict wvf, 
				    const float *restrict opd, const float *restrict amp, 
				    const float wvl, const int nxin, const int nxout){
    const int isa=blockIdx.x;
    const int pad=(nxout-nxin)>>1;
    const int skipin=isa*nxin*nxin;
    const int skipout=isa*nxout*nxout+pad;
    const float pi2l=2.f*M_PI/wvl;
    for(int iy=threadIdx.y; iy<nxin; iy+=blockDim.y){
	const int skipin2=skipin+iy*nxin;
	const int skipout2=skipout+(iy+pad)*nxout;
	for(int ix=threadIdx.x; ix<nxin; ix+=blockDim.x){
	    /*test sinpi later */
	    float s,c;
	    sincosf(pi2l*opd[skipin2+ix], &s, &c);
	    wvf[skipout2+ix]=make_hipComplex(amp[skipin2+ix]*c, amp[skipin2+ix]*s);
	}
    }
}

/**
   Embed or crop an array to another array. Preserve corner.
*/
__global__ static void sa_cpcorner_do(fcomplex *restrict out, int noutx,  int nouty,
				   const fcomplex *restrict in, int ninx, int niny){
    int nx,ny;
    ny=MIN(niny, nouty)>>1;
    nx=MIN(ninx, noutx)>>1;
    const int isa=blockIdx.x;
    out+=isa*noutx*nouty;
    in+=isa*ninx*niny;
    for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    out[iy*noutx+ix]                   = in[iy*ninx+ix];
	    out[iy*noutx+(noutx-1-ix)]         = in[iy*ninx+(ninx-1-ix)];
	    out[(nouty-1-iy)*noutx+(noutx-1-ix)] = in[(niny-1-iy)*ninx+(ninx-1-ix)];
	    out[(nouty-1-iy)*noutx+(ix)]       = in[(niny-1-iy)*ninx+(ix)];
	}
    }
}

/**
   Embed or crop an array to another array. Preserve center. 
*/
__global__ void sa_cpcenter_do(fcomplex *restrict out, int noutx, int nouty,
			    const fcomplex *restrict in, int ninx, int niny, float scale){
    int nx, ny, nskipoutx, nskipouty, nskipinx, nskipiny;
    if(noutx<ninx){
	nx=noutx;
	nskipoutx=0;
	nskipinx=(ninx-noutx)>>1;
    }else{
	nx=ninx;
	nskipoutx=(noutx-ninx)>>1;
	nskipinx=0;
    }
    if(nouty<niny){
	ny=nouty;
	nskipouty=0;
	nskipiny=(niny-nouty)>>1;
    }else{
	ny=niny;
	nskipouty=(nouty-niny)>>1;
	nskipiny=0;
    }
    const int isa=blockIdx.x;
    out+=isa*noutx*nouty+nskipouty*(noutx)+nskipoutx;
    in+=isa*ninx*niny+nskipiny*(ninx)+nskipinx;
    for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    out[iy*noutx+ix]=make_hipFloatComplex(scale*hipCrealf(in[iy*ninx+ix]),scale*hipCimagf(in[iy*ninx+ix]));
	}
    }
}
/**
   abs2 to real.
*/
__global__ static void sa_abs2real_do(fcomplex *wvf, const int nx, float alpha){
    const int isa=blockIdx.x;
    wvf+=nx*nx*isa;
    for(int iy=threadIdx.y; iy<nx; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    float r=hipCrealf(wvf[iy*nx+ix]);
	    float i=hipCimagf(wvf[iy*nx+ix]);
	    wvf[iy*nx+ix]=make_hipFloatComplex((r*r+i*i)*alpha, 0);
	}
    }
}
/**
   FFT Shift.
*/
__global__ static void sa_fftshift_do(fcomplex *wvf, const int nx, const int ny){
    const int isa=blockIdx.x;
    wvf+=nx*ny*isa;
    int nx2=nx>>1;
    int ny2=ny>>1;
    for(int iy=threadIdx.y; iy<ny2; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx2; ix+=blockDim.x){
	    fcomplex tmp;
	    tmp=wvf[ix+iy*nx];
	    wvf[ix+iy*nx]=wvf[(ix+nx2)+(iy+ny2)*nx];
	    wvf[(ix+nx2)+(iy+ny2)*nx]=tmp;
	    tmp=wvf[ix+(iy+ny2)*nx];
	    wvf[ix+(iy+ny2)*nx]=wvf[(ix+nx2)+iy*nx];
	    wvf[(ix+nx2)+iy*nx]=tmp;
	}
    }
}
/**
   FFT Shift from complex to real.
*/
__global__ static void sa_acc_real_fftshift_do(float *restrict out, const fcomplex *restrict wvf, 
					    int nx, int ny, float alpha){
    const int isa=blockIdx.x;
    wvf+=nx*ny*isa;
    out+=nx*ny*isa;
    int nx2=nx>>1;
    int ny2=ny>>1;
    for(int iy=threadIdx.y; iy<ny2; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx2; ix+=blockDim.x){
	    out[ix+iy*nx]+=alpha*hipCrealf(wvf[(ix+nx2)+(iy+ny2)*nx]);
	    out[(ix+nx2)+(iy+ny2)*nx]+=alpha*hipCrealf(wvf[ix+iy*nx]);
	    out[ix+(iy+ny2)*nx]+=alpha*hipCrealf(wvf[(ix+nx2)+iy*nx]);
	    out[(ix+nx2)+iy*nx]+=alpha*hipCrealf(wvf[ix+(iy+ny2)*nx]);
	}
    }
}
/**
   Rotate and embed.
 */
__global__ static void sa_embed_rot_do(fcomplex *restrict out, const int noutx, const int nouty,
				    const fcomplex *restrict in, const int ninx, const int niny, const float* srot){
    const int isa=blockIdx.x;
    out+=isa*noutx*nouty;
    in+=isa*ninx*niny;
    float theta=srot[isa];
    float sx, cx;
    sincosf(theta, &sx, &cx);
    const int niny2=niny/2;
    const int ninx2=ninx/2;
    const int nouty2=nouty/2;
    const int noutx2=noutx/2;

    for(int iy=threadIdx.y; iy<nouty; iy+=blockDim.y){
	int y0=iy-nouty2;
	for(int ix=threadIdx.x; ix<noutx; ix+=blockDim.x){
	    int x0=ix-noutx2;
	    float x=(cx*x0-sx*y0)+ninx2;
	    float y=(sx*x0+cx*y0)+niny2;
	    int jx=floorf(x); x=x-jx;
	    int jy=floorf(y); y=y-jy;
	    if(jx>=0 && jx<ninx-1 && jy>=0 && jy<niny-1){
		out[iy*noutx+ix]=
		    make_hipFloatComplex((hipCrealf(in[jy*ninx+jx])*(1-x)
					 +hipCrealf(in[jy*ninx+jx+1])*x)*(1-y)
					+(hipCrealf(in[(jy+1)*ninx+jx])*(1-x)
					  +hipCrealf(in[(jy+1)*ninx+jx+1])*x)*y, 0);
	    }
	}
    }
} 
/**
   Multiple each OTF with another. 
*/
__global__ static void sa_ccwm_do(fcomplex *otf, const int notfx, const int notfy, 
			       fcomplex **lotfcs, int each){
    const int isa=blockIdx.x;
    otf+=notfx*notfy*isa;
    const fcomplex *restrict lotfc=each?(fcomplex*)lotfcs:lotfcs[isa];
    for(int iy=threadIdx.y; iy<notfy; iy+=blockDim.y){
	const int skip=iy*notfx;
	fcomplex *restrict otf2=otf+skip;
	const fcomplex *restrict lotfc2=lotfc+skip; 
	for(int ix=threadIdx.x; ix<notfx; ix+=blockDim.x){
	    otf2[ix]=hipCmulf(otf2[ix], lotfc2[ix]);
	}
    }
}
/**
   Multiple an otf with another 1-d otf along each column
*/
__global__ static void sa_ccwmcol_do(fcomplex *otf, const int notfx, const int notfy,
				  fcomplex *const *etfs, int each){
    const int isa=blockIdx.x;
    otf+=notfy*notfx*isa;
    const fcomplex *restrict etf=etfs[each?0:isa];
    for(int iy=threadIdx.y; iy<notfy; iy+=blockDim.y){
	fcomplex *restrict otf2=otf+iy*notfx;
	for(int ix=threadIdx.x; ix<notfx; ix+=blockDim.x){
	    otf2[ix]=hipCmulf(otf2[ix], etf[ix]);
	}
    }
}
/**
   Take the real part. Notice we are using = instead of +=
*/
__global__ static void sa_realpart_do(float *out, const fcomplex*restrict in, int ninx, int niny){
    const int isa=blockIdx.x;
    in+=isa*ninx*niny;
    out+=isa*ninx*niny;
    for(int iy=threadIdx.y; iy<niny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<ninx; ix+=blockDim.x){
	    out[ix+iy*ninx]=hipCrealf(in[ix+iy*ninx]);
	}
    }
}
/**
   Take the real part and accumulate to output
*/
__global__ static void sa_acc_real_do(float *out, const fcomplex*restrict in, int ninx, int niny, float alpha){
    const int isa=blockIdx.x;
    in+=isa*ninx*niny;
    out+=isa*ninx*niny;
    for(int iy=threadIdx.y; iy<niny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<ninx; ix+=blockDim.x){
	    out[ix+iy*ninx]+=hipCrealf(in[ix+iy*ninx])*alpha;
	}
    }
}
/**
   Do the role of si. input psfr is sampled with notfx*notfy, with sampling dtheta.
   Output ints is sampled with pixpsax*pixpsay, at pixtheta.
*/
__global__ static void sa_si_rot_do(float *restrict ints, int pixpsax, int pixpsay, 
				 int pixoffx, int pixoffy, float pixthetax, float pixthetay,
				 const float *restrict psfr, float dtheta, int notfx, int notfy,
				 const float *restrict srot, float alpha){
    float pxo=-(pixpsax*0.5-0.5+pixoffx)*pixthetax;
    float pyo=-(pixpsay*0.5-0.5+pixoffy)*pixthetay;
    int isa=blockIdx.x;
    float dispx=notfx/2;
    float dispy=notfy/2;
    float dtheta1=1.f/dtheta;
    float sx=0, cx=1.f;
    if(srot){
	sincosf(srot[isa], &sx, &cx);
    }
    ints+=isa*pixpsax*pixpsay;
    psfr+=isa*notfx*notfy;
    for(int iy=threadIdx.y; iy<pixpsay; iy+=blockDim.y){
	float y0=iy*pixthetay+pyo;
	for(int ix=threadIdx.x; ix<pixpsax; ix+=blockDim.x){
	    float x0=ix*pixthetax+pxo;
	    float x=(cx*x0-sx*y0)*dtheta1+dispx;
	    float y=(sx*x0+cx*y0)*dtheta1+dispy;
	    int jx=floorf(x); x=x-jx;
	    int jy=floorf(y); y=y-jy;
	    if(jx>=0 && jx<notfx-1 && jy>=0 && jy<notfy-1){
		ints[iy*pixpsax+ix]+=alpha*((psfr[jy*notfx+jx]*(1.-x)+psfr[jy*notfx+jx+1]*x)*(1.-y)
					    +(psfr[(jy+1)*notfx+jx]*(1.-x)+psfr[(jy+1)*notfx+jx+1]*x)*y);
	    }
	}
    }
}

/**
   Add tip/tilt to the OTF for each subaps. exp(-2*pi*sx/nx)*exp(-2*pi*sy/ny).
   peak of otf is in corner.
 */
__global__ static void sa_add_otf_tilt_corner_do(fcomplex *restrict otf, int nx, int ny, 
					      float *restrict gx, float *restrict gy, float gscale){
    int isa=blockIdx.x;
    float sx=gx[isa]*gscale;
    float sy=gy[isa]*gscale;
    fcomplex *restrict otfi=otf+isa*nx*ny;
    for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    float phase=-2.f*M_PI*(+(float)(ix-(ix*2>=nx?nx:0))/(float)nx*sx
				   +(float)(iy-(iy*2>=ny?ny:0))/(float)ny*sy);
	    float s,c;
	    sincosf(phase, &s, &c);
	    fcomplex otfii=otfi[ix+iy*nx];
	    otfi[ix+iy*nx]=make_hipFloatComplex(hipCrealf(otfii)*c - hipCimagf(otfii)*s,
					       hipCrealf(otfii)*s + hipCimagf(otfii)*c);
	}
    }
}
/**
   Do physical wfs images in GPU. please check wfsints() in CPU code for comments.
*/
void gpu_wfsints(SIM_T *simu, float *phiout, curmat *gradref, int iwfs, int isim, hipStream_t stream){
    TIC;tic;
    cuwloc_t *cupowfs=cudata->powfs;
    cuwfs_t *cuwfs=cudata->wfs;
    const PARMS_T *parms=simu->parms;
    const POWFS_T *powfs=simu->powfs;
    const int ipowfs=parms->wfs[iwfs].powfs;
    const int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
    const float hs=parms->powfs[ipowfs].hs;
    const float dtisim=parms->sim.dt*isim;
    const int nsa=powfs[ipowfs].pts->nsa;
    const int ncompx=powfs[ipowfs].ncompx;/*necessary size to build detector image. */
    const int ncompy=powfs[ipowfs].ncompy;
    const int notf=MAX(ncompx,ncompy);
    const int nx=powfs[ipowfs].pts->nx;
    const int nwvf=nx*parms->powfs[ipowfs].embfac;
    const int pixpsax=powfs[ipowfs].pixpsax;
    const int pixpsay=powfs[ipowfs].pixpsay;
    const float pixthetax=parms->powfs[ipowfs].radpixtheta;
    const float pixthetay=parms->powfs[ipowfs].pixtheta;
    const float siglev=parms->wfs[iwfs].siglevsim;
    const float *restrict const srot1=parms->powfs[ipowfs].radrot?cuwfs[iwfs].srot:NULL;
    const int multi_dtf=(parms->powfs[ipowfs].llt&&!parms->powfs[ipowfs].radrot 
			 && parms->powfs[ipowfs].radpix);
    const float *restrict const srot2=multi_dtf?cuwfs[iwfs].srot:NULL;
    const int nwvl=parms->powfs[ipowfs].nwvl;
    curcell *restrict const ints=cuwfs[iwfs].ints;
    curmat *lltopd=NULL;
    curcell *pistatout=NULL;
    if(parms->powfs[ipowfs].pistatout && isim>=parms->powfs[ipowfs].pistatstart){
	pistatout=cuwfs[iwfs].pistatout;
    }
    cuccell *wvfout=NULL;
    const int wvf_n=notf/2+2;//was notf/2
    if(parms->powfs[ipowfs].psfout){
	wvfout=cuccellnew(nsa, nwvl, wvf_n, wvf_n);
    }
    float norm_psf=sqrt(powfs[ipowfs].areascale)/((float)powfs[ipowfs].pts->nx*nwvf);
    float norm_pistat=norm_psf*norm_psf/((float)notf*notf);
    float norm_ints=siglev*norm_psf*norm_psf/((float)ncompx*ncompy);
    /* Do msa subapertures in a batch to avoid using too much memory.*/
    fcomplex *psf, *wvf, *otf, *psfout=NULL, *psfstat=NULL;

    fcomplex *lotfc=NULL;
    fcomplex *lwvf=NULL;

    if(powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
	int nlx=powfs[ipowfs].llt->pts->nx;
	lltopd=curnew(nlx, nlx);
	if(cuwfs[iwfs].lltncpa){
	    hipMemcpyAsync(lltopd->p, cuwfs[iwfs].lltncpa, 
			    sizeof(float)*nlx*nlx, MEMCPY_D2D, stream);
	}else{
	    hipMemsetAsync(lltopd->p, 0, sizeof(float)*nlx*nlx, stream);
	}
	const int illt=parms->powfs[ipowfs].llt->i[wfsind];
	const double thetaxl=parms->wfs[iwfs].thetax-parms->powfs[ipowfs].llt->ox[illt]/hs;
	const double thetayl=parms->wfs[iwfs].thetay-parms->powfs[ipowfs].llt->oy[illt]/hs;
	gpu_atm2loc(lltopd->p, cupowfs[ipowfs].llt->loc, cupowfs[ipowfs].llt->nloc, 
		    hs, thetaxl, thetayl, 
		    parms->powfs[ipowfs].llt->misreg[0], 
		    parms->powfs[ipowfs].llt->misreg[1], 
		    dtisim, 1, stream);
	if((simu->uptreal && simu->uptreal->p[iwfs]) ||pistatout||parms->sim.uptideal){
	    float ttx,tty;
	    if(pistatout||parms->sim.uptideal){
		//warning("Remove tip/tilt in uplink ideally\n");
		float *lltg;
		cudaCallocHost(lltg, 2*sizeof(float), stream);
		cuztilt<<<1,dim3(16,16),0,stream>>>(lltg, lltopd->p, 1, cupowfs[ipowfs].llt->dx, 
						    cupowfs[ipowfs].llt->nxsa, cuwfs[iwfs].lltimcc,
						    cupowfs[ipowfs].llt->pts, cuwfs[iwfs].lltamp, 1.f);
		CUDA_SYNC_STREAM;
		ttx=-lltg[0];
		tty=-lltg[1];
		hipHostFree(lltg);
	    }else{
		ttx=simu->uptreal->p[iwfs]->p[0];
		tty=simu->uptreal->p[iwfs]->p[1];
	    }
	    /* copy uptreal to output  */
	    PDMAT(simu->uptcmds->p[iwfs], puptcmds);
	    puptcmds[isim][0]=ttx;
	    puptcmds[isim][1]=tty;
	    /* add tip/tilt to opd  */
	    const double dx=powfs[ipowfs].llt->pts->dx;
	    const double ox=powfs[ipowfs].llt->pts->origx[0];
	    const double oy=powfs[ipowfs].llt->pts->origy[0];
	    add_tilt_do<<<1, dim3(16,16), 0, stream>>>(lltopd->p, nlx, nlx, ox, oy, dx, ttx, tty);
	}/*if upt */
	ctoc("llt opd");
	int nlwvf=nlx*parms->powfs[ipowfs].embfac;
	hipMalloc(&lwvf, nlwvf*nlwvf*sizeof(fcomplex));
	if(nlwvf != notf){
	    hipMalloc(&lotfc, notf*notf*sizeof(fcomplex));
	}else{
	    lotfc=lwvf;
	}
	if(parms->save.wfsopd[iwfs]){
	    cellarr_cur(simu->save->wfslltopd[iwfs], isim, lltopd, stream);
	}
    }/*if has llt */

    /* Now begin physical optics preparation*/
    int isotf=(lltopd || pistatout);
    int msa=cuwfs[iwfs].msa;/* number of subaps to process at each time.*/
    hipMalloc(&wvf, sizeof(fcomplex)*nwvf*nwvf*msa);
    if(nwvf==notf){
	psf=wvf;
    }else{
	hipMalloc(&psf, sizeof(fcomplex)*notf*notf*msa);
    }
    if(srot1 || ncompx!=notf || ncompy!=notf){
	hipMalloc(&otf, sizeof(fcomplex)*ncompx*ncompy*msa);
	if(isotf){/*There is an additional pair of FFT.*/
	    norm_ints/=((float)notf*notf);
	}
    }else{
	otf=psf;
    }
    if(wvfout){
	hipMalloc(&psfout, sizeof(fcomplex)*notf*notf*msa);
    }
    if(pistatout){
	hipMalloc(&psfstat, sizeof(fcomplex)*notf*notf*msa);
    }
    /* Now begin physical optics  */
    for(int iwvl=0; iwvl<nwvl; iwvl++){
	float wvl=parms->powfs[ipowfs].wvl[iwvl];
	float dtheta=wvl/(nwvf*powfs[ipowfs].pts->dx);
	if(lltopd){ /*First calculate LOTF */
	    int nlx=powfs[ipowfs].llt->pts->nx;
	    int nlwvf=nlx*parms->powfs[ipowfs].embfac;
	    hipMemsetAsync(lwvf, 0, sizeof(fcomplex)*nlwvf*nlwvf, stream);
	    if(nlwvf<notf){
		hipMemsetAsync(lotfc, 0, sizeof(fcomplex)*notf*notf, stream);
	    }
	    sa_embed_wvf_do<<<1,dim3(16,16),0,stream>>>
		(lwvf, lltopd->p, cuwfs[iwfs].lltamp, wvl, nlx, nlwvf);
	    /*Turn to PSF. peak in corner */
	    CUFFT(cuwfs[iwfs].lltplan_wvf, lwvf, HIPFFT_FORWARD);
	    sa_abs2real_do<<<1,dim3(16,16),0,stream>>>(lwvf, nlwvf, 1./(float)(nlwvf*nlwvf));
	    /*Turn to OTF. peak in corner*/
	    /*Use backward to make lotfc the conjugate of otf. peak is in corner. */
	    CUFFT(cuwfs[iwfs].lltplan_wvf, lwvf, HIPFFT_BACKWARD);
	    if(lwvf!=lotfc){
		sa_cpcorner_do<<<1, dim3(16,16),0,stream>>>(lotfc, notf, notf, lwvf, nlwvf, nlwvf);
	    }
	}
	ctoc("llt otf");

	for(int isa=0; isa<nsa; isa+=msa){
	    int ksa=MIN(msa, nsa-isa);/*total number of subapertures left to do. */
	    /*embed amp/opd to wvf */
	    hipMemsetAsync(wvf, 0, sizeof(fcomplex)*ksa*nwvf*nwvf, stream);
	    if(notf>nwvf){
		hipMemsetAsync(psf, 0, sizeof(fcomplex)*ksa*notf*notf, stream);
	    }
	    sa_embed_wvf_do<<<ksa, dim3(16,16),0,stream>>>
		(wvf, phiout+isa*nx*nx, cuwfs[iwfs].amp+isa*nx*nx, wvl, nx, nwvf);
	    ctoc("embed");
	    /* turn to complex psf, peak in corner */
	    CUFFT(cuwfs[iwfs].plan1, wvf, HIPFFT_FORWARD);
	    /* copy big psf to smaller psf covering detector focal plane. */
	    if(psf!=wvf){
		sa_cpcorner_do<<<ksa, dim3(16,16),0,stream>>>
		    (psf, notf, notf, wvf, nwvf, nwvf);
	    }
	    //gpu_write(psf, notf, notf*ksa, "psf_out_1");
	    ctoc("psf");
	    if(wvfout){
		hipMemsetAsync(psfout, 0, sizeof(fcomplex)*ksa*notf*notf, stream);
		CUFFT2(cuwfs[iwfs].plan2, psf, psfout, HIPFFT_BACKWARD);
		sa_cpcenter_do<<<ksa,dim3(16,16),0,stream>>>
		    (wvfout->p[isa+nsa*iwvl]->p, wvf_n, wvf_n, 
		     psfout, notf, notf, norm_psf/(notf*notf));
	    }
	    /* abs2 part to real, peak in corner */
	    sa_abs2real_do<<<ksa,dim3(16,16),0,stream>>>(psf, notf, 1);
	    ctoc("abs2real");
	    //gpu_write(psf, notf, notf*ksa, "psf_out_2");
	    if(isotf){
		/* turn to otf. peak in corner */
		CUFFT(cuwfs[iwfs].plan2, psf, HIPFFT_FORWARD);
		ctoc("fft to otf");
		if(pistatout){
		    hipMemcpyAsync(psfstat, psf, sizeof(fcomplex)*notf*notf*ksa, 
				    MEMCPY_D2D, stream);
		    sa_add_otf_tilt_corner_do<<<ksa,dim3(16,16),0,stream>>>
			(psfstat, notf,notf, gradref->p+isa, gradref->p+nsa+isa, -1.f/dtheta);
		    CUFFT(cuwfs[iwfs].plan2, psfstat, HIPFFT_BACKWARD);/*back to PSF. peak in corner*/
		    if(parms->sim.skysim){/*want peak in corner*/
			sa_acc_real_do<<<ksa,dim3(16,16),0,stream>>>
			    (pistatout->p[isa+nsa*iwvl]->p, psfstat, notf, notf, norm_pistat);
		    }else{/*want peak in center*/
			sa_acc_real_fftshift_do<<<ksa,dim3(16,16),0,stream>>>
			    (pistatout->p[isa+nsa*iwvl]->p, psfstat, notf, notf, norm_pistat);
		    }
		}
		if(lltopd){/*multiply with uplink otf. */
		    sa_ccwm_do<<<ksa,dim3(16,16),0,stream>>>(psf, notf, notf, (fcomplex**)lotfc, 1);
		    ctoc("ccwm with lotfc");
		}
		/* is OTF now. */
	    }
	    //gpu_write(psf, notf, notf*ksa, "psf_out_3");
	    ctoc("before ints");
	    if(ints){
		if(!isotf || otf!=psf){/*rotate PSF, or turn to OTF first time. */
		    if(isotf){/*srot1 is true. turn otf back to psf for rotation. */
			CUFFT(cuwfs[iwfs].plan2, psf, HIPFFT_BACKWARD);
			ctoc("fft to psf");
		    }
		    if(otf!=psf){
			hipMemsetAsync(otf, 0, sizeof(fcomplex)*ksa*ncompx*ncompy, stream);
		    }
		    if(srot1){/*rotate and embed psf*/
			sa_fftshift_do<<<ksa, dim3(16,16),0,stream>>>
			    (psf, notf, notf);/*shift to center */
			sa_embed_rot_do<<<ksa, dim3(16,16), 0, stream>>>
			    (otf, ncompx, ncompy, psf, notf, notf, srot1?srot1+isa:NULL);
			sa_fftshift_do<<<ksa, dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy);/*shift back to corner */
		    }else if(otf!=psf){/*copy the psf corner*/
			sa_cpcorner_do<<<ksa, dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy, psf, notf, notf);
			ctoc("cpcorner");
		    }
		    /*Turn PSF to OTF. */
		    CUFFT(cuwfs[iwfs].plan3, otf,HIPFFT_FORWARD);
		    ctoc("fft to otf");
		}
		/*now we have otf. multiple with etf, dtf. */
		if(cuwfs[iwfs].dtf[iwvl].etf){
		    if(cuwfs[iwfs].dtf[iwvl].etfis1d){
			sa_ccwmcol_do<<<ksa,dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy, cuwfs[iwfs].dtf[iwvl].etf+isa, 0);
		    }else{
			ctoc("before ccwm");
			sa_ccwm_do<<<ksa,dim3(16,16),0,stream>>>
			    (otf, ncompx, ncompy, cuwfs[iwfs].dtf[iwvl].etf+isa, 0);
			ctoc("ccwm");
		    }
		}
		/*multiple with nominal */
		if(cuwfs[iwfs].dtf[iwvl].nominal){
		    sa_ccwm_do<<<ksa,dim3(16,16),0,stream>>>
			(otf, ncompx, ncompy, cuwfs[iwfs].dtf[iwvl].nominal+isa, 0);
		    ctoc("nominal");
		}
		/*back to spatial domain. */
		CUFFT(cuwfs[iwfs].plan3, otf, HIPFFT_BACKWARD);
		ctoc("fft");
		float *psfr;
		DO(hipMalloc(&psfr, sizeof(float)*ncompx*ncompy*ksa));
		sa_realpart_do<<<ksa, dim3(16,16),0,stream>>>(psfr, otf, ncompx, ncompy);
		ctoc("realpart");
		sa_si_rot_do<<<ksa, dim3(16,16),0,stream>>>
		    (ints->p[isa]->p, pixpsax, pixpsay, 
		     parms->powfs[ipowfs].pixoffx, parms->powfs[ipowfs].pixoffy,
		     pixthetax, pixthetay, psfr, dtheta, ncompx, ncompy, srot2?srot2+isa:NULL, 
		     norm_ints*parms->wfs[iwfs].wvlwts[iwvl]);
		ctoc("final");
		DO(hipFree(psfr));
	    }/*if ints. */
	}/*for isa block loop */
    }/*for iwvl */
    if(lltopd){
	curfree(lltopd);
	if(lwvf!=lotfc) hipFree(lotfc);
	hipFree(lwvf);
    }
    if(otf!=psf) hipFree(otf);
    if(psf!=wvf) hipFree(psf);
    hipFree(wvf);
    if(psfstat)  hipFree(psfstat);
    if(parms->powfs[ipowfs].psfout){
	cellarr_cuccell(simu->save->wfspsfout[iwfs], isim, wvfout, stream);
	cuccellfree(wvfout);
	hipFree(psfout);
    }
}
