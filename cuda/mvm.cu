#include "hip/hip_runtime.h"
/*
  Copyright 2009-2012 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "utils.h"
#include "curmat.h"
#include "cucmat.h"
extern "C"{
#include "../lib/sys/scheduler_server.h"
#include "../maos/mvm_client.h"
#include <sys/file.h>
#include <netinet/tcp.h> /*SOL_TCP */
#include <netinet/in.h>
#include <errno.h>
}

/*Read into double array until we get all*/
#define READ_ARR(p,n,type)					\
    nleft=(n)*sizeof(type);					\
    start=(char*)(p);						\
    do{								\
	int nread=read(sock_mvm, start, nleft);			\
	nleft-=nread;						\
	start+=nread;						\
	if(nread<=0){						\
	    warning2("nread=%d, nleft=%d\n", nread, nleft);	\
	    return -1;						\
	}							\
    }while(nleft>0);

#define WRITE_ARR(p,n,type)						\
    if((ndone=write(sock_mvm, p, sizeof(type)*(n)))!=sizeof(type)*(n)){	\
	perror("write");						\
	warning("error writing. want %ld wrote %d\n",(n)*sizeof(type), ndone); \
	return -1;							\
    }

#define READ_CMD(p) READ_ARR(p,N_CMD,int)
#define WRITE_CMD(p) WRITE_ARR(p,N_CMD,int)



/**
 * 2012-06-22: Serious bug found:
 * Initially was using curnew. But later replaced by new while should be using hipMalloc
 * This caused misterious bugs, such as no answer or even kernel panic.
 */
pthread_t thread_init;
typedef struct mvm_g_mul_t{
    int nact;
    int ngtot;
    int icol;
    int ngpu;
    int k;
    GTYPE *g;/*full g*/
    ATYPE **ac;/*a for each gpu.*/
    ATYPE *a;/*final a*/
    pthread_mutex_t mutex;
}mvm_g_mul_t;

typedef struct mvm_t{
    mvm_g_mul_t data;
    thread_t *mvm_g_mul;
    thread_t *mvm_a_cp;
    thread_t *mvm_a_sum;
    thread_t *mvm_a_cp_sum;
}mvm_t;
mvm_t *mvm_data=NULL;

int mp_count;/*number multi-processors on each GPU*/
int mp_core;/*number of cuda cores per multi-processor*/

int ndone;
int nleft;
char *start;

int sock_mvm;

__global__ static void mvm_g_mul_do(float *restrict mvm, ATYPE *restrict a, const GTYPE *restrict g, int nact, int ng){
    extern __shared__ float acc[];
    int iact=threadIdx.x+blockIdx.x*blockDim.x;
    if(iact<nact){
	acc[threadIdx.x]=0;
	for(int ig=0; ig<ng; ig++){
	    register float mvmi=mvm[nact*ig+iact];
	    acc[threadIdx.x]+=mvmi*(float)(g[ig]);
	}
	a[iact]+=(ATYPE)acc[threadIdx.x];
    }
}
/* multiply mvm against g for each GPU. each gpu get an equal slice of the g.*/
static void mvm_g_mul(thread_t *info){
    int igpu=info->ithread;
    gpu_set(igpu);
    mvm_g_mul_t *data=(mvm_g_mul_t *)info->data;
    int ki=(data->k+NGPU-1)/NGPU;
    int icol=data->icol+ki*igpu;
    int ki2=data->k+data->icol-icol;
    int k=MIN(ki, ki2);
    int m=cudata->mvm_m->nx;

    hipMemcpyAsync(cudata->mvm_g+icol, data->g+icol, k*sizeof(GTYPE), 
		    hipMemcpyHostToDevice, *cudata->mvm_stream);
    int neach=(m+mp_count-1)/mp_count;
    mvm_g_mul_do<<<mp_count, neach, sizeof(float)*neach, *cudata->mvm_stream>>>
	(cudata->mvm_m->p+m*icol, cudata->mvm_a, cudata->mvm_g+icol, m, k);
}
/*
  copy data from each gpu to cpu and clear accumulation. 
  Tested adding act commands in each thread after copying with a mutex lock. Slower.
*/
static void mvm_a_cp(thread_t *info){
    int igpu=info->ithread;
    gpu_set(igpu);
    mvm_g_mul_t *data=(mvm_g_mul_t *)info->data;
    int nact=data->nact;
    hipMemcpyAsync(data->ac[igpu], cudata->mvm_a, nact*sizeof(ATYPE),
		    hipMemcpyDeviceToHost, *cudata->mvm_stream);
    hipStreamSynchronize(*cudata->mvm_stream);
    hipMemsetAsync(cudata->mvm_a, 0, nact*sizeof(ATYPE), *cudata->mvm_stream);
}
/*sum the DM commands from different GPUs together.*/
static void mvm_a_sum(thread_t *info){
    mvm_g_mul_t *data=(mvm_g_mul_t *)info->data;
    ATYPE *restrict pout=data->a;
    for(int igpu=0; igpu<NGPU; igpu++){
	const ATYPE *restrict pin=data->ac[igpu];
	for(int i=info->start; i<info->end; i++){
	    pout[i]+=pin[i];
	}
    }
}
static void mvm_copy_m(thread_t *info){
    smat *mvm=(smat *)info->data;
    int igpu=info->ithread;
    gpu_set(igpu);
    pthread_mutex_init(&cudata->mvm_mutex, NULL);
    cudata->mvm_stream=new stream_t;
    cp2gpu(&cudata->mvm_m, mvm, *cudata->mvm_stream);
    if(cudata->mvm_a){
	hipFree(cudata->mvm_a); cudata->mvm_a=NULL;
	hipFree(cudata->mvm_g); cudata->mvm_g=NULL;
    }
    hipMalloc(&cudata->mvm_a, mvm->nx*sizeof(ATYPE));
    hipMalloc(&cudata->mvm_g, mvm->ny*sizeof(GTYPE));
    hipMemsetAsync(cudata->mvm_a, 0, mvm->nx*sizeof(ATYPE), *cudata->mvm_stream);
    hipStreamSynchronize(*cudata->mvm_stream);
    if(cudata->mvm_a2){
	free(cudata->mvm_a2);
    }
    cudata->mvm_a2=(ATYPE**)calloc(NGPU, sizeof(ATYPE*));
}
static void mvm_data_free(void){
    hipHostFree(mvm_data->data.g);
    hipHostFree(mvm_data->data.a);
    for(int i=0; i<mvm_data->data.ngpu; i++){
	hipHostFree(mvm_data->data.ac[i]);
    }
    free(mvm_data->data.ac);
    free(mvm_data->mvm_g_mul);
    free(mvm_data->mvm_a_cp);
    free(mvm_data->mvm_a_sum);
    free(mvm_data->mvm_a_cp_sum);
    free(mvm_data);
}
double tim_cmd=0, tim_gsend=0, tim_gcp=0, tim_dmcp=0, tim_queue=0, tim_dmsum=0, tim_dmsend=0;
static int respond(int sock){
    TIC;tic;
    sock_mvm=sock;
    int cmd[N_CMD];
    READ_CMD(cmd);
    static int ksave=0;
    static double tim_gfirst=0;
    switch(cmd[0]){
    case GPU_MVM_M:{/*maos sends M matrix*/
	int nact=cmd[2];
	int ngtot=cmd[3];
	info("Receiving mvm %dx%d\n", nact, ngtot);
	smat *mvm=snew(nact, ngtot);
	READ_ARR(mvm->p, (nact*ngtot), float);
	pthread_join(thread_init, NULL);
	toc22("Read mvm");tic;
	thread_t info[NGPU];
	thread_prep(info, 0, NGPU, NGPU, mvm_copy_m, mvm);
	CALL_THREAD(info, NGPU, 0);
	toc22("copy mvm to gpu");
	if(mvm_data){
	    mvm_data_free();
	}
	mvm_data=(mvm_t*)calloc(1, sizeof(mvm_t));
	mvm_data->data.nact=mvm->nx;
	mvm_data->data.ngtot=mvm->ny;
	mvm_data->data.ngpu=NGPU;
	mvm_data->data.ac=new ATYPE*[NGPU];
	hipHostMalloc(&mvm_data->data.g, sizeof(GTYPE)*ngtot);
	hipHostMalloc(&mvm_data->data.a, sizeof(ATYPE)*nact);
	memset(mvm_data->data.a, 0., nact*sizeof(ATYPE));
	for(int ig=0; ig<NGPU; ig++){
	    hipHostMalloc(&mvm_data->data.ac[ig], sizeof(ATYPE)*nact);
	}
	pthread_mutex_init(&mvm_data->data.mutex, NULL);
	mvm_data->mvm_g_mul=new thread_t[NGPU];
	thread_prep(mvm_data->mvm_g_mul, 0, ngtot, NGPU, mvm_g_mul, &mvm_data->data);
	mvm_data->mvm_a_cp=new thread_t[NGPU];
	mvm_data->mvm_a_sum=new thread_t[NCPU];
	thread_prep(mvm_data->mvm_a_cp, 0, NGPU, NGPU, mvm_a_cp, &mvm_data->data);
	thread_prep(mvm_data->mvm_a_sum, 0, nact, NCPU, mvm_a_sum, &mvm_data->data);
	info2("done");
	sfree(mvm);
    }
	break;
    case GPU_MVM_G:{/*maos sends gradients*/
	int icol=cmd[1];/*starting column*/
	assert(icol==0);
	int ngeach=cmd[2];
	tim_cmd+=toc3; tic;
	int ngtot=mvm_data->data.ngtot;
	int nact=mvm_data->data.nact;
	tim_gfirst=myclockd();
	for(int icol=cmd[1]; icol<ngtot; icol+=ngeach){
	    int k=MIN(ngeach, ngtot-icol);
	    READ_ARR(mvm_data->data.g+icol, k, GTYPE);
	    tim_gsend+=toc3;tic;
	    if(cmd[2]<1800){//part of grads
		gpu_set(gpu_next());//use next GPUs
		hipMemcpyAsync(cudata->mvm_g+icol, mvm_data->data.g+icol, k*sizeof(GTYPE), 
				hipMemcpyHostToDevice, cudata->mvm_stream[0]);
		tim_gcp+=toc3; tic;
		int naeach=(nact+mp_count-1)/mp_count;
		mvm_g_mul_do<<<mp_count, naeach, sizeof(float)*naeach, *cudata->mvm_stream>>>
		    (cudata->mvm_m->p+nact*icol, cudata->mvm_a, cudata->mvm_g+icol, nact, k);
		tim_queue+=toc3;tic;
	    }else{ //Send to different gpus
		mvm_data->data.icol=icol;
		mvm_data->data.k=k;
		CALL_THREAD(mvm_data->mvm_g_mul, NGPU, 1);
		tim_queue+=toc3;tic;
	    }
	}

	CALL_THREAD(mvm_data->mvm_a_cp, NGPU, 0);
	tim_dmcp+=toc3;tic;
	CALL_THREAD(mvm_data->mvm_a_sum, NCPU, 0);
	tim_dmsum+=toc3;tic;
	WRITE_ARR(mvm_data->data.a, mvm_data->data.nact, ATYPE);
	tim_dmsend+=toc3;tic;
	memset(mvm_data->data.a, 0., mvm_data->data.nact*sizeof(ATYPE));
	info2("k=%4d CMD %1.0f, gsend %2.0f, gcp %3.0f, queue %3.0f, sync %3.0f sum %3.0f, send %2.0f, total %4.0f\n", ngeach,
	      tim_cmd*1e6, tim_gsend*1e6, tim_gcp*1e6, tim_queue*1e6, tim_dmcp*1e6, 
	      tim_dmsum*1e6, tim_dmsend*1e6, (myclockd()-tim_gfirst)*1e6);
	tim_cmd=tim_gsend=tim_gcp=tim_dmcp=tim_queue=tim_dmsum=tim_dmsend=0;
	
    }
	break;
    }
    return 0;
}
void* gpu_mvm_gpu_init(void* A){
    (void)A;
    gpu_init(NULL, 0);
    DO(hipFuncSetCacheConfig(reinterpret_cast<const void*>(mvm_g_mul_do), hipFuncCachePreferShared));
    struct hipDeviceProp_t prop;
    DO(hipGetDeviceProperties(&prop, 0));
    mp_count=prop.multiProcessorCount;
    switch(prop.major){
    case 1:
	mp_core=8;
	break;
    case 2:{
	switch(prop.minor){
	case 0:
	    mp_core=32;
	    break;
	case 1:
	    mp_core=48;
	    break;
	default:
	    error("Please fill in this");
	}
	break;
    }
    default:
	error("Please fill on this");
    }
    THREAD_POOL_INIT(NCPU);
    /*Creating stream for the first time is slow. So do it here to avoid latency
      later. Do not init stream_t in multithread. It is slower.*/
    TIC;tic;
    for(int igpu=0; igpu<NGPU; igpu++){
	gpu_set(igpu);
	stream_t temp;
	toc22("init gpu");
    }
    return NULL;
}
void gpu_mvm_daemon(int port){
    info2("Starting MVM daemon at port %d\n", port);
    pthread_create(&thread_init, NULL, gpu_mvm_gpu_init, NULL);
    listen_port(port, respond, 0, NULL);
}
