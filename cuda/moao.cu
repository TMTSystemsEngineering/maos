/*
  Copyright 2009-2012 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "recon.h"
#include "accphi.h"
#include "pcg.h"
#define CALL_ONCE\
    do {static int count=0; count++; if(count>1) warning("This function should only be called once\n");} while(0)
extern int *wfsgpu;
extern int *evlgpu;
void gpu_setup_moao(const PARMS_T *parms, RECON_T *recon){
    gpu_set(0);
    if(!curecon){
	curecon=(curecon_t*)calloc(1, sizeof(curecon_t));
    }
    curecon->moao=(cumoao_t*)calloc(parms->nmoao,sizeof(cumoao_t));
    for(int imoao=0; imoao<parms->nmoao; imoao++){
	cumoao_t* cumoao=&curecon->moao[imoao];
	gpu_dcell2cu(&cumoao->fitNW,recon->moao[imoao].NW);
	gpu_spcell2dev(&cumoao->actslave, recon->moao[imoao].actslave);
	if(parms->moao[imoao].cubic){
	    cumoao->cubic_cc=gpu_dmcubic_cc(parms->moao[imoao].iac);
	}
	cumoao->dxa=recon->moao[imoao].amap->dx;
	cumoao->oxa=recon->moao[imoao].amap->ox;
	cumoao->oya=recon->moao[imoao].amap->oy;
	cumoao->nxa=recon->moao[imoao].amap->nx;
	cumoao->nya=recon->moao[imoao].amap->ny;
	
	cumoao->dxf=recon->fmap->dx;
	cumoao->oxf=recon->fmap->ox;
	cumoao->oyf=recon->fmap->oy;
	cumoao->nxf=recon->fmap->nx;
	cumoao->nyf=recon->fmap->ny;
	if(curecon->W01){
	    cumoao->W01=curecon->W01;/*same configuration.*/
	}else{
	    cumoao->W01=gpu_get_W01(recon->W0, recon->W1);
	}
    }
    if(!curecon->moao_wfs){
	int nwfs=parms->nwfs;
	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    int ipowfs=parms->wfs[iwfs].powfs;
	    int imoao=parms->powfs[ipowfs].moao;
	    if(imoao<0) continue;
	    cumoao_t *cumoao=&curecon->moao[imoao];
	    if(!curecon->moao_wfs){
		curecon->moao_wfs=curcellnew(nwfs, 1);
	    }
	    curecon->moao_wfs->p[iwfs]=new cumap_t(cumoao->nxa, cumoao->nya, 
						cumoao->oxa, cumoao->oya,
						cumoao->dxa, 0, 0, 0);
	    if(parms->gpu.wfs){
		gpu_set(wfsgpu[iwfs]);
	    }
	    if(!cudata->moao_wfs){
		cudata->moao_wfs=curcellnew(nwfs, 1);
	    }
	    if(parms->sim.closeloop){
		cudata->moao_wfs->p[iwfs]=new cumap_t(cumoao->nxa, cumoao->nya, 
						      cumoao->oxa, cumoao->oya,
						      cumoao->dxa, 0, 0, 0);
	    }else{
		cudata->moao_wfs->p[iwfs]=curecon->moao_wfs->p[iwfs]->ref(); 
	    }
	    gpu_set(0);
	}
    }
    if(!curecon->moao_evl && parms->evl.moao!=-1){
	const int imoao=parms->evl.moao;
	const int nevl=parms->evl.nevl;
	curecon->moao_evl=curcellnew(nevl, 1);
	cumoao_t *cumoao=&curecon->moao[imoao];
	for(int ievl=0; ievl<nevl; ievl++){
	    gpu_set(0);
	    curecon->moao_evl->p[ievl]=new cumap_t(cumoao->nxa, cumoao->nya,
						   cumoao->oxa, cumoao->oya,
						   cumoao->dxa, 0, 0, 0);
	    if(parms->gpu.evl){
		gpu_set(evlgpu[ievl]);
	    }
	    if(!cudata->moao_evl){
		cudata->moao_evl=curcellnew(nevl,1);
	    }
	    if(parms->sim.closeloop){
		cudata->moao_evl->p[ievl]=new cumap_t(cumoao->nxa, cumoao->nya,
						      cumoao->oxa, cumoao->oya,
						      cumoao->dxa, 0, 0, 0);
	    }else{
		cudata->moao_evl->p[ievl]=curecon->moao_evl->p[ievl]->ref();
	    }
	}
    }
}

#define DO_W								\
    gpu_inn(pis, xp->p, cumoao->W01->W1->p, np, stream);			\
    add2_do<<<DIM(np, 256), 0, stream>>>(xp2->p, cumoao->W01->W1->p, pis, -1.f, np); \
    cuspmul(xp2->p, cumoao->W01->W0p, xp->p, 1.f, sphandle);			\
    if(cumoao->W01->nW0f){							\
	apply_W_do<<<DIM(np, 256),0,stream>>>(xp2->p, xp->p, cumoao->W01->W0f, cumoao->W01->W0v, \
					      cumoao->nxf, cumoao->W01->nW0f); \
    }

#define DO_HAT /*Apply HAT, from xp2 to xout.*/\
    if(!*xout) *xout=curcellnew(1,1);					\
    if(!(*xout)->p[0]) (*xout)->p[0]=curnew(cumoao->nxa, cumoao->nya);	\
    if(cumoao->cubic_cc){						\
	gpu_prop_grid_cubic(xp2, cumoao->oxf,cumoao->oyf, cumoao->dxf,	\
			    (*xout)->p[0], cumoao->oxa, cumoao->oya, cumoao->dxa, \
			    0,0, cumoao->cubic_cc, alpha, 't', stream);	\
    }else{\
	gpu_prop_grid(xp2, cumoao->oxf, cumoao->oyf, cumoao->dxf,	\
		      (*xout)->p[0], cumoao->oxa, cumoao->oya, cumoao->dxa, \
		      0,0, alpha, 't', stream);				\
    }

/*Right hand size vector.*/
void gpu_moao_FitR(curcell **xout, SIM_T *simu, cumoao_t *cumoao, float thetax, float thetay, float hs, const float alpha){
    const PARMS_T *parms=simu->parms;
    const RECON_T *recon=simu->recon;
    const int npsr=recon->npsr;
    const int np=cumoao->nxf*cumoao->nyf;
    curmat *xp=curnew(cumoao->nxf, cumoao->nyf);
    curmat *xp2=curnew(cumoao->nxf, cumoao->nyf);
    hipStream_t stream;
    hipsparseHandle_t sphandle;
    STREAM_NEW(stream);
    SPHANDLE_NEW(sphandle, stream);
    float *pis; 
    hipMalloc(&pis, sizeof(float));
    /*do HX operation, from curecon->opdr to xp. */
    for(int ips=0; ips<npsr; ips++){
	const float ht = (float)recon->ht->p[ips];
	const float scale=1.f-ht/hs;
	const float dispx=thetax*ht;
	const float dispy=thetay*ht;
	gpu_prop_grid(xp, cumoao->oxf*scale, cumoao->oyf*scale, cumoao->dxf*scale, 
		      curecon->opdr->p[ips], recon->xmap[ips]->ox, recon->xmap[ips]->oy, recon->xmap[ips]->dx,
		      dispx, dispy, 1.f,'n', stream);
    }
    /*do HA operation, from curecon->dmfit to xp */
    for(int idm=0; idm<recon->ndm; idm++){
	const float ht = (float)parms->dm[idm].ht;
	const float scale=1.f-ht/hs;
	const float dispx=thetax*ht;
	const float dispy=thetay*ht;
	if(curecon->cubic_cc[idm]){
	    gpu_prop_grid_cubic
		(xp, cumoao->oxf*scale,cumoao->oyf*scale, cumoao->dxf*scale, 
		 curecon->dmfit->p[idm], recon->amap[idm]->ox, recon->amap[idm]->oy, recon->amap[idm]->dx,
		 dispx, dispy, curecon->cubic_cc[idm], -1.f, 'n', stream);
	}else{
	    gpu_prop_grid
		(xp, cumoao->oxf*scale, cumoao->oyf*scale, cumoao->dxf*scale, 
		 curecon->dmfit->p[idm], recon->amap[idm]->ox, recon->amap[idm]->oy, recon->amap[idm]->dx,
		 dispx, dispy, -1.f, 'n', stream);
	}
    }
    /*apply W, from xp to xp2*/
    DO_W;
    /*do HAT operation, from xp2 to xout*/
    DO_HAT;
    STREAM_DONE(stream);
    SPHANDLE_DONE(sphandle);
    hipFree(pis);
    curfree(xp);
    curfree(xp2);
}

void gpu_moao_FitL(curcell **xout, const void *A, const curcell *xin, const float alpha){
    cumoao_t *cumoao=(cumoao_t*)A;
    hipStream_t stream;
    hipsparseHandle_t sphandle;
    hipblasHandle_t handle;
    STREAM_NEW(stream);
    SPHANDLE_NEW(sphandle, stream);
    HANDLE_NEW(handle, stream);
    float *pis; 
    hipMalloc(&pis, sizeof(float));
    const int np=cumoao->nxf*cumoao->nyf;
    curmat *xp=curnew(cumoao->nxf, cumoao->nyf);
    curmat *xp2=curnew(cumoao->nxf, cumoao->nyf);

    /*Apply HA, from xin to xp.*/
    if(cumoao->cubic_cc){
	gpu_prop_grid_cubic(xp, cumoao->oxf,cumoao->oyf, cumoao->dxf, 
			    xin->p[0], cumoao->oxa, cumoao->oya, cumoao->dxa,
			    0,0, cumoao->cubic_cc, 1.f, 'n', stream);
    }else{
	gpu_prop_grid(xp, cumoao->oxf, cumoao->oyf, cumoao->dxf, 
		      xin->p[0], cumoao->oxa, cumoao->oya, cumoao->dxa,
		      0,0, 1.f, 'n', stream);	
    }
    /*Apply W, from xp to xp2*/
    DO_W;
    
    /*Apply Hat, from xp2 to xout*/
    DO_HAT;
    /*Additional terms*/
    curmat *tmp=NULL;
    if(cumoao->fitNW){
	tmp=curnew(cumoao->fitNW->p[0]->nx, 1);
	curmv(tmp->p, 0, cumoao->fitNW->p[0], xin->p[0]->p, 't', 1, handle);
	curmv((*xout)->p[0]->p, 1, cumoao->fitNW->p[0], tmp->p, 'n', alpha, handle);
    }
    if(cumoao->actslave){
	cuspmul((*xout)->p[0]->p, cumoao->actslave->p[0], xin->p[0]->p, alpha, sphandle);
    }
    STREAM_DONE(stream);
    SPHANDLE_DONE(sphandle);
    HANDLE_DONE(handle);
    curfree(tmp);
    hipFree(pis);
    curfree(xp);
    curfree(xp2);
}
/**
   MOAO reconstruction.

   Do not output directly to cudata->moao since that is being used by wfsgrad
and perfevl. The new result is supposed to be used next time step. The input
based on opdr, dmfit is on gradients from last time step. So two cycle delay is
maintained.  */
void gpu_moao_recon(SIM_T *simu){
    gpu_set(0);
    curcell *dmcommon=NULL;
    const PARMS_T *parms=simu->parms;
    const int nwfs=parms->nwfs;
    const int nevl=parms->evl.nevl;
    if(parms->gpu.fit){
	dmcommon=curecon->dmfit;
    }else{
	gpu_dcell2cu(&dmcommon, simu->dmfit_hi);
    }
    hipStream_t wfs_stream[nwfs];
    hipStream_t evl_stream[nevl];
    curcell *wfs_dmfit[nwfs];
    curcell *evl_dmfit[nevl];
    curcell *wfs_rhs[nwfs];
    curcell *evl_rhs[nevl];
    if(curecon->moao_wfs){/*There is MOAO DM for WFS */
	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    int ipowfs=parms->wfs[iwfs].powfs;
	    int imoao=parms->powfs[ipowfs].moao;
	    wfs_rhs[iwfs]=NULL;
	    wfs_stream[iwfs]=NULL;
	    wfs_dmfit[iwfs]=NULL;
	    if(imoao<0){
		continue;
	    }
	    STREAM_NEW(wfs_stream[iwfs]);
	    cumoao_t *cumoao=&curecon->moao[imoao];
	    wfs_dmfit[iwfs]=curcellnew(1,1);
	    wfs_dmfit[iwfs]->p[0]=curref(curecon->moao_wfs->p[iwfs]);
	    gpu_moao_FitR(&wfs_rhs[iwfs], simu, cumoao,
			  parms->wfs[iwfs].thetax, parms->wfs[iwfs].thetay, 
			  parms->powfs[ipowfs].hs, 1);
	    if(gpu_pcg(&wfs_dmfit[iwfs], gpu_moao_FitL, cumoao, NULL, NULL, wfs_rhs[iwfs],
		       simu->parms->recon.warm_restart, parms->fit.maxit, wfs_stream[iwfs])){
		error("PCG failed\n");
	    }
	}
    }
    if(curecon->moao_evl){
	int imoao=parms->evl.moao;
	if(imoao==-1) error("Inconsistent\n");
	cumoao_t *cumoao=&curecon->moao[imoao];
	for(int ievl=0; ievl<nevl; ievl++){
	    evl_stream[ievl]=NULL;
	    evl_dmfit[ievl]=curcellnew(1,1);
	    evl_dmfit[ievl]->p[0]=curref(curecon->moao_evl->p[ievl]);
	    STREAM_NEW(evl_stream[ievl]);
	    evl_rhs[ievl]=NULL;
	    gpu_moao_FitR(&evl_rhs[ievl], simu, cumoao,
			  parms->evl.thetax[ievl], parms->evl.thetay[ievl], 
			  parms->evl.hs[ievl], 1);
	    if(gpu_pcg(&evl_dmfit[ievl], gpu_moao_FitL, cumoao, NULL, NULL, evl_rhs[ievl],
		       simu->parms->recon.warm_restart, parms->fit.maxit, evl_stream[ievl])){
		error("PCG failed\n");
	    }
	}
    }

    if(curecon->moao_wfs){
	for(int iwfs=0;iwfs<nwfs; iwfs++){
	    if(wfs_stream[iwfs]){
		STREAM_DONE(wfs_stream[iwfs]);
		curcellfree(wfs_dmfit[iwfs]);
		curcellfree(wfs_rhs[iwfs]);
	    }
	}
    }
    if(curecon->moao_evl){
	for(int ievl=0; ievl<nevl; ievl++){
	    if(evl_stream[ievl]){
		STREAM_DONE(evl_stream[ievl]);
		curcellfree(evl_dmfit[ievl]);
		curcellfree(evl_rhs[ievl]);
	    }
	}
    }
    if(dmcommon!=curecon->dmfit){
	curcellfree(dmcommon);
    }
}
void gpu_moao_filter(SIM_T *simu){
    gpu_set(0);
    const PARMS_T *parms=simu->parms;
    const int nwfs=parms->nwfs;
    const int nevl=parms->evl.nevl;
    if(curecon->moao_wfs){
	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    int ipowfs=parms->wfs[iwfs].powfs;
	    int imoao=parms->powfs[ipowfs].moao;
	    if(imoao<0) continue;
	    double g=parms->moao[imoao].gdm;
	    if(parms->gpu.wfs) {
		gpu_set(wfsgpu[iwfs]);
	    }
	    curmat *temp=NULL;
	    if(wfsgpu[iwfs]!=0){//copy between GPUs
		curcp(&temp, curecon->moao_wfs->p[iwfs]);
	    }else{
		temp=curref(curecon->moao_wfs->p[iwfs]);
	    }
	    if(parms->sim.closeloop){
		curadd(&cudata->moao_wfs->p[iwfs], 1.-g, temp, g, 0);
	    }
	    if(!parms->gpu.wfs ){
		gpu_cur2d(&simu->moao_wfs->p[iwfs], 0, temp, 1, 0);
	    }
	    hipStreamSynchronize(0);
	    curfree(temp);
	}
    }

    if(curecon->moao_evl){
	int imoao=parms->evl.moao;
	double g=parms->moao[imoao].gdm;
	for(int ievl=0; ievl<nevl; ievl++){
	    if(parms->gpu.evl){
		gpu_set(evlgpu[ievl]);
	    }
	    curmat *temp=NULL;
	    if(evlgpu[ievl]!=0){//copy between GPUs
		curcp(&temp, curecon->moao_evl->p[ievl]);
	    }else{
		temp=curref(curecon->moao_evl->p[ievl]);
	    }
	    if(parms->sim.closeloop){
		curadd(&cudata->moao_evl->p[ievl], 1.-g, temp, g, 0);
	    }
	    if(!parms->gpu.evl){
		gpu_cur2d(&simu->moao_evl->p[ievl], 0, temp, 1, 0);
	    }
	    hipStreamSynchronize(0);
	    curfree(temp);
	}
    }
    gpu_set(0);
}
/**
   Copy MOAO DM commands from CPU to GPU.*/
void gpu_moao_2gpu(SIM_T *simu){
    const PARMS_T *parms=simu->parms;
    const RECON_T *recon=simu->recon;
    if(parms->gpu.moao){
	error("Invalid use\n");
    }
    const int nwfs=parms->nwfs;
    const int nevl=parms->evl.nevl;
    if(parms->gpu.wfs && simu->moao_wfs){
	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    int ipowfs=parms->wfs[iwfs].powfs;
	    int imoao=parms->powfs[ipowfs].moao;
	    if(imoao<0) continue;
	    gpu_set(wfsgpu[iwfs]);
	    if(!cudata->moao_wfs){
		cudata->moao_wfs=curcellnew(nwfs, 1);
	    }
	    gpu_dmat2cu(&cudata->moao_wfs->p[iwfs], simu->moao_wfs->p[iwfs]);
	}
    }
    if(parms->gpu.evl && simu->moao_evl){
	int imoao=parms->evl.moao;
	for(int ievl=0; ievl<nevl; ievl++){
	    gpu_set(evlgpu[ievl]);
	    if(!cudata->moao_evl){
		cudata->moao_evl=curcellnew(nevl, 1);
	    }
	    if(!cudata->moao_evl->p[ievl]){
		double dxa=recon->moao[imoao].amap->dx;
		double oxa=recon->moao[imoao].amap->ox;
		double oya=recon->moao[imoao].amap->oy;
		int nxa=recon->moao[imoao].amap->nx;
		int nya=recon->moao[imoao].amap->ny;
		cudata->moao_evl->p[ievl]=new cumap_t(nxa, nya,
						      oxa, oya,
						      dxa, 0, 0, 0);
	    }
	    gpu_dmat2cu(&cudata->moao_evl->p[ievl], simu->moao_evl->p[ievl]);
	}
    }
}
