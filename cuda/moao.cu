/*
  Copyright 2009-2015 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include "gpu.h"
}
#include "moao.h"
#include "utils.h"
#include "recon.h"
#include "accphi.h"
#include "pcg.h"
#include "cudata.h"
#include "perf.h"

namespace cuda_recon{
cumoao_t::cumoao_t(const PARMS_T *parms, MOAO_T *moao, dir_t *dir, int _ndir, curecon_geom *_grid)
    :cucg_t(parms?parms->fit.maxit:0, parms?parms->recon.warm_restart:0),grid(_grid),
     NW(0),dotNW(0),amap(0),actslave(0),opdfit(0),opdfit2(0),ha(0),ndir(_ndir){

    amap=new cugrid_t(moao->amap->p[0]);
    if(moao->NW){
	cp2gpu(&NW, moao->NW->p[0]);
	dotNW=curnew(NW->ny, 1);
    }
    if(moao->actslave){
	actslave=new cusp(moao->actslave->p[0], 1);
    }

    dir_t dir0={0,0,INFINITY,0};
    ha=new map_l2d(grid->fmap, &dir0, 1, amap, 1);
    opdfit=curcellnew(1,1,grid->fmap.nx,grid->fmap.ny);
    opdfit2=curcellnew(1,1,grid->fmap.nx,grid->fmap.ny);

    hxp=new map_ray*[ndir];
    hap=new map_ray*[ndir];
    for(int idir=0; idir<ndir; idir++){
	hxp[idir]=new map_l2d(grid->fmap, dir+idir, 1, grid->xmap, grid->npsr);
	hap[idir]=new map_l2d(grid->fmap, dir+idir, 1, grid->amap, grid->ndm);
    }
    rhs=curcellnew(1,1,amap->nx,amap->ny);
}
Real cumoao_t::moao_solve(curcell **xout, const curcell *xin, const curcell *ain, stream_t &stream){
    for(int idir=0; idir<ndir; idir++){
	opdfit->m->zero(stream);
	hxp[idir]->forward(opdfit->pm, xin->pm, 1.f, NULL, stream);//tomography	
	hap[idir]->forward(opdfit->pm, ain->pm, -1.f, NULL, stream);//minus common DM.
	grid->W01->apply(opdfit2->m->p, opdfit->m->p, opdfit->nx, stream);
	rhs->m->zero(stream);
	ha->backward(opdfit2->pm, rhs->pm, 1, NULL, stream);
	solve(&xout[idir], rhs, stream);
	/*{
	    static int ic=-1; ic++;
	    curcellwrite(xout[idir], "xout_%d", ic);
	    curcellwrite(rhs, "rhs_%d", ic);
	    curcellwrite(xin, "xin_%d", ic);
	    curcellwrite(ain, "ain_%d", ic);
	    curcellwrite(opdfit, "opd_%d", ic);
	    }*/
    }
    return 0;
}
void cumoao_t::L(curcell **xout, Real beta, const curcell *xin, Real alpha, stream_t &stream){
    if(!*xout){
	*xout=curcellnew(1, 1, amap->nx, amap->ny);
    }else{
	curscale((*xout)->m, beta, stream);
    }
    opdfit->m->zero(stream);
    ha->forward(opdfit->pm, xin->pm, 1, NULL, stream);
    grid->W01->apply(opdfit2->m->p, opdfit->m->p, opdfit->nx, stream);
    ha->backward(opdfit2->pm, (*xout)->pm, alpha, NULL, stream);
    if(NW){
	curmv(dotNW->p, 0, NW, xin->m->p, 't', 1, stream);
	curmv((*xout)->m->p, 1, NW, dotNW->p, 'n', alpha, stream);
    }
    if(actslave){
	cuspmul((*xout)->m->p, actslave, xin->m->p, 1,'n', alpha, stream);
    }
}
}//namespace
/*
  Embed and copy DM commands to GPU.
*/
static void gpu_dm2gpu_embed(curmat *dmgpu, dmat *dmcpu, loc_t *loc, int nx, int ny){
    assert(dmcpu->ny==1);
    Real *pout=(Real*)calloc(nx*ny, sizeof(Real));
    map_t *map=loc->map;
    double *pin=dmcpu->p-1;
    for(long i=0; i<map->nx*map->ny; i++){
	long iphi=map->p[i];
	if(iphi){
	    pout[i]=pin[iphi];
	}
    }
    DO(hipMemcpy(dmgpu->p, pout, nx*ny*sizeof(Real), hipMemcpyHostToDevice));
    free(pout);
}

/**
   Copy MOAO DM commands from CPU to GPU.*/
void gpu_moao_2gpu(SIM_T *simu){
    const PARMS_T *parms=simu->parms;
    const RECON_T *recon=simu->recon;
    if(parms->gpu.moao){
	error("Invalid use\n");
    }
    const int nwfs=parms->nwfs;
    const int nevl=parms->evl.nevl;
    if(parms->gpu.wfs && simu->dm_wfs){
	for(int iwfs=0; iwfs<nwfs; iwfs++){
	    int ipowfs=parms->wfs[iwfs].powfs;
	    int imoao=parms->powfs[ipowfs].moao;
	    if(imoao<0) continue;
	    MOAO_T *moao=recon->moao+imoao;
	    gpu_set(cudata_t::wfsgpu[iwfs]);
	    if(!cudata->dm_wfs){
		cudata->dm_wfs=(cumap_t**)calloc(nwfs, sizeof(cumap_t*));
	    }
	    if(!cudata->dm_wfs[iwfs]){
		cudata->dm_wfs[iwfs]=new cumap_t(recon->moao[imoao].amap->p[0]); 
	    }
	    if(parms->fit.square){
		cp2gpu(&cudata->dm_wfs[iwfs]->p, simu->dm_wfs->p[iwfs]);
	    }else{
		gpu_dm2gpu_embed(cudata->dm_wfs[iwfs]->p, simu->dm_wfs->p[iwfs],
				 moao->aloc->p[0], moao->amap->p[0]->nx, moao->amap->p[0]->ny);
	    }
	}
    }
    if(parms->gpu.evl && simu->dm_evl){
	int imoao=parms->evl.moao;
	MOAO_T *moao=recon->moao+imoao;
	for(int ievl=0; ievl<nevl; ievl++){
	    gpu_set(cudata_t::evlgpu[ievl]);
	    if(!cudata->dm_evl){
		cudata->dm_evl=(cumap_t**)calloc(nevl, sizeof(cumap_t*));
	    }
	    if(!cudata->dm_evl[ievl]){
		cudata->dm_evl[ievl]=new cumap_t(recon->moao[imoao].amap->p[0]); 
	    }
	    if(parms->fit.square){
		cp2gpu(&cudata->dm_evl[ievl]->p, simu->dm_evl->p[ievl]);
	    }else{
		gpu_dm2gpu_embed(cudata->dm_evl[ievl]->p, simu->dm_evl->p[ievl],
				 moao->aloc->p[0], moao->amap->p[0]->nx, moao->amap->p[0]->ny);
	    }
	}
    }
}

