/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include "../lib/aos.h"
#include "gpu.h"
}
#include "kernel.h"
#include "utils.h"
#include "curmat.h"
#include "prop_wrap.h"
/**
   \file test.cu
   Routines for testing cuda. Execute from test/test_cuda
   
*/

void test_sum(){
    TIC;tic;
    const int BS=128;
    int N=120*120*6000;
    X(mat) *As=X(new)(N,1);
    //rand_t srand;
    //seed_rand(&srand, 1);
    //srandu(As, 1, &srand);
    X(set)(As, 1);
    curmat *Ap=NULL;
    cp2gpu(&Ap, As);
    Real *res_gpu;
    hipMalloc(&res_gpu, sizeof(Real));
    hipMemset(res_gpu, 0, sizeof(Real));
    Real res[4];
    hipStream_t stream;
    STREAM_NEW(stream);
    toc("malloc");tic;
    sum_wrap(res_gpu, Ap->p, N, stream);
    hipMemcpyAsync(res, res_gpu, sizeof(Real), hipMemcpyDeviceToHost);
    CUDA_SYNC_STREAM;
    toc("sum_wrap");tic;
    sum_do<<<DIM_REDUCE, BS, DIM_REDUCE*sizeof(Real), stream>>>(res_gpu, Ap->p, N);
    hipMemcpyAsync(res+1, res_gpu, sizeof(Real), hipMemcpyDeviceToHost);
    CUDA_SYNC_STREAM;
    double tim=toc3*1024*1024*1024;
    toc("sum_wrap");tic;
    sum2_wrap(res_gpu, Ap->p, N, stream);
    hipMemcpyAsync(res+2, res_gpu, sizeof(Real), hipMemcpyDeviceToHost);
    CUDA_SYNC_STREAM;
    toc("sum2_wrap");tic;
    sum2_do<<<DIM_REDUCE, BS, 0, stream>>>(res_gpu, Ap->p, N);
    //sum2_wrap(res_gpu, Ap->p, N, stream);
    hipMemcpyAsync(res+3, res_gpu, sizeof(Real), hipMemcpyDeviceToHost);
    CUDA_SYNC_STREAM;
    double tim2=toc3*1024*1024*1024;
    toc("sum2_wrap");

    info("sum_wrap  %.2f GB/s\n", N*sizeof(Real)/tim);
    info("sum2_wrap %.2f GB/s\n", N*sizeof(Real)/tim2);
    info("Result: %g %g %g %g\n", res[0], res[1]-res[0], res[2]-res[1], res[3]-res[2]);
}
/*Test ray tracing*/
void test_prop(){
    map_t *mapin=mapnew(30, 30, 0.5, 0.5, 0);
    map_t *mapout=mapnew(200, 200, 0.1, 0.1, 0);
    dset((dmat*)mapin, 1);
    writebin(mapin, "prop_mapin_cpu");
    cumap_t cumapin(mapin);
    cumap_t cumapout(mapout);
    cp2gpu(&cumapin.p, (dmat*)mapin);
    curwrite(cumapin.p, "prop_mapin");
    Real alpha=1;
    Real dispx=0;
    Real dispy=0;
    gpu_prop_grid(cumapout, cumapin, dispx, dispy, alpha, 0, 'n');
    curwrite(cumapout.p, "prop_mapout");
    prop_grid_map(mapin, mapout, alpha, dispx, dispy, 1, 0, 0, 0);
    writebin(mapout, "prop_mapout_cpu");
    mapin->cubic=1;
    mapin->iac=0.3;
}

int main(int argc, char** argv){
    const char *cmd=0;
    if(argc>1){
	cmd=argv[1];
    }
    if(!cmd){
	test_prop();
    }else if(!strcmp(cmd, "sum")){
	test_sum();
    }else if(!strcmp(cmd, "prop")){
	test_prop();
    }
}
