#include "hip/hip_runtime.h"
/*
  Copyright 2009, 2010, 2011 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipfft/hipfft.h>
#include "wfs.h"

#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define ctoc(A)
#else
#define ctoc(A) CUDA_SYNC_STREAM; toc2(A)
#endif
extern char *dirskysim;
/*
  Notice that both blocks and threads are partitioning isa
 */
__global__ static void add_geom_noise_do(float *restrict g, const float *restrict nea, 
				      int nsa, curandStat *restrict rstat){
    const int id=threadIdx.x + blockIdx.x * blockDim.x;
    curandStat lstat=rstat[id];
    const int nstep=blockDim.x * gridDim.x;
    for(int i=id; i<nsa; i+=nstep){
	float n1=hiprand_normal(&lstat);
	float n2=hiprand_normal(&lstat);
	g[i]+=n1*nea[i];
	g[i+nsa]+=n2*nea[i+nsa]+n1*nea[i+nsa*2];/*cross term. */
    }
    rstat[id]=lstat;
}

/**
   Compute ztilt
*/
__global__ void cuztilt(float *restrict g, float *restrict opd, 
			const int nsa, const float dx, const int nx, float (**imcc)[3],
			const float (*orig)[2], const float*restrict amp, float alpha){
    __shared__ float a[3];
    if(threadIdx.x<3 && threadIdx.y==0){
	a[threadIdx.x]=0.f;
    }
    __syncthreads();
    const int isa=blockIdx.x;
    float b0=0.f;
    float b1=0.f;
    float b2=0.f;
    const int skip=isa*nx*nx;
    const float ox=orig[isa][0];
    const float oy=orig[isa][1];
    for(int iy=threadIdx.y; iy<nx; iy+=blockDim.y){
	const int skip2=skip+iy*nx;
	const float y=iy*dx+oy;
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    const int ind=skip2+ix;
	    const float tmp=amp[ind]*opd[ind];
	    b0+=tmp;
	    b1+=tmp*(dx*ix+ox);
	    b2+=tmp*y;
	}
    }
    atomicAdd(&a[0],b0);
    atomicAdd(&a[1],b1);
    atomicAdd(&a[2],b2);
    __syncthreads();/*Wait until all threads in this block is done. */
    if(threadIdx.x<3 && threadIdx.y==0){
	float (*restrict A)[3]=imcc[isa];
	atomicAdd(&g[isa],     alpha*(A[threadIdx.x][1]*a[threadIdx.x]));
	atomicAdd(&g[isa+nsa], alpha*(A[threadIdx.x][2]*a[threadIdx.x]));
    }
    /*
      if(threadIdx.x==0 && threadIdx.y==0){
      g[isa]    +=alpha*(A[0][1]*a[0]+A[1][1]*a[1]+A[2][1]*a[2]);
      g[isa+nsa]+=alpha*(A[0][2]*a[0]+A[1][2]*a[1]+A[2][2]*a[2]);
      }*/
}
/**
   Apply matched filter. \todo this implementation relies on shared variable. It is probably causing competition.
*/
__global__ static void mtche_do(float *restrict grad, float (*restrict *restrict mtches)[2], 
				const float *restrict ints, int pixpsa, int nsa){
    __shared__ float g[2];/*shared by threads in the same block (with the same isa). */
    if(threadIdx.x<2){
	g[threadIdx.x]=0.f;
    }
    __syncthreads();
    int isa=blockIdx.x;
    ints+=isa*pixpsa;
    const float (*const restrict mtche)[2]=mtches[isa];
 
    float gp[2]={0.f,0.f};
    for (int ipix=threadIdx.x; ipix<pixpsa; ipix+=blockDim.x){
	gp[0]+=mtche[ipix][0]*ints[ipix];
	gp[1]+=mtche[ipix][1]*ints[ipix];
    }
    if(fabsf(gp[0])>1e-5 || fabsf(gp[1])>1e-5){
	printf("gp=%g %g. g=%g %g\n", gp[0], gp[1], g[0], g[1]);
    }
    atomicAdd(&g[0], gp[0]);
    atomicAdd(&g[1], gp[1]);
    __syncthreads();
    if(threadIdx.x<2){
	grad[isa+nsa*threadIdx.x]=g[threadIdx.x];
    }
}

/**
   Poisson random generator.
*/
__device__ static float curandp(curandStat *rstat, float xm){
    float g, t, xmu;
    int x=0, xu;
    if(xm>200){
	x=(int)round(xm+hiprand_normal(rstat)*sqrt(xm));
    }else{
	while(xm>0){
	    xmu = xm > 12.f ? 12.f : xm;
	    xm -= xmu;
	    g   = __expf(-xmu);
	    xu  = -1;
	    t   = 1.f;
	    while(t>g){
		xu++;
		t *= hiprand_uniform(rstat);
	    }
	    x += xu;
	}
    }
    return x;
}
/**
   Add noise to pix images.
*/
__global__ static void addnoise_do(float *restrict ints0, int nsa, int pixpsa, float bkgrnd, float pcalib, 
				   float *const restrict *restrict bkgrnd2s, float *const restrict *restrict bkgrnd2cs,
				   float rne, curandStat *rstat){
    const int id=threadIdx.x + blockIdx.x * blockDim.x;
    const int nstep=blockDim.x * gridDim.x;
    curandStat lstat=rstat[id];
    for(int isa=id; isa<nsa; isa+=nstep){
	float *restrict ints=ints0+isa*pixpsa;
	const float *restrict bkgrnd2=bkgrnd2s?bkgrnd2s[isa]:NULL;
	const float *restrict bkgrnd2c=bkgrnd2cs?bkgrnd2cs[isa]:NULL;
	for(int ipix=0; ipix<pixpsa; ipix++){
	    if(bkgrnd2){
		ints[ipix]=curandp(&lstat, ints[ipix]+bkgrnd+bkgrnd2[ipix])+rne*hiprand_normal(&lstat)-bkgrnd*pcalib;
	    }else{
		ints[ipix]=curandp(&lstat, ints[ipix]+bkgrnd)+rne*hiprand_normal(&lstat)-bkgrnd*pcalib;
	    }
	    if(bkgrnd2c){
		ints[ipix]-=bkgrnd2c[ipix];
	    }
	}
    }
    rstat[id]=lstat;
}
/**
   Collect the noise we actually applied.
 */
__global__ static void collect_noise_do(float *restrict neareal, const float *restrict gnf, const float *restrict gny, int nsa){
    for(int isa=threadIdx.x+blockIdx.x*blockDim.x; isa<nsa; isa+=blockDim.x*gridDim.x){
        float dx=gny[isa]-gnf[isa];
	float dy=gny[isa+nsa]-gnf[isa+nsa];
	float *restrict nea=neareal+isa*4;
	nea[0]+=dx*dx;
	nea[1]+=dx*dy;
	nea[2]+=dx*dy;
	nea[3]+=dy*dy;
    }
}

/**
   Ray tracing and gradient computation for WFS. \todo Expand to do gradients in GPU without transfering
   data back to CPU.
*/
extern int *wfsgpu;
void gpu_wfsgrad(thread_t *info){
    const int iwfs=info->start;
    gpu_set(wfsgpu[iwfs]);
    cuwloc_t *cupowfs=cudata->powfs;
    cuwfs_t *cuwfs=cudata->wfs;
    TIC;tic;
    SIM_T *simu=(SIM_T*)info->data;
    const PARMS_T *parms=simu->parms;
    assert(info->end==info->start+1);/*only 1 WFS. */
    assert(iwfs<parms->nwfs);
    const POWFS_T *powfs=simu->powfs;
    const RECON_T *recon=simu->recon;
    /*output */
    const int CL=parms->sim.closeloop;
    const int isim=simu->isim;
    /*The following are truly constants for this powfs */
    const int ipowfs=parms->wfs[iwfs].powfs;
    const int imoao=parms->powfs[ipowfs].moao;
    const int nsa=powfs[ipowfs].pts->nsa;
    const int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
    const float hs=parms->powfs[ipowfs].hs;
    const int dtrat=parms->powfs[ipowfs].dtrat;
    const int save_grad=parms->save.grad[iwfs];
    const int save_gradgeom=parms->save.gradgeom[iwfs];
    const int save_ints=parms->save.ints[iwfs];
    const int noisy=parms->powfs[ipowfs].noisy;
    /*The following depends on isim */
    const int dtrat_output=((isim+1)%dtrat==0);
    const int do_phy=(parms->powfs[ipowfs].usephy && isim>=parms->powfs[ipowfs].phystep);
    const int do_pistatout=parms->powfs[ipowfs].pistatout&&isim>=parms->powfs[ipowfs].pistatstart;
    const int do_geom=!do_phy || save_gradgeom || do_pistatout;
    const float thetax=parms->wfs[iwfs].thetax;
    const float thetay=parms->wfs[iwfs].thetay;
    const float mispx=powfs[ipowfs].misreg[wfsind][0];
    const float mispy=powfs[ipowfs].misreg[wfsind][1];
    const float dtisim=parms->sim.dt*isim;
    float (*loc)[2]=cupowfs[ipowfs].loc;
    const int nloc=cupowfs[ipowfs].nloc;
    /*Out to host for now. \todo : keep grad in device when do reconstruction on device. */
    hipStream_t stream=cuwfs[iwfs].stream;
    dmat *gradout=simu->gradcl->p[iwfs];
    curmat *phiout=curnew(nloc, 1);
    curzero(phiout, stream);
    curmat *gradacc=cuwfs[iwfs].gradacc;
    curmat *gradref=NULL;
    if(cuwfs[iwfs].opdadd){ /*copy to phiout. */
	curcp(&phiout, cuwfs[iwfs].opdadd, stream);
    }
    if(parms->sim.idealwfs){
	gpu_dm2loc(phiout->p, loc, nloc, cudata->dmproj, hs, thetax, thetay, mispx, mispy, 1, stream);
    }else{
	gpu_atm2loc(phiout->p, loc, nloc, hs, thetax, thetay, mispx, mispy, dtisim, 1, stream);
	if(parms->sim.wfsalias){
	    gpu_dm2loc(phiout->p, loc, nloc, cudata->dmproj, hs, thetax, thetay, mispx, mispy, -1, stream);
	}
    }
    if(CL){
	gpu_dm2loc(phiout->p, loc, nloc, cudata->dmreal, hs, thetax, thetay, mispx, mispy, -1, stream);
    }
    /*CUDA_SYNC_STREAM; */
    
    if(imoao>-1){
	TO_IMPLEMENT;
    }
    if(simu->telws){
	float tt=simu->telws->p[isim];
	float angle=simu->winddir?simu->winddir->p[0]:0;
	curaddptt(phiout, loc, 0, tt*cosf(angle), tt*sinf(angle), stream);
    }
   
    if(powfs[ipowfs].focus){
	TO_IMPLEMENT;
    }
    if(parms->powfs[ipowfs].llt && simu->focusint && simu->focusint->p[iwfs]){
	TO_IMPLEMENT;
    }
    if(do_geom){
	float *gradcalc;
	if(do_pistatout && !parms->powfs[ipowfs].pistatstc){
	    if(dtrat>1){
		gradref=curnew(nsa*2,1);
	    }else{
		gradref=curref(gradacc);
	    }
	    gradcalc=gradref->p;
	}else{
	    gradcalc=gradacc->p;
	}
	if(parms->powfs[ipowfs].gtype_sim==1){
	    cuztilt<<<nsa, dim3(16,16), 0, stream>>>
		(gradcalc, phiout->p, cupowfs[ipowfs].nsa, 
		 cupowfs[ipowfs].dx, 
		 cupowfs[ipowfs].nxsa, cuwfs[iwfs].imcc,
		 cupowfs[ipowfs].pts, cuwfs[iwfs].amp, 
		 1.f/(float)dtrat);
	}else{
	    cusp *GS0=cuwfs[iwfs].GS0t;
	    cusptmul(gradcalc, GS0, phiout->p, 1.f/(float)dtrat, cuwfs[iwfs].sphandle);
	}
	if(gradcalc!=gradacc->p){
	    curadd(&gradacc, 1, gradref, 1, stream);
	}
    }   
    if(parms->powfs[ipowfs].psfout){
	cellarr_cur(simu->save->ztiltout[iwfs], gradacc, stream);
    }
    /*CUDA_SYNC_STREAM; */
    if(do_phy || parms->powfs[ipowfs].psfout || do_pistatout){/*physical optics */
	gpu_wfsints(simu, phiout->p, gradref, iwfs, isim, stream);
    }/*do phy */
    /*CUDA_SYNC_STREAM; */
    ctoc("grad");
    if(dtrat_output){
	if(do_phy){
	    /*signal level was already multiplied in ints. */
	    curcell *ints=cuwfs[iwfs].ints;
	    curmat *gradnf=curnew(nsa*2, 1);
	    curmat *gradny=NULL;
	    const int pixpsa=powfs[ipowfs].pixpsax*powfs[ipowfs].pixpsay;
	    switch(parms->powfs[ipowfs].phytypesim){
	    case 1:
		/*use 32 instead of pixpsa here. using pixpsa causes random
		  error in g. is this due to lack of ECC?*/
		mtche_do<<<nsa, 32,0,stream>>>(gradnf->p, cuwfs[iwfs].mtche, ints->p[0]->p, pixpsa, nsa);
		break;
	    default:
		TO_IMPLEMENT;
	    }
	    if(save_ints){
		cellarr_curcell(simu->save->intsnf[iwfs], ints, stream);
	    }
	    /*CUDA_SYNC_STREAM; */
	    ctoc("mtche");
	    if(noisy){
		float rne=parms->powfs[ipowfs].rne;
		float bkgrnd=parms->powfs[ipowfs].bkgrnd*dtrat;
		addnoise_do<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt, 0, stream>>>
		    (ints->p[0]->p, nsa, pixpsa, bkgrnd, parms->powfs[ipowfs].bkgrndc,
		     cuwfs[iwfs].bkgrnd2, cuwfs[iwfs].bkgrnd2c, 
		     rne, cuwfs[iwfs].custat);
		ctoc("noise");
		gradny=curnew(nsa*2, 1);
		switch(parms->powfs[ipowfs].phytypesim){
		case 1:
		    mtche_do<<<nsa, 16, 0, stream>>>(gradny->p, cuwfs[iwfs].mtche, ints->p[0]->p, pixpsa, nsa);
		    break;
		default:
		    TO_IMPLEMENT;
		}
		collect_noise_do<<<DIM(nsa,256), 0, stream>>>
		    (cuwfs[iwfs].neareal->p, gradnf->p, gradny->p, nsa);
		if(save_ints){
		    cellarr_curcell(simu->save->intsny[iwfs], ints, stream);
		}
		if(save_grad){
		    cellarr_cur(simu->save->gradnf[iwfs], gradnf, stream);
		}
		ctoc("mtche");
	    }
	    /*send grad to CPU. */
	    gpu_dev2dbl(&gradout->p, gradny?gradny->p:gradnf->p, nsa*2, stream);
	    ctoc("dev2dbl");
	    curcellzero(ints, stream);
	    CUDA_SYNC_STREAM;/*necessary. */
	    ctoc("sync");
	    curfree(gradny);
	    curfree(gradnf);
	    ctoc("send");
	    if(parms->powfs[ipowfs].llt && parms->powfs[ipowfs].trs){
		if(!recon->PTT){
		    error("powfs %d has llt, but recon->PTT is NULL",ipowfs);
		}
		dmat *PTT=NULL;
		if(parms->recon.glao){
		    PTT=recon->PTT->p[ipowfs+ipowfs*parms->npowfs];
		}else{
		    PTT=recon->PTT->p[iwfs+iwfs*parms->nwfs];
		}
		if(!PTT){
		    error("powfs %d has llt, but TT removal is empty\n", ipowfs);
		}
		/* Compute LGS Uplink error. */
		dzero(simu->upterr->p[iwfs]);
		dmm(&simu->upterr->p[iwfs], PTT, gradout, "nn", 1);
		/* copy upterr to output. */
		PDMAT(simu->upterrs->p[iwfs], pupterrs);
		pupterrs[isim][0]=simu->upterr->p[iwfs]->p[0];
		pupterrs[isim][1]=simu->upterr->p[iwfs]->p[1];
	    }
	    if(save_gradgeom){
		if(dtrat!=1) curscale(gradacc, 1./dtrat, stream);
		cellarr_cur(simu->save->gradgeom[iwfs], gradacc, stream);
	    }
	    curzero(gradacc, stream);
	}else{
	    if(noisy){
		if(save_grad){
		    cellarr_cur(simu->save->gradnf[iwfs], gradacc, stream);
		}
		if(!parms->powfs[ipowfs].usephy){
		    add_geom_noise_do<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt, 0, stream>>>
			(gradacc->p, cuwfs[iwfs].neasim, nsa,cuwfs[iwfs].custat);
		    ctoc("noise");
		}
	    }
	    gpu_cur2d(&gradout, gradacc, stream);
	    ctoc("dev2dbl");
	    curzero(gradacc, stream);
	    ctoc("zero");
	}
	CUDA_SYNC_STREAM;
	if(powfs[ipowfs].ncpa_grad){
	    warning("Applying ncpa_grad to gradout\n");
	    dadd(&gradout, 1., powfs[ipowfs].ncpa_grad->p[wfsind], -1.);
	}
	if(save_grad){
	    cellarr_dmat(simu->save->gradcl[iwfs], gradout);
	}
    }/*dtrat_output */
    ctoc("done");
    CUDA_SYNC_STREAM;
    curfree(phiout);
    curfree(gradref);
}
void gpu_wfsgrad_save(SIM_T *simu){
    const PARMS_T *parms=simu->parms;
    const int isim=simu->isim;
    const int seed=simu->seed;
    if((isim % 50 ==0) || isim+1==parms->sim.end){
	for(int iwfs=0; iwfs<simu->parms->nwfs; iwfs++){
	    gpu_set(wfsgpu[iwfs]);
	    cuwfs_t *cuwfs=cudata->wfs;
	    const PARMS_T *parms=simu->parms;
	    const int ipowfs=simu->parms->wfs[iwfs].powfs;
	    hipStream_t stream=cuwfs[iwfs].stream;
	    if(cuwfs[iwfs].neareal){
		const int dtrat=parms->powfs[ipowfs].dtrat;
		float scale;
		if(parms->powfs[ipowfs].usephy){
		    scale=(simu->isim+1-simu->parms->powfs[ipowfs].phystep)/dtrat;
		}else{
		    scale=(simu->isim+1)/dtrat;
		}
		if(scale>0){
		    scale=1.f/floor(scale);/*only multiple of dtrat is recorded. */
		    curmat *sanea=NULL;
		    curadd(&sanea, 0, cuwfs[iwfs].neareal, scale, stream);
		    curwrite(sanea,"sanea_sim_wfs%d_%d.bin",iwfs,seed);
		    curfree(sanea);
		}
	    }
	    if(cuwfs[iwfs].pistatout){
		int nstep=isim+1-parms->powfs[ipowfs].pistatstart;
		if(nstep>0){
		    curcell* tmp=NULL;
		    curcelladd(&tmp, 0, cuwfs[iwfs].pistatout, 1.f/(float)nstep, stream);
		    if(parms->sim.skysim){
			TO_IMPLEMENT;//fftshift in gpu.
			curcellwrite(tmp, "%s/pistat/pistat_seed%d_sa%d_x%g_y%g.bin",
				     dirskysim,simu->seed,
				     parms->powfs[ipowfs].order,
				     parms->wfs[iwfs].thetax*206265,
				     parms->wfs[iwfs].thetay*206265);
		    }else{
			curcellwrite(tmp,"pistat_seed%d_wfs%d.bin", simu->seed,iwfs);
		    }
		    curcellfree(tmp);
		}
	    }
	    CUDA_SYNC_STREAM;
	}
    }
}
