#include "hip/hip_runtime.h"
/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
#include "../maos/sim.h"
}
#include "utils.h"
#include "accphi.h"
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipfft/hipfft.h>
#include "wfs.h"
#include "cudata.h"
#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define ctoc(A)
#else
#define ctoc(A) CUDA_SYNC_STREAM; toc2(A)
#endif
extern const char *dirskysim;
/*
  Notice that both blocks and threads are partitioning isa
 */
__global__ static void add_geom_noise_do(float *restrict g, const float *restrict nea, 
				      int nsa, hiprandState *restrict rstat){
    const int id=threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState lstat=rstat[id];
    const int nstep=blockDim.x * gridDim.x;
    for(int i=id; i<nsa; i+=nstep){
	float n1=hiprand_normal(&lstat);
	float n2=hiprand_normal(&lstat);
	g[i]+=n1*nea[i];
	g[i+nsa]+=n2*nea[i+nsa]+n1*nea[i+nsa*2];/*cross term. */
    }
    rstat[id]=lstat;
}

/**
   Compute ztilt
*/
__global__ void cuztilt(float *restrict g, float *restrict opd, 
			const int nsa, const float dx, const int nx, float (**imcc)[3],
			const float (*orig)[2], const float*restrict amp, float alpha){
    __shared__ float a[3];
    if(threadIdx.x<3 && threadIdx.y==0){
	a[threadIdx.x]=0.f;
    }
    __syncthreads();
    const int isa=blockIdx.x;
    float b0=0.f;
    float b1=0.f;
    float b2=0.f;
    const int skip=isa*nx*nx;
    const float ox=orig[isa][0];
    const float oy=orig[isa][1];
    for(int iy=threadIdx.y; iy<nx; iy+=blockDim.y){
	const int skip2=skip+iy*nx;
	const float y=iy*dx+oy;
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    const int ind=skip2+ix;
	    const float tmp=amp[ind]*opd[ind];
	    b0+=tmp;
	    b1+=tmp*(dx*ix+ox);
	    b2+=tmp*y;
	}
    }
    atomicAdd(&a[0],b0);
    atomicAdd(&a[1],b1);
    atomicAdd(&a[2],b2);
    __syncthreads();/*Wait until all threads in this block is done. */
    if(threadIdx.x<3 && threadIdx.y==0){
	float (*restrict A)[3]=imcc[isa];
	atomicAdd(&g[isa],     alpha*(A[threadIdx.x][1]*a[threadIdx.x]));
	atomicAdd(&g[isa+nsa], alpha*(A[threadIdx.x][2]*a[threadIdx.x]));
    }
}
/**
   Apply matched filter. \todo this implementation relies on shared variable. It
is probably causing competition.  */
__global__ static void mtche_do(float *restrict grad, float (*restrict *restrict mtches)[2], 
				const float *restrict ints, const float *restrict i0sum, int pixpsa, int nsa){
    __shared__ float g[3];/*shared by threads in the same block (with the same isa). */
    if(threadIdx.x<3){
	g[threadIdx.x]=0.f;
    }
    __syncthreads();//is this necessary?
    int isa=blockIdx.x;
    ints+=isa*pixpsa;
    const float (*const restrict mtche)[2]=mtches[isa];
 
    float gp[3]={0.f,0.f,0.f};
    for (int ipix=threadIdx.x; ipix<pixpsa; ipix+=blockDim.x){
	gp[0]+=mtche[ipix][0]*ints[ipix];
	gp[1]+=mtche[ipix][1]*ints[ipix];
	gp[2]+=ints[ipix];
    }
    atomicAdd(&g[0], gp[0]);
    atomicAdd(&g[1], gp[1]);
    atomicAdd(&g[2], gp[2]);
    __syncthreads();
    if(threadIdx.x<2){
	if(i0sum){
	    /*normalize gradients according to siglev.*/
	    g[threadIdx.x]*=i0sum[isa]/g[2];
	}
	grad[isa+nsa*threadIdx.x]=g[threadIdx.x];
    }
}/*
static inline __device__ uint32_t float2int(uint32_t *f){
    //if *tmp is positive, mask is 0x800000000. If *tmp is negative, mask is 0xFFFFFFFF since -1 is 0xFFFFFFFF.
    uint32_t mask = (-(int32_t)(*f >> 31)) | 0x80000000;
    return (*f) ^ mask;
}
static inline __device__ uint32_t int2float(uint32_t f){
    uint32_t mask = ((f >> 31) - 1) | 0x80000000;
    return f ^ mask;
    }*/
/**
   Apply tCoG.
*/
__global__ static void tcog_do(float *grad, const float *restrict ints, 
			       int nx, int ny, float pixthetax, float pixthetay, int nsa, float (*cogcoeff)[2], float rne, float *srot){
    __shared__ float sum[3];
    if(threadIdx.x<3 && threadIdx.y==0) sum[threadIdx.x]=0.f;
    __syncthreads();//is this necessary?
    int isa=blockIdx.x;
    ints+=isa*nx*ny;
    float cogthres=cogcoeff[isa][0]*rne;
    float cogoff=cogcoeff[isa][1]*rne;
    for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
	for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
	    float im=ints[ix+iy*nx]-cogoff;
	    if(im>cogthres){
		atomicAdd(&sum[0], im);
		atomicAdd(&sum[1], im*ix);
		atomicAdd(&sum[2], im*iy);
	    }
	}
    }
    __syncthreads();
    if(threadIdx.x==0 && threadIdx.y==0){
	if(fabsf(sum[0])>0){
	    float gx=(sum[1]/sum[0]-(nx-1)*0.5)*pixthetax;
	    float gy=(sum[2]/sum[0]-(ny-1)*0.5)*pixthetay;
	    if(srot){
		float s,c;
		sincos(srot[isa], &s, &c);
		float tmp=gx*c-gy*s;
		gy=gx*s+gy*c;
		gx=tmp;
	    }
	    grad[isa]=gx;
	    grad[isa+nsa]=gy;
	}else{
	    grad[isa]=0;
	    grad[isa+nsa]=0;
	}
    }
}
/**
   Poisson random generator.
*/
__device__ static float curandp(hiprandState *rstat, float xm){
    float g, t, xmu;
    int x=0, xu;
    if(xm>200){
	x=(int)round(xm+hiprand_normal(rstat)*sqrt(xm));
    }else{
	while(xm>0){
	    xmu = xm > 12.f ? 12.f : xm;
	    xm -= xmu;
	    g   = __expf(-xmu);
	    xu  = -1;
	    t   = 1.f;
	    while(t>g){
		xu++;
		t *= hiprand_uniform(rstat);
	    }
	    x += xu;
	}
    }
    return x;
}
/**
   Add noise to pix images.
*/
__global__ static void addnoise_do(float *restrict ints0, int nsa, int pixpsa, float bkgrnd, float bkgrndc, 
				   float *const restrict *restrict bkgrnd2s, float *const restrict *restrict bkgrnd2cs,
				   float rne, hiprandState *rstat){
    const int id=threadIdx.x + blockIdx.x * blockDim.x;
    const int nstep=blockDim.x * gridDim.x;
    hiprandState lstat=rstat[id];
    for(int isa=id; isa<nsa; isa+=nstep){
	float *restrict ints=ints0+isa*pixpsa;
	const float *restrict bkgrnd2=bkgrnd2s?bkgrnd2s[isa]:NULL;
	const float *restrict bkgrnd2c=bkgrnd2cs?bkgrnd2cs[isa]:NULL;
	for(int ipix=0; ipix<pixpsa; ipix++){
	    float corr=bkgrnd2c?(bkgrnd2c[ipix]+bkgrndc):bkgrndc;
	    if(bkgrnd2){
		ints[ipix]=curandp(&lstat, ints[ipix]+bkgrnd+bkgrnd2[ipix])+rne*hiprand_normal(&lstat)-corr;
	    }else{
		ints[ipix]=curandp(&lstat, ints[ipix]+bkgrnd)+rne*hiprand_normal(&lstat)-corr;
	    }
	}
    }
    rstat[id]=lstat;
}
void gpu_fieldstop(curmat *opd, float *amp, int *embed, int nembed, 
		   curmat* fieldstop, float wvl, hipfftHandle fftplan, hipStream_t stream){
    cucmat wvf(nembed, nembed);
    embed_wvf_do<<<DIM(opd->nx, 256), 0, stream>>>
	(wvf.p, opd->p, amp, embed, opd->nx, wvl);
    CUFFT(fftplan, wvf.p, HIPFFT_FORWARD);
    cwm_do<<<DIM(wvf.nx*wvf.ny, 256),0,stream>>>
      (wvf.p, fieldstop->p, wvf.nx*wvf.ny);
    CUFFT(fftplan, wvf.p, HIPFFT_BACKWARD);
    unwrap_phase_do<<<DIM2(wvf.nx, wvf.ny, 16),0,stream>>>
	(wvf.p, opd->p, embed, opd->nx, wvl);
}
__global__ static void
dither_acc_do(float *restrict *im0, float *restrict *imx, float *restrict *imy, 
	      float *restrict const *pints, float cd, float sd, int pixpsa, int nsa){
    for(int isa=blockIdx.x; isa<nsa; isa+=gridDim.x){
	const float *ints=pints[isa];
	float *restrict acc_ints=im0[isa];
	float *restrict acc_intsx=imx[isa];
	float *restrict acc_intsy=imy[isa];
	for(int ipix=threadIdx.x; ipix<pixpsa; ipix+=blockDim.x){
	    acc_ints[ipix]+=ints[ipix];
	    acc_intsx[ipix]+=ints[ipix]*cd;
	    acc_intsy[ipix]+=ints[ipix]*sd;
	}
    }
}
dither_t::dither_t(int nsa, int pixpsax, int pixpsay):imc(0){
    im0=curcellnew(nsa,1,pixpsax,pixpsay);
    imx=curcellnew(nsa,1,pixpsax,pixpsay);
    imy=curcellnew(nsa,1,pixpsax,pixpsay);
}
void dither_t::reset(){
    imc=0;
    curcellzero(im0);
    curcellzero(imx);
    curcellzero(imy);
}
/**Accumulate for matched filter updating*/
void dither_t::acc(curcell *ints, float angle, hipStream_t stream){
    const int nsa=ints->nx*ints->ny;
    const int pixpsa=ints->p[0]->nx*ints->p[0]->ny;
    dither_acc_do<<<ints->nx, pixpsa, 0, stream>>>
	(im0->pm, imx->pm, imy->pm, ints->pm, cosf(angle), sinf(angle), pixpsa, nsa);
    imc++;
}
/**Output for matched filter updating*/
void dither_t::output(float a2m, int iwfs, int isim, hipStream_t stream){
    curcellscale(im0, 1./(imc), stream);
    curcellscale(imx, 2./(a2m*imc), stream);
    curcellscale(imy, 2./(a2m*imc), stream);
    CUDA_SYNC_STREAM;
    curcellwrite(im0, "wfs%d_i0_%d", iwfs, isim);
    curcellwrite(imx, "wfs%d_gx_%d", iwfs, isim);
    curcellwrite(imy, "wfs%d_gy_%d", iwfs, isim);
    reset();
}

/**
   Ray tracing and gradient computation for WFS. \todo Expand to do gradients in GPU without transfering
   data back to CPU.
*/
extern int *wfsgpu;
void gpu_wfsgrad_iwfs(SIM_T *simu, int iwfs){
    gpu_set(wfsgpu[iwfs]);
    cuwloc_t *cupowfs=cudata->powfs;
    cuwfs_t *cuwfs=cudata->wfs;
    TIC;tic;
    const PARMS_T *parms=simu->parms;
    assert(iwfs<parms->nwfs);
    const POWFS_T *powfs=simu->powfs;
    const RECON_T *recon=simu->recon;
    /*output */
    const int CL=parms->sim.closeloop;
    const int isim=simu->isim;
    /*The following are truly constants for this powfs */
    const int ipowfs=parms->wfs[iwfs].powfs;
    const int imoao=parms->powfs[ipowfs].moao;
    const int nsa=powfs[ipowfs].pts->nsa;
    const int wfsind=parms->powfs[ipowfs].wfsind[iwfs];
    const float hs=parms->powfs[ipowfs].hs;
    const int dtrat=parms->powfs[ipowfs].dtrat;
    const int save_gradgeom=parms->save.gradgeom[iwfs];
    const int save_opd =parms->save.wfsopd[iwfs];
    const int save_ints=parms->save.ints[iwfs];
    const int noisy=parms->powfs[ipowfs].noisy;
    /*The following depends on isim */
    const int dtrat_output=((isim+1)%dtrat==0);
    const int do_phy=(parms->powfs[ipowfs].usephy && isim>=parms->powfs[ipowfs].phystep);
    const int do_pistatout=parms->powfs[ipowfs].pistatout&&isim>=parms->powfs[ipowfs].pistatstart;
    const int do_geom=!do_phy || save_gradgeom || do_pistatout;
    const float thetax=parms->wfs[iwfs].thetax;
    const float thetay=parms->wfs[iwfs].thetay;
    const float dtisim=parms->sim.dt*isim;
    float (*loc)[2]=cupowfs[ipowfs].loc->p;
    const int nloc=cupowfs[ipowfs].loc->nloc;
    /*Out to host for now. \todo : keep grad in device when do reconstruction on device. */
    stream_t &stream=*cuwfs[iwfs].stream;
    dmat *gradcl=simu->gradcl->p[iwfs];
    curmat *phiout=curnew(nloc, 1);
    curmat *gradacc=cuwfs[iwfs].gradacc;
    curmat *gradcalc=NULL;
    if(cuwfs[iwfs].opdadd){ /*copy to phiout. */
	curcp(&phiout, cuwfs[iwfs].opdadd, stream);
    }else{
	curzero(phiout, stream);
    }
    if(parms->sim.idealwfs){
	gpu_dm2loc(phiout->p, cuwfs[iwfs].loc_dm, cudata->dmproj, cudata->ndm,
		   hs, thetax, thetay, 0, 0, 1, stream);
    }else{
	if(simu->atm){
	    gpu_atm2loc(phiout->p, cuwfs[iwfs].loc_tel, hs, thetax, thetay, 0, 0, dtisim, 1, stream);
	}
	if(parms->sim.wfsalias){
	    gpu_dm2loc(phiout->p, cuwfs[iwfs].loc_dm, cudata->dmproj, cudata->ndm,
		       hs, thetax, thetay, 0, 0, -1, stream);
	}
    }
    if(save_opd){
	cellarr_cur(simu->save->wfsopdol[iwfs], simu->isim, phiout, stream);
    }
    if(CL){
	gpu_dm2loc(phiout->p, cuwfs[iwfs].loc_dm, cudata->dmreal, cudata->ndm,
		   hs, thetax, thetay, 0, 0, -1, stream);
    }
    if(parms->tomo.ahst_idealngs && parms->powfs[ipowfs].skip){
	const double *cleNGSm=simu->cleNGSm->p+isim*recon->ngsmod->nmod;
	gpu_ngsmod2science(phiout, cupowfs[ipowfs].loc->p, recon->ngsmod, cleNGSm, 
			   parms->wfs[iwfs].thetax, parms->wfs[iwfs].thetay, 
			   -1, stream);
    }
    /*CUDA_SYNC_STREAM; */
    
    if(imoao>-1){
	gpu_dm2loc(phiout->p, cuwfs[iwfs].loc_dm, cudata->dm_wfs[iwfs], 1,
		   INFINITY, 0, 0, 0, 0, -1, stream);
    }
    if(simu->telws){
	float tt=simu->telws->p[isim];
	float angle=simu->winddir?simu->winddir->p[0]:0;
	curaddptt(phiout, loc, 0, tt*cosf(angle), tt*sinf(angle), stream);
    }
    if(parms->powfs[ipowfs].llt){
	float focus=(float)wfsfocusadj(simu, iwfs);
	if(fabsf(focus)>1e-20){
	    add_focus_do<<<DIM(nloc, 256), 0, stream>>>(phiout->p, loc, nloc, focus);
	}
    }
    if(parms->powfs[ipowfs].fieldstop){
	gpu_fieldstop(phiout, cuwfs[iwfs].amp, cupowfs[ipowfs].embed, cupowfs[ipowfs].nembed, 
		      cupowfs[ipowfs].fieldstop, parms->powfs[ipowfs].wvl[0], cuwfs[iwfs].plan_fs, stream);
    }
    if(save_opd){
	cellarr_cur(simu->save->wfsopd[iwfs], simu->isim, phiout, stream);
    }
    if(parms->plot.run>1){
	const double *realamp=powfs[ipowfs].realamp->p[wfsind]->p;
	dmat *tmp=NULL;
	cp2cpu(&tmp, phiout, stream);
	drawopdamp("wfsopd",powfs[ipowfs].loc,tmp->p,realamp,NULL,
		   "WFS OPD","x (m)", "y (m)", "WFS %d", iwfs);
	dfree(tmp);
    }
    if(do_geom){
	float ratio;
	if(!do_pistatout || parms->powfs[ipowfs].pistatstc || dtrat==1){
	    gradcalc=gradacc->ref();
	    ratio=1.f/(float)dtrat;
	}else{ //calculate first to gradcalc then add to gradacc
	    gradcalc=curnew(nsa*2, 1);	
	    ratio=1;
	}
	if(parms->powfs[ipowfs].gtype_sim==1){
	    cuztilt<<<nsa, dim3(16,16), 0, stream>>>
		(gradcalc->p, phiout->p, 
		 cupowfs[ipowfs].pts->nloc, 
		 cupowfs[ipowfs].pts->dxsa, 
		 cupowfs[ipowfs].pts->nxsa, cuwfs[iwfs].imcc,
		 cupowfs[ipowfs].pts->p, cuwfs[iwfs].amp, 
		 ratio);
	}else{
	    cusp *GS0=cuwfs[iwfs].GS0;
	    cuspmul(gradcalc->p, GS0, phiout->p, 1, 'n', ratio, stream);
	}
	
	if(gradcalc->p!=gradacc->p){
	    curadd(&gradacc, 1, gradcalc, 1.f/(float)dtrat, stream);
	}
    }   
    if(parms->powfs[ipowfs].psfout){
	cellarr_cur(simu->save->ztiltout[iwfs], simu->isim, gradacc, stream);
    }
    if(do_phy || parms->powfs[ipowfs].psfout || do_pistatout){/*physical optics */
	gpu_wfsints(simu, phiout->p, gradcalc, iwfs, isim, stream);
    }/*do phy */
    ctoc("grad");
    if(dtrat_output){
	if(do_phy){
	    /*signal level was already multiplied in ints. */
	    curcell *ints=cuwfs[iwfs].ints;
	    curmat *gradgpu=NULL;
	    const int pixpsa=powfs[ipowfs].pixpsax*powfs[ipowfs].pixpsay;
	    if(save_ints){
		cellarr_curcell(simu->save->intsnf[iwfs], simu->isim, ints, stream);
	    }
	    /*CUDA_SYNC_STREAM; */
	    ctoc("mtche");
	    float rne=0, bkgrnd=0;
	    if(noisy){
		rne=parms->powfs[ipowfs].rne;
		bkgrnd=parms->powfs[ipowfs].bkgrnd*dtrat;
		addnoise_do<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt, 0, stream>>>
		    (ints->p[0]->p, nsa, pixpsa, bkgrnd, bkgrnd*parms->powfs[ipowfs].bkgrndc,
		     cuwfs[iwfs].bkgrnd2, cuwfs[iwfs].bkgrnd2c, 
		     rne, cuwfs[iwfs].custat);
		ctoc("noise");
	    }
	    if(parms->powfs[ipowfs].dither && isim>=parms->powfs[ipowfs].dither_nskip){
		double angle=M_PI*0.5*isim/parms->powfs[ipowfs].dtrat;
		angle+=simu->dither[iwfs]->deltam;
		cuwfs[iwfs].dither->acc(ints, angle, stream);
		int nstat=parms->powfs[ipowfs].dither_nstat;
		int dtrat=parms->powfs[ipowfs].dtrat;
		if((isim-parms->powfs[ipowfs].dither_nskip+1)%(nstat*dtrat)==0){
		    warning2("Dither step%d, wfs%d: output statistics\n", isim, iwfs);
		    cuwfs[iwfs].dither->output((float)simu->dither[iwfs]->a2m, iwfs, isim, stream);
		}
	    }
	    gradgpu=curnew(nsa*2, 1);
	    switch(parms->powfs[ipowfs].phytypesim){
	    case 1:
		mtche_do<<<nsa, 16, 0, stream>>>(gradgpu->p, cuwfs[iwfs].mtche, ints->p[0]->p, 
						 parms->powfs[ipowfs].mtchscl?cuwfs[iwfs].i0sum:NULL,
						 pixpsa, nsa);
		break;
	    case 2:{
		float pixthetax=(float)parms->powfs[ipowfs].radpixtheta;
		float pixthetay=(float)parms->powfs[ipowfs].pixtheta;
		int pixpsax=powfs[ipowfs].pixpsax;
		int pixpsay=powfs[ipowfs].pixpsay;
		float *srot=parms->powfs[ipowfs].radpix?cuwfs[iwfs].srot:NULL;
		float rnee=sqrt(rne*rne+bkgrnd);
		tcog_do<<<nsa, dim3(pixpsax, pixpsay),0,stream>>>
		    (gradgpu->p, ints->p[0]->p, 
		     pixpsax, pixpsay, pixthetax, pixthetay, nsa, (float(*)[2])cuwfs[iwfs].cogcoeff, rnee, srot);
	    }
		break;
	    case 3:{
		dcell *cints=NULL;
		cp2cpu(&cints, ints, stream);
		CUDA_SYNC_STREAM;
		double geach[3];
		for(int isa=0; isa<nsa; isa++){
		    geach[0]=gradcl->p[isa];
		    geach[1]=gradcl->p[isa+nsa];
		    geach[2]=1;
		    maxapriori(geach, cints->p[isa], parms, powfs, iwfs, isa, 1, bkgrnd, rne);
		    gradcl->p[isa]=geach[0];
		    gradcl->p[isa+nsa]=geach[1];
		}
		dcellfree(cints);
	    }
		break;
	    default:
		TO_IMPLEMENT;
	    }
	    /*if(parms->powfs[ipowfs].mtchupdate){
		cuwfs[iwfs].mtchu->acc(ints, gradgpu, stream);
		}*/
	    if(save_ints){
		cellarr_curcell(simu->save->intsny[iwfs], simu->isim, ints, stream);
	    }
	    ctoc("mtche");
	    
	    /*send grad to CPU. */
	    if(parms->powfs[ipowfs].phytypesim!=3){//3 is handled in cpu.
		cp2cpu(&gradcl->p, 0, gradgpu->p, 1, nsa*2, stream);
	    }
	    ctoc("dev2dbl");
	    curcellzero(ints, stream);
	    CUDA_SYNC_STREAM;/*necessary. */
	    curfree(gradgpu);
	    ctoc("sync");

	    if(save_gradgeom){//also do geom grad during phy grad sims
		cellarr_cur(simu->save->gradgeom[iwfs], simu->isim, gradacc, stream);
	    }
	}else{
	    if(noisy && !parms->powfs[ipowfs].usephy){
		add_geom_noise_do<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt, 0, stream>>>
		    (gradacc->p, cuwfs[iwfs].neasim, nsa,cuwfs[iwfs].custat);
		ctoc("noise");
	    }
	    cp2cpu(&gradcl, gradacc, stream);
	    if(save_gradgeom){
		cellarr_cur(simu->save->gradgeom[iwfs], simu->isim, NULL, stream);
	    }
	    ctoc("dev2dbl");
	}
	if(do_geom){
	    curzero(gradacc, stream);
	}
	CUDA_SYNC_STREAM;
    }/*dtrat_output */
    ctoc("done");
    CUDA_SYNC_STREAM;
    curfree(phiout);
    curfree(gradcalc);
}
void gpu_wfsgrad_save(SIM_T *simu){
    const PARMS_T *parms=simu->parms;
    const int isim=simu->isim;
    if((isim % 50 ==0) || isim+1==parms->sim.end){
	for(int iwfs=0; iwfs<simu->parms->nwfs; iwfs++){
	    gpu_set(wfsgpu[iwfs]);
	    cuwfs_t *cuwfs=cudata->wfs;
	    const PARMS_T *parms=simu->parms;
	    const int ipowfs=simu->parms->wfs[iwfs].powfs;
	    stream_t &stream=*cuwfs[iwfs].stream;
	    if(cuwfs[iwfs].pistatout){
		int nstep=isim+1-parms->powfs[ipowfs].pistatstart;
		if(nstep>0){
		    curcell* tmp=cuwfs[iwfs].pistatout;
		    curcellscale(tmp, 1.f/(float)nstep, stream);
		    if(parms->sim.skysim){
			curcellwrite(tmp, "%s/pistat/pistat_seed%d_sa%d_x%g_y%g.bin",
				     dirskysim,simu->seed,
				     parms->powfs[ipowfs].order,
				     parms->wfs[iwfs].thetax*206265,
				     parms->wfs[iwfs].thetay*206265);
		    }else{
			curcellwrite(tmp,"pistat_seed%d_wfs%d.bin", simu->seed,iwfs);
		    }
		    curcellscale(tmp, nstep, stream);
		}
	    }
	}
    }
}
