#include "hip/hip_runtime.h"
/*
  Copyright 2009, 2010, 2011 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "curmat.h"
#include "pcg.h"

#include "recon.h" /*for  debugging */

#define PRINT_RES 0
#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define toc(A)
#endif
__global__ static void assign_do(float *dest, const float *restrict res){
    dest[0]=res[0];
}
__global__ static void div_do(float *dest, const float *a, const float *b){
    dest[0]=a[0]/b[0];
}
#if PRINT_RES
__global__ static void div_sqrt_do(float *dest, const float *a,  const float *b){
    dest[0]=sqrt(a[0]/b[0]);
}
#endif
__global__ static void scale_do(float *dest,  float b){
    dest[0]*=b;
}
/**
   The PCG algorithm. Copy of lib/pcg.c, but replacing dcell with curcell.
   Timing: 
   curcellinn implemented with blas takes 0.457 ms each call.
   curcellinn implemented with kernel takes 0.193 ms each call with 256x32 split. 0.158 with 128x16 split. 0.137 with 64x64 split.
   curcelladd takes 0.048 ms per call.
   return non-zero during unconvergence.
 */

int gpu_pcg(curcell **px, 
	    G_CGFUN Amul, const void *A, 
	    G_PREFUN Mmul, const void *M, 
	    const curcell *b, int warm, int maxiter,
	    hipStream_t stream){
    TIC;tic;
    int ans=0;
    curcell *r0=NULL;
    curcell *x0=NULL;/*The initial vector. equals to *px*/
    curcell *z0=NULL;/*Is reference or preconditioned value. */
    typedef struct{
	float r0z0;
	float r0z1;
	float r0z2;
	float ak;
	float bk;
	float tmp;
    }CGRES_T;
    CGRES_T *res;
    /*structure that contains temporary scalars. */
    DO(hipMalloc((float**)&res, sizeof(CGRES_T)));
    /*computes r0=b-A*x0 */
    curcellcp(&r0, b, stream);
    if(!*px){
	*px=curcellnew(b);
    }
    x0=*px;
    if(warm){
	Amul(&r0, 1, A, x0, -1);/*r0=r0+(-1)*A*x0 */
    }else{
	curcellzero(x0, stream);
    }

    curcell *p0=NULL;
    if(Mmul){
	CUDA_SYNC_STREAM;
	Mmul(&z0,M,r0);
    }else{
	z0=r0;
    }
    curcellcp(&p0, z0, stream);
    curcellinn2(&res->r0z1, r0, z0, stream);
    curcell *Ap=NULL;
#if PRINT_RES == 1
    curcellinn2(&res->r0z0, b, b, stream);
    float diff[maxiter+1];
    if(Mmul){
	/*res->tmp=r0'*r0; */
	curcellinn2(&res->tmp, r0, r0, stream);
	/*tmp=sqrt(rmp/r0z0); */
	div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->tmp, &res->r0z0);
    }else{
	/*tmp=sqrt(r0z1/r0z0); */
	div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->r0z1, &res->r0z0);
    }
    hipMemcpyAsync(&diff[0], &res->tmp, sizeof(float), hipMemcpyDefault, stream);
#endif
    for(int k=0; k<maxiter; k++){
	Amul(&Ap, 0, A, p0, 1);
	/*ak=r0z1/(p0'*Ap); */
	curcellinn2(&res->ak, p0, Ap, stream);
	div_do<<<1,1,0,stream>>>(&res->ak, &res->r0z1, &res->ak);
	CUDA_SYNC_STREAM;/*put here helps to remove the spikes in performance/wfs */
	curcelladd2(&r0, Ap, &res->ak, -1, stream);/*r0=r0-ak*Ap */
	curcelladd2(&x0, p0, &res->ak, 1, stream);/*x0=x0+ak*p0 */
	if(Mmul){
	    CUDA_SYNC_STREAM;
	    Mmul(&z0,M,r0);
	}
	/*r0z2=r0'*z0 */
	curcellinn2(&res->r0z2, r0, z0, stream);
#if PRINT_RES == 1
	if(Mmul){ 
	    /*diff[k+1]=sqrt(r0'*r0/r0z0) */
	    curcellinn2(&res->tmp, r0, r0, stream);
	    div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->tmp, &res->r0z0);
	}else{ 
	    /*diff[k+1]=sqrt(r0z2/r0z0); */
	    div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->r0z2, &res->r0z0);
	}
	hipMemcpyAsync(&diff[k+1], &res->tmp, sizeof(float), hipMemcpyDefault,stream);
#endif
	/*bk=r0z2/r0z1; */
	div_do<<<1,1,0,stream>>>(&res->bk, &res->r0z2, &res->r0z1);
	/*p0=bk*p0+z0 */
	curcelladd3(&p0, &res->bk, z0, stream);
	/*r0z1=r0z2; */
	assign_do<<<1,1,0,stream>>>(&res->r0z1, &res->r0z2);
	toc("cg");
    }
    /* Instead of check in the middle, we only copy the last result. Improves performance by 20 nm !!!*/
//curcellcp(px, x0, stream);
    CUDA_SYNC_STREAM;
#if PRINT_RES == 1
    if(diff[maxiter]>0.02 && curecon->reconisim>5){
	ans=1;
    }
#endif
    curcellfree(r0); 
    if(Mmul){
	curcellfree(z0);
    }
    curcellfree(Ap);
    curcellfree(p0);
    hipFree(res);
    return ans;
}
