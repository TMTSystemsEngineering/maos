#include "hip/hip_runtime.h"
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "curmat.h"
#include "pcg.h"

#include "recon.h" //for  debugging

#define PRINT_RES 0
#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define toc(A)
#endif
__global__ static void assign_do(float *dest, const float *restrict res){
    dest[0]=res[0];
}
__global__ static void div_do(float *dest, const float *a, const float *b){
    dest[0]=a[0]/b[0];
}
#if PRINT_RES
__global__ static void div_sqrt_do(float *dest, const float *a,  const float *b){
    dest[0]=sqrt(a[0]/b[0]);
}
#endif
__global__ static void scale_do(float *dest,  float b){
    dest[0]*=b;
}
/**
   The PCG algorithm. Copy of lib/pcg.c, but replacing dcell with curcell.
   Timing: 
   curcellinn implemented with blas takes 0.457 ms each call.
   curcellinn implemented with kernel takes 0.193 ms each call with 256x32 split. 0.158 with 128x16 split. 0.137 with 64x64 split.
   curcelladd takes 0.048 ms per call.
   return non-zero during unconvergence.
 */

int gpu_pcg(curcell **px, 
	    G_CGFUN Amul, const void *A, 
	    G_PREFUN Mmul, const void *M, 
	    const curcell *b, int warm, int maxiter,
	    hipStream_t stream){
    TIC;tic;
    int ans=0;
    curcell *r0=NULL;
    curcell *x0=NULL;//The initial vector.
    curcell *z0=NULL;//Is reference or preconditioned value.
    typedef struct{
	float r0z0;
	float r0z1;
	float r0z2;
	float ak;
	float bk;
	float tmp;
    }CGRES_T;
    CGRES_T *res;
    //structure that contains temporary scalars.
    DO(hipMalloc((float**)&res, sizeof(CGRES_T)));
    //computes r0=b-A*x0
    curcellcp(&r0, b, stream);
    if(!*px || !warm){//start from zero guess.
	x0=curcellnew2(b);
	if(!*px) *px=curcellnew(x0->nx, x0->ny);
    }else{
	curcellcp(&x0, *px, stream);
	CUDA_SYNC_STREAM;
	Amul(&r0, 1, A, x0, -1);//r0=r0+(-1)*A*x0
    }
    curcell *p0=NULL;
    if(Mmul){
	CUDA_SYNC_STREAM;
	Mmul(&z0,M,r0);
    }else{
	z0=r0;
    }
    curcellcp(&p0, z0, stream);
    curcellinn2(&res->r0z1, r0, z0, stream);
    curcell *Ap=NULL;
#if PRINT_RES == 1
    curcellinn2(&res->r0z0, b, b, stream);
    float diff[maxiter+1];
    if(Mmul){
	//res->tmp=r0'*r0;
	curcellinn2(&res->tmp, r0, r0, stream);
	//tmp=sqrt(rmp/r0z0);
	div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->tmp, &res->r0z0);
    }else{
	//tmp=sqrt(r0z1/r0z0);
	div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->r0z1, &res->r0z0);
    }
    hipMemcpyAsync(&diff[0], &res->tmp, sizeof(float), hipMemcpyDefault, stream);
#endif
    for(int k=0; k<maxiter; k++){
	Amul(&Ap, 0, A, p0, 1);
	//ak=r0z1/(p0'*Ap);
	curcellinn2(&res->ak, p0, Ap, stream);
	div_do<<<1,1,0,stream>>>(&res->ak, &res->r0z1, &res->ak);
	CUDA_SYNC_STREAM;//put here helps to remove the spikes in performance/wfs
	curcelladd2(&r0, Ap, &res->ak, -1, stream);//r0=r0-ak*Ap
	curcelladd2(&x0, p0, &res->ak, 1, stream);//x0=x0+ak*p0
	if(Mmul){
	    CUDA_SYNC_STREAM;
	    Mmul(&z0,M,r0);
	}
	//r0z2=r0'*z0
	curcellinn2(&res->r0z2, r0, z0, stream);
#if PRINT_RES == 1
	if(Mmul){ 
	    //diff[k+1]=sqrt(r0'*r0/r0z0)
	    curcellinn2(&res->tmp, r0, r0, stream);
	    div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->tmp, &res->r0z0);
	}else{ 
	    //diff[k+1]=sqrt(r0z2/r0z0);
	    div_sqrt_do<<<1,1,0,stream>>>(&res->tmp, &res->r0z2, &res->r0z0);
	}
	hipMemcpyAsync(&diff[k+1], &res->tmp, sizeof(float), hipMemcpyDefault,stream);
	if(curecon->reconisim>10){
	    if(diff[k+1]>diff[k]){
		warning("CG%d  %d: Step %d %.5f --> %.5f\n", maxiter,
			curecon->reconisim, k+1, diff[k], diff[k+1]);
		if(diff[k+1]>0.1){
		    curcellwrite(Ap, "CG%d_Ap_%d_%d", maxiter,curecon->reconisim, k+1);
		    curcellwrite(p0, "CG%d_p0_%d_%d", maxiter,curecon->reconisim, k+1);
		    curcellwrite(x0, "CG%d_x0_%d_%d", maxiter,curecon->reconisim, k+1);
		    curcellwrite(r0, "CG%d_r0_%d_%d", maxiter,curecon->reconisim, k+1);
		    Amul(&Ap, 0, A, p0, 1);
		    curcellwrite(Ap, "CG%d_Ap2_%d_%d", maxiter,curecon->reconisim, k+1);
		    ans=1;
		}
	    }

	    for(int ips=0; ips<x0->nx; ips++){
		float max=0;
		if((max=curmax(x0->p[ips], curecon->psstream[ips]))>2e-5){
		    warning("CG%d  %d: Step %d max(x0)=%g\n", maxiter, curecon->reconisim, k+1, max);
		    curcellwrite(Ap, "CG%d_Ap_%d_%d", maxiter,curecon->reconisim, k+1);
		    curcellwrite(p0, "CG%d_p0_%d_%d", maxiter,curecon->reconisim, k+1);
		    curcellwrite(x0, "CG%d_x0_%d_%d", maxiter,curecon->reconisim, k+1);
		    curcellwrite(r0, "CG%d_r0_%d_%d", maxiter,curecon->reconisim, k+1);
		    curcellwrite(*px, "CG%d_px_%d_%d", maxiter,curecon->reconisim, k+1);
		    Amul(&Ap, 0, A, p0, 1);
		    curcellwrite(Ap, "CG%d_Ap2_%d_%d", maxiter,curecon->reconisim, k+1);
		    ans=1;
		}
	    }
	}
#endif
	//bk=r0z2/r0z1;
	div_do<<<1,1,0,stream>>>(&res->bk, &res->r0z2, &res->r0z1);
	//p0=bk*p0+z0
	curcelladd3(&p0, &res->bk, z0, stream);
	//r0z1=r0z2;
	assign_do<<<1,1,0,stream>>>(&res->r0z1, &res->r0z2);
	toc("cg");
    }
    /* Instead of check in the middle, we only copy the last result. Improves performance by 20 nm !!!*/
    curcellcp(px, x0, stream);
    CUDA_SYNC_STREAM;
#if PRINT_RES == 1
    if(diff[maxiter]>0.02 && curecon->reconisim>5){
	if(maxiter>20){//tomo
	    warning2("Tomo %d: PCG: %.5f --> %.5f\n", curecon->reconisim, diff[0], diff[maxiter]);
	    for(int i=0; i<=maxiter; i++){
		info("Tomo %d: PCG: Step %d: res=%.5f\n", curecon->reconisim, i, diff[i]);
	    }
	    curcellwrite(curecon->gradin, "tomo_cugrad_%d", curecon->reconisim);
	    curcellwrite(b, "tomo_b_%d", curecon->reconisim);
	    curcellwrite(*px, "tomo_x_%d",  curecon->reconisim);
	}else{
	    warning2("Fit  %d: PCG: %.5f --> %.5f\n", curecon->reconisim, diff[0], diff[maxiter]);
	    for(int i=0; i<=maxiter; i++){
		info("Fit  %d: PCG: Step %d: res=%.5f\n", curecon->reconisim, i, diff[i]);
	    }	
	    curcellwrite(b, "fit_b_%d", curecon->reconisim);
	    curcellwrite(*px, "fit_x_%d",  curecon->reconisim);
	}
	ans=1;
    }

    for(int ips=0; ips<x0->nx; ips++){
	float max=0;
	if((max=curmax((*px)->p[ips], curecon->psstream[ips]))>2e-5){
	    int k=31;
	    warning("CG%d  %d: Step %d max(x0)=%g\n", maxiter, curecon->reconisim, k+1, max);
	    curcellwrite(Ap, "CG%d_Ap_%d_%d", maxiter,curecon->reconisim, k+1);
	    curcellwrite(p0, "CG%d_p0_%d_%d", maxiter,curecon->reconisim, k+1);
	    curcellwrite((*px), "CG%d_x0_%d_%d", maxiter,curecon->reconisim, k+1);
	    Amul(&Ap, 0, A, p0, 1);
	    curcellwrite(Ap, "CG%d_Ap2_%d_%d", maxiter,curecon->reconisim, k+1);
	    ans=1;
	}
    }
#endif
    curcellfree(r0); 
    if(Mmul){
	curcellfree(z0);
    }
    curcellfree(x0);
    curcellfree(Ap);
    curcellfree(p0);
    hipFree(res);
    return ans;
}
