#include "hip/hip_runtime.h"
/*
  Copyright 2009-2022 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "curmat.h"
#include "cucmat.h"
#include "utils.h"
#include "kernel.h"

void curset(curmat& A, Real alpha, hipStream_t stream){
	if(A&&A()){
		set_do<<<DIM(A.Nx()*A.Ny(), 256), 0, stream>>>(A(), alpha, A.Nx()*A.Ny());
	}
}

/**
   out=out*beta+in*alpha;
*/
void curadd(curmat& out, Real alpha, const curmat& in, Real beta, hipStream_t stream){
	if(!in) return;
	if(!out||alpha==0){
		cucp(out, in, stream);
		if(Z(fabs)(beta-(Real)1)>EPS){
			scale_do<<<DIM(in.Nx()*in.Ny(), 256), 0, stream>>>
				(out(), in.Nx()*in.Ny(), beta);
		}
	} else{
		assert(in.N()==out.N());
		add_do<<<DIM(in.Nx()*in.Ny(), 256), 0, stream>>>
			(out(), NULL, alpha, in(), NULL, beta, in.Nx()*in.Ny());
	}
}
/**
   out=out*beta+abs2(in)*alpha;
*/
void curaddcabs2(curmat& out, Real alpha, const cucmat& in, Real beta, hipStream_t stream){
	if(!out){
		out=curmat(in.Nx(), in.Ny());
	} else if(alpha==0){
		cuzero(out, stream);
	}
	addcabs2_do<<<DIM(in.Nx()*in.Ny(), 256), 0, stream>>>
		(out(), alpha, in(), beta, in.Nx()*in.Ny());
}
/**
   out=out+abs2(in)*alpha;
*/
void curaddcabs2(curmat &out, const cucmat &in, Real beta, hipStream_t stream){
	if(!out){
		out=curmat(in.Nx(), in.Ny());
	}
	addcabs2_do<<<DIM(in.Nx()*in.Ny(), 256), 0, stream>>>
		(out(), in(), beta, in.Nx()*in.Ny());
}
void curscale(curmat &in, Real alpha, hipStream_t stream){
	if(!in) return;
	if(alpha==0){
		in.Zero(stream);
	} else if(Z(fabs)(alpha-(Real)1)>EPS){
		scale_do<<<DIM(in.N(), 256), 0, stream>>>(in(), in.N(), alpha);
	}
}


/**
   Computes C = alpha * C + beta * op(A) * B ;
*/
void curmv(Real* c, Real alpha, const curmat& A, const Real* b, char trans, Real beta, stream_t& stream){
	CUBL(gemv)(stream.blas(), (trans=='t'||trans==1)?HIPBLAS_OP_T:HIPBLAS_OP_N, A.Nx(), A.Ny(), &beta, A(), A.Nx(), b, 1, &alpha, c, 1);
}
void curcellmm(curcell& C, Real alpha, const curcell& A, const curcell& B,
	const char trans[2], const Real beta, stream_t& stream){
	if(!A||!B) return;
	int ax, az;
	int nx, ny, nz;
	int bz, by;
	if(trans[0]=='n'||trans[0]=='N'){
		nx=A.Nx();
		ax=1; az=A.Nx();
		nz=A.Ny();
	} else{
		nx=A.Ny();
		az=1; ax=A.Nx();
		nz=A.Nx();
	}
	if(trans[1]=='n'||trans[0]=='N'){
		ny=B.Ny();
		bz=1; by=B.Nx();
		if(nz!=B.Nx()) error("mismatch\n");
	} else{
		ny=B.Nx();
		by=1; bz=B.Nx();
		if(nz!=B.Ny()) error("mismatch\n");
	}
	if(!C){
		C=curcell(nx, ny);
	} else{
		assert(C.Nx()==nx&&C.Ny()==ny);
		if(alpha==0){
			cuzero(C, stream);
		} else if(Z(fabs)(alpha-(Real)1)>EPS){
			curcellscale(C, alpha, stream);
		}
	}
	for(int iy=0; iy<ny; iy++){
		for(int ix=0; ix<nx; ix++){
			for(int iz=0; iz<nz; iz++){
				if(A[ix*ax+iz*az]&&B[iz*bz+iy*by]){
					cugemm(C[ix+iy*nx], (Real)1., A[ix*ax+iz*az],
						B[iz*bz+iy*by], trans, beta, stream);
				}
			}
		}
	}
}
/*Transpose a matrix in naive way. Faster way is to use shared memory and handle
  a block each time.*/
__global__ void transpose(Real* restrict out, const Real* restrict in, int nx, int ny){
	const int stepx=blockDim.x*gridDim.x;
	const int stepy=blockDim.y*gridDim.y;
	const int ix0=threadIdx.x+blockDim.x*blockIdx.x;
	const int iy0=threadIdx.y+blockDim.y*blockIdx.y;
	for(int iy=iy0; iy<ny; iy+=stepy){
		for(int ix=ix0; ix<nx; ix+=stepx){
			out[iy+ix*ny]=in[ix+iy*nx];
		}
	}
}
/*Transpose a matrix*/
template <>
curmat curmat::trans(stream_t& stream){
	curmat B=curmat(ny, nx);
	transpose<<<dim3(16, 16), dim3(16, 16), 0, stream>>>
		(B(), p, nx, ny);
	return B;
}

/*
  A=A*beta+B*alpha;
*/
void curcelladd(curcell& A, Real beta, const curcell& B, Real alpha, hipStream_t stream){
	if(!B) return;
	if(!A){
		A=B.New();
	} else{
		assert(A.Nx()==B.Nx()&&A.Ny()==B.Ny());
	}
	if(A.M()&&B.M()){
		curadd(A.M(), beta, B.M(), alpha, stream);
	} else{
		for(int i=0; i<B.Nx()*B.Ny(); i++){
			curadd(A[i], beta, B[i], alpha, stream);
		}
	}
}

void curadd(curmat& A, Real beta, hipStream_t stream){
	const int n=A.Nx()*A.Ny();
	add_do<<<DIM(n, 256), 0, stream>>>(A(), beta, n);
}
/**
   add a vector to another, scaled by alpha and beta. all in device memory.
   a=a+b*alpha*beta;
*/

/**
   out=out+in*alpha; beta, alpha lives in device memory.
*/
void curadd(curmat& out, const curmat& in, Real* alpha, Real alpha2, hipStream_t stream){
	if(!out){
		out=curmat(in.Nx(), in.Ny());
	}
	add_do<<<DIM(in.Nx()*in.Ny(), 256), 0, stream>>>
		(out(), in(), alpha, alpha2, in.Nx()*in.Ny());
}


/**
   A=A*beta+B*alpha; beta, alpha lives in device memory.
*/
void curcelladd(curcell& A, const curcell& B, Real* alpha, Real alpha2, hipStream_t stream){
	if(!B) return;
	if(!A){
		A=B.New();
	} else{
		assert(A.Nx()==B.Nx()&&A.Ny()==B.Ny());
	}
	if(A.M()&&B.M()){
		curadd(A.M(), B.M(), alpha, alpha2, stream);
	} else{
		for(int i=0; i<B.N(); i++){
			curadd(A[i], B[i], alpha, alpha2, stream);
		}
	}
}

/**
   out=out*beta+in; beta, alpha lives in device memory.
*/
void curadd(curmat& out, Real* alpha1, const curmat& in, hipStream_t stream){
	if(!out){
		out=curmat(in.Nx(), in.Ny());
	}
	add_do<<<DIM(in.Nx()*in.Ny(), 256), 0, stream>>>
		(out(), alpha1, 1.f, in(), in.Nx()*in.Ny());
}

/**
   A=A*alpha1+B*alpha; alpha1, alpha lives in device memory.
*/
void curcelladd(curcell& A, Real* alpha1, const curcell& B, hipStream_t stream){
	if(!B) return;
	if(!A){
		A=B.New();
	} else{
		assert(A.Nx()==B.Nx()&&A.Ny()==B.Ny());
	}
	if(A.M()&&B.M()){
		curadd(A.M(), alpha1, B.M(), stream);
	} else{
		for(int i=0; i<B.Nx()*B.Ny(); i++){
			curadd(A[i], alpha1, B[i], stream);
		}
	}
}



Real curinn(const curmat& a, const curmat& b, hipStream_t stream){
	curmat res(1, 1);
	Real out;
	inn_wrap(res(), a(), b(), a.Nx()*a.Ny(), stream);
	DO(hipMemcpyAsync(&out, res(), sizeof(Real), D2H, stream));
	CUDA_SYNC_STREAM;
	return out;
}

/**
   Sum all the elements in an array.
 */
void cursum2(Real* restrict res,/**<Result in GPU*/
	const curmat& a,   /**<Source in GPU*/
	hipStream_t stream){
	DO(hipMemsetAsync(res, 0, sizeof(Real), stream));
	sum_wrap(res, a(), a.Nx()*a.Ny(), stream);
}
/**
   Sum all the elements in an array, and return a value.
*/
Real cursum(const curmat& a, hipStream_t stream){
	Real out;//result in CPU.
	curmat res(1, 1);
	sum_wrap(res, a(), a.Nx()*a.Ny(), stream);
	DO(hipMemcpyAsync(&out, res(), sizeof(Real), D2H, stream));
	CUDA_SYNC_STREAM;
	return out;
}

/**
   Find the maximum value
*/
Real curmax(const curmat& a, hipStream_t stream){
	Real out;
	curmat res(1, 1);
	max_wrap(res, a(), a.N(), stream);
	DO(hipMemcpyAsync(&out, res(), sizeof(Real), D2H, stream));
	CUDA_SYNC_STREAM;
	return out;
}

/**
   Find the maximum value
*/
Real curmaxabs(const curmat& a, hipStream_t stream){
	Real out;
	curmat res(1, 1);
	maxabs_wrap(res, a(), a.N(), stream);
	DO(hipMemcpyAsync(&out, res(), sizeof(Real), D2H, stream));
	CUDA_SYNC_STREAM;
	return out;
}
/**
   Find the maximum value
*/
Real curcellmax(const curcell& a, hipStream_t stream){
	int n=a.Nx()*a.Ny();
	Real out;
	curmat res(n+1, 1);
	for(int i=0; i<n; i++){
		int m=a[i].N();
		max_wrap(&res[i], a[i](), m, stream);
	}
	if(n>1){
		max_wrap(&res[n], res, n, stream);
	}
	DO(hipMemcpyAsync(&out, &res[n>1?n:0], sizeof(Real), D2H, stream));
	CUDA_SYNC_STREAM;
	return out;
}
/**
   Find the maximum value
*/
Real curcellmaxabs(const curcell& a, hipStream_t stream){
	int n=a.N();
	Real out;
	curmat res(n+1, 1);
	for(int i=0; i<n; i++){
		int m=a[i].N();
		maxabs_wrap(&res[i], a[i](), m, stream);
	}
	if(n>1){
		maxabs_wrap(&res[n], res, n, stream);
	}
	DO(hipMemcpyAsync(&out, &res[n>1?n:0], sizeof(Real), D2H, stream));
	CUDA_SYNC_STREAM;
	return out;
}
/**
   Scale elements
*/
void curcellscale(curcell& A, Real alpha, hipStream_t stream){
	if(!A) return;
	if(A.M()){
		curscale(A.M(), alpha, stream);
	} else{
		for(int i=0; i<A.Nx()*A.Ny(); i++){
			curscale(A[i], alpha, stream);
		}
	}
}



void cucscale(cucmat& in, Real alpha, hipStream_t stream){
	if(!in) return;
	if(alpha==0){
		cuzero(in, stream);
	} else if(Z(fabs)(alpha-1.f)>EPS){
		int n=in.Nx()*in.Ny();
		scale_do<<<DIM(n, 256), 0, stream>>>(in(), n, alpha);
	}
}
