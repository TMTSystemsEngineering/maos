/*
  Copyright 2009-2022 Lianqi Wang <lianqiw-at-tmt-dot-org>

  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * 
 * Wraps cuda cusparse, cusolver routines
 * */
#include "hipblas.h"
#include "curmat.h"
#include "cucmat.h"
#include <hipsolver.h>
int NULL_STREAM=0;
static hipsparseMatDescr_t spdesc=NULL;
static __attribute((constructor)) void init(){
	DO(hipsparseCreateMatDescr(&spdesc));
	hipsparseSetMatType(spdesc, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(spdesc, HIPSPARSE_INDEX_BASE_ZERO);
	char *tmp=getenv("CUDA_LAUNCH_BLOCKING");
	if(tmp){
		int blocking=strtol(tmp, NULL, 10);
		if(blocking){
			warning("CUDA_LAUNCH_BLOCKING is enabled. Use only NULL stream\n");
			NULL_STREAM=1; //Use only default stream
		}
	}
}


static const char *scsrmv_err[]={
	"Success",
	"Not initialized",
	"Allocation failed",
	"Invalid value",
	"Archtecture mismatch",
	"Mapping error",
	"Execution failed",
	"Internal error",
	"Matrix type not supported"
};

/*
	y=A*x where A is sparse. x, y are vectors or matrices. (Slow for GS0.)
	Y = α op (A) ⋅ X + β Y or Y = α op (A) ⋅ X + β Y with β=1
	Converted cuSparse generic API for CUDA version 11 which removed old API.
*/

void cuspmul(Real *y, const cusp &A, const Real *x, int ncolvec, char trans, Real alpha, stream_t &stream){
	hipsparseOperation_t opr;
	int istrans=(trans=='t'||trans==1);
	if(A.Type()==SP_CSC){
		istrans=!istrans;
	}
	if(istrans){
		opr=HIPSPARSE_OPERATION_TRANSPOSE;
	} else{
		opr=HIPSPARSE_OPERATION_NON_TRANSPOSE;
	}
	int ncol=0, nrow=0;
	switch(A.Type()){
	case SP_CSR:
		nrow=A.Nx(); ncol=A.Ny(); break;
	case SP_CSC:
		nrow=A.Ny(); ncol=A.Nx(); break;
	default:
		error("Invalid type: %d", A.Type());
	}
	int status;
	Real one=1.f;

#if CUDA_VERSION >= 10000
	/*
hipsparseStatus_t
hipsparseSpMV(hipsparseHandle_t     handle,
					hipsparseOperation_t  opA,
					const void*          alpha,
					hipsparseSpMatDescr_t matA,
					hipsparseDnVecDescr_t vecX,
					const void*          beta,
					hipsparseDnVecDescr_t vecY,
					hipDataType         computeType,
					hipsparseSpMVAlg_t    alg,
					void*                externalBuffer)

hipsparseStatus_t
hipsparseSpMM(hipsparseHandle_t     handle,
			hipsparseOperation_t  opA,
			hipsparseOperation_t  opB,
			const void*          alpha,
			hipsparseSpMatDescr_t matA,
			hipsparseDnMatDescr_t matB,
			const void*          beta,
			hipsparseDnMatDescr_t matC,
			hipDataType         computeType,
			hipsparseSpMMAlg_t    alg,
			void*                externalBuffer)

	*/
	size_t bsize;
	void *buffer;
	size_t ny=istrans?ncol:nrow;
	size_t nx=istrans?nrow:ncol;
	if(ncolvec==1){
		hipsparseDnVecDescr_t xv, yv;
		hipsparseCreateDnVec(&xv, nx, (void *)x, CUDA_R);
		hipsparseCreateDnVec(&yv, ny, (void *)y, CUDA_R);
#if CUDA_VERSION > 11010 
		hipsparseSpMVAlg_t alg=HIPSPARSE_SPMV_ALG_DEFAULT;
#else
		hipsparseSpMVAlg_t alg=HIPSPARSE_MV_ALG_DEFAULT;
#endif
		DO(status=hipsparseSpMV_bufferSize(stream.sparse(), opr, &alpha, A.Desc(), xv, &one, yv, CUDA_R, alg, &bsize));
		DO(hipMalloc(&buffer, bsize));
		DO(status=hipsparseSpMV(stream.sparse(), opr, &alpha, A.Desc(), xv, &one, yv, CUDA_R, alg, buffer));
		DO(hipFree(buffer));
		hipsparseDestroyDnVec(yv);
		hipsparseDestroyDnVec(xv);
	} else{
		hipsparseDnMatDescr_t Bm, Cm;
		hipsparseCreateDnMat(&Bm, nx, ncolvec, nx, (void *)x, CUDA_R, HIPSPARSE_ORDER_COL);
		hipsparseCreateDnMat(&Cm, ny, ncolvec, ny, (void *)y, CUDA_R, HIPSPARSE_ORDER_COL);
		hipsparseOperation_t opB=HIPSPARSE_OPERATION_NON_TRANSPOSE;
#if CUDA_VERSION > 11010
		hipsparseSpMMAlg_t alg=HIPSPARSE_SPMM_ALG_DEFAULT;
#else
		hipsparseSpMMAlg_t alg=HIPSPARSE_MM_ALG_DEFAULT;
#endif
		DO(status=hipsparseSpMM_bufferSize(stream.sparse(), opr, opB, &alpha,
										  A.Desc(), Bm, &one, Cm, CUDA_R, alg, &bsize));
		DO(hipMalloc(&buffer, bsize));

		DO(status=hipsparseSpMM(stream.sparse(), opr, opB, &alpha,
							   A.Desc(), Bm, &one, Cm, CUDA_R, alg, buffer));
		DO(hipFree(buffer));
		hipsparseDestroyDnMat(Bm);
		hipsparseDestroyDnMat(Cm);
	}
#else		
	if(ncolvec==1){
		status=CUSP(csrmv)(stream.sparse(), opr,
			nrow, ncol, A.Nzmax(), &alpha, spdesc,
			A.Px(), A.Pp(), A.Pi(), x, &one, y);

	} else{
		int nleadx=istrans?nrow:ncol;
		int nleady=istrans?ncol:nrow;
		status=CUSP(csrmm)(stream.sparse(), opr,
			nrow, ncolvec, ncol, A.Nzmax(), &alpha, spdesc,
			A.Px(), A.Pp(), A.Pi(), x, nleadx, &one, y, nleady);
	}
#endif
	if(status!=0){
		error("cuspmul failed with status '%s'\n", scsrmv_err[status]);
	}
}

template<>
hipblasStatus_t cublasGdgmm(hipblasHandle_t handle, hipblasSideMode_t mode,
						  int m, int n,
						  const float *A, int lda,
						  const float *x, int incx,
						  float *C, int ldc){
	return hipblasSdgmm(handle, mode, m, n, A, lda, x, incx, C, ldc);
}
hipblasStatus_t cublasGdgmm(hipblasHandle_t handle, hipblasSideMode_t mode,
						  int m, int n,
						  const double *A, int lda,
						  const double *x, int incx,
						  double *C, int ldc){
	return hipblasDdgmm(handle, mode, m, n, A, lda, x, incx, C, ldc);
}
template<>
hipblasStatus_t cublasGgemm(hipblasHandle_t handle,
						   hipblasOperation_t transa, hipblasOperation_t transb,
						   int m, int n, int k,
						   const float *alpha,
						   const float *A, int lda,
						   const float *B, int ldb,
						   const float *beta,
						   float *C, int ldc){
	return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
template<>
hipblasStatus_t cublasGgemm(hipblasHandle_t handle,
						   hipblasOperation_t transa, hipblasOperation_t transb,
						   int m, int n, int k,
						   const double *alpha,
						   const double *A, int lda,
						   const double *B, int ldb,
						   const double *beta,
						   double *C, int ldc){
	return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
