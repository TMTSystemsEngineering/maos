/*
  Copyright 2009-2013 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "wfs.h"
#include "recon.h"
#include "pcg.h"
#include "curmat.h"
#include "cucmat.h"
#include "accphi.h"


typedef struct MVM_IGPU_T{
    const PARMS_T *parms;
    RECON_T *recon;
    POWFS_T *powfs;
    curcell *mvmig; /*intermediate TomoL result*/
    curcell *mvmfg; /*intermediate FitR result*/
    smat *mvmt;     /*result: tranpose of MVM calculated by this GPU.*/
    float *FLI;
    smat *residual;
    smat *residualfit;
    long (*curp)[2];
    int ntotact;
    int ntotgrad;
    int load_mvmf; /*intermediate FitR result is for 1) loading, 0) saving.*/
}MVM_IGPU_T;
static void mvm_trans_igpu(thread_t *info){
    TIC;tic;
    double tk_prep=0, tk_fitL=0, tk_fitR=0, tk_TomoL=0, tk_TomoR=0, tk_cp=0;
    MVM_IGPU_T *data=(MVM_IGPU_T*)info->data;
    const PARMS_T *parms=data->parms;
    RECON_T *recon=data->recon;
    smat *residual=data->residual;
    smat *residualfit=data->residualfit;
    long (*curp)[2]=data->curp;
    const int ntotact=data->ntotact;
    const int ntotgrad=data->ntotgrad;
    const int load_mvmf=data->load_mvmf;
    int igpu=info->ithread;
    gpu_set(igpu);
    curecon_t *curecon=cudata->recon;
    curmat *mvmi=data->mvmig?data->mvmig->p[igpu]:NULL;/*Tomography output, for warm restart*/
    curmat *mvmf=data->mvmfg?data->mvmfg->p[igpu]:NULL;/*loaded FitR output.*/
    /*Tomography*/
    G_PREFUN prefun=NULL;
    void *predata=NULL;
    if(parms->tomo.precond==1){
	prefun=gpu_Tomo_fdprecond;
	predata=(void*)recon;
    }
  
    curcell *eyec=NULL;/* Only use eyec for CG.*/
    float eye2c[2]={0,1.};
    float *eye2;
    hipMalloc(&eye2, sizeof(float)*2);
    hipMemcpy(eye2, eye2c, sizeof(float)*2, hipMemcpyHostToDevice);
    //const int nwfs=parms->nwfsr;
    const int ndm=parms->ndm;
    /*fit*/
    G_CGFUN cg_fun;
    void *cg_data;
    if(parms->gpu.fit==1){//sparse matrix
	cg_fun=(G_CGFUN) cumuv;
	cg_data=&curecon->FL;
    }else{
	cg_fun=(G_CGFUN) gpu_FitL;
	cg_data=recon;
    }
    const float *FLI=data->FLI;
    if(!FLI && !load_mvmf){
	if(parms->fit.square){
	    eyec=curcellnew(ndm, 1, recon->anx, recon->any);
	}else{
	    eyec=curcellnew(ndm, 1, recon->anloc, (long*)0);
	}
    }
 
    curcell *dmfit=load_mvmf?NULL:curcellnew(curecon->dmfit);
    curcell *opdx=curcellnew(recon->npsr, 1, recon->xnx, recon->xny, (float*)(mvmf?1L:0L));
    curcell *opdr=curcellnew(recon->npsr, 1, recon->xnx, recon->xny, (float*)(mvmi?1L:0L));
    curcell *grad=curcellnew(parms->nwfsr, 1, recon->ngrad, (long*)0, (float*)1);
    if(ntotact==0){
	error("ntotact=0;\n");
    }
    curmat *mvmt=curnew(ntotgrad, info->end-info->start);/*contains result*/
    tk_prep+=toc3;tic;
    stream_t &stream=curecon->cgstream;
    for(int iact=info->start; iact<info->end; iact++){
	int curdm=curp[iact][0];
	int curact=curp[iact][1];
	if(info->ithread==0){
	    if(!detached){
		info2("%6d of %6d\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b", iact*NGPU, ntotact);
	    }else if(iact % 100==0){
		info2("%6d of %6d\n", iact*NGPU, ntotact);
	    }
	}
	if(eyec){
	    if(iact){
		hipMemcpyAsync(eyec->m->p+iact-1, eye2, 2*sizeof(float),
				hipMemcpyDeviceToDevice, stream);
	    }else{
		hipMemcpyAsync(eyec->m->p+iact, eye2+1, sizeof(float), 
				hipMemcpyDeviceToDevice, stream);
	    }
	}
	if(!recon->actcpl || recon->actcpl->p[curdm]->p[curact]>EPS){
	    if(mvmf) opdx->replace(mvmf->p+(iact-info->start)*mvmf->nx, 0, stream);
	    if(!load_mvmf){
		if(eyec){
		    /*Fitting operator*/
		    curcellzero(dmfit, stream);//temp
		    if((residualfit->p[iact]=gpu_pcg(dmfit, (G_CGFUN)cg_fun, cg_data, NULL, NULL, eyec, &curecon->cgtmp_fit,
						    parms->recon.warm_restart, parms->fit.maxit, stream))>1.){
		    warning("Fit CG residual is %.2f for %d.\n",
			    residualfit->p[iact], iact);
		    }
		}else{
		    hipMemcpyAsync(dmfit->m->p, FLI+iact*ntotact, sizeof(float)*ntotact, 
				    hipMemcpyHostToDevice, stream);
		}
    		tk_fitL+=toc3; tic;
		/*Transpose of fitting operator*/
		if(parms->gpu.fit==1){//sparse matrix
		    cumuv_trans(&opdx, 0, &curecon->FR, dmfit, 1, stream);
		}else{
		    gpu_FitRt(&opdx, 0, recon, dmfit, 1, stream);
		}
	    }
	    tk_fitR+=toc3; tic;
	    switch(parms->tomo.alg){
	    case 0:
		if(!opdr->m || !opdx->m){
		    error("opdr and opdx must be continuous\n");
		}
		cuchol_solve(opdr->m->p, curecon->RCl, curecon->RCp, opdx->m->p, stream);
		if(curecon->RUp){
		    curmat *tmp=curnew(curecon->RVp->ny, 1);
		    curmv(tmp->p, 0, curecon->RVp, opdx->m->p, 't', -1, stream);
		    curmv(opdr->m->p, 1, curecon->RUp, tmp->p, 'n', 1, stream);
		    curfree(tmp);
		}
		break;
	    case 1:{
		if(mvmi){
		    opdr->replace(mvmi->p+(iact-info->start)*mvmi->nx, 0, stream);
		}
		/*disable the t/t removal lrt in split tomo that creats problem in fdpcg mode*/
		if((residual->p[iact]=gpu_pcg(opdr, gpu_TomoL, recon, prefun, predata, opdx, &curecon->cgtmp_tomo,
					      parms->recon.warm_restart, parms->tomo.maxit,
					      stream, parms->tomo.cgthres))>0.5){
		    warning2("Tomo CG residual is %.2f for %d\n", residual->p[iact], iact);
		}
	    }
		break;
	    case 2:
		curmv(opdr->m->p, 0, curecon->RMI, opdx->m->p, 'n', 1, stream);
		break;
	    default:
		error("Invalid");
	    }
	    tk_TomoL+=toc3; tic;
	    /*Right hand side. output directly to mvmt*/
	    grad->replace(mvmt->p+(iact-info->start)*ntotgrad, 0, stream);
	    gpu_TomoRt(&grad, 0, recon, opdr, 1, stream);
	    tk_TomoR+=toc3; tic;
	}
    }//for iact
    int nn=ntotgrad*(info->end-info->start)*sizeof(float);
    float *mvmtc=data->mvmt->p+info->start*ntotgrad;
    hipMemcpyAsync(mvmtc, mvmt->p, nn, hipMemcpyDeviceToHost, curecon->cgstream);
    hipStreamSynchronize(curecon->cgstream);
    curcellfree(dmfit);
    curcellfree(opdx);
    curcellfree(opdr);
    curcellfree(grad);
    curcellfree(eyec);
    curfree(mvmt);
    hipFree(eye2);
    tk_cp+=toc3;tic;
    info2("GPU %d: Prep %.2f FitL %.2f FitR %.2f TomoL %.1f TomoR %.1f cp %.2f\n", 
	  igpu, tk_prep, tk_fitL, tk_fitR, tk_TomoL, tk_TomoR, tk_cp);
}

void gpu_setup_recon_mvm_trans(const PARMS_T *parms, RECON_T *recon, POWFS_T *powfs){
    TIC;tic;
    if(parms->recon.alg!=0){
	error("Please adapt to LSR\n");
    } 
    if(!parms->load.mvm){
	info2("Assembling MVR MVM (transpose) in GPU\n");
	int ntotact=0;
	int ntotgrad=0;
	int ntotxloc=0;
	const int ndm=parms->ndm;
	for(int idm=0; idm<ndm; idm++){
	    ntotact+=recon->anloc[idm];
	} 
	for(int ips=0; ips<recon->npsr; ips++){
	    ntotxloc+=recon->xloc[ips]->nloc;
	}
	for(int iwfs=0; iwfs<parms->nwfsr; iwfs++){
	    ntotgrad+=recon->ngrad[iwfs];
	}
	

	long (*curp)[2]=(long(*)[2])malloc(ntotact*2*sizeof(long));
	int nact=0;
	for(int idm=0; idm<ndm; idm++){
	    for(int iact=0; iact<recon->anloc[idm]; iact++){
		curp[nact+iact][0]=idm;
		curp[nact+iact][1]=iact;
	    }
	    nact+=recon->anloc[idm];
	}   

	smat *residual=NULL;
	smat *residualfit=NULL;
	if(parms->tomo.alg==1){
	    residual=snew(ntotact, 1);
	}
	if(parms->fit.alg==1){
	    residualfit=snew(ntotact, 1);
	}
	dmat *FLId=NULL; /* MI is inv(FL) for direct methods*/
	float *FLI=NULL;

	/* Loading or saving intermediate TomoL result. */
	smat *mvmi=NULL; 
	if(parms->load.mvmi){
	    mvmi=sread("%s", parms->load.mvmi);
	    if(mvmi->nx!=ntotxloc || mvmi->ny!=ntotact){
		error("loaded mvmi has dimension (%ld, %ld) but we expect (%d, %d)",
		      mvmi->nx, mvmi->ny, ntotxloc, ntotact);
	    }
       	}else if(parms->save.mvmi){
	    mvmi=snew(ntotxloc, ntotact);
	}
	curcell *mvmig=NULL;
	if(mvmi){
	    mvmig=curcellnew(NGPU, 1);
	}

	/* Loading or saving intermediate FitR Result.*/
	smat *mvmf=NULL;
	if(parms->load.mvmf){
	    /*Load FitR FitL results from file. Resembling warm restart case
	      where mvmf is kept in memory*/
	    mvmf=sread("%s", parms->load.mvmf);
	    if(mvmf->nx!=ntotxloc || mvmf->ny!=ntotact){
		error("loaded mvmf has dimension (%ld, %ld) but we expect (%d, %d)",
		      mvmf->nx, mvmf->ny, ntotxloc, ntotact);
	    }
	}else if(parms->save.mvmf){
	    /*save FitR FitL resutls to file, for later loading.*/
	    mvmf=snew(ntotxloc, ntotact);
	}
	curcell *mvmfg=NULL;
	if(mvmf){
	    mvmfg=curcellnew(NGPU, 1);
	}
	if(!parms->load.mvmf){
	    /*Prepare FitR, FitL is don't load fitting results using mvmf*/
	    switch(parms->fit.alg){
	    case 0:{//Use CPU to handle CBS.
		dmat *eye=dnew(ntotact, ntotact);
		daddI(eye, 1);
		FLId=dnew(ntotact, ntotact);
		muv_direct_solve(&FLId, &recon->FL, eye);
		dfree(eye);
		toc("Fit CBS");tic;
	    }
		break;
	    case 1://Use GPU.
		break;
	    case 2:
		FLId=dref(recon->FL.MI);
		break;
	    default:
		error("Invalid fit.alg=%d\n", parms->fit.alg);
	    }
	    if(FLId){
		FLI=(float*)malloc4async(sizeof(float)*ntotact*ntotact);
		for(long i=0; i<ntotact*ntotact; i++){
		    FLI[i]=(float)FLId->p[i];
		}
		dfree(FLId);
	    }
	}
    	smat *mvmt=snew(ntotgrad, ntotact);
	MVM_IGPU_T data={parms, recon, powfs, mvmig, mvmfg, mvmt, FLI, residual, residualfit, curp, ntotact, ntotgrad, parms->load.mvmf?1:0};
	int nthread=NGPU;
	thread_t info[nthread];
	thread_prep(info, 0, ntotact, nthread, mvm_trans_igpu, &data);

	/*Initialyze intermediate TomoL result array in GPU. Send intermediate
	  TomoL results to GPU if load.mvmi is set.*/
	if(mvmi){
	    TIC;tic;
	    for(int i=0; i<NGPU; i++){
		gpu_set(i);
		mvmig->p[i]=curnew(ntotxloc, info[i].end-info[i].start);
		if(parms->load.mvmi){
		    hipMemcpy(mvmig->p[i]->p, mvmi->p+info[i].start*ntotxloc, 
			       sizeof(float)*ntotxloc*(info[i].end-info[i].start), hipMemcpyHostToDevice);
		}
	    }
	    if(parms->load.mvmi){
		toc2("copy mvmi to gpu");
	    }
	}
	/*Initialyze intermediate FitL/FitR result array in GPU. Send
	  intermediate FitL/FitR results to GPU if load.mvmf is set.*/
	if(mvmf){
	    TIC;tic;
	    for(int i=0; i<NGPU; i++){
		gpu_set(i);
		mvmfg->p[i]=curnew(ntotxloc, info[i].end-info[i].start);
		if(parms->load.mvmf){
		    hipMemcpy(mvmfg->p[i]->p, mvmf->p+info[i].start*ntotxloc, 
			       sizeof(float)*ntotxloc*(info[i].end-info[i].start), hipMemcpyHostToDevice);
		}
	    }
	    if(parms->load.mvmf){
		toc2("copy mvmf to gpu");
	    }
	}
	/*Do real MVM control matrix assemble in multiply CPU/GPU*/
	CALL_THREAD(info, nthread, 1);
	/*Copy MVM control matrix results back*/
	{
	    TIC;tic;
	    int ndm=parms->ndm;
	    int nwfs=parms->nwfsr;
	    recon->MVM=dcellnew(ndm, nwfs);
	    for(int iwfs=0; iwfs<nwfs; iwfs++){
		int ipowfs=parms->wfsr[iwfs].powfs;
		if(!parms->powfs[ipowfs].skip){
		    for(int idm=0; idm<ndm; idm++){
			recon->MVM->p[idm+ndm*iwfs]=dnew(recon->anloc[idm], powfs[ipowfs].saloc->nloc*2);
		    }
		}
	    }
	    dmat *mvmtt=dnew(mvmt->ny, mvmt->nx);
	    for(int iy=0; iy<mvmtt->ny; iy++){
		for(int ix=0; ix<mvmtt->nx; ix++){
		    mvmtt->p[ix+iy*mvmtt->nx]=(double)mvmt->p[iy+ix*mvmt->nx];
		}
	    }
	    toc2("MVM Reshape in CPU 1");
	    sfree(mvmt);
	    d2cell(&recon->MVM, mvmtt, NULL);
	    dfree(mvmtt);
	    toc2("MVM Reshape in CPU 2");
	}
	swrite(residual, "MVM_RL_residual");
	swrite(residualfit, "MVM_FL_residual");
	
	if(parms->save.mvmi){
	    for(int i=0; i<NGPU; i++){
		gpu_set(i);
		hipMemcpy(mvmi->p+info[i].start*ntotxloc, mvmig->p[i]->p,  
			   sizeof(float)*ntotxloc*(info[i].end-info[i].start), hipMemcpyDeviceToHost);
	    }
	    swrite(mvmi, "MVM_Tomo.bin");
	}
	if(parms->save.mvmf){
	    for(int i=0; i<NGPU; i++){
		gpu_set(i);
		hipMemcpy(mvmf->p+info[i].start*ntotxloc, mvmfg->p[i]->p,  
			   sizeof(float)*ntotxloc*(info[i].end-info[i].start), hipMemcpyDeviceToHost);
	    }
	    swrite(mvmf, "MVM_FitL.bin");
	}
	if(mvmig){
	    for(int i=0; i<NGPU; i++){
		gpu_set(i);
		curfree(mvmig->p[i]);
	    }
	    curcellfree(mvmig);
	}
	if(mvmfg){
	    for(int i=0; i<NGPU; i++){
		gpu_set(i);
		curfree(mvmfg->p[i]);
	    }
	    curcellfree(mvmfg);
	}
	sfree(mvmi);
	sfree(mvmf);
	sfree(residual);
	sfree(residualfit);
	free(curp);
	if(FLI) free4async(FLI);
    }//if assemble in gpu
}
