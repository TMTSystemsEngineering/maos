#include "hip/hip_runtime.h"
/*
  Copyright 2009-2012 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "wfs.h"
#include "recon.h"
#include "accphi.h"
#include "cucmat.h"
#include "pcg.h"

#define TIMING 0
#if TIMING==1
#define CTIC TIC
#define ctic tic
#define ctoc(A) toc2(A);tic
#define ctocs(A) CUDA_SYNC_STREAM; toc2(A);tic
#else
#define CTIC
#define ctic
#define ctoc(A)
#endif
/*
  If merge the operation in to gpu_prop_grid_adaptive_do, need to do atomic
operation because previous retracing starts at offo, not 0.  */
__global__ void gpu_laplacian_do(GPU_PROP_GRID_T *data, float **inall, float **in2all, int nwfs, float alpha){
    int ips=blockIdx.z;
    float *restrict in=in2all[ips];
    float *restrict out=inall[ips];
    GPU_PROP_GRID_T *datai=data+nwfs*ips;
    int nx=datai->nxi;
    int ny=datai->nyi;
    float alpha2=datai->l2c*alpha;
    int stepx=blockDim.x*gridDim.x;
    int stepy=blockDim.y*gridDim.y;
    int nx1=nx-1;
    int ny1=ny-1;
    const int ix0=blockIdx.x*blockDim.x+threadIdx.x;
    const int iy0=blockIdx.y*blockDim.y+threadIdx.y;
    for(int iy=iy0; iy<ny; iy+=stepy){
	int iy1 =iy+1; if(iy1>ny1) iy1-=ny;
	int iy2 =iy+2; if(iy2>ny1) iy2-=ny;
	int iy1p=iy-1; if(iy1p<0) iy1p+=ny;
	int iy2p=iy-2; if(iy2p<0) iy2p+=ny;
	for(int ix=ix0; ix<nx; ix+=stepx){
	    int ix1 =ix+1; if(ix1>nx1) ix1-=nx;
	    int ix2 =ix+2; if(ix2>nx1) ix2-=nx;
	    int ix1p=ix-1; if(ix1p<0) ix1p+=nx;
	    int ix2p=ix-2; if(ix2p<0) ix2p+=nx;
	    out[ix+iy*nx]+=alpha2*(20.f*in[ix+iy*nx]
				   -8.f*(in[ix1p+iy*nx]+in[ix1+iy*nx]+in[ix+iy1p*nx]+in[ix+iy1*nx])
				   +2.f*(in[ix1p+iy1p*nx]+in[ix1+iy1p*nx]+in[ix1p+iy1*nx]+in[ix1+iy1*nx])
				   +(in[ix+iy2p*nx]+in[ix2p+iy*nx]+in[ix2+iy*nx]+in[ix+iy2*nx]));
	}
    }
    if(datai->zzi>-1){/*piston constaint*/
	if(threadIdx.x==0 && threadIdx.y==0 && blockIdx.x==0 && blockIdx.y==0){
	    out[datai->zzi]+=in[datai->zzi]*datai->zzv*alpha;
	}
    }
}
__global__ void gpu_prop_grid_adaptive_do(GPU_PROP_GRID_T *data, float **outall, float **inall, int nwfs, int nps, 
					  float alpha, char trans){
    /*
      Each block handles a specific part of some wfs
    */
    int iwfs, ips, nn;
    if(trans=='t'){
	ips=blockIdx.z;
	nn=nwfs;
    }else{
	iwfs=blockIdx.z;
	nn=nps;
    }
    const int ix0=blockIdx.x*blockDim.x+threadIdx.x;
    const int iy0=blockIdx.y*blockDim.y+threadIdx.y;
    const int stepx=blockDim.x*gridDim.x;
    const int stepy=blockDim.y*gridDim.y;

    for(int ii=0; ii<nn; ii++){
	if(trans=='t'){
	    iwfs=ii;
	}else{
	    ips=ii;
	}
	GPU_PROP_GRID_T *datai=data+iwfs+nwfs*ips;
	const int nx=datai->nx;
	const int ny=datai->ny;
	if(nx==0) continue;//skip empty wfs
	const int nxo=datai->nxo;
	const int nxi=datai->nxi;
	
	float *restrict out;
	float *restrict in;
	if(datai->trans=='r'){/*reverse input/output*/
	    out=inall[ips]+datai->offo;
	    in=outall[iwfs]+datai->offi;
	}else{
	    out=outall[iwfs]+datai->offo;
	    in=inall[ips]+datai->offi;
	}
	const float ratio=datai->ratio;

	if(fabsf(ratio-1.f)<EPS){//Matched. always forward prop.
	    const float fracx=datai->dispx;
	    const float fracy=datai->dispy;
	    const float fracx1=1.f-fracx;
	    const float fracy1=1.f-fracy;
	    /*During reverse operation, for different iwfs, the offo is
	      different causing same thread to handle different memory in out
	      for different iwfs. This causes problem without synchronization/atomic operation*/
	    if(datai->trans=='r'){
		for(int iy=iy0; iy<ny; iy+=stepy){
		    for(int ix=ix0; ix<nx; ix+=stepx){
			atomicAdd(&out[ix+iy*nxo], 
				  alpha*(+(in[ix+    iy*nxi]*fracx1+in[ix+1+    iy*nxi]*fracx)*fracy1
					 +(in[ix+(iy+1)*nxi]*fracx1+in[ix+1+(iy+1)*nxi]*fracx)*fracy));
		    }
		}
	    }else{
		for(int iy=iy0; iy<ny; iy+=stepy){
		    for(int ix=ix0; ix<nx; ix+=stepx){
			out[ix+iy*nxo]+=
			    alpha*(+(in[ix+    iy*nxi]*fracx1+in[ix+1+    iy*nxi]*fracx)*fracy1
				   +(in[ix+(iy+1)*nxi]*fracx1+in[ix+1+(iy+1)*nxi]*fracx)*fracy);
		    }
		}
	    }
	}else{//Generic
	    const float dispx=datai->dispx;
	    const float dispy=datai->dispy;
	    if(datai->trans=='t'){
		for(int iy=iy0; iy<ny; iy+=stepy){
		    float jy;
		    float fracy=modff(dispy+iy*ratio, &jy);
		    int ky=(int)jy;
		    for(int ix=ix0; ix<nx; ix+=stepx){
			float jx;
			float fracx=modff(dispx+ix*ratio, &jx);
			int kx=(int)jx;
			float temp=out[ix+iy*nxo]*alpha;
			atomicAdd(&in[kx+      ky*nxi], temp*(1.f-fracx)*(1.f-fracy));
			atomicAdd(&in[kx+1    +ky*nxi], temp*fracx*(1.f-fracy));
			atomicAdd(&in[kx+  (ky+1)*nxi], temp*(1.f-fracx)*fracy);
			atomicAdd(&in[kx+1+(ky+1)*nxi], temp*fracx*fracy);
		    }
		}
	    }else{
		for(int iy=iy0; iy<ny; iy+=stepy){
		    float jy;
		    float fracy=modff(dispy+iy*ratio, &jy);
		    int ky=(int)jy;
		    for(int ix=ix0; ix<nx; ix+=stepx){
			float jx;
			float fracx=modff(dispx+ix*ratio, &jx);
			int kx=(int)jx;
			out[ix+iy*nxo]+=
			    alpha*(+(in[kx+      ky*nxi]*(1.f-fracx)+
				     in[kx+1+    ky*nxi]*fracx)*(1.f-fracy)
			       +(in[kx  +(ky+1)*nxi]*(1.f-fracx)+
				 in[kx+1+(ky+1)*nxi]*fracx)*fracy);
		    }
		}
	    }
	}
    }
}

/*
  The third grid dimension tells the wfs to handle. 
*/
#define DIM_GP 128
__global__ static void gpu_gp_do(GPU_GP_T *data, float **gout, float *ttout, float *dfout, float **wfsopd, int ptt){
    __shared__ float gx[DIM_GP];
    __shared__ float gy[DIM_GP];
    __shared__ float gdf[DIM_GP];
    const int iwfs=blockIdx.z;
    const int nwfs=gridDim.z;
    GPU_GP_T *datai=data+iwfs;
    const int pos=datai->pos;
    if(!pos) return;
    const int nsa=datai->nsa;
    const int step=blockDim.x * gridDim.x;
    int (*restrict saptr)[2]=datai->saptr;
    float *restrict g=gout[iwfs];
    if(wfsopd){
	const float *restrict map=wfsopd[iwfs];
	const float *pxy=datai->GPp;
	int nx=datai->nxp;
	/*GP operation.*/
	if(pos==1){
	    for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){
		int ix=saptr[isa][0];
		int iy=saptr[isa][1];
	
		const float *restrict pxy2=pxy+isa*8;
		g[isa]=
		    +map[iy*nx+ix  ]*pxy2[0]
		    +map[iy*nx+ix+1]*pxy2[1]
		    +map[(iy+1)*nx+ix ] *pxy2[2]
		    +map[(iy+1)*nx+ix+1]*pxy2[3];

		g[isa+nsa]=
		    +map[iy*nx+ix  ]*pxy2[4]
		    +map[iy*nx+ix+1]*pxy2[5]
		    +map[(iy+1)*nx+ix ] *pxy2[6]
		    +map[(iy+1)*nx+ix+1]*pxy2[7];
	    }/*for isa */
	}else{
	    for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){
		int ix=saptr[isa][0];
		int iy=saptr[isa][1];
		const float *restrict pxy2=pxy+isa*18;
		g[isa]=
		    +map[iy*nx+ix  ]*pxy2[0]
		    +map[iy*nx+ix+1]*pxy2[1]
		    +map[iy*nx+ix+2]*pxy2[2]
		    +map[(iy+1)*nx+ix ] *pxy2[3]
		    +map[(iy+1)*nx+ix+1]*pxy2[4]
		    +map[(iy+1)*nx+ix+2]*pxy2[5]
		    +map[(iy+2)*nx+ix ] *pxy2[6]
		    +map[(iy+2)*nx+ix+1]*pxy2[7]
		    +map[(iy+2)*nx+ix+2]*pxy2[8];
		g[isa+nsa]=
		    +map[iy*nx+ix  ]*pxy2[9]
		    +map[iy*nx+ix+1]*pxy2[10]
		    +map[iy*nx+ix+2]*pxy2[11]
		    +map[(iy+1)*nx+ix ] *pxy2[12]
		    +map[(iy+1)*nx+ix+1]*pxy2[13]
		    +map[(iy+1)*nx+ix+2]*pxy2[14]
		    +map[(iy+2)*nx+ix ] *pxy2[15]
		    +map[(iy+2)*nx+ix+1]*pxy2[16]
		    +map[(iy+2)*nx+ix+2]*pxy2[17];
	    }/*for isa */
	}
    }
    /* Global TT, Diff-Focus projection. Modifed from previous kernel so that
       each thread handle the same subaperture as previous gradient operation to
       avoid synchronization */
    if(datai->PTT && ptt){ //temp
	float (*restrict PTT)[2]=(float(*)[2])datai->PTT;
	gx[threadIdx.x]=0;
	gy[threadIdx.x]=0;
	for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){/*ng is nsa*2. */
	    gx[threadIdx.x]+=PTT[isa][0]*g[isa];
	    gy[threadIdx.x]+=PTT[isa][1]*g[isa];
	    gx[threadIdx.x]+=PTT[isa+nsa][0]*g[isa+nsa];
	    gy[threadIdx.x]+=PTT[isa+nsa][1]*g[isa+nsa];
	}
	for(int step=(DIM_GP>>1); step>0; step>>=1){
	    __syncthreads();
	    if(threadIdx.x<step){
		gx[threadIdx.x]+=gx[threadIdx.x+step];
		gy[threadIdx.x]+=gy[threadIdx.x+step];
	    }
	}
	if(threadIdx.x==0){
	    atomicAdd(&ttout[iwfs*2], -gx[0]);
	    atomicAdd(&ttout[iwfs*2+1], -gy[0]);
	}
    }
    if(datai->PDF && ptt){
	for(int irow=0; irow<nwfs; irow++){
	    float *restrict PDF=datai->PDF[irow];
	    if(!PDF) continue;
	    gdf[threadIdx.x]=0;
	    for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){/*ng is nsa*2. */
		gdf[threadIdx.x]+=PDF[isa]*g[isa]+PDF[isa+nsa]*g[isa+nsa];
	    }
	    for(int step=(DIM_GP>>1); step>0; step>>=1){
		__syncthreads();
		if(threadIdx.x<step){
		    gdf[threadIdx.x]+=gdf[threadIdx.x+step];
		}
	    }
	    if(threadIdx.x==0){
		atomicAdd(&dfout[irow], -gdf[0]);
	    }
	}
    }
}
__global__ static void gpu_gpt_do(GPU_GP_T *data, float **wfsopd, float *ttin, float *dfin, float **gin, int ptt){
    const int iwfs=blockIdx.z;
    const int nwfs=gridDim.z;
    GPU_GP_T *datai=data+iwfs;
    const int pos=datai->pos;
    if(!pos) return;
    const int step=blockDim.x * gridDim.x;
    const int nsa=datai->nsa;
    int (*saptr)[2]=datai->saptr;
    const float (*restrict neai)[3]=datai->neai;
    float dxp=datai->dxp;
    float oxp=datai->oxp;
    float oyp=datai->oyp;
    float focus=0;
    if(datai->PDF && ptt){
	if(iwfs==0){
	    for(int id=1; id<nwfs; id++){
		focus+=dfin[id];
	    }
	}else{
	    focus=-dfin[iwfs];
	}
    }
    const float *restrict g=gin[iwfs];
    float *restrict map=wfsopd[iwfs];
    const float *pxy=datai->GPp;
    float ttx=0, tty=0;
    if(datai->PTT && ptt){
	ttx=ttin[iwfs*2+0];
	tty=ttin[iwfs*2+1];
    }
    const int nx=datai->nxp;
    if(pos==1){
	for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){
	    int ix=saptr[isa][0];
	    int iy=saptr[isa][1];
	    float cx=neai[isa][0];
	    float cy=neai[isa][1];
	    float cxy=neai[isa][2];
	    float gx=g[isa    ]+ttx+focus*(ix*dxp+oxp);
	    float gy=g[isa+nsa]+tty+focus*(iy*dxp+oyp);
	    float tmp=cxy*gx;
	    gx=cx*gx+cxy*gy;
	    gy=tmp+cy*gy;
	    const float *restrict pxy2=pxy+isa*8;
	    atomicAdd(&map[iy    *nx+ix],   gx*pxy2[0] + gy*pxy2[4]);
	    atomicAdd(&map[iy    *nx+ix+1], gx*pxy2[1] + gy*pxy2[5]);
	    atomicAdd(&map[(iy+1)*nx+ix],   gx*pxy2[2] + gy*pxy2[6]);
	    atomicAdd(&map[(iy+1)*nx+ix+1], gx*pxy2[3] + gy*pxy2[7]);
	}
    }else if(pos==2){
	for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){
	    int ix=saptr[isa][0];
	    int iy=saptr[isa][1];
	    float cx=neai[isa][0];
	    float cy=neai[isa][1];
	    float cxy=neai[isa][2];
	    float gx=g[isa    ]+ttx+focus*(ix*dxp+oxp);
	    float gy=g[isa+nsa]+tty+focus*(iy*dxp+oyp);
	    float tmp=cxy*gx;
	    gx=cx*gx+cxy*gy;
	    gy=tmp+cy*gy;
	    const float *restrict pxy2=pxy+isa*18;
	    atomicAdd(&map[iy    *nx+ix],   gx*pxy2[0] + gy*pxy2[9]);
	    atomicAdd(&map[iy    *nx+ix+1], gx*pxy2[1] + gy*pxy2[10]);
	    atomicAdd(&map[iy    *nx+ix+2], gx*pxy2[2] + gy*pxy2[11]);
	    atomicAdd(&map[(iy+1)*nx+ix],   gx*pxy2[3] + gy*pxy2[12]);
	    atomicAdd(&map[(iy+1)*nx+ix+1], gx*pxy2[4] + gy*pxy2[13]);
	    atomicAdd(&map[(iy+1)*nx+ix+2], gx*pxy2[5] + gy*pxy2[14]);
	    atomicAdd(&map[(iy+2)*nx+ix],   gx*pxy2[6] + gy*pxy2[15]);
	    atomicAdd(&map[(iy+2)*nx+ix+1], gx*pxy2[7] + gy*pxy2[16]);
	    atomicAdd(&map[(iy+2)*nx+ix+2], gx*pxy2[8] + gy*pxy2[17]);
	}
    }
}
/*Only do tt, NEA. do not to GP'*/
__global__ static void gpu_nea_do(GPU_GP_T *data, float *ttin, float *dfin, float **gin){
    const int iwfs=blockIdx.z;
    const int nwfs=gridDim.z;
    GPU_GP_T *datai=data+iwfs;
    const int pos=datai->pos;
    if(!pos) return;
    const int step=blockDim.x * gridDim.x;
    const int nsa=datai->nsa;
    int (*saptr)[2]=datai->saptr;
    const float (*restrict neai)[3]=datai->neai;
    float dxp=datai->dxp;
    float oxp=datai->oxp;
    float oyp=datai->oyp;
    float focus=0;
    if(datai->PDF){
	if(iwfs==0){
	    for(int id=1; id<nwfs; id++){
		focus+=dfin[id];
	    }
	}else{
	    focus=-dfin[iwfs];
	}
    }
    float *restrict g=gin[iwfs];
    float ttx=ttin[iwfs*2+0];
    float tty=ttin[iwfs*2+1];
    int addfocus=fabsf(focus)>1e-7?1:0;
    for(int isa=blockIdx.x * blockDim.x + threadIdx.x; isa<nsa; isa+=step){
	int ix=saptr[isa][0];
	int iy=saptr[isa][1];
	float cx=neai[isa][0];
	float cy=neai[isa][1];
	float cxy=neai[isa][2];
	float gx=g[isa]+ttx;
	float gy=g[isa+nsa]+tty;
	if(addfocus){
	    gx+=focus*(ix*dxp+oxp);
	    gy+=focus*(iy*dxp+oyp);
	}
	float tmp=cxy*gx;
	g[isa]=cx*gx+cxy*gy;
	g[isa+nsa]=tmp+cy*gy;
    }
}

/*
  Tomography right hand size matrix. Computes xout = xout *beta + alpha * Hx' G' C * xin.
  xout is zeroed out before accumulation.
*/
void gpu_TomoR(curcell **xout, float beta, const void *A, const curcell *grad, float alpha, stream_t &stream){
    CTIC;ctic;
    curecon_t *curecon=cudata->recon;
    const RECON_T *recon=(const RECON_T *)A;
    if(!*xout){
	*xout=curcellnew(recon->npsr, 1, recon->xnx, recon->xny);
    }
    curcell *opdx=*xout;
    const int nwfs=grad->nx;
    curcell *opdwfs=curecon->opdwfs;
    curmat *ttf=curecon->ttf;
    curzero(ttf, stream);
    gpu_gp_do<<<dim3(24,1,nwfs), dim3(DIM_GP,1), 0, stream>>>
	(curecon->gpdata, grad->pm, ttf->p, ttf->p+nwfs*2, NULL, 1);
    curzero(opdwfs->m, stream);
    gpu_gpt_do<<<dim3(24,1,nwfs), dim3(DIM_GP,1), 0, stream>>>
	(curecon->gpdata, opdwfs->pm, ttf->p, ttf->p+nwfs*2, grad->pm, 1);
    if(fabsf(beta)<EPS){
	curzero(opdx->m, stream);
    }else if(fabsf(beta-1)>EPS){
	curscale(opdx->m, beta, stream);
    }
    gpu_prop_grid_adaptive_do<<<dim3(3,3,recon->npsr), dim3(16,16), 0, stream>>>
	(curecon->hxtdata, opdwfs->pm, opdx->pm, nwfs, recon->npsr, alpha, 't');
    ctoc("TomoR");
}

void gpu_TomoRt(curcell **gout, float beta, const void *A, const curcell *xin, float alpha, stream_t &stream){
    curecon_t *curecon=cudata->recon;
    const RECON_T *recon=(const RECON_T *)A;
    const PARMS_T *parms=recon->parms;
    const int nwfs=parms->nwfsr;
    if(!*gout){
	*gout=curcellnew(parms->nwfs, 1, recon->ngrad, (long*)NULL);
    }
    curcell *grad=*gout;
    curcell *opdwfs=curecon->opdwfs;
    curmat *ttf=curecon->ttf;
    curzero(opdwfs->m, stream);
    gpu_prop_grid_adaptive_do<<<dim3(3,3, nwfs), dim3(16,16), 0, stream>>>
      (curecon->hxdata, opdwfs->pm, xin->pm, nwfs, recon->npsr, alpha, 'n');
    curzero(ttf, stream);
    gpu_gp_do<<<dim3(24,1,nwfs), dim3(DIM_GP,1), 0, stream>>>
	(curecon->gpdata, grad->pm, ttf->p, ttf->p+nwfs*2, opdwfs->pm, 1);
    gpu_nea_do<<<dim3(24,1,nwfs), dim3(DIM_GP,1), 0, stream>>>
	(curecon->gpdata, ttf->p, ttf->p+nwfs*2, grad->pm);
}

/*
  Tomography left hand size matrix. Computes xout = beta*xout + alpha * Hx' G' C Gp Hx * xin.
  xout is zeroed out before accumulation.
*/
void gpu_TomoL(curcell **xout, float beta, const void *A, const curcell *xin, float alpha, stream_t &stream){
    curecon_t *curecon=cudata->recon;
    const RECON_T *recon=(const RECON_T *)A;
    const PARMS_T *parms=recon->parms;
    const int nwfs=parms->nwfsr;
    curcell *grad=curecon->grad;
    if(!*xout){
	*xout=curcellnew(recon->npsr, 1, recon->xnx, recon->xny);
    }
    curcell *opdx=*xout;
    curcell *opdwfs=curecon->opdwfs;
    int ptt=!parms->recon.split || parms->dbg.splitlrt; 
    curmat *ttf=curecon->ttf;
    curzero(opdwfs->m, stream);
    gpu_prop_grid_adaptive_do<<<dim3(3,3, nwfs), dim3(16,16), 0, stream>>>
	(curecon->hxdata, opdwfs->pm, xin->pm, nwfs, recon->npsr, 1, 'n');

    curzero(ttf, stream);
    gpu_gp_do<<<dim3(24,1,nwfs), dim3(DIM_GP,1), 0, stream>>>
	(curecon->gpdata, grad->pm, ttf->p, ttf->p+nwfs*2, opdwfs->pm, ptt);

    curzero(opdwfs->m, stream);
    gpu_gpt_do<<<dim3(24,1,nwfs), dim3(DIM_GP,1), 0, stream>>>
	(curecon->gpdata, opdwfs->pm, ttf->p, ttf->p+nwfs*2, grad->pm, ptt);

    if(fabsf(beta)<EPS){
	curzero(opdx->m, stream);
    }else if(fabsf(beta-1.f)>EPS){
	curscale(opdx->m, beta, stream);
    }
    gpu_prop_grid_adaptive_do<<<dim3(3,3,recon->npsr), dim3(16,16), 0, stream>>>
	(curecon->hxtdata, opdwfs->pm, opdx->pm, nwfs, recon->npsr, alpha, 't');
    gpu_laplacian_do<<<dim3(3,3,recon->npsr),dim3(16,16), 0, stream>>>
	(curecon->hxdata, opdx->pm, xin->pm, nwfs, alpha);
    //overhead of TomoL 27 micro-seconds (timing without synchornization).
}
/**
   Wrap of the tomography operation

   grad is the gradient input.
   opdx is the right hand side vector computed from grad. Allow NULL.
   opdr is the tomography result.
*/
double gpu_tomo_do(const PARMS_T *parms,const RECON_T *recon, curcell *opdr, curcell *opdx, curcell *grad, stream_t &stream){
    curcell *rhs=NULL;
    if(opdx){
	rhs=opdx;
    }
    double res=0;
    curecon_t *curecon=cudata->recon;
    curmat *tmp=NULL;
    gpu_TomoR(&rhs, 0, recon, grad, 1, stream);
    switch(parms->tomo.alg){
    case 0:
	if(!opdr->m){
	    error("opdr must be continuous\n");
	}
	if(!rhs->m){
	    error("rhs must be continuous\n");
	}
	cuchol_solve(opdr->m->p, curecon->RCl, curecon->RCp, rhs->m->p, stream);
	if(curecon->RUp){
	    tmp=curnew(curecon->RVp->ny, 1);
	    curmv(tmp->p, 0, curecon->RVp, rhs->m->p, 't', -1, stream);
	    curmv(opdr->m->p, 1, curecon->RUp, tmp->p, 'n', 1, stream);
	}
	break;
    case 1:{
	G_PREFUN prefun=NULL;
	void *predata=NULL;
	if(parms->tomo.precond==1){
	    prefun=gpu_Tomo_fdprecond;
	    predata=(void*)recon;
	}
	if((res=gpu_pcg(&opdr, gpu_TomoL, recon, prefun, predata, rhs, &curecon->cgtmp_tomo, 
			parms->recon.warm_restart, parms->tomo.maxit, stream))>1){
	    warning("Tomo CG not converge.\n");
	}
    }break;
    case 2:
	curmv(opdr->m->p, 0, curecon->RMI, rhs->m->p, 'n', 1, stream);
	break;
    default:
	error("Invalid");
    }
    if(!opdx){
	curcellfree(rhs);
    }
    if(tmp) curfree(tmp);
    return res;
}
void gpu_tomo_test(SIM_T *simu){
    gpu_set(gpu_recon);
    curecon_t *curecon=cudata->recon;
    const PARMS_T *parms=simu->parms;
    stream_t &stream=curecon->cgstream[0];//had to be cgstream because fdpcg uses it intrisically
    RECON_T *recon=simu->recon;
    /*Debugging. */
    dcell *rhsc=NULL;
    dcell *lc=NULL;
    dcell *rtc=NULL;
    curcell *rhsg=NULL;
    curcell *lg=NULL;
    curcell *rtg=NULL;
    if(1){
	muv(&rhsc, &recon->RR, simu->gradlastol, 1);
    }else{
	rhsc=dcellread("../load_rhs");
	cp2gpu(&rhsg, rhsc);
	dcellscale(rhsc, 1.e12);
	for(int i=0; i<rhsc->nx; i++){
	    if(rhsc->p[i]->nx != recon->xnx[i]*recon->xny[i]){
		error("Loaded RHS has wrong dimension\n");
	    }
	    rhsc->p[i]->nx=rhsc->p[i]->nx*rhsc->p[i]->ny;
	    rhsc->p[i]->ny=1;
	    rhsg->p[i]->nx=recon->xnx[i];
	    rhsg->p[i]->ny=recon->xny[i];
	}
    }
    dcellwrite(rhsc, "CPU_TomoR");
    muv_trans(&rtc, &recon->RR, rhsc, 1);
    dcellwrite(rtc, "CPU_TomoRt");
    muv(&lc, &recon->RL, rhsc, 1);
    dcellwrite(lc, "CPU_TomoL");
    muv(&lc, &recon->RL, rhsc, -1);
    dcellwrite(lc, "CPU_TomoL2");
    dcellzero(lc);
    muv(&lc, &recon->RL, rhsc, 1);
    dcellwrite(lc, "CPU_TomoL3");
    if(parms->tomo.alg==1 && parms->tomo.precond==1){
	dcell *lp=NULL;
	fdpcg_precond(&lp, recon, lc);
	dcellwrite(lp, "CPU_TomoP");
	fdpcg_precond(&lp, recon, lc);
	dcellwrite(lp, "CPU_TomoP2");
    }
    dcellzero(lc);
    recon->desplitlrt=1;//temporary.
    for(int i=0; i<10; i++){
	muv_solve(&lc, &recon->RL, NULL, rhsc);
	dcellwrite(lc, "CPU_Tomo_%d", i);
    }
	
    if(!rhsg){
	cp2gpu(&curecon->gradin, simu->gradlastol);
	gpu_TomoR(&rhsg, 0, recon, curecon->gradin, 1, stream);
    }
    curcellwrite(rhsg, "GPU_TomoR");
    gpu_TomoR(&rhsg, 0, recon, curecon->gradin, 1, stream);
    curcellwrite(rhsg, "GPU_TomoR1");
    gpu_TomoRt(&rtg, 0, recon, rhsg, 1, stream);
    curcellwrite(rtg, "GPU_TomoRt");
    gpu_TomoL(&lg, 0, recon, rhsg, 1,stream);
    curcellwrite(lg, "GPU_TomoL");
    gpu_TomoL(&lg, 0, recon, rhsg, 1,stream);
    curcellwrite(lg, "GPU_TomoL1"); 
    gpu_TomoL(&lg, 1, recon, rhsg, -1,stream);
    curcellwrite(lg, "GPU_TomoL2");
    gpu_TomoL(&lg, 0, recon, rhsg, 1,stream);
    curcellwrite(lg, "GPU_TomoL3");
    if(parms->tomo.alg==1 && parms->tomo.precond==1){
	curcell *lp=NULL;
	gpu_Tomo_fdprecond(&lp, recon, lg, stream);
	curcellwrite(lp, "GPU_TomoP");
	gpu_Tomo_fdprecond(&lp, recon, lg, stream);
	curcellwrite(lp, "GPU_TomoP2");
	//exit(0);
    }
    G_PREFUN prefun=NULL;
    void *predata=NULL;
    if(parms->tomo.precond==1){
	prefun=gpu_Tomo_fdprecond;
	predata=(void*)recon;
    }
    curcellzero(lg, 0);
    for(int i=0; i<10; i++){
	gpu_pcg(&lg, (G_CGFUN)gpu_TomoL, (void*)recon, prefun, predata, rhsg, &curecon->cgtmp_tomo,
		simu->parms->recon.warm_restart, parms->tomo.maxit, stream);
	CUDA_SYNC_STREAM;
	curcellwrite(lg, "GPU_Tomo_%d", i);
    }
    CUDA_SYNC_DEVICE;
    exit(0);
}
