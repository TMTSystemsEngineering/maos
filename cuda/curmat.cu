#include "hip/hip_runtime.h"
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "utils.h"
#include "curmat.h"

/**
   Createa curmat object.
*/
curmat *curnew(int nx, int ny){
    curmat *out;
    out=(curmat*)calloc(1, sizeof(curmat));
    out->ref=0;
    DO(hipMalloc(&(out->p), nx*ny*sizeof(float)));
    DO(hipMemset(out->p, 0, nx*ny*sizeof(float)));
    out->nx=nx;
    out->ny=ny;
    return out;
}
void curfree(curmat *A){
    if(A){
	if(A->p){
	    hipFree(A->p);
	}
	free(A);
    }
}
void curzero(curmat *A, hipStream_t stream){
    if(A && A->p){
	DO(hipMemsetAsync(A->p, 0, A->nx*A->ny*sizeof(float), stream));
    }
}
__global__ static void set_do(float *a, float alpha, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=alpha;
    }
}
void curset(curmat *A, float alpha, hipStream_t stream){
    if(A && A->p){
	set_do<<<DIM(A->nx*A->ny,256),0,stream>>>(A->p, alpha, A->nx*A->ny);
    }
}
__global__ static void show_do(float *a, int nx, int ny){
    const int stepx=blockDim.x * gridDim.x;
    const int stepy=blockDim.y * gridDim.y;
    for(int iy=blockIdx.y * blockDim.y + threadIdx.y; iy<ny; iy+=stepy){
	for(int ix=blockIdx.x * blockDim.x + threadIdx.x; ix<nx; ix+=stepx){
	    printf("a(%d,%d)=%g\n", ix, iy, a[ix+iy*nx]);
	}
    }
}
/**< Show the content of an array*/
void curshow(curmat *A, hipStream_t stream){
    info("curshow: %dx%d\n", A->nx, A->ny);
    show_do<<<1,1,0,stream>>>(A->p, A->nx, A->ny);
}
void curcp(curmat **out, const curmat *in, hipStream_t stream){
    if(!in){
	curzero(*out, stream);
    }else{
	if(!*out){
	    *out=curnew(in->nx, in->ny);
	}else{
	    assert((*out)->nx * (*out)->ny==in->nx * in->ny);
	}
	hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(float), hipMemcpyDefault, stream);
	
    }
}
void curwritedata(const curmat *A, file_t *fp){
    if(A && A->nx >0 && A->ny>0){
	hipDeviceSynchronize();
	float *tmp=(float*)malloc(A->nx*A->ny*sizeof(float));
	hipMemcpy(tmp, A->p, A->nx*A->ny*sizeof(float), hipMemcpyDefault);
	hipDeviceSynchronize();
	do_write(fp, 0, sizeof(float), M_FLT, tmp, A->nx, A->ny);
	free(tmp);
    }else{
	do_write(fp, 0, sizeof(float), M_FLT, NULL, 0, 0);
    }
}
void curwrite(const curmat *A, const char *format, ...){
    format2fn;
    file_t *fp=zfopen(fn, "wb");
    curwritedata(A, fp);
    zfclose(fp);
}
/**
   out=out*beta+in*alpha;
*/
void curadd(curmat **out, float beta, curmat *in, float alpha, hipblasHandle_t handle){
    if(*out){
	if(fabsf(beta-1)>1e-6){
	    hipblasSscal(handle, (*out)->nx*(*out)->ny, &beta, (*out)->p, 1);
	}
    }else{
	*out=curnew(in->nx, in->ny);
    }
    hipblasSaxpy(handle, in->nx*in->ny, &alpha, in->p, 1, (*out)->p, 1);
}
__global__ static void scale_do(float *restrict in, int n, float alpha){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	in[i]*=alpha;
    }
}
void curscale(curmat *in, float alpha, hipStream_t stream){
    int n=in->nx*in->ny;
    scale_do<<<MAX(MIN(n/256, 32), 1), MIN(n, 256), 0, stream>>>(in->p, n, alpha); 
}

/**
   Computes C = alpha * C + beta * op(A) * B ;
*/
void curmm(curmat **C, float alpha, const curmat *A, const curmat *B, char trans[2], float beta, hipblasHandle_t handle){
    if(!*C){
	*C=curnew(trans[0]=='t'?A->ny:A->nx, trans[1]=='t'?B->nx:B->ny);
    }else{
	assert((*C)->nx==(trans[0]=='t'?A->ny:A->nx) && (*C)->ny==(trans[1]=='t'?B->nx:B->ny));
    }

    if(B->ny==1){
	hipblasSgemv(handle, trans[0]=='t'?HIPBLAS_OP_T:HIPBLAS_OP_N, A->nx, A->ny, &beta, A->p, A->nx, B->p, 1, &alpha, (*C)->p, 1);
    }else{
	hipblasSgemm(handle, trans[0]=='t'?HIPBLAS_OP_T:HIPBLAS_OP_N, trans[1]=='t'?HIPBLAS_OP_T:HIPBLAS_OP_N, A->nx, B->ny, A->ny, &beta, A->p, A->nx, B->p, B->nx, &alpha, (*C)->p, (*C)->nx);
    }
}
/**
   Computes C = alpha * C + beta * op(A) * B ;
*/
void curmv(curmat **C, float alpha, const curmat *A, const curmat *B, char trans, float beta, hipblasHandle_t handle){
    if(!*C){
	*C=curnew(trans=='t'?A->ny:A->nx, 1);
    }else{
	assert((*C)->nx==(trans=='t'?A->ny:A->nx) && (*C)->ny==1);
    }
    hipblasSgemv(handle, trans=='t'?HIPBLAS_OP_T:HIPBLAS_OP_N, A->nx, A->ny, &beta, A->p, A->nx, B->p, 1, &alpha, (*C)->p, 1);
}

curcell* curcellnew(int nx, int ny){
    curcell *out=(curcell*)calloc(1, sizeof(curcell));
    out->p=(curmat**)calloc(nx*ny, sizeof(void*));
    out->nx=nx;
    out->ny=ny;
    return out;
}

cuspcell* cuspcellnew(int nx, int ny){
    cuspcell *out=(cuspcell*)calloc(1, sizeof(cuspcell));
    out->p=(cusp**)calloc(nx*ny, sizeof(void*));
    out->nx=nx;
    out->ny=ny;
    return out;
}
curcell *curcellnew2(const curcell *in){
    curcell *out=(curcell*)calloc(1, sizeof(curcell));
    out->p=(curmat**)calloc(in->nx*in->ny, sizeof(void*));
    out->nx=in->nx;
    out->ny=in->ny;
    for(int i=0; i<in->nx*in->ny; i++){
	out->p[i]=curnew(in->p[i]->nx, in->p[i]->ny);
    }
    return out;
}

void curcellfree(curcell *A){
    if(!A) return;
    if(A->p){
	for(int i=0; i<A->nx*A->ny; i++){
	    curfree(A->p[i]);
	}
	free(A->p);
    }
    free(A);
}

void curcellwrite(const curcell *A, const char *format, ...){
    format2fn;
    file_t *fp=zfopen(fn, "wb");
    write_magic(MCC_ANY, fp);
    if(A){
	uint64_t nx=A->nx;
	uint64_t ny=A->ny;
	zfwritelarr(fp, 2, &nx, &ny);
	for(int i=0; i<A->nx*A->ny; i++){
	    curwritedata(A->p[i], fp);
	}
    }else{
	uint64_t zero=0;
	zfwritelarr(fp, 2, &zero, &zero);
    }
    zfclose(fp);	
}
void curcellzero(curcell *A, hipStream_t stream){
    if(!A) return;
    for(int i=0; i<A->nx*A->ny; i++){
	curzero(A->p[i], stream);
    }
}
void curcellcp(curcell **A, const curcell *B, hipStream_t stream){
    if(!B)
	curcellzero(*A, stream);
    else{
	if(!*A){
	    *A=curcellnew2(B);
	}else{
	    assert((*A)->nx * (*A)->ny == B->nx * B->ny);
	}
	for(int i=0; i<B->nx*B->ny; i++){
	    curcp(&(*A)->p[i], B->p[i], stream);
	}
    }
}
/*
  A=A*beta+B*alpha;
*/
void curcelladd(curcell **A, float beta, const curcell *B, float alpha, hipblasHandle_t handle){
    if(!B) return;
    if(!*A){
	*A=curcellnew2(B);
    }else{
	assert((*A)->nx==B->nx && (*A)->ny==B->ny);
    }
    for(int i=0; i<B->nx*B->ny; i++){
	curadd(&((*A)->p[i]), beta, B->p[i], alpha, handle);
    }
}


/**
   add a scalar alpha, scaled by beta to a vector. all in device memory.
 */
__global__ void adds_do(float *vec, float *palpha, float beta, int n){
    __shared__ float alpha;
    if(threadIdx.x==0) alpha=beta**palpha;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	vec[i]+=alpha;
    }
}
/**
   add a vector to another, scaled by alpha and beta. all in device memory.
   a=a+b*alpha*beta;
*/
__global__ void add_do(float *restrict a, const float *restrict b, const float *restrict b_sc1, float b_sc2, int n){
    float alpha=*b_sc1*b_sc2;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]+=b[i]*alpha;
    }
}

__global__ static void add2_do(float *restrict a, const float *restrict a_sc, const float a_sc2, const float *restrict b, int n){
    float alpha=*a_sc*a_sc2;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=a[i]*alpha+b[i];
    }
}

/**
   out=out+in*alpha; beta, alpha lives in device memory.
*/
void curadd2(curmat **out, const curmat *in, float *alpha, hipStream_t stream){
    if(!*out){
	*out=curnew(in->nx, in->ny);
    }
    add_do<<<DIM(in->nx*in->ny, 256),0,stream>>>
	((*out)->p, in->p, alpha, 1.f, in->nx*in->ny);
}


/**
  A=A*beta+B*alpha; beta, alpha lives in device memory.
*/
void curcelladd2(curcell **A, const curcell *B, float* alpha, hipStream_t stream){
    if(!B) return;
    if(!*A){
	*A=curcellnew2(B);
    }else{
	assert((*A)->nx==B->nx && (*A)->ny==B->ny);
    }
    for(int i=0; i<B->nx*B->ny; i++){
	curadd2(&((*A)->p[i]), B->p[i], alpha, stream);
    }
}

/**
   out=out*beta+in; beta, alpha lives in device memory.
*/
void curadd3(curmat **out, float *beta, const curmat *in, hipStream_t stream){
    if(!*out){
	*out=curnew(in->nx, in->ny);
    }
    add2_do<<<DIM(in->nx*in->ny, 256),0,stream>>>
	((*out)->p, beta, 1.f, in->p, in->nx*in->ny);
}

/**
   A=A*beta+B*alpha; beta, alpha lives in device memory.
*/
void curcelladd3(curcell **A, float* beta, const curcell *B, hipStream_t stream){
    if(!B) return;
    if(!*A){
	*A=curcellnew2(B);
    }else{
	assert((*A)->nx==B->nx && (*A)->ny==B->ny);
    }
    for(int i=0; i<B->nx*B->ny; i++){
	curadd3(&((*A)->p[i]), beta, B->p[i],  stream);
    }
}
__global__ void inn_do(float *restrict res, const float *a, const float *b, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i]*b[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	atomicAdd(res, sb[0]);
    }
}
__global__ void sum_do(float *restrict res, const float *a, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	atomicAdd(res, sb[0]);
    }
}

inline static void inn_wrap(float *res, const float *restrict a, const float *restrict b, 
			    const int n, hipStream_t stream){
    inn_do<<<DIM(n, DIM_REDUCE), DIM_REDUCE*sizeof(float), stream>>>(res,a,b,n);
}
inline static void sum_wrap(float *res, const float *restrict a, const int n, hipStream_t stream){
    sum_do<<<DIM(n, DIM_REDUCE), DIM_REDUCE*sizeof(float), stream>>>(res,a,n);
}
float curinn(const curmat *a, const curmat *b, hipStream_t stream){
    float *res;
    hipMalloc(&res, sizeof(float));
    curinn2(res,a,b,stream);
    float out;
    hipMemcpyAsync(&out, res, sizeof(float), hipMemcpyDefault, stream);
    CUDA_SYNC_STREAM;
    return out;
}
float curcellinn(const curcell *A, const curcell *B, hipStream_t stream){
    float out;
    static float *res=NULL;
    if(!res) hipMalloc(&res, sizeof(float));
    curcellinn2(res, A, B, stream);
    hipMemcpyAsync(&out, res, sizeof(float), hipMemcpyDefault, stream);
    hipStreamSynchronize(stream);
    return out;
}

void curinn2(float *restrict res, const curmat *a, const curmat *b, hipStream_t stream){
    hipMemsetAsync(res, 0,sizeof(float), stream);
    inn_wrap(res, a->p, b->p, a->nx*a->ny, stream);
}
/**
   res points to a scalar in device memory.
*/
void curcellinn2(float *restrict res, const curcell *A, const curcell *B, hipStream_t stream){
    hipMemsetAsync(res, 0,sizeof(float), stream);
    for(int i=0; i<A->nx*A->ny; i++){
	const curmat *a=A->p[i];
	const curmat *b=B->p[i];
	const int n=a->nx*a->ny;
	inn_wrap(res, a->p, b->p, n, stream);
    }
}

/**
   Sum all the elements in an array.
 */
void cursum2(float *restrict res, const curmat *a, hipStream_t stream){
    hipMemsetAsync(res, 0,sizeof(float), stream);
    sum_wrap(res, a->p, a->nx*a->ny, stream);
}
