#include "hip/hip_runtime.h"
/*
  Copyright 2009, 2010, 2011 Lianqi Wang <lianqiw@gmail.com> <lianqiw@tmt.org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
extern "C"
{
#include <hip/hip_runtime.h>
#include "gpu.h"
}
#include "curmat.h"
#include "utils.h"
#include "kernel.h"
/**
   Createa curmat object.
*/
curmat *curnew(int nx, int ny){
    curmat *out;
    out=(curmat*)calloc(1, sizeof(curmat));
    out->nref=(int*)calloc(1, sizeof(int));
    out->nref[0]=1;
    DO(hipMalloc(&(out->p), nx*ny*sizeof(float)));
    DO(hipMemset(out->p, 0, nx*ny*sizeof(float)));
    out->nx=nx;
    out->ny=ny;
    return out;
}

/**
   Create curmat object with existing vector. own=1: we own the pointer. own=0: we don't own the pointer.
*/
curmat *curnew(int nx, int ny, float *p, int own){
    curmat *out;
    out=(curmat*)calloc(1, sizeof(curmat));
    if(own){
	out->nref=(int*)calloc(1, sizeof(int));
	out->nref[0]=1;
    }
    out->p=p;
    out->nx=nx;
    out->ny=ny;
    return out;
}
/**
   Createa curmat object.
*/
curmat *curnew(int nx, int ny, hipStream_t stream){
    curmat *out;
    out=(curmat*)calloc(1, sizeof(curmat));
    out->nref=(int*)calloc(1, sizeof(int));
    out->nref[0]=1;
    DO(hipMalloc(&(out->p), nx*ny*sizeof(float)));
    DO(hipMemsetAsync(out->p, 0, nx*ny*sizeof(float), stream));
    out->nx=nx;
    out->ny=ny;
    return out;
}
curmat *curref(curmat *A){
    if(!A) return NULL;
    curmat *out=(curmat*)calloc(1, sizeof(curmat));
    memcpy(out, A, sizeof(curmat));
    A->nref[0]++;
    return out;
}

void curfree(curmat *A){
    if(A){
	if(A->nref){
	    if(A->nref[0]==1){
		hipFree(A->p);
		free(A->nref);
	    }else{
		A->nref[0]--;
		if(A->nref[0]<0){
		error("Invalid nref=%d\n", A->nref[0]);
		}
	    }
	}
	free(A);
    }
}
void curzero(curmat *A, hipStream_t stream){
    if(A && A->p){
	DO(hipMemsetAsync(A->p, 0, A->nx*A->ny*sizeof(float), stream));
    }
}
void curset(curmat *A, float alpha, hipStream_t stream){
    if(A && A->p){
	set_do<<<DIM(A->nx*A->ny,256),0,stream>>>(A->p, alpha, A->nx*A->ny);
    }
}
/**< Show the content of an array*/
void curshow(curmat *A, hipStream_t stream){
    info("curshow: %dx%d\n", A->nx, A->ny);
    show_do<<<1,1,0,stream>>>(A->p, A->nx, A->ny);
}
void curcp(curmat **out, const curmat *in, hipStream_t stream){
    if(!in){
	curzero(*out, stream);
    }else{
	if(!*out){
	    *out=curnew(in->nx, in->ny);
	}else{
	    assert((*out)->nx * (*out)->ny==in->nx * in->ny);
	}
	hipMemcpyAsync((*out)->p, in->p, in->nx*in->ny*sizeof(float), hipMemcpyDeviceToDevice, stream);
    }
}
void curwritedata(const curmat *A, file_t *fp){
    if(A && A->nx>0 && A->ny>0){
	hipDeviceSynchronize();
	float *tmp=(float*)malloc(A->nx*A->ny*sizeof(float));
	hipMemcpy(tmp, A->p, A->nx*A->ny*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	do_write(fp, 0, sizeof(float), M_FLT, tmp, A->nx, A->ny);
	free(tmp);
    }else{
	do_write(fp, 0, sizeof(float), M_FLT, NULL, 0, 0);
    }
}
void curwrite(const curmat *A, const char *format, ...){
    format2fn;
    file_t *fp=zfopen(fn, "wb");
    curwritedata(A, fp);
    zfclose(fp);
}

/**
   add a vector to another, scaled by alpha and beta. all in device memory.
   a=a*alpha+b*beta;
*/
__global__ void add_do(float *restrict a, float alpha, 
		       const float * b, float beta, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=a[i]*alpha+b[i]*beta;
    }
}

__global__ static void scale_do(float *restrict in, int n, float alpha){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	in[i]*=alpha;
    }
}

/**
   out=out*beta+in*alpha;
*/
void curadd(curmat **out, float alpha, curmat *in, float beta, hipStream_t stream){
    if(!in) return;
    if(!*out || fabsf(alpha)<1e-7){
	curcp(out, in, stream);
	scale_do<<<DIM(in->nx*in->ny, 256),0,stream>>>
	    ((*out)->p, in->nx*in->ny, beta);
    }else{
	add_do<<<DIM(in->nx*in->ny, 256),0,stream>>>
	    ((*out)->p, alpha, in->p, beta, in->nx*in->ny);
    }
}
__global__ void addcabs2_do(float *restrict a, float alpha, 
			    const fcomplex *restrict b, float beta, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=a[i]*alpha+CABS2(b[i])*beta;
    }
}
/**
   out=out*beta+abs2(in)*alpha;
*/
void curaddcabs2(curmat **out, float alpha, cucmat *in, float beta, hipStream_t stream){
    if(!*out){
	*out=curnew(in->nx,in->ny);
    }else if(fabsf(alpha)<1e-7){
	curzero(*out, stream);
    }
    addcabs2_do<<<DIM(in->nx*in->ny, 256),0,stream>>>
	((*out)->p, alpha, in->p, beta, in->nx*in->ny);
}
void curscale(curmat *in, float alpha, hipStream_t stream){
    if(!in) return;
    if(fabsf(alpha)<1e-5) {
	curzero(in, stream);
    }else if(fabsf(alpha-1.f)>1e-5){
	int n=in->nx*in->ny;
	scale_do<<<DIM(n,256), 0, stream>>>(in->p, n, alpha); 
    }
}

/**
   Computes C = alpha * C + beta * op(A) * B ;
*/
void curmm(curmat **C, float alpha, const curmat *A, const curmat *B, char trans[2], float beta, hipblasHandle_t handle){
    int m,n,k,k2;
    hipblasOperation_t transa, transb;
    if(trans[0]=='t'){
	m=A->ny;
	k=A->nx;
	transa=HIPBLAS_OP_T;
    }else{
	m=A->nx;
	k=A->ny;
	transa=HIPBLAS_OP_N;
    }
    if(trans[1]=='t'){
	n=B->nx;
	k2=B->ny;
	transb=HIPBLAS_OP_T;
    }else{
	n=B->ny;
	k2=B->nx;
	transb=HIPBLAS_OP_N;
    }
    if(!*C){
	*C=curnew(m,n);
    }else{
	assert((*C)->nx==m && (*C)->ny==n);
    }
    assert(k==k2);
    DO(hipblasSgemm(handle, transa, transb, m,n,k,
		       &beta, A->p, A->nx, B->p, B->nx, &alpha, (*C)->p, (*C)->nx));
}
/**
   Computes C = alpha * C + beta * op(A) * B ;
*/
void curmv(curmat **C, float alpha, const curmat *A, const curmat *B, char trans, float beta, hipblasHandle_t handle){
    if(!*C){
	*C=curnew(trans=='t'?A->ny:A->nx, 1);
    }else{
	assert((*C)->nx==(trans=='t'?A->ny:A->nx) && (*C)->ny==1);
    }
    hipblasSgemv(handle, trans=='t'?HIPBLAS_OP_T:HIPBLAS_OP_N, A->nx, A->ny, &beta, A->p, A->nx, B->p, 1, &alpha, (*C)->p, 1);
}

curcell* curcellnew(int nx, int ny){
    curcell *out=(curcell*)calloc(1, sizeof(curcell));
    out->p=(curmat**)calloc(nx*ny, sizeof(void*));
    out->nx=nx;
    out->ny=ny;
    return out;
}
/** Allocate continuous memory for blocks of the same size*/
curcell *curcellnew(int nx, int ny, int mx, int my){
    curcell *out=curcellnew(nx, ny);
    out->m=curnew(mx, my*nx*ny);
    for(int i=0; i<nx*ny; i++){
	out->p[i]=curnew(mx, my, out->m->p+i*(mx*my), 0);
    }
    return out;
}
cuspcell* cuspcellnew(int nx, int ny){
    cuspcell *out=(cuspcell*)calloc(1, sizeof(cuspcell));
    out->p=(cusp**)calloc(nx*ny, sizeof(void*));
    out->nx=nx;
    out->ny=ny;
    return out;
}
curcell *curcellnew2(const curcell *in){
    curcell *out=(curcell*)calloc(1, sizeof(curcell));
    out->p=(curmat**)calloc(in->nx*in->ny, sizeof(void*));
    out->nx=in->nx;
    out->ny=in->ny;
    for(int i=0; i<in->nx*in->ny; i++){
	out->p[i]=curnew(in->p[i]->nx, in->p[i]->ny);
    }
    return out;
}

void curcellfree(curcell *A){
    if(!A) return;
    if(A->p){
	for(int i=0; i<A->nx*A->ny; i++){
	    curfree(A->p[i]);
	}
	free(A->p);
    }
    curfree(A->m);
    free(A);
}

void curcellwrite(const curcell *A, const char *format, ...){
    format2fn;
    file_t *fp=zfopen(fn, "wb");
    write_magic(MCC_ANY, fp);
    if(A){
	uint64_t nx=A->nx;
	uint64_t ny=A->ny;
	zfwritelarr(fp, 2, &nx, &ny);
	for(int i=0; i<A->nx*A->ny; i++){
	    curwritedata(A->p[i], fp);
	}
    }else{
	uint64_t zero=0;
	zfwritelarr(fp, 2, &zero, &zero);
    }
    zfclose(fp);	
}
void curcellzero(curcell *A, hipStream_t stream){
    if(!A) return;
    if(A->m){
	curzero(A->m, stream);
    }else{
	for(int i=0; i<A->nx*A->ny; i++){
	    curzero(A->p[i], stream);
	}
    }
}
void curcellcp(curcell **A, const curcell *B, hipStream_t stream){
    if(!B)
	curcellzero(*A, stream);
    else{
	if(!*A){
	    *A=curcellnew2(B);
	}else{
	    assert((*A)->nx * (*A)->ny == B->nx * B->ny);
	}
	for(int i=0; i<B->nx*B->ny; i++){
	    curcp(&(*A)->p[i], B->p[i], stream);
	}
    }
}
/*
  A=A*beta+B*alpha;
*/
void curcelladd(curcell **A, float beta, const curcell *B, float alpha, hipStream_t stream){
    if(!B) return;
    if(!*A){
	*A=curcellnew2(B);
    }else{
	assert((*A)->nx==B->nx && (*A)->ny==B->ny);
    }
    for(int i=0; i<B->nx*B->ny; i++){
	curadd(&((*A)->p[i]), beta, B->p[i], alpha,stream);
    }
}


/**
   add a beta to a vector. 
*/
__global__ void adds_do(float *vec, float beta, int n){
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	vec[i]+=beta;
    }
}
void curadds(curmat *A, float beta, hipStream_t stream){
    const int n=A->nx*A->ny;
    adds_do<<<DIM(n, 256), 0, stream>>>(A->p, beta, n);
}
/**
   add a vector to another, scaled by alpha and beta. all in device memory.
   a=a+b*alpha*beta;
*/
__global__ void add2_do(float *restrict a, const float * b, const float *restrict b_sc1, float b_sc2, int n){
    float alpha=*b_sc1*b_sc2;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]+=b[i]*alpha;
    }
}


/**
   out=out+in*alpha; beta, alpha lives in device memory.
*/
void curadd2(curmat **out, const curmat *in, float *alpha, float alpha2, hipStream_t stream){
    if(!*out){
	*out=curnew(in->nx, in->ny);
    }
    add2_do<<<DIM(in->nx*in->ny, 256),0,stream>>>
	((*out)->p, in->p, alpha, alpha2, in->nx*in->ny);
}


/**
   A=A*beta+B*alpha; beta, alpha lives in device memory.
*/
void curcelladd2(curcell **A, const curcell *B, float* alpha, float alpha2, hipStream_t stream){
    if(!B) return;
    if(!*A){
	*A=curcellnew2(B);
    }else{
	assert((*A)->nx==B->nx && (*A)->ny==B->ny);
    }
    for(int i=0; i<B->nx*B->ny; i++){
	curadd2((*A)->p+i, B->p[i], alpha, alpha2,  stream);
    }
}

__global__ static void add3_do(float *restrict a, const float *restrict a_sc, const float a_sc2, const float *restrict b, int n){
    float alpha=*a_sc*a_sc2;
    const int step=blockDim.x * gridDim.x;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	a[i]=a[i]*alpha+b[i];
    }
}
/**
   out=out*beta+in; beta, alpha lives in device memory.
*/
void curadd3(curmat **out, float *beta, const curmat *in, hipStream_t stream){
    if(!*out){
	*out=curnew(in->nx, in->ny);
    }
    add3_do<<<DIM(in->nx*in->ny, 256),0,stream>>>
	((*out)->p, beta, 1.f, in->p, in->nx*in->ny);
}

/**
   A=A*beta+B*alpha; beta, alpha lives in device memory.
*/
void curcelladd3(curcell **A, float* beta, const curcell *B, hipStream_t stream){
    if(!B) return;
    if(!*A){
	*A=curcellnew2(B);
    }else{
	assert((*A)->nx==B->nx && (*A)->ny==B->ny);
    }
    for(int i=0; i<B->nx*B->ny; i++){
	curadd3(&((*A)->p[i]), beta, B->p[i],  stream);
    }
}
__global__ void inn_do(float *restrict res, const float *a, const float *b, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i]*b[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	atomicAdd(res, sb[0]);
    }
}
__global__ void sum_do(float *restrict res, const float *a, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	sb[threadIdx.x]+=a[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    sb[threadIdx.x]+=sb[threadIdx.x+step];
	}
    }
    if(threadIdx.x==0){
	atomicAdd(res, sb[0]);
    }
}

inline static void inn_wrap(float *res, const float * a, const float * b, 
			    const int n, hipStream_t stream){
    inn_do<<<DIM(n, DIM_REDUCE), DIM_REDUCE*sizeof(float), stream>>>(res,a,b,n);
}
inline static void sum_wrap(float *res, const float * a, const int n, hipStream_t stream){
    sum_do<<<DIM(n, DIM_REDUCE), DIM_REDUCE*sizeof(float), stream>>>(res,a,n);
}
float curinn(const curmat *a, const curmat *b, hipStream_t stream){
    float *res;
    hipMalloc(&res, sizeof(float));
    curinn2(res,a,b,stream);
    float out;
    hipMemcpyAsync(&out, res, sizeof(float), hipMemcpyDeviceToHost, stream);
    CUDA_SYNC_STREAM;
    return out;
}
float curcellinn(const curcell *A, const curcell *B, hipStream_t stream){
    float out;
    static float *res=NULL;
    if(!res) hipMalloc(&res, sizeof(float));
    curcellinn2(res, A, B, stream);
    hipMemcpyAsync(&out, res, sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    return out;
}

void curinn2(float *restrict res, const curmat *a, const curmat *b, hipStream_t stream){
    hipMemsetAsync(res, 0,sizeof(float), stream);
    inn_wrap(res, a->p, b->p, a->nx*a->ny, stream);
}
/**
   res points to a scalar in device memory.
*/
void curcellinn2(float *restrict res, const curcell *A, const curcell *B, hipStream_t stream){
    hipMemsetAsync(res, 0,sizeof(float), stream);
    for(int i=0; i<A->nx*A->ny; i++){
	const curmat *a=A->p[i];
	const curmat *b=B->p[i];
	const int n=a->nx*a->ny;
	inn_wrap(res, a->p, b->p, n, stream);
    }
}

/**
   Sum all the elements in an array.
 */
void cursum2(float *restrict res, const curmat *a, hipStream_t stream){
    hipMemsetAsync(res, 0,sizeof(float), stream);
    sum_wrap(res, a->p, a->nx*a->ny, stream);
}

__global__ void max_do(float *restrict res, const float *a, const int n){
    extern __shared__ float sb[];
    sb[threadIdx.x]=0;
    int step=blockDim.x * gridDim.x ;
    for(int i=blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=step){
	if(sb[threadIdx.x]<a[i]) sb[threadIdx.x]=a[i];
    }
    for(step=(blockDim.x>>1);step>0;step>>=1){
	__syncthreads();
	if(threadIdx.x<step){
	    if(sb[threadIdx.x]<sb[threadIdx.x+step]){
		sb[threadIdx.x]=sb[threadIdx.x+step];
	    }
	}
    }
    if(threadIdx.x==0){
	*res=sb[0];
    }
}
/**
   Find the maximum value
*/
float curmax(const curmat *a, hipStream_t stream){
    float *res;
    hipHostMalloc(&res, sizeof(float));
    int n=a->nx*a->ny;
    max_do<<<DIM(n, DIM_REDUCE), DIM_REDUCE*sizeof(float), stream>>> (res, a->p, n);
    CUDA_SYNC_STREAM;
    float out=res[0];
    hipHostFree(res);
    return out;
}

/**
   Scale elements
*/
void curcellscale(curcell *A, float alpha, hipStream_t stream){
    if(!A) return;
    for(int i=0; i<A->nx*A->ny; i++){
	curscale(A->p[i], alpha, stream);
    }
}
